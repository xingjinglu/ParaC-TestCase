#include "hip/hip_runtime.h"
__global__ void MorphDilition_cuda(unsigned char* src, 
    unsigned char *dst, int width, int height, 
    int structWidth, int structHeight){


  int x = threadIdx.x + blockIdx.x * blockDim.x;
  int y = threadIdx.y + blockIdx.y * blockDim.y;

  if(x >= width || y >= height)
    return;

  dst[y*width + x] = src[y*width+x];
  int mid = (structWidth+1)/2-1;

  if( (y < mid) || (Y >= height - mid -1) )
    return;
  if( (x < mid) || (x >= width - mid - 1) )
    return;

  for( int m = 0; m < structWidth; m++){
    for( int n = 0; n < structHeight; n++){
      val |= src[(y+m)*width+x+n];
    }
  }

  dst[y*width+x] = val;
}
