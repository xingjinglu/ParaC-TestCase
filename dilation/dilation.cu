#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>
#include <helper_functions.h>

const char *sSDKname = "erosion";
hipStream_t stream;

__global__ void MorphDilation_cuda(unsigned char*src, 
     unsigned char*dst, int width,int height, 
     int structWidth, int structHeight){

  int x = threadIdx.x + blockIdx.x*blockDim.x;
  int y = threadIdx.y + blockIdx.y*blockDim.y;

  if(x >= width || y >= height)
    return;

  dst[y*width+x] = src[y*width+x];
  int mid = (structWidth+1)/2 -1 ; // structWidth = structHeight = 3,4.

  if( (y < mid) || (y>=height-mid-1) )
    return;
  if( (x < mid) || (x >= width - mid -1 ))
    return;

  unsigned char val = 0;

#if 1
  for(int m = 0; m < structWidth; m++){
    for(int n = 0; n < structHeight; n++){
      val |= src[(y+n)*width+x+m];
    }
  }
#endif
#if 0
  val = val & src[y*width+x];
  val = val & src[y*width+x+1];
  val = val & src[y*width+x+2];
  val = val & src[(y+1)*width+x];
  val = val & src[(y+1)*width+x+1];
  val = val & src[(y+1)*width+x+2];
  val = val & src[(y+2)*width+x];
  val = val & src[(y+2)*width+x+1];
  val = val & src[(y+2)*width+x+2];
  val = val & src[(y+1)*width+x+2];
#endif

  dst[y*width+x] = val;
}
//
#define VEC 1
#define TYPE char
#define STYPE char
#define MAKETYPE char

#define STREAM_COUNT 4





hipEvent_t start, stop;




int N = 1 << 22;
int nreps = 10;
int inner_reps = 5;
int memsize;

dim3 block(1024);
dim3 grid;

int thread_blocks;

void cudaDeviceInfoSetup(int argc, char *argv[]);

main(int argc, char *argv[])
{
  
  cudaDeviceInfoSetup(argc, argv);

  if( argc < 3 ){
    std::cout<<"Please input Height and Width \n";
    return 0;
  }

  int Height = atoi(argv[1]);
  int Width = atoi(argv[2]);

  memsize = sizeof(unsigned char)*Width*Height;

  // Init on the host.
  unsigned char *src = (unsigned char*) malloc(memsize);
  unsigned char *dst = (unsigned char*) malloc(memsize);

  for(int i = 0; i < Height; i++)
    for( int j = 0; j < Width; j++){
      src[i*Width+j] = rand()%255;
    }


  unsigned char *d_src, *d_dst;

  //
  checkCudaErrors(hipStreamCreate(&stream));

  //


  hipEventCreate(&start);
  hipEventCreate(&stop);

  checkCudaErrors(hipMalloc(&d_src, memsize)); 
  checkCudaErrors(hipMalloc(&d_dst, memsize));

  // Time copies and kernel.
  checkCudaErrors( hipMemcpy(d_src, src, memsize, hipMemcpyHostToDevice ) );

  thread_blocks = N / block.x;
  dim3 threadPerBlock(16, 16);
  dim3 numBlocks(Width/threadPerBlock.x, Height/threadPerBlock.y);

  // Kernel wazrmup.
  MorphDilation_cuda<<<numBlocks, threadPerBlock>>>(d_src, d_dst, Width, Height, 3, 3);

  // Real do.
  float kernel_time;
  float total_time = 0.0;
  for( int i = 0; i < 10; i++){
    hipEventRecord(start, 0);
    MorphDilation_cuda<<<numBlocks, threadPerBlock>>>(d_src, d_dst, Width, Height, 3, 3);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&kernel_time, start, stop);
    total_time += kernel_time;
  }

  printf(" Kernel\t\t\t: %f ms \n", total_time/10);


  checkCudaErrors( hipMemcpy(dst, d_dst, memsize, hipMemcpyDeviceToHost ) );

    // verify the results, we will use the results for final output.
   // bool bResults ; //= test();

    // Free resources

      hipFree(d_src);
      hipFree(d_dst);
      free(src);
      free(dst);


      hipStreamDestroy(stream);
      hipEventDestroy(start);
      hipEventDestroy(stop);


      hipDeviceReset();
      //if(bResults) printf("OK\n");
      //else printf("Error \n");
}


void cudaDeviceInfoSetup(int argc, char *argv[] )
{

  int cuda_device = 0;
  float scale_factor;
  hipDeviceProp_t deviceProp;

  printf("[%s] - Starting ...\n", sSDKname);

  if(checkCmdLineFlag(argc, (const char**)argv, "device"))
  {
    cuda_device = getCmdLineArgumentInt(argc, (const char**)argv, "device=");

    if(cuda_device < 0){
      printf("Invalid command line parameters\n");
      exit(EXIT_FAILURE);
    }
    else{
      printf("cuda_device  %d\n", cuda_device);
      cuda_device = gpuDeviceInit(cuda_device);
      if(cuda_device < 0 ){
        printf("No CUDA  Capable devices found, exiting ...\n");
        exit(EXIT_SUCCESS);
      }

    }
  }
  else{

    // Otherwise pick the device with the highest Gflops/s
    cuda_device = gpuGetMaxGflopsDeviceId();
    checkCudaErrors(hipSetDevice(cuda_device));
    checkCudaErrors(hipGetDeviceProperties(&deviceProp, cuda_device));
    printf("> Using CUDA device [ %d ]: %s \n", cuda_device, deviceProp.name);
  }

  checkCudaErrors(hipGetDeviceProperties(&deviceProp, cuda_device));
  printf("[ %s] has %d MP(s) x %d (Cores/MP) = %d (Cores)\n",
      deviceProp.name, deviceProp.multiProcessorCount,
      _ConvertSMVer2Cores(deviceProp.major, deviceProp.minor),
      _ConvertSMVer2Cores(deviceProp.major, deviceProp.minor) * deviceProp.multiProcessorCount);

  // Anything that is less that 32 Cores will have scaled down workload.
  scale_factor = max((32.0f /(_ConvertSMVer2Cores(deviceProp.major, deviceProp.minor)*(float)deviceProp.multiProcessorCount)), 1.0f);
  N = (int)((float)N/scale_factor);
  printf("> Device name: %s\n", deviceProp.name);
  printf("> CUDA Capability %d.%d hardware with %d multi-processors\n",
      deviceProp.major, deviceProp.minor,
      deviceProp.multiProcessorCount);
  printf("> scale_factor = %.2f\n", 1.0f/scale_factor);
  printf("> array_size = %d\n\n", N);

  // --------------- End Platform information --------------

}

