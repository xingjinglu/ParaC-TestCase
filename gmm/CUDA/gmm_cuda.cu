
#include <hip/hip_runtime.h>
#include<iostream>
#include<cstring>
#include<cstdlib>

#define GMM_MAX_COMPONT 3	
#define GMM_LEARN_ALPHA 0.005  
#define GMM_THRESHOD_SUMW 0.7
#define HEIGHT 1080
#define WIDTH 1920

using namespace std;

__global__ void trainGMM_CUDA(unsigned char *_image, unsigned char *mask, float *modelW, float *modelS, unsigned char *modelM, int height, int width)
{
  const int x = blockIdx.x * blockDim.x + threadIdx.x;
  const int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x >= height || y >= width)
    return; 
 
  unsigned char *_imageData = _image + x * width + y;
 
  float *modelWData = modelW + GMM_MAX_COMPONT * (x * width + y);
  float *modelSData = modelS + GMM_MAX_COMPONT * (x * width + y);
  unsigned char *modelMData = modelM + GMM_MAX_COMPONT * (x * width + y);

	int num_fit = 0;
	
  for (int k = 0; k < GMM_MAX_COMPONT; k++)
	{
		int delm = abs(_imageData[0] - modelMData[k]);
		long dist = delm * delm;
		if (dist < 3.0 * modelSData[k])
		{
			modelWData[k] += GMM_LEARN_ALPHA * (1 - modelWData[k]);
			modelMData[k] += (GMM_LEARN_ALPHA / modelWData[k]) * delm;
			modelSData[k] += (GMM_LEARN_ALPHA / modelWData[k]) * (dist - modelSData[k]);
		}
		else
		{
			modelWData[k] += GMM_LEARN_ALPHA * (0 - modelWData[k]);
			num_fit++;
		}
	}

	for (int kk = 0; kk < GMM_MAX_COMPONT; kk++)
	{
		for (int rr = kk; rr< GMM_MAX_COMPONT; rr++)
		{
			if (modelWData[rr] / modelSData[rr] > modelWData[kk] / modelSData[kk])
			{
				float temp_weight = modelWData[rr];
				modelWData[rr] = modelWData[kk];
				modelWData[kk] = temp_weight;
				unsigned char temp_mean = modelMData[rr];
			  modelMData[rr] = modelMData[kk];
				modelMData[kk] = temp_mean;

				float temp_sigma = modelSData[rr];
				modelSData[rr] = modelSData[kk];
				modelSData[kk] = temp_sigma;
			}
		}
	}
	
  if (num_fit == GMM_MAX_COMPONT && modelWData[GMM_MAX_COMPONT - 1] == 0)
	{
		for (int k = 0; k < GMM_MAX_COMPONT; k++)
		{
			if (0 == modelWData[k])
			{
				if (k == 0)
				  modelWData[k] = 1;
				else
				  modelWData[k] = GMM_LEARN_ALPHA;
				
        modelMData[k] = _imageData[0];
				modelSData[k] = 15.0;

				for (int q = 0; q < GMM_MAX_COMPONT && q != k; q++)
				{
					modelWData[q] *= (1 - GMM_LEARN_ALPHA);
				}
				break;
			}
		}
	}
	else if (num_fit == GMM_MAX_COMPONT && modelWData[GMM_MAX_COMPONT - 1] != 0)
	{
		modelMData[GMM_MAX_COMPONT - 1] = _imageData[0];
		modelSData[GMM_MAX_COMPONT - 1] = 15.0;
	}
}

__global__ void testGMM_CUDA(unsigned char *_image, unsigned char *mask, float *modelW, float *modelS, unsigned char *modelM, int height, int width)
{
  const int x = blockIdx.x * blockDim.x + threadIdx.x;
  const int y = blockIdx.y * blockDim.y + threadIdx.y;
  
  if (x >= height || y >= width)
    return; 
 
  unsigned char *_imageData = _image + x * width + y;
  unsigned char *maskData = mask + x * width + y;
 
  float *modelWData = modelW + GMM_MAX_COMPONT * (x * width + y);
  float *modelSData = modelS + GMM_MAX_COMPONT * (x * width + y);
  unsigned char *modelMData = modelM + GMM_MAX_COMPONT * (x * width + y);
	
	float sum_w = 0.0;
	for (unsigned char k = 0; k < GMM_MAX_COMPONT; k++)
	{
		if (abs(_imageData[0] - modelMData[k]) < (unsigned char)(2.5 * modelSData[k]))
		{
			maskData[0] = 0;
			break;
		}
			sum_w += modelWData[k];
		if (sum_w >= GMM_THRESHOD_SUMW)
		{
			maskData[0] = 255;
			break;
		}
	}
}

void trainGMM(unsigned char *_image, unsigned char *mask, float *modelW, float *modelS, unsigned char *modelM, int height, int width)
{
	/**************************** Train ******************************************/
	for (int i = 0; i < height; i++)
	{
		float *modelWData = modelW + i * width * GMM_MAX_COMPONT;
		float *modelSData = modelS + i * width * GMM_MAX_COMPONT;
		unsigned char *modelMData = modelM + i * width * GMM_MAX_COMPONT;
		const unsigned char *_imageData = _image + i * width;
		for (int j = 0; j < width; j++)
		{
			int num_fit = 0;

			/**************************** Update parameters Start ******************************************/
			for (int k = 0; k < GMM_MAX_COMPONT; k++)
			{
				int delm = abs(_imageData[j] - modelMData[k]);
				long dist = delm * delm;
				if (dist < 3.0 * modelSData[k])
				{
					modelWData[k] += GMM_LEARN_ALPHA * (1 - modelWData[k]);
					modelMData[k] += (GMM_LEARN_ALPHA / modelWData[k]) * delm;
					modelSData[k] += (GMM_LEARN_ALPHA / modelWData[k]) * (dist - modelSData[k]);
				}
				else
				{
					modelWData[k] += GMM_LEARN_ALPHA * (0 - modelWData[k]);
					num_fit++;
				}
			}
			/**************************** Update parameters End ******************************************/

			/*********************** Sort Gaussian component by 'weight / sigma' Start ****************************/
			for (int kk = 0; kk < GMM_MAX_COMPONT; kk++)
			{
				for (int rr = kk; rr< GMM_MAX_COMPONT; rr++)
				{
					if (modelWData[rr] / modelSData[rr] > modelWData[kk] / modelSData[kk])
					{
						float temp_weight = modelWData[rr];
						modelWData[rr] = modelWData[kk];
						modelWData[kk] = temp_weight;

						unsigned char temp_mean = modelMData[rr];
						modelMData[rr] = modelMData[kk];
						modelMData[kk] = temp_mean;

						float temp_sigma = modelSData[rr];
						modelSData[rr] = modelSData[kk];
						modelSData[kk] = temp_sigma;
					}
				}
			}
			/*********************** Sort Gaussian model by 'weight / sigma' End ****************************/

			/*********************** Create new Gaussian component Start ****************************/
			if (num_fit == GMM_MAX_COMPONT && modelWData[GMM_MAX_COMPONT - 1] == 0)
			{
				//if there is no exit component fit,then start a new component
				for (int k = 0; k < GMM_MAX_COMPONT; k++)
				{
					if (0 == modelWData[k])
					{
						if (k == 0)
							modelWData[k] = 1;
						else
							modelWData[k] = GMM_LEARN_ALPHA;
						
            modelMData[k] = _imageData[j];
						modelSData[k] = 15.0;

						//normalization the weight,let they sum to 1
						for (int q = 0; q < GMM_MAX_COMPONT && q != k; q++)
						{
							/****update the other unfit's weight,u and sigma remain unchanged****/
							modelWData[q] *= (1 - GMM_LEARN_ALPHA);
						}
						break;
					}
				}
			}
			else if (num_fit == GMM_MAX_COMPONT && modelWData[GMM_MAX_COMPONT - 1] != 0)
			{
				modelMData[GMM_MAX_COMPONT - 1] = _imageData[j];
				modelSData[GMM_MAX_COMPONT - 1] = 15.0;
			}
			/*********************** Create new Gaussian component End ****************************/

			modelWData += GMM_MAX_COMPONT;
			modelSData += GMM_MAX_COMPONT;
			modelMData += GMM_MAX_COMPONT;
		}
	}
}

void testGMM(unsigned char* _image, unsigned char* mask, float* modelW, float* modelS, unsigned char* modelM, int height, int width)
{
	/*********************** Predict ****************************/
	for (int i = 0; i < height; i++)
	{
		float *modelWData = modelW + i * width * GMM_MAX_COMPONT;
		float *modelSData = modelS + i * width * GMM_MAX_COMPONT;
		unsigned char *modelMData = modelM + i * width * GMM_MAX_COMPONT;
		const unsigned char *_imageData = _image + i * width;
		unsigned char *maskData = mask + i * width;
		for (int j = 0; j < width; j++)
		{
			float sum_w = 0.0;
			for (unsigned char k = 0; k < GMM_MAX_COMPONT; k++)
			{
				if (abs(_imageData[j] - modelMData[k]) < (unsigned char)(2.5 * modelSData[k]))
				{
					maskData[j] = 0;
					break;
				}
				sum_w += modelWData[k];
				if (sum_w >= GMM_THRESHOD_SUMW)
				{
					maskData[j] = 255;
					break;
				}
			}
			modelWData += GMM_MAX_COMPONT;
			modelSData += GMM_MAX_COMPONT;
			modelMData += GMM_MAX_COMPONT;
		}
	}
}

int main()
{	
  float *modelW = new float[HEIGHT * GMM_MAX_COMPONT * WIDTH];
  float *modelS = new float[HEIGHT * GMM_MAX_COMPONT * WIDTH];
  unsigned char *modelM = new unsigned char[HEIGHT * GMM_MAX_COMPONT * WIDTH];;
  unsigned char *frame = new unsigned char[HEIGHT * WIDTH];
  unsigned char *mask = new unsigned char[HEIGHT * WIDTH];
  unsigned char *mask2 = new unsigned char[HEIGHT * WIDTH];
  int height = HEIGHT;
  int width = WIDTH;  

  memset(modelW, 0, sizeof(modelW));
  memset(modelS, 0, sizeof(modelM));
  memset(modelM, 0, sizeof(modelS));
  memset(mask, 0, sizeof(mask));

  // For GPU  
  float *dev_modelW, *dev_modelS;
  unsigned char *dev_frame, *dev_mask, *dev_modelM;

  hipMalloc((void**)&dev_modelW, (HEIGHT * GMM_MAX_COMPONT * WIDTH) * sizeof(float));
  hipMalloc((void**)&dev_modelS, (HEIGHT * GMM_MAX_COMPONT * WIDTH) * sizeof(float));
  hipMalloc((void**)&dev_modelM, (HEIGHT * GMM_MAX_COMPONT * WIDTH) * sizeof(unsigned char));
  hipMalloc((void**)&dev_frame, (HEIGHT * WIDTH) * sizeof(unsigned char));
  hipMalloc((void**)&dev_mask, (HEIGHT * WIDTH) * sizeof(unsigned char));

  hipMemcpy(dev_modelW,
             modelW,
             HEIGHT * GMM_MAX_COMPONT * WIDTH * sizeof(float),
             hipMemcpyHostToDevice);
  hipMemcpy(dev_modelS,
             modelS,
             HEIGHT * GMM_MAX_COMPONT * WIDTH * sizeof(float),
             hipMemcpyHostToDevice);
  hipMemcpy(dev_modelM,
             modelM,
             HEIGHT * GMM_MAX_COMPONT * WIDTH * sizeof(unsigned char),
             hipMemcpyHostToDevice);
  
  dim3 block(32,8);
  dim3 grid((HEIGHT + block.x - 1) / block.x, 
					  (WIDTH + block.y - 1) / block.y);

  clock_t start, end;
  clock_t start2, end2;
	double sum_t = 0;
	double sum_t2 = 0;
	 
  srand(unsigned(time(0)));  

  int frame_num = 100;
  for (int t = 0; t < frame_num; t++)
  {
    for (int i = 0; i < height; i++)
      for (int j = 0; j < width; j++)
        *(frame + i * width + j) = rand() % 255;
    
    start = clock();  
    trainGMM(frame, mask, modelW, modelS, modelM, height, width);
    testGMM(frame, mask, modelW, modelS, modelM, height, width);
    end = clock();
    sum_t += end - start;

    start2 = clock();
    hipMemcpy(dev_frame,
               frame,
               HEIGHT * WIDTH * sizeof(unsigned char),
               hipMemcpyHostToDevice);
    trainGMM_CUDA<<<grid, block>>>(dev_frame, dev_mask,  dev_modelW, dev_modelS, dev_modelM, height, width);    
    testGMM_CUDA<<<grid, block>>>(dev_frame, dev_mask,  dev_modelW, dev_modelS, dev_modelM, height, width);    
    hipMemcpy(mask2,
               dev_mask,
               HEIGHT * WIDTH * sizeof(unsigned char),
               hipMemcpyDeviceToHost);
    end2 = clock();
    sum_t2 += end2 - start2;
  
  // TEST
  /*
  int diff_count = 0;
  for (int i = 0; i < height * width; i++)
    if (mask[i] != mask2[i])
      diff_count++;
  
  cout <<"Diff: " << diff_count << endl;
  */
  }
	cout <<"CPU: " << (sum_t / frame_num) / CLOCKS_PER_SEC << endl;
	cout <<"GPU: " << (sum_t2 / frame_num) / CLOCKS_PER_SEC << endl;
}
