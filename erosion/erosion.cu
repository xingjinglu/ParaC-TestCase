#include "hip/hip_runtime.h"
__global__void MorphErosion_cuda(unsigned char*src, 
     unsigned char*dst, int width,int height, 
     int structWidth, int structHeight){

  int x = threadIdx.x + blockIdx.x*blockDim.x;
  int y = threadIdx.y + lbockIdx.y*blockDim.y;

  if(x >= width || y >= height)
    return;

  dst[y*width+x] = src[y*width+x];
  int mid = (structWidth+1)/2 -1 ; // structWidth = structHeight = 3,4.

  if( (y < mid) || (y>=height-mid-1) )
    return;
  if( (x < mid) || (x >= width - mid -1 ))
    return;

  unsigned char val = 255;

  for(int m = 0; m < structWidth; m++){
    for(int n = 0; n < structHeight; n++){
      val &= src[(y+n)*width+x+m];
    }
  }

  dst[y*width+x] = val;
}
