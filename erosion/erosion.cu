#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>
#include <helper_functions.h>

const char *sSDKname = "erosion";

__global__ void MorphErosion_cuda(unsigned char*src, 
     unsigned char*dst, int width,int height, 
     int structWidth, int structHeight){

  int x = threadIdx.x + blockIdx.x*blockDim.x;
  int y = threadIdx.y + blockIdx.y*blockDim.y;

  if(x >= width || y >= height)
    return;

  dst[y*width+x] = src[y*width+x];
  int mid = (structWidth+1)/2 -1 ; // structWidth = structHeight = 3,4.

  if( (y < mid) || (y>=height-mid-1) )
    return;
  if( (x < mid) || (x >= width - mid -1 ))
    return;

  unsigned char val = 255;

  for(int m = 0; m < structWidth; m++){
    for(int n = 0; n < structHeight; n++){
      val &= src[(y+n)*width+x+m];
    }
  }

  dst[y*width+x] = val;
}
//
#define VEC 1
#define TYPE char
#define STYPE char
#define MAKETYPE char

#define STREAM_COUNT 4





hipEvent_t start, stop;




int N = 1 << 22;
int nreps = 10;
int inner_reps = 5;
int memsize;

dim3 block(1024);
dim3 grid;

int thread_blocks;

void cudaDeviceInfoSetup(int argc, char *argv[]);

main(int argc, char *argv[])
{
  
  cudaDeviceInfoSetup(argc, argv);

  if( argc < 3 ){
    std::cout<<"Please input Height and Width \n";
    return 0;
  }

  int Height = atoi(argv[1]);
  int Width = atoi(argv[2]);

  memsize = sizeof(unsigned char)*Width*Height;

  // Init on the host.
  unsigned char *src = (unsigned char*) malloc(memsize);
  unsigned char *dst = (unsigned char*) malloc(memsize);

  for(int i = 0; i < Height; i++)
    for( int j = 0; j < Width; j++){
      src[i*Width+j] = rand()%255;
    }


  unsigned char *d_src, *d_dst;

  hipEventCreate(&start);
  hipEventCreate(&stop);

  checkCudaErrors(hipMalloc(&d_src, memsize)); 
  checkCudaErrors(hipMalloc(&d_dst, memsize));

  // Time copies and kernel.
  checkCudaErrors( hipMemcpy(d_src, src, memsize, hipMemcpyHostToDevice ) );

  thread_blocks = N / block.x;
  dim3 threadPerBlock(16, 16);
  dim3 numBlocks(Width/threadPerBlock.x, Height/threadPerBlock.y);

  // Kernel wazrmup.
  MorphErosion_cuda<<<numBlocks, threadPerBlock>>>(d_src, d_dst, Width, Height, 3, 3);

  // Real do.
  hipEventRecord(start, 0);
  MorphErosion_cuda<<<grid, block>>>(d_src, d_dst, Width, Height, 3, 3);
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);

  float kernel_time;
  hipEventElapsedTime(&kernel_time, start, stop);
  printf(" Kernel\t\t\t: %f ms \n", kernel_time);


  checkCudaErrors( hipMemcpy(dst, d_dst, memsize, hipMemcpyDeviceToHost ) );

    // verify the results, we will use the results for final output.
   // bool bResults ; //= test();

    // Free resources

      hipFree(d_src);
      hipFree(d_dst);
      free(src);
      free(dst);


      hipEventDestroy(start);
      hipEventDestroy(stop);


      hipDeviceReset();
      //if(bResults) printf("OK\n");
      //else printf("Error \n");
}


void cudaDeviceInfoSetup(int argc, char *argv[] )
{

  int cuda_device = 0;
  float scale_factor;
  hipDeviceProp_t deviceProp;

  printf("[%s] - Starting ...\n", sSDKname);

  if(checkCmdLineFlag(argc, (const char**)argv, "device"))
  {
    cuda_device = getCmdLineArgumentInt(argc, (const char**)argv, "device=");

    if(cuda_device < 0){
      printf("Invalid command line parameters\n");
      exit(EXIT_FAILURE);
    }
    else{
      printf("cuda_device  %d\n", cuda_device);
      cuda_device = gpuDeviceInit(cuda_device);
      if(cuda_device < 0 ){
        printf("No CUDA  Capable devices found, exiting ...\n");
        exit(EXIT_SUCCESS);
      }

    }
  }
  else{

    // Otherwise pick the device with the highest Gflops/s
    cuda_device = gpuGetMaxGflopsDeviceId();
    checkCudaErrors(hipSetDevice(cuda_device));
    checkCudaErrors(hipGetDeviceProperties(&deviceProp, cuda_device));
    printf("> Using CUDA device [ %d ]: %s \n", cuda_device, deviceProp.name);
  }

  checkCudaErrors(hipGetDeviceProperties(&deviceProp, cuda_device));
  printf("[ %s] has %d MP(s) x %d (Cores/MP) = %d (Cores)\n",
      deviceProp.name, deviceProp.multiProcessorCount,
      _ConvertSMVer2Cores(deviceProp.major, deviceProp.minor),
      _ConvertSMVer2Cores(deviceProp.major, deviceProp.minor) * deviceProp.multiProcessorCount);

  // Anything that is less that 32 Cores will have scaled down workload.
  scale_factor = max((32.0f /(_ConvertSMVer2Cores(deviceProp.major, deviceProp.minor)*(float)deviceProp.multiProcessorCount)), 1.0f);
  N = (int)((float)N/scale_factor);
  printf("> Device name: %s\n", deviceProp.name);
  printf("> CUDA Capability %d.%d hardware with %d multi-processors\n",
      deviceProp.major, deviceProp.minor,
      deviceProp.multiProcessorCount);
  printf("> scale_factor = %.2f\n", 1.0f/scale_factor);
  printf("> array_size = %d\n\n", N);

  // --------------- End Platform information --------------

}

