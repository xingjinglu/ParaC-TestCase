#include "hip/hip_runtime.h"
__global__ void  change_size_c3_cuda(unsigned char* dst,
    unsigned char* src, int width, int height,
    float rate){

  int x = threadIdx.x + blockIdx.x * blockDim.x;
  int y = threadIdx.y + blockIdx.y * blockDim.y;


  if( x >= width || y >= height)
    return;

  int new_x = (int)(x/rate);
  int new_y = (int)(y/rate);

  for( int k = 0; k < 3; k++){
    dst[ (y*width+x)*3+k] = src[((int)(new_y*(width/rate) + new_x))*3+k];
  }

}

__global__ void change_size_c1_cuda(unsigned char*dst,
    unsigned char *src, int width, int height, float rate){

  int x = threadIdx.x + blockIdx.x * blockDim.x;
  int y = threadIdx.y + blockIdx.y * blockDim.y;

  if( x >= width || y >= height)
    return;

  int new_x = (int)(x/rate);
  int new_y = (int)(y/rate);
  dst[y*width+x] = src[(int)(new_y * width/rate) + new_x)];


}






