#include "hip/hip_runtime.h"

#include <stdio.h>

#ifndef __HIPCC__
#include <stdlib.h>
#include <math.h>

#include <hmpprt/Grouplet.h>
#include <hmpprt/HostTypes.h>
#include <hmpprt/Context.h>
#include <hmpprt/CUDAGrid.h>
#include <hmpprt/CUDAModule.h>
#include <hmpprt/DeviceManager.h>
#include <hmpperr/hmpperr.h>

#include <openacci/openacci_c.h>

#ifdef _WIN32
#  define CDLT_API __declspec(dllexport)
#else /* ! _WIN32 */
#  define CDLT_API
#endif /* _WIN32 */



#else // ! __HIPCC__

#include <hmpprt/HostTypes.h>
#include <hmpprt/CUDAIntrinsics.h>

extern __shared__ int64_t hmpp_sharedmem[];
#endif // __HIPCC__



#ifndef __HIPCC__

#else


#endif

#define HMPPCG_SIMD_LENGTH 32

# 369 "svd_fuse.cpp"

#ifdef __HIPCC__
__device__ double reduce_double_add_x(double priv, double* tmpa, double orig)
;
#endif // __HIPCC__



# 369 "svd_fuse.cpp"

#ifndef __HIPCC__
extern "C" CDLT_API  void hmpp_acc_region_svd_369(double* A_H_4, double* B_H_1, double* S_H_5)
;
#endif // __HIPCC__



# 369 "svd_fuse.cpp"

#ifndef __HIPCC__
void hmpp_acc_region_svd_369_internal_1(hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  A_H_7, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  B_H_2, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  S_H_1)
;
#endif // __HIPCC__



# 330 "svd_fuse.cpp"

#ifndef __HIPCC__
static hmpprt::CUDAGrid * hmpp_acc_region_svd_369_parallel_region_1 = 0;
#else

extern "C" __global__ void hmpp_acc_region_svd_369_parallel_region_1(double* A_H_1, double* B_H, double* S_H_3);
#endif // __HIPCC__




# 330 "svd_fuse.cpp"

#ifndef __HIPCC__
extern "C" CDLT_API  void hmpp_acc_region_svd_338(double* A_H_5, double* S_H, double* ci_3)
;
#endif // __HIPCC__



# 330 "svd_fuse.cpp"

#ifndef __HIPCC__
void hmpp_acc_region_svd_338_internal_1(hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  A_H_8, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  S_H_4, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  ci_5)
;
#endif // __HIPCC__



# 60 "svd_fuse.cpp"

#ifndef __HIPCC__
static hmpprt::CUDAGrid * hmpp_acc_region_svd_338_parallel_region_1 = 0;
#else

extern "C" __global__ void hmpp_acc_region_svd_338_parallel_region_1(double* A_H_2, double* S_H_2, double* ci_1);
#endif // __HIPCC__




# 60 "svd_fuse.cpp"

#ifndef __HIPCC__
extern "C" CDLT_API  void hmpp_acc_region_svd_129(hmpprt::s32* pass_3, double* A_H_6, double* V_H_2, hmpprt::s32* rotate_col_idx_2, double* ci_4, double* cj, double* vi_1, double* vj_1, hmpprt::s32 start_index_1)
;
#endif // __HIPCC__



# 60 "svd_fuse.cpp"

#ifndef __HIPCC__
void hmpp_acc_region_svd_129_internal_1(hmpprt::s32* pass, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  A_H, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  V_H_1, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s32>  rotate_col_idx_1, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  ci_2, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  cj_2, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  vi_2, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  vj_2, hmpprt::s32 start_index_2)
;
#endif // __HIPCC__



# 60 "svd_fuse.cpp"

#ifndef __HIPCC__
static hmpprt::CUDAGrid * hmpp_acc_region_svd_129_parallel_region_1 = 0;
#else

extern "C" __global__ void hmpp_acc_region_svd_129_parallel_region_1(double* A_H_3, double* V_H, double* ci, double* cj_1, hmpprt::s32* pass_1, hmpprt::s32* rotate_col_idx, hmpprt::s32 start_index, double* value_2, double* vi, double* vj);
#endif // __HIPCC__




# 60 "svd_fuse.cpp"

#ifdef __HIPCC__

extern "C" __global__ void hmpp_acc_region_svd_129_parallel_region_1(double* A_H_3, double* V_H, double* ci, double* cj_1, hmpprt::s32* pass_1, hmpprt::s32* rotate_col_idx, hmpprt::s32 start_index, double* value_2, double* vi, double* vj)
{
 
 double * tmpr_4 = (double *)(((char *)hmpp_sharedmem + 0));
 
 # 62 "svd_fuse.cpp"
 {
  # 132 "svd_fuse.cpp"
  hmpprt::s32 iter_per_gang_3;
  # 132 "svd_fuse.cpp"
  hmpprt::s32 first_gang_iter_3;
  # 132 "svd_fuse.cpp"
  hmpprt::s32 last_gang_iter_3;
  # 132 "svd_fuse.cpp"
  iter_per_gang_3 = ((1 + ((256 - start_index) / 2 - 1) / 192) > 8LL ? (1 + ((256 - start_index) / 2 - 1) / 192) : 8LL);
  # 132 "svd_fuse.cpp"
  first_gang_iter_3 = (hmpprt::gr_gbidx()) * iter_per_gang_3;
  # 132 "svd_fuse.cpp"
  last_gang_iter_3 = ((first_gang_iter_3 + iter_per_gang_3 - 1) < ((256 - start_index) / 2 - 1) ? (first_gang_iter_3 + iter_per_gang_3 - 1) : ((256 - start_index) / 2 - 1));
  # 132 "svd_fuse.cpp"
  hmpprt::s32 j_2;
  # 132 "svd_fuse.cpp"
  # 133 "svd_fuse.cpp"
  for (j_2 = first_gang_iter_3 + (hmpprt::gr_btidy()) ; j_2 <= last_gang_iter_3 ; j_2 = j_2 + (hmpprt::gr_btnumy()))
  {
   # 138 "svd_fuse.cpp"
   hmpprt::s32 col_p;
   # 138 "svd_fuse.cpp"
   col_p = *(rotate_col_idx + (j_2 * 2 + (start_index + 1) - 1));
   # 139 "svd_fuse.cpp"
   hmpprt::s32 col_q;
   # 139 "svd_fuse.cpp"
   col_q = *(rotate_col_idx + (j_2 * 2 + (start_index + 1)));
   # 140 "svd_fuse.cpp"
   double real1;
   # 141 "svd_fuse.cpp"
   double imag1;
   # 147 "svd_fuse.cpp"
   hmpprt::s32 k_18;
   # 147 "svd_fuse.cpp"
   # 148 "svd_fuse.cpp"
   for (k_18 = (hmpprt::gr_btidx()) ; k_18 <= 255 ; k_18 = k_18 + (hmpprt::gr_btnumx()))
   {
    # 149 "svd_fuse.cpp"
    *(ci + (j_2 * 2 + (start_index + 1)) / 2 * 256 * 2 + k_18) = *(A_H_3 + (col_p * 256 + k_18));
    # 150 "svd_fuse.cpp"
    *(ci + (j_2 * 2 + (start_index + 1)) / 2 * 256 * 2 + (256 + k_18)) = *(A_H_3 + (65536 + col_p * 256 + k_18));
   }
   # 155 "svd_fuse.cpp"
   # 155 "svd_fuse.cpp"
   hmpprt::s32 k_19;
   # 155 "svd_fuse.cpp"
   # 156 "svd_fuse.cpp"
   for (k_19 = (hmpprt::gr_btidx()) ; k_19 <= 255 ; k_19 = k_19 + (hmpprt::gr_btnumx()))
   {
    # 157 "svd_fuse.cpp"
    *(cj_1 + (j_2 * 2 + (start_index + 1)) / 2 * 256 * 2 + k_19) = *(A_H_3 + (col_q * 256 + k_19));
    # 158 "svd_fuse.cpp"
    *(cj_1 + (j_2 * 2 + (start_index + 1)) / 2 * 256 * 2 + (256 + k_19)) = *(A_H_3 + (65536 + col_q * 256 + k_19));
   }
   # 163 "svd_fuse.cpp"
   # 163 "svd_fuse.cpp"
   hmpprt::s32 k_20;
   # 163 "svd_fuse.cpp"
   # 164 "svd_fuse.cpp"
   for (k_20 = (hmpprt::gr_btidx()) ; k_20 <= 255 ; k_20 = k_20 + (hmpprt::gr_btnumx()))
   {
    # 165 "svd_fuse.cpp"
    *(vi + (j_2 * 2 + (start_index + 1)) / 2 * 256 * 2 + k_20) = *(V_H + (col_p * 256 + k_20));
    # 166 "svd_fuse.cpp"
    *(vi + (j_2 * 2 + (start_index + 1)) / 2 * 256 * 2 + (256 + k_20)) = *(V_H + (65536 + col_p * 256 + k_20));
   }
   # 171 "svd_fuse.cpp"
   # 171 "svd_fuse.cpp"
   hmpprt::s32 k_21;
   # 171 "svd_fuse.cpp"
   # 172 "svd_fuse.cpp"
   for (k_21 = (hmpprt::gr_btidx()) ; k_21 <= 255 ; k_21 = k_21 + (hmpprt::gr_btnumx()))
   {
    # 173 "svd_fuse.cpp"
    *(vj + (j_2 * 2 + (start_index + 1)) / 2 * 256 * 2 + k_21) = *(V_H + (col_q * 256 + k_21));
    # 174 "svd_fuse.cpp"
    *(vj + (j_2 * 2 + (start_index + 1)) / 2 * 256 * 2 + (256 + k_21)) = *(V_H + (65536 + col_q * 256 + k_21));
   }
   # 177 "svd_fuse.cpp"
   # 177 "svd_fuse.cpp"
   hmpprt::s32 skip_trans;
   # 177 "svd_fuse.cpp"
   skip_trans = 0;
   # 141 "svd_fuse.cpp"
   double imag1_1;
   # 140 "svd_fuse.cpp"
   imag1_1 = 0;
   # 140 "svd_fuse.cpp"
   double real1_1;
   # 181 "svd_fuse.cpp"
   real1_1 = 0;
   # 181 "svd_fuse.cpp"
   hmpprt::s32 k_22;
   # 181 "svd_fuse.cpp"
   # 182 "svd_fuse.cpp"
   for (k_22 = (hmpprt::gr_btidx()) ; k_22 <= 255 ; k_22 = k_22 + (hmpprt::gr_btnumx()))
   {
    # 183 "svd_fuse.cpp"
    real1_1 = real1_1 + (*(ci + (j_2 * 2 + (start_index + 1)) / 2 * 256 * 2 + k_22) * *(cj_1 + (j_2 * 2 + (start_index + 1)) / 2 * 256 * 2 + k_22) + *(ci + (j_2 * 2 + (start_index + 1)) / 2 * 256 * 2 + (256 + k_22)) * *(cj_1 + (j_2 * 2 + (start_index + 1)) / 2 * 256 * 2 + (256 + k_22)));
    # 184 "svd_fuse.cpp"
    imag1_1 = imag1_1 + (*(ci + (j_2 * 2 + (start_index + 1)) / 2 * 256 * 2 + k_22) * *(cj_1 + (j_2 * 2 + (start_index + 1)) / 2 * 256 * 2 + (256 + k_22)) - *(ci + (j_2 * 2 + (start_index + 1)) / 2 * 256 * 2 + (256 + k_22)) * *(cj_1 + (j_2 * 2 + (start_index + 1)) / 2 * 256 * 2 + k_22));
   }
   # 186 "svd_fuse.cpp"
   # 186 "svd_fuse.cpp"
   real1 = (reduce_double_add_x(real1_1, tmpr_4, (double) 0.));
   # 186 "svd_fuse.cpp"
   imag1 = (reduce_double_add_x(imag1_1, tmpr_4, (double) 0.));
   # 186 "svd_fuse.cpp"
   *(value_2 + (j_2 * 2 + (start_index + 1)) / 2) = sqrt(real1 * real1 + imag1 * imag1);
   # 188 "svd_fuse.cpp"
   if (*(value_2 + (j_2 * 2 + (start_index + 1)) / 2) < (double) 1.0000000000000000818e-05)
   {
    # 189 "svd_fuse.cpp"
    skip_trans = 1;
   }
   # 191 "svd_fuse.cpp"
   if (skip_trans == 0)
   {
    # 193 "svd_fuse.cpp"
    *pass_1 = 0;
    # 197 "svd_fuse.cpp"
    double result1;
    # 198 "svd_fuse.cpp"
    double result2;
    # 199 "svd_fuse.cpp"
    double result3;
    # 200 "svd_fuse.cpp"
    double result4;
    # 201 "svd_fuse.cpp"
    double result5;
    # 202 "svd_fuse.cpp"
    double result6;
    # 203 "svd_fuse.cpp"
    double result7;
    # 204 "svd_fuse.cpp"
    double result8;
    # 197 "svd_fuse.cpp"
    double result1_1;
    # 207 "svd_fuse.cpp"
    result1_1 = 0;
    # 207 "svd_fuse.cpp"
    hmpprt::s32 k_23;
    # 207 "svd_fuse.cpp"
    # 208 "svd_fuse.cpp"
    for (k_23 = (hmpprt::gr_btidx()) ; k_23 <= 255 ; k_23 = k_23 + (hmpprt::gr_btnumx()))
    {
     # 208 "svd_fuse.cpp"
     result1_1 = result1_1 + *(ci + (j_2 * 2 + (start_index + 1)) / 2 * 256 * 2 + k_23) * *(cj_1 + (j_2 * 2 + (start_index + 1)) / 2 * 256 * 2 + (k_23 + 256));
    }
    # 198 "svd_fuse.cpp"
    # 198 "svd_fuse.cpp"
    result1 = (reduce_double_add_x(result1_1, tmpr_4, (double) 0.));
    # 198 "svd_fuse.cpp"
    double result2_1;
    # 213 "svd_fuse.cpp"
    result2_1 = 0;
    # 213 "svd_fuse.cpp"
    hmpprt::s32 k_24;
    # 213 "svd_fuse.cpp"
    # 214 "svd_fuse.cpp"
    for (k_24 = (hmpprt::gr_btidx()) ; k_24 <= 255 ; k_24 = k_24 + (hmpprt::gr_btnumx()))
    {
     # 214 "svd_fuse.cpp"
     result2_1 = result2_1 + *(ci + (j_2 * 2 + (start_index + 1)) / 2 * 256 * 2 + (k_24 + 256)) * *(cj_1 + (j_2 * 2 + (start_index + 1)) / 2 * 256 * 2 + k_24);
    }
    # 199 "svd_fuse.cpp"
    # 199 "svd_fuse.cpp"
    result2 = (reduce_double_add_x(result2_1, tmpr_4, (double) 0.));
    # 199 "svd_fuse.cpp"
    double result3_1;
    # 219 "svd_fuse.cpp"
    result3_1 = 0;
    # 219 "svd_fuse.cpp"
    hmpprt::s32 k_25;
    # 219 "svd_fuse.cpp"
    # 220 "svd_fuse.cpp"
    for (k_25 = (hmpprt::gr_btidx()) ; k_25 <= 255 ; k_25 = k_25 + (hmpprt::gr_btnumx()))
    {
     # 220 "svd_fuse.cpp"
     result3_1 = result3_1 + *(ci + (j_2 * 2 + (start_index + 1)) / 2 * 256 * 2 + k_25) * *(cj_1 + (j_2 * 2 + (start_index + 1)) / 2 * 256 * 2 + k_25);
    }
    # 200 "svd_fuse.cpp"
    # 200 "svd_fuse.cpp"
    result3 = (reduce_double_add_x(result3_1, tmpr_4, (double) 0.));
    # 200 "svd_fuse.cpp"
    double result4_1;
    # 225 "svd_fuse.cpp"
    result4_1 = 0;
    # 225 "svd_fuse.cpp"
    hmpprt::s32 k_26;
    # 225 "svd_fuse.cpp"
    # 226 "svd_fuse.cpp"
    for (k_26 = (hmpprt::gr_btidx()) ; k_26 <= 255 ; k_26 = k_26 + (hmpprt::gr_btnumx()))
    {
     # 226 "svd_fuse.cpp"
     result4_1 = result4_1 + *(ci + (j_2 * 2 + (start_index + 1)) / 2 * 256 * 2 + (k_26 + 256)) * *(cj_1 + (j_2 * 2 + (start_index + 1)) / 2 * 256 * 2 + (k_26 + 256));
    }
    # 229 "svd_fuse.cpp"
    # 229 "svd_fuse.cpp"
    result4 = (reduce_double_add_x(result4_1, tmpr_4, (double) 0.));
    # 229 "svd_fuse.cpp"
    double a;
    # 229 "svd_fuse.cpp"
    a = result1 - result2;
    # 230 "svd_fuse.cpp"
    double b;
    # 230 "svd_fuse.cpp"
    b = result3 + result4;
    # 232 "svd_fuse.cpp"
    double s_a2b2;
    # 50 "svd_fuse.cpp"
    hmpprt::s32 rvalue_1;
    # 50 "svd_fuse.cpp"
    if (b > (double) 0.)
    {
     # 53 "svd_fuse.cpp"
     rvalue_1 = 1;
     # 53 "svd_fuse.cpp"
     goto endf_2;
    }
    # 53 "svd_fuse.cpp"
    if (b == (double) 0.)
    {
     # 232 "svd_fuse.cpp"
     rvalue_1 = 0;
     # 232 "svd_fuse.cpp"
     goto endf_2;
    }
    # 232 "svd_fuse.cpp"
    rvalue_1 =  -1;
    # 232 "svd_fuse.cpp"
    endf_2:;
    # 232 "svd_fuse.cpp"
    s_a2b2 = (double ) (rvalue_1) * sqrt(a * a + b * b);
    # 233 "svd_fuse.cpp"
    double sin_alpha;
    # 233 "svd_fuse.cpp"
    sin_alpha = a / s_a2b2;
    # 234 "svd_fuse.cpp"
    double cos_alpha;
    # 234 "svd_fuse.cpp"
    cos_alpha = b / s_a2b2;
    # 235 "svd_fuse.cpp"
    double m;
    # 235 "svd_fuse.cpp"
    m =  - (b * cos_alpha + a * sin_alpha);
    # 201 "svd_fuse.cpp"
    double result5_1;
    # 240 "svd_fuse.cpp"
    result5_1 = 0;
    # 240 "svd_fuse.cpp"
    hmpprt::s32 k_27;
    # 240 "svd_fuse.cpp"
    # 241 "svd_fuse.cpp"
    for (k_27 = (hmpprt::gr_btidx()) ; k_27 <= 255 ; k_27 = k_27 + (hmpprt::gr_btnumx()))
    {
     # 241 "svd_fuse.cpp"
     result5_1 = result5_1 + *(cj_1 + (j_2 * 2 + (start_index + 1)) / 2 * 256 * 2 + k_27) * *(cj_1 + (j_2 * 2 + (start_index + 1)) / 2 * 256 * 2 + k_27);
    }
    # 202 "svd_fuse.cpp"
    # 202 "svd_fuse.cpp"
    result5 = (reduce_double_add_x(result5_1, tmpr_4, (double) 0.));
    # 202 "svd_fuse.cpp"
    double result6_1;
    # 246 "svd_fuse.cpp"
    result6_1 = 0;
    # 246 "svd_fuse.cpp"
    hmpprt::s32 k_28;
    # 246 "svd_fuse.cpp"
    # 247 "svd_fuse.cpp"
    for (k_28 = (hmpprt::gr_btidx()) ; k_28 <= 255 ; k_28 = k_28 + (hmpprt::gr_btnumx()))
    {
     # 247 "svd_fuse.cpp"
     result6_1 = result6_1 + *(cj_1 + (j_2 * 2 + (start_index + 1)) / 2 * 256 * 2 + (k_28 + 256)) * *(cj_1 + (j_2 * 2 + (start_index + 1)) / 2 * 256 * 2 + (k_28 + 256));
    }
    # 203 "svd_fuse.cpp"
    # 203 "svd_fuse.cpp"
    result6 = (reduce_double_add_x(result6_1, tmpr_4, (double) 0.));
    # 203 "svd_fuse.cpp"
    double result7_1;
    # 252 "svd_fuse.cpp"
    result7_1 = 0;
    # 252 "svd_fuse.cpp"
    hmpprt::s32 k_29;
    # 252 "svd_fuse.cpp"
    # 253 "svd_fuse.cpp"
    for (k_29 = (hmpprt::gr_btidx()) ; k_29 <= 255 ; k_29 = k_29 + (hmpprt::gr_btnumx()))
    {
     # 253 "svd_fuse.cpp"
     result7_1 = result7_1 + *(ci + (j_2 * 2 + (start_index + 1)) / 2 * 256 * 2 + k_29) * *(ci + (j_2 * 2 + (start_index + 1)) / 2 * 256 * 2 + k_29);
    }
    # 204 "svd_fuse.cpp"
    # 204 "svd_fuse.cpp"
    result7 = (reduce_double_add_x(result7_1, tmpr_4, (double) 0.));
    # 204 "svd_fuse.cpp"
    double result8_1;
    # 258 "svd_fuse.cpp"
    result8_1 = 0;
    # 258 "svd_fuse.cpp"
    hmpprt::s32 k_30;
    # 258 "svd_fuse.cpp"
    # 259 "svd_fuse.cpp"
    for (k_30 = (hmpprt::gr_btidx()) ; k_30 <= 255 ; k_30 = k_30 + (hmpprt::gr_btnumx()))
    {
     # 259 "svd_fuse.cpp"
     result8_1 = result8_1 + *(ci + (j_2 * 2 + (start_index + 1)) / 2 * 256 * 2 + (k_30 + 256)) * *(ci + (j_2 * 2 + (start_index + 1)) / 2 * 256 * 2 + (k_30 + 256));
    }
    # 262 "svd_fuse.cpp"
    # 262 "svd_fuse.cpp"
    result8 = (reduce_double_add_x(result8_1, tmpr_4, (double) 0.));
    # 262 "svd_fuse.cpp"
    double n_1;
    # 262 "svd_fuse.cpp"
    n_1 = (double) 0.5 * (result5 + result6 - result7 - result8);
    # 263 "svd_fuse.cpp"
    double w;
    # 50 "svd_fuse.cpp"
    hmpprt::s32 rvalue_2;
    # 50 "svd_fuse.cpp"
    if (n_1 > (double) 0.)
    {
     # 53 "svd_fuse.cpp"
     rvalue_2 = 1;
     # 53 "svd_fuse.cpp"
     goto endf_3;
    }
    # 53 "svd_fuse.cpp"
    if (n_1 == (double) 0.)
    {
     # 263 "svd_fuse.cpp"
     rvalue_2 = 0;
     # 263 "svd_fuse.cpp"
     goto endf_3;
    }
    # 263 "svd_fuse.cpp"
    rvalue_2 =  -1;
    # 263 "svd_fuse.cpp"
    endf_3:;
    # 263 "svd_fuse.cpp"
    w = (double ) (rvalue_2) * m / sqrt(m * m + n_1 * n_1);
    # 265 "svd_fuse.cpp"
    double sin_x;
    # 265 "svd_fuse.cpp"
    sin_x = w / sqrt((double) 2. * ((double) 1. + sqrt((double) 1. - w * w)));
    # 266 "svd_fuse.cpp"
    double cos_x;
    # 266 "svd_fuse.cpp"
    cos_x = sqrt((double) 1. - sin_x * sin_x);
    # 269 "svd_fuse.cpp"
    double real2;
    # 270 "svd_fuse.cpp"
    double imag2;
    # 271 "svd_fuse.cpp"
    double real3;
    # 272 "svd_fuse.cpp"
    double imag3;
    # 273 "svd_fuse.cpp"
    double real4;
    # 274 "svd_fuse.cpp"
    double imag4;
    # 275 "svd_fuse.cpp"
    double real5;
    # 276 "svd_fuse.cpp"
    double imag5;
    # 279 "svd_fuse.cpp"
    hmpprt::s32 k_31;
    # 279 "svd_fuse.cpp"
    # 280 "svd_fuse.cpp"
    for (k_31 = (hmpprt::gr_btidx()) ; k_31 <= 255 ; k_31 = k_31 + (hmpprt::gr_btnumx()))
    {
     # 281 "svd_fuse.cpp"
     real2 = *(ci + (j_2 * 2 + (start_index + 1)) / 2 * 256 * 2 + k_31) * cos_x + *(cj_1 + (j_2 * 2 + (start_index + 1)) / 2 * 256 * 2 + k_31) * cos_alpha * sin_x + *(cj_1 + (j_2 * 2 + (start_index + 1)) / 2 * 256 * 2 + (k_31 + 256)) * sin_alpha * sin_x;
     # 282 "svd_fuse.cpp"
     imag2 = *(ci + (j_2 * 2 + (start_index + 1)) / 2 * 256 * 2 + (k_31 + 256)) * cos_x + *(cj_1 + (j_2 * 2 + (start_index + 1)) / 2 * 256 * 2 + (k_31 + 256)) * cos_alpha * sin_x - *(cj_1 + (j_2 * 2 + (start_index + 1)) / 2 * 256 * 2 + k_31) * sin_alpha * sin_x;
     # 284 "svd_fuse.cpp"
     *(A_H_3 + (col_p * 256 + k_31)) = real2;
     # 285 "svd_fuse.cpp"
     *(A_H_3 + (65536 + col_p * 256 + k_31)) = imag2;
    }
    # 290 "svd_fuse.cpp"
    # 290 "svd_fuse.cpp"
    hmpprt::s32 k_32;
    # 290 "svd_fuse.cpp"
    # 291 "svd_fuse.cpp"
    for (k_32 = (hmpprt::gr_btidx()) ; k_32 <= 255 ; k_32 = k_32 + (hmpprt::gr_btnumx()))
    {
     # 292 "svd_fuse.cpp"
     real3 = *(cj_1 + (j_2 * 2 + (start_index + 1)) / 2 * 256 * 2 + k_32) * cos_x +  - *(ci + (j_2 * 2 + (start_index + 1)) / 2 * 256 * 2 + k_32) * cos_alpha * sin_x + *(ci + (j_2 * 2 + (start_index + 1)) / 2 * 256 * 2 + (k_32 + 256)) * sin_alpha * sin_x;
     # 293 "svd_fuse.cpp"
     imag3 = *(cj_1 + (j_2 * 2 + (start_index + 1)) / 2 * 256 * 2 + (k_32 + 256)) * cos_x +  - *(ci + (j_2 * 2 + (start_index + 1)) / 2 * 256 * 2 + (k_32 + 256)) * cos_alpha * sin_x - *(ci + (j_2 * 2 + (start_index + 1)) / 2 * 256 * 2 + k_32) * sin_alpha * sin_x;
     # 295 "svd_fuse.cpp"
     *(A_H_3 + (col_q * 256 + k_32)) = real3;
     # 296 "svd_fuse.cpp"
     *(A_H_3 + (65536 + col_q * 256 + k_32)) = imag3;
    }
    # 301 "svd_fuse.cpp"
    # 301 "svd_fuse.cpp"
    hmpprt::s32 k_33;
    # 301 "svd_fuse.cpp"
    # 302 "svd_fuse.cpp"
    for (k_33 = (hmpprt::gr_btidx()) ; k_33 <= 255 ; k_33 = k_33 + (hmpprt::gr_btnumx()))
    {
     # 303 "svd_fuse.cpp"
     real4 = *(vi + (j_2 * 2 + (start_index + 1)) / 2 * 256 * 2 + k_33) * cos_x + *(vj + (j_2 * 2 + (start_index + 1)) / 2 * 256 * 2 + k_33) * cos_alpha * sin_x + *(vj + (j_2 * 2 + (start_index + 1)) / 2 * 256 * 2 + (k_33 + 256)) * sin_alpha * sin_x;
     # 304 "svd_fuse.cpp"
     imag4 = *(vi + (j_2 * 2 + (start_index + 1)) / 2 * 256 * 2 + (k_33 + 256)) * cos_x + *(vj + (j_2 * 2 + (start_index + 1)) / 2 * 256 * 2 + (k_33 + 256)) * cos_alpha * sin_x - *(vj + (j_2 * 2 + (start_index + 1)) / 2 * 256 * 2 + k_33) * sin_alpha * sin_x;
     # 306 "svd_fuse.cpp"
     *(V_H + (col_p * 256 + k_33)) = real4;
     # 307 "svd_fuse.cpp"
     *(V_H + (65536 + col_p * 256 + k_33)) = imag4;
    }
    # 312 "svd_fuse.cpp"
    # 312 "svd_fuse.cpp"
    hmpprt::s32 k_34;
    # 312 "svd_fuse.cpp"
    # 313 "svd_fuse.cpp"
    for (k_34 = (hmpprt::gr_btidx()) ; k_34 <= 255 ; k_34 = k_34 + (hmpprt::gr_btnumx()))
    {
     # 314 "svd_fuse.cpp"
     real5 = *(vj + (j_2 * 2 + (start_index + 1)) / 2 * 256 * 2 + k_34) * cos_x +  - *(vi + (j_2 * 2 + (start_index + 1)) / 2 * 256 * 2 + k_34) * cos_alpha * sin_x + *(vi + (j_2 * 2 + (start_index + 1)) / 2 * 256 * 2 + (k_34 + 256)) * sin_alpha * sin_x;
     # 315 "svd_fuse.cpp"
     imag5 = *(vj + (j_2 * 2 + (start_index + 1)) / 2 * 256 * 2 + (k_34 + 256)) * cos_x +  - *(vi + (j_2 * 2 + (start_index + 1)) / 2 * 256 * 2 + (k_34 + 256)) * cos_alpha * sin_x - *(vi + (j_2 * 2 + (start_index + 1)) / 2 * 256 * 2 + k_34) * sin_alpha * sin_x;
     # 317 "svd_fuse.cpp"
     *(V_H + (col_q * 256 + k_34)) = real5;
     # 318 "svd_fuse.cpp"
     *(V_H + (65536 + col_q * 256 + k_34)) = imag5;
    }
    # 322 "svd_fuse.cpp"
   }
   # 322 "svd_fuse.cpp"
   *(rotate_col_idx + (j_2 * 2 + (start_index + 1) - 1)) = col_q;
   # 323 "svd_fuse.cpp"
   *(rotate_col_idx + (j_2 * 2 + (start_index + 1))) = col_p;
  }
  # 60 "svd_fuse.cpp"
 }
}
#endif // __HIPCC__



# 60 "svd_fuse.cpp"

#ifndef __HIPCC__
void hmpp_acc_region_svd_129_internal_1(hmpprt::s32* pass, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  A_H, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  V_H_1, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s32>  rotate_col_idx_1, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  ci_2, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  cj_2, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  vi_2, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  vj_2, hmpprt::s32 start_index_2)
{
 # 60 "svd_fuse.cpp"
 hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s32>  pass_2;
 # 60 "svd_fuse.cpp"
 hmpprt::Context::getInstance()->allocate((void **) (&pass_2), hmpprt::MS_CUDA_GLOB, 4);
 # 60 "svd_fuse.cpp"
 *pass_2 = *pass;
 # 60 "svd_fuse.cpp"
 hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  value_1;
 # 60 "svd_fuse.cpp"
 hmpprt::Context::getInstance()->allocate((void **) (&value_1), hmpprt::MS_CUDA_GLOB, 1024uLL);
 # 60 "svd_fuse.cpp"
 hmpprt::DevicePtr<hmpprt::MS_CUDA_SHARED,double>  tmpr_2;
 # 60 "svd_fuse.cpp"
 hmpprt::Context::getInstance()->allocate((void **) (&tmpr_2), hmpprt::MS_CUDA_SHARED, 2048LL);
 # 60 "svd_fuse.cpp"
 if (1)
 {
  hmpprt::CUDAGridCall __hmppcg_call;
  __hmppcg_call.setSizeX(192);
  __hmppcg_call.setSizeY(1);
  __hmppcg_call.setBlockSizeX(32);
  __hmppcg_call.setBlockSizeY(8LL);
  __hmppcg_call.addLocalParameter(&A_H, 8, "A_H_3");
  __hmppcg_call.addLocalParameter(&V_H_1, 8, "V_H");
  __hmppcg_call.addLocalParameter(&ci_2, 8, "ci");
  __hmppcg_call.addLocalParameter(&cj_2, 8, "cj_1");
  __hmppcg_call.addLocalParameter(&pass_2, 8, "pass_1");
  __hmppcg_call.addLocalParameter(&rotate_col_idx_1, 8, "rotate_col_idx");
  __hmppcg_call.addLocalParameter((hmpprt::s32) (start_index_2), "start_index");
  __hmppcg_call.addSharedParameter(tmpr_2, "tmpr_4");
  __hmppcg_call.addLocalParameter(&value_1, 8, "value_2");
  __hmppcg_call.addLocalParameter(&vi_2, 8, "vi");
  __hmppcg_call.addLocalParameter(&vj_2, 8, "vj");
  __hmppcg_call.launch(hmpp_acc_region_svd_129_parallel_region_1, hmpprt::Context::getInstance()->getCUDADevice());
 }
 ;
 # 60 "svd_fuse.cpp"
 *pass = *pass_2;
 # 60 "svd_fuse.cpp"
 hmpprt::Context::getInstance()->free((void **) (&pass_2));
 # 60 "svd_fuse.cpp"
 hmpprt::Context::getInstance()->free((void **) (&value_1));
 # 60 "svd_fuse.cpp"
 hmpprt::Context::getInstance()->free((void **) (&tmpr_2));
}
#endif // __HIPCC__



# 60 "svd_fuse.cpp"

#ifndef __HIPCC__
extern "C" CDLT_API  void hmpp_acc_region_svd_129(hmpprt::s32* pass_3, double* A_H_6, double* V_H_2, hmpprt::s32* rotate_col_idx_2, double* ci_4, double* cj, double* vi_1, double* vj_1, hmpprt::s32 start_index_1)
{
 # 330 "svd_fuse.cpp"
 (hmpp_acc_region_svd_129_internal_1(pass_3, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double> (A_H_6), hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double> (V_H_2), hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s32> (rotate_col_idx_2), hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double> (ci_4), hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double> (cj), hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double> (vi_1), hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double> (vj_1), start_index_1));
}
#endif // __HIPCC__



# 330 "svd_fuse.cpp"

#ifdef __HIPCC__

extern "C" __global__ void hmpp_acc_region_svd_338_parallel_region_1(double* A_H_2, double* S_H_2, double* ci_1)
{
 
 double * tmpr_3 = (double *)(((char *)hmpp_sharedmem + 0));
 
 # 332 "svd_fuse.cpp"
 {
  # 341 "svd_fuse.cpp"
  hmpprt::s32 first_gang_iter_2;
  # 341 "svd_fuse.cpp"
  hmpprt::s32 last_gang_iter_2;
  # 341 "svd_fuse.cpp"
  first_gang_iter_2 = (hmpprt::gr_gbidx()) * 8;
  # 341 "svd_fuse.cpp"
  last_gang_iter_2 = ((first_gang_iter_2 + 7) < 255 ? (first_gang_iter_2 + 7) : 255);
  # 341 "svd_fuse.cpp"
  hmpprt::s32 i_2;
  # 341 "svd_fuse.cpp"
  # 342 "svd_fuse.cpp"
  for (i_2 = first_gang_iter_2 + (hmpprt::gr_btidy()) ; i_2 <= last_gang_iter_2 ; i_2 = i_2 + (hmpprt::gr_btnumy()))
  {
   # 343 "svd_fuse.cpp"
   double real;
   # 344 "svd_fuse.cpp"
   double imag;
   # 347 "svd_fuse.cpp"
   hmpprt::s32 k_35;
   # 347 "svd_fuse.cpp"
   # 348 "svd_fuse.cpp"
   for (k_35 = (hmpprt::gr_btidx()) ; k_35 <= 255 ; k_35 = k_35 + (hmpprt::gr_btnumx()))
   {
    # 349 "svd_fuse.cpp"
    *(ci_1 + i_2 * 256 * 2 + k_35) = *(A_H_2 + (i_2 * 256 + k_35));
    # 350 "svd_fuse.cpp"
    *(ci_1 + i_2 * 256 * 2 + (256 + k_35)) = *(A_H_2 + (65536 + i_2 * 256 + k_35));
   }
   # 344 "svd_fuse.cpp"
   # 344 "svd_fuse.cpp"
   double imag_1;
   # 343 "svd_fuse.cpp"
   imag_1 = 0;
   # 343 "svd_fuse.cpp"
   double real_1;
   # 355 "svd_fuse.cpp"
   real_1 = 0;
   # 355 "svd_fuse.cpp"
   hmpprt::s32 l_1;
   # 355 "svd_fuse.cpp"
   # 356 "svd_fuse.cpp"
   for (l_1 = (hmpprt::gr_btidx()) ; l_1 <= 255 ; l_1 = l_1 + (hmpprt::gr_btnumx()))
   {
    # 357 "svd_fuse.cpp"
    real_1 = real_1 + (*(ci_1 + i_2 * 256 * 2 + l_1) * *(ci_1 + i_2 * 256 * 2 + l_1) + *(ci_1 + i_2 * 256 * 2 + (256 + l_1)) * *(ci_1 + i_2 * 256 * 2 + (256 + l_1)));
    # 358 "svd_fuse.cpp"
    imag_1 = imag_1 + (*(ci_1 + i_2 * 256 * 2 + l_1) * *(ci_1 + i_2 * 256 * 2 + (256 + l_1)) - *(ci_1 + i_2 * 256 * 2 + (256 + l_1)) * *(ci_1 + i_2 * 256 * 2 + l_1));
   }
   # 361 "svd_fuse.cpp"
   # 361 "svd_fuse.cpp"
   real = (reduce_double_add_x(real_1, tmpr_3, (double) 0.));
   # 361 "svd_fuse.cpp"
   imag = (reduce_double_add_x(imag_1, tmpr_3, (double) 0.));
   # 361 "svd_fuse.cpp"
   double norm_A;
   # 361 "svd_fuse.cpp"
   norm_A = sqrt(sqrt(real * real + imag * imag));
   # 362 "svd_fuse.cpp"
   *(S_H_2 + i_2) = norm_A;
  }
  # 330 "svd_fuse.cpp"
 }
}
#endif // __HIPCC__



# 330 "svd_fuse.cpp"

#ifndef __HIPCC__
void hmpp_acc_region_svd_338_internal_1(hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  A_H_8, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  S_H_4, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  ci_5)
{
 # 330 "svd_fuse.cpp"
 hmpprt::DevicePtr<hmpprt::MS_CUDA_SHARED,double>  tmpr_1;
 # 330 "svd_fuse.cpp"
 hmpprt::Context::getInstance()->allocate((void **) (&tmpr_1), hmpprt::MS_CUDA_SHARED, 2048LL);
 # 330 "svd_fuse.cpp"
 if (1)
 {
  hmpprt::CUDAGridCall __hmppcg_call;
  __hmppcg_call.setSizeX(192);
  __hmppcg_call.setSizeY(1);
  __hmppcg_call.setBlockSizeX(32);
  __hmppcg_call.setBlockSizeY(8LL);
  __hmppcg_call.addLocalParameter(&A_H_8, 8, "A_H_2");
  __hmppcg_call.addLocalParameter(&S_H_4, 8, "S_H_2");
  __hmppcg_call.addLocalParameter(&ci_5, 8, "ci_1");
  __hmppcg_call.addSharedParameter(tmpr_1, "tmpr_3");
  __hmppcg_call.launch(hmpp_acc_region_svd_338_parallel_region_1, hmpprt::Context::getInstance()->getCUDADevice());
 }
 ;
 # 330 "svd_fuse.cpp"
 hmpprt::Context::getInstance()->free((void **) (&tmpr_1));
}
#endif // __HIPCC__



# 330 "svd_fuse.cpp"

#ifndef __HIPCC__
extern "C" CDLT_API  void hmpp_acc_region_svd_338(double* A_H_5, double* S_H, double* ci_3)
{
 # 369 "svd_fuse.cpp"
 (hmpp_acc_region_svd_338_internal_1(hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double> (A_H_5), hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double> (S_H), hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double> (ci_3)));
}
#endif // __HIPCC__



# 369 "svd_fuse.cpp"

#ifdef __HIPCC__

extern "C" __global__ void hmpp_acc_region_svd_369_parallel_region_1(double* A_H_1, double* B_H, double* S_H_3)
{
 # 371 "svd_fuse.cpp"
 {
  # 372 "svd_fuse.cpp"
  hmpprt::s32 first_gang_iter_1;
  # 372 "svd_fuse.cpp"
  hmpprt::s32 last_gang_iter_1;
  # 372 "svd_fuse.cpp"
  first_gang_iter_1 = (hmpprt::gr_gbidx()) * 342;
  # 372 "svd_fuse.cpp"
  last_gang_iter_1 = ((first_gang_iter_1 + 341) < 65535 ? (first_gang_iter_1 + 341) : 65535);
  # 372 "svd_fuse.cpp"
  hmpprt::s32 j_4;
  # 372 "svd_fuse.cpp"
  # 375 "svd_fuse.cpp"
  for (j_4 = first_gang_iter_1 + (hmpprt::gr_btidy()) ; j_4 <= last_gang_iter_1 ; j_4 = j_4 + (hmpprt::gr_btnumy()))
  {
   # 372 "svd_fuse.cpp"
   hmpprt::s32 j_3;
   # 374 "svd_fuse.cpp"
   hmpprt::s32 i_3;
   # 376 "svd_fuse.cpp"
   i_3 = j_4 % 256;
   # 376 "svd_fuse.cpp"
   j_3 = j_4 / 256;
   # 376 "svd_fuse.cpp"
   *(B_H + (j_3 * 256 + i_3)) = *(A_H_1 + (j_3 * 256 + i_3)) / *(S_H_3 + j_3);
   # 377 "svd_fuse.cpp"
   *(B_H + (65536 + j_3 * 256 + i_3)) = *(A_H_1 + (65536 + j_3 * 256 + i_3)) / *(S_H_3 + j_3);
  }
  # 369 "svd_fuse.cpp"
 }
}
#endif // __HIPCC__



# 369 "svd_fuse.cpp"

#ifndef __HIPCC__
void hmpp_acc_region_svd_369_internal_1(hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  A_H_7, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  B_H_2, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  S_H_1)
{
 # 369 "svd_fuse.cpp"
 if (1)
 {
  hmpprt::CUDAGridCall __hmppcg_call;
  __hmppcg_call.setSizeX(192);
  __hmppcg_call.setSizeY(1);
  __hmppcg_call.setBlockSizeX(1);
  __hmppcg_call.setBlockSizeY(256);
  __hmppcg_call.addLocalParameter(&A_H_7, 8, "A_H_1");
  __hmppcg_call.addLocalParameter(&B_H_2, 8, "B_H");
  __hmppcg_call.addLocalParameter(&S_H_1, 8, "S_H_3");
  __hmppcg_call.launch(hmpp_acc_region_svd_369_parallel_region_1, hmpprt::Context::getInstance()->getCUDADevice());
 }
 ;
}
#endif // __HIPCC__



# 369 "svd_fuse.cpp"

#ifndef __HIPCC__
extern "C" CDLT_API  void hmpp_acc_region_svd_369(double* A_H_4, double* B_H_1, double* S_H_5)
{
 # 1 "<preprocessor>"
 (hmpp_acc_region_svd_369_internal_1(hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double> (A_H_4), hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double> (B_H_1), hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double> (S_H_5)));
}
#endif // __HIPCC__



# 1 "<preprocessor>"

#ifdef __HIPCC__
__device__ double reduce_double_add_x(double priv, double* tmpa, double orig)
{
 # 1 "<preprocessor>"
 *(tmpa + (hmpprt::gr_btidy()) + (hmpprt::gr_btnumy()) * (hmpprt::gr_btidx())) = priv;
 # 1 "<preprocessor>"
 hmpprt::s32 n;
 # 1 "<preprocessor>"
 # 1 "<preprocessor>"
 for (n = ((hmpprt::gr_btnumx()) - 1 >> 1) + 1 ; n > 0 ; n = (n >> 1))
 {
  # 1 "<preprocessor>"
  hmpprt::s32 next;
  # 1 "<preprocessor>"
  next = (hmpprt::gr_btidx()) + n;
  # 1 "<preprocessor>"
  if ((hmpprt::gr_btidx()) < n && next < (hmpprt::gr_btnumx()))
  {
   # 1 "<preprocessor>"
   *(tmpa + (hmpprt::gr_btidy()) + (hmpprt::gr_btnumy()) * (hmpprt::gr_btidx())) = *(tmpa + (hmpprt::gr_btidy()) + (hmpprt::gr_btnumy()) * (hmpprt::gr_btidx())) + *(tmpa + (hmpprt::gr_btidy()) + (hmpprt::gr_btnumy()) * next);
  }
 }
 # 1 "<preprocessor>"
 # 1 "<preprocessor>"
 return *(tmpa + (hmpprt::gr_btidy())) + orig;
}
#endif // __HIPCC__




#ifndef __HIPCC__
extern "C" const char * hmpprt_cuda_get_gpu_code();

static hmpprt::CUDAModule * hmpprt_module = 0;
static int hmpprt_uses = 0;

extern "C" CDLT_API void * hmpprt_init()
{
  try
  {
    if (hmpprt_uses++ == 0)
    {
      hmpprt_module = new hmpprt::CUDAModule(hmpprt_cuda_get_gpu_code());
      hmpp_acc_region_svd_129_parallel_region_1 = new hmpprt::CUDAGrid(hmpprt_module, "hmpp_acc_region_svd_129_parallel_region_1");
      hmpp_acc_region_svd_338_parallel_region_1 = new hmpprt::CUDAGrid(hmpprt_module, "hmpp_acc_region_svd_338_parallel_region_1");
      hmpp_acc_region_svd_369_parallel_region_1 = new hmpprt::CUDAGrid(hmpprt_module, "hmpp_acc_region_svd_369_parallel_region_1");

    }
    hmpprt::Context::getInstance()->getGrouplet()->setTarget(hmpprt::CUDA);
    hmpprt::Context::getInstance()->getGrouplet()->addSignature("hmpp_acc_region_svd_129", "prototype hmpp_acc_region_svd_129(pass: ^host s32, A_H: ^cudaglob double, V_H: ^cudaglob double, rotate_col_idx: ^cudaglob s32, ci: ^cudaglob double, cj: ^cudaglob double, vi: ^cudaglob double, vj: ^cudaglob double, start_index: s32)");
    hmpprt::Context::getInstance()->getGrouplet()->addSignature("hmpp_acc_region_svd_338", "prototype hmpp_acc_region_svd_338(A_H: ^cudaglob double, S_H: ^cudaglob double, ci: ^cudaglob double)");
    hmpprt::Context::getInstance()->getGrouplet()->addSignature("hmpp_acc_region_svd_369", "prototype hmpp_acc_region_svd_369(A_H: ^cudaglob double, B_H: ^cudaglob double, S_H: ^cudaglob double)");

  }
  catch (hmpperr::Error & e)
  {
    return e.clone();
  }
  catch(...)
  {
    fprintf(stderr,"Unexpected error in hmpprt_init()\n");
    abort();
  }
  return 0;
}
#endif // __HIPCC__

#ifndef __HIPCC__
extern "C" CDLT_API void * hmpprt_fini()
{
  try
  {
    if (--hmpprt_uses == 0)
    {
      delete hmpp_acc_region_svd_129_parallel_region_1;
      delete hmpp_acc_region_svd_338_parallel_region_1;
      delete hmpp_acc_region_svd_369_parallel_region_1;

      delete hmpprt_module;
      hmpprt_module = 0;
    }
  }
  catch (hmpperr::Error & e)
  {
    return e.clone();
  }
  catch(...)
  {
    fprintf(stderr,"Unexpected error in hmpprt_fini()\n");
    abort();
  }
  return 0;
}
#endif // __HIPCC__

// footer
