#include "hip/hip_runtime.h"

#include <stdio.h>

#ifndef __HIPCC__
#include <stdlib.h>
#include <math.h>

#include <hmpprt/Grouplet.h>
#include <hmpprt/HostTypes.h>
#include <hmpprt/Context.h>
#include <hmpprt/CUDAGrid.h>
#include <hmpprt/CUDAModule.h>
#include <hmpprt/DeviceManager.h>
#include <hmpperr/hmpperr.h>

#include <openacci/openacci_c.h>

#ifdef _WIN32
#  define CDLT_API __declspec(dllexport)
#else /* ! _WIN32 */
#  define CDLT_API
#endif /* _WIN32 */



#else // ! __HIPCC__

#include <hmpprt/HostTypes.h>
#include <hmpprt/CUDAIntrinsics.h>

extern __shared__ int64_t hmpp_sharedmem[];
#endif // __HIPCC__



#ifndef __HIPCC__

#else


#endif

#define HMPPCG_SIMD_LENGTH 32

# 98 "laplacian_fuse.cpp"

#ifndef __HIPCC__
extern "C" CDLT_API  void hmpp_acc_region_main_94(hmpprt::s32 height_8, hmpprt::s32 width_11, hmpprt::s32* pSrc_padding2_4, hmpprt::u08* pBufL_cp_4)
;
#endif // __HIPCC__



# 98 "laplacian_fuse.cpp"

#ifndef __HIPCC__
void hmpp_acc_region_main_94_internal_1(hmpprt::s32 height_19, hmpprt::s32 width_20, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s32>  pSrc_padding2_7, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  pBufL_cp_7)
;
#endif // __HIPCC__



# 75 "laplacian_fuse.cpp"

#ifndef __HIPCC__
static hmpprt::CUDAGrid * hmpp_acc_region_main_94_parallel_region_1 = 0;
#else

extern "C" __global__ void hmpp_acc_region_main_94_parallel_region_1(hmpprt::s32 height_1, hmpprt::u08* pBufL_cp_1, hmpprt::s32* pSrc_padding2_1, hmpprt::s32 width_1);
#endif // __HIPCC__




# 75 "laplacian_fuse.cpp"

#ifndef __HIPCC__
extern "C" CDLT_API  void hmpp_acc_region_main_80(hmpprt::s32 height_9, hmpprt::s32 width_12, hmpprt::s32* pSrc_padding_3, hmpprt::u08* pBufL_cp_5)
;
#endif // __HIPCC__



# 75 "laplacian_fuse.cpp"

#ifndef __HIPCC__
void hmpp_acc_region_main_80_internal_1(hmpprt::s32 height_20, hmpprt::s32 width_21, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s32>  pSrc_padding_5, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  pBufL_cp_8)
;
#endif // __HIPCC__



# 12 "laplacian_fuse.cpp"

#ifndef __HIPCC__
static hmpprt::CUDAGrid * hmpp_acc_region_main_80_parallel_region_1 = 0;
#else

extern "C" __global__ void hmpp_acc_region_main_80_parallel_region_1(hmpprt::s32 height_2, hmpprt::u08* pBufL_cp_2, hmpprt::s32* pSrc_padding_1, hmpprt::s32 width_2);
#endif // __HIPCC__




# 12 "laplacian_fuse.cpp"

#ifndef __HIPCC__
extern "C" CDLT_API  void hmpp_acc_region_main_55(hmpprt::s32 height_10, hmpprt::s32 width_13, hmpprt::u08* Source_3, hmpprt::s32* pSrc_padding_2)
;
#endif // __HIPCC__



# 12 "laplacian_fuse.cpp"

#ifndef __HIPCC__
void hmpp_acc_region_main_55_internal_1(hmpprt::s32 height_15, hmpprt::s32 width_22, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  Source_5, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s32>  pSrc_padding_4)
;
#endif // __HIPCC__



# 233 "laplacian_fuse.cpp"

#ifndef __HIPCC__
static hmpprt::CUDAGrid * hmpp_acc_region_main_55_parallel_region_1 = 0;
#else

extern "C" __global__ void hmpp_acc_region_main_55_parallel_region_1(hmpprt::u08* Source_1, hmpprt::s32 height_3, hmpprt::s32* pSrc_padding, hmpprt::s32 width_3);
#endif // __HIPCC__




# 233 "laplacian_fuse.cpp"

#ifndef __HIPCC__
extern "C" CDLT_API  void hmpp_acc_region_main_239(hmpprt::s32 height_11, hmpprt::s32 width_14, hmpprt::u08* Source_2, hmpprt::s16* LaplacianLayer_1, hmpprt::u08* UpsampleDst_5)
;
#endif // __HIPCC__



# 233 "laplacian_fuse.cpp"

#ifndef __HIPCC__
void hmpp_acc_region_main_239_internal_1(hmpprt::s32 height_16, hmpprt::s32 width_23, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  Source_4, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s16>  LaplacianLayer_2, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  UpsampleDst_9)
;
#endif // __HIPCC__



# 211 "laplacian_fuse.cpp"

#ifndef __HIPCC__
static hmpprt::CUDAGrid * hmpp_acc_region_main_239_parallel_region_1 = 0;
#else

extern "C" __global__ void hmpp_acc_region_main_239_parallel_region_1(hmpprt::s16* LaplacianLayer, hmpprt::u08* Source, hmpprt::u08* UpsampleDst_1, hmpprt::s32 height_4, hmpprt::s32 width_4);
#endif // __HIPCC__




# 211 "laplacian_fuse.cpp"

#ifndef __HIPCC__
extern "C" CDLT_API  void hmpp_acc_region_main_206(hmpprt::s32 width_15, hmpprt::s32 halfWidth_5, hmpprt::s32 halfHeight_7, hmpprt::u08* DownsampleDst_9, hmpprt::u08* UpsampleDst_6)
;
#endif // __HIPCC__



# 211 "laplacian_fuse.cpp"

#ifndef __HIPCC__
void hmpp_acc_region_main_206_internal_1(hmpprt::s32 width_24, hmpprt::s32 halfWidth_9, hmpprt::s32 halfHeight_4, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  DownsampleDst_5, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  UpsampleDst_10)
;
#endif // __HIPCC__



# 196 "laplacian_fuse.cpp"

#ifndef __HIPCC__
static hmpprt::CUDAGrid * hmpp_acc_region_main_206_parallel_region_1 = 0;
#else

extern "C" __global__ void hmpp_acc_region_main_206_parallel_region_1(hmpprt::u08* DownsampleDst_1, hmpprt::u08* UpsampleDst_2, hmpprt::s32 halfHeight_1, hmpprt::s32 halfWidth_1, hmpprt::s32 width_5);
#endif // __HIPCC__




# 196 "laplacian_fuse.cpp"

#ifndef __HIPCC__
extern "C" CDLT_API  void hmpp_acc_region_main_192(hmpprt::s32 width_6, hmpprt::s32 halfWidth_6, hmpprt::u08* DownsampleDst_10, hmpprt::u08* UpsampleDst_11)
;
#endif // __HIPCC__



# 196 "laplacian_fuse.cpp"

#ifndef __HIPCC__
void hmpp_acc_region_main_192_internal_1(hmpprt::s32 width_28, hmpprt::s32 halfWidth_10, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  DownsampleDst_6, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  UpsampleDst_7)
;
#endif // __HIPCC__



# 169 "laplacian_fuse.cpp"

#ifndef __HIPCC__
static hmpprt::CUDAGrid * hmpp_acc_region_main_192_parallel_region_1 = 0;
#else

extern "C" __global__ void hmpp_acc_region_main_192_parallel_region_1(hmpprt::u08* DownsampleDst_2, hmpprt::u08* UpsampleDst_3, hmpprt::s32 halfWidth_2);
#endif // __HIPCC__




# 169 "laplacian_fuse.cpp"

#ifndef __HIPCC__
extern "C" CDLT_API  void hmpp_acc_region_main_183(hmpprt::s32 height_12, hmpprt::s32 width_16, hmpprt::s32 halfWidth_11, hmpprt::s32 halfHeight_5, hmpprt::u08* DownsampleDst_7, hmpprt::u08* UpsampleDst_4)
;
#endif // __HIPCC__



# 169 "laplacian_fuse.cpp"

#ifndef __HIPCC__
void hmpp_acc_region_main_183_internal_1(hmpprt::s32 height_17, hmpprt::s32 width_25, hmpprt::s32 halfWidth_7, hmpprt::s32 halfHeight_8, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  DownsampleDst_11, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  UpsampleDst_8)
;
#endif // __HIPCC__



# 148 "laplacian_fuse.cpp"

#ifndef __HIPCC__
static hmpprt::CUDAGrid * hmpp_acc_region_main_183_parallel_region_1 = 0;
#else

extern "C" __global__ void hmpp_acc_region_main_183_parallel_region_1(hmpprt::u08* DownsampleDst_3, hmpprt::u08* UpsampleDst, hmpprt::s32 halfHeight_2, hmpprt::s32 halfWidth_3, hmpprt::s32 height_5, hmpprt::s32 width_7);
#endif // __HIPCC__




# 148 "laplacian_fuse.cpp"

#ifndef __HIPCC__
extern "C" CDLT_API  void hmpp_acc_region_main_153(hmpprt::s32 width_17, hmpprt::s32 halfWidth_4, hmpprt::s32 halfHeight_3, hmpprt::u08* FilterDst_1, hmpprt::u08* DownsampleDst_4)
;
#endif // __HIPCC__



# 148 "laplacian_fuse.cpp"

#ifndef __HIPCC__
void hmpp_acc_region_main_153_internal_1(hmpprt::s32 width_26, hmpprt::s32 halfWidth_8, hmpprt::s32 halfHeight_6, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  FilterDst_4, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  DownsampleDst_8)
;
#endif // __HIPCC__



# 126 "laplacian_fuse.cpp"

#ifndef __HIPCC__
static hmpprt::CUDAGrid * hmpp_acc_region_main_153_parallel_region_1 = 0;
#else

extern "C" __global__ void hmpp_acc_region_main_153_parallel_region_1(hmpprt::u08* DownsampleDst, hmpprt::u08* FilterDst_3, hmpprt::s32 halfHeight, hmpprt::s32 halfWidth, hmpprt::s32 width_8);
#endif // __HIPCC__




# 126 "laplacian_fuse.cpp"

#ifndef __HIPCC__
extern "C" CDLT_API  void hmpp_acc_region_main_130(hmpprt::s32 height_6, hmpprt::s32 width_18, hmpprt::u08* FilterDst_2, hmpprt::s32* pSrc_padding2_8)
;
#endif // __HIPCC__



# 126 "laplacian_fuse.cpp"

#ifndef __HIPCC__
void hmpp_acc_region_main_130_internal_1(hmpprt::s32 height_18, hmpprt::s32 width_27, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  FilterDst_5, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s32>  pSrc_padding2_5)
;
#endif // __HIPCC__



# 114 "laplacian_fuse.cpp"

#ifndef __HIPCC__
static hmpprt::CUDAGrid * hmpp_acc_region_main_130_parallel_region_1 = 0;
#else

extern "C" __global__ void hmpp_acc_region_main_130_parallel_region_1(hmpprt::u08* FilterDst, hmpprt::s32 height_13, hmpprt::s32* pSrc_padding2_2, hmpprt::s32 width_9);
#endif // __HIPCC__




# 114 "laplacian_fuse.cpp"

#ifndef __HIPCC__
extern "C" CDLT_API  void hmpp_acc_region_main_108(hmpprt::s32 height_14, hmpprt::s32 width_10, hmpprt::s32* pSrc_padding2_3, hmpprt::u08* pBufL_cp)
;
#endif // __HIPCC__



# 114 "laplacian_fuse.cpp"

#ifndef __HIPCC__
void hmpp_acc_region_main_108_internal_1(hmpprt::s32 height, hmpprt::s32 width_19, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s32>  pSrc_padding2_6, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  pBufL_cp_6)
;
#endif // __HIPCC__



# 114 "laplacian_fuse.cpp"

#ifndef __HIPCC__
static hmpprt::CUDAGrid * hmpp_acc_region_main_108_parallel_region_1 = 0;
#else

extern "C" __global__ void hmpp_acc_region_main_108_parallel_region_1(hmpprt::s32 height_7, hmpprt::u08* pBufL_cp_3, hmpprt::s32* pSrc_padding2, hmpprt::s32 width);
#endif // __HIPCC__




# 114 "laplacian_fuse.cpp"

#ifdef __HIPCC__

extern "C" __global__ void hmpp_acc_region_main_108_parallel_region_1(hmpprt::s32 height_7, hmpprt::u08* pBufL_cp_3, hmpprt::s32* pSrc_padding2, hmpprt::s32 width)
{
 # 116 "laplacian_fuse.cpp"
 {
  # 111 "laplacian_fuse.cpp"
  hmpprt::s32 iter_per_gang_10;
  # 111 "laplacian_fuse.cpp"
  hmpprt::s32 first_gang_iter_10;
  # 111 "laplacian_fuse.cpp"
  hmpprt::s32 last_gang_iter_10;
  # 111 "laplacian_fuse.cpp"
  iter_per_gang_10 = ((1 + (width - 1) / 192) > 256 ? (1 + (width - 1) / 192) : 256);
  # 111 "laplacian_fuse.cpp"
  first_gang_iter_10 = (hmpprt::gr_gbidx()) * iter_per_gang_10;
  # 111 "laplacian_fuse.cpp"
  last_gang_iter_10 = ((first_gang_iter_10 + iter_per_gang_10 - 1) < (width - 1) ? (first_gang_iter_10 + iter_per_gang_10 - 1) : (width - 1));
  # 111 "laplacian_fuse.cpp"
  hmpprt::s32 w_5;
  # 111 "laplacian_fuse.cpp"
  # 112 "laplacian_fuse.cpp"
  for (w_5 = first_gang_iter_10 + (hmpprt::gr_btidy()) ; w_5 <= last_gang_iter_10 ; w_5 = w_5 + (hmpprt::gr_btnumy()))
  {
   # 113 "laplacian_fuse.cpp"
   *(pSrc_padding2 + (width + 4 + (w_5 + 2))) = 2 * (hmpprt::s32 ) (*(pBufL_cp_3 + (width + (w_5 + 2) - 2))) - (hmpprt::s32 ) (*(pBufL_cp_3 + (3 * width + (w_5 + 2) - 2)));
   # 114 "laplacian_fuse.cpp"
   *(pSrc_padding2 + (w_5 + 2)) = 4 * (hmpprt::s32 ) (*(pBufL_cp_3 + (3 * width + (w_5 + 2) - 2))) - 4 * (hmpprt::s32 ) (*(pBufL_cp_3 + (width + (w_5 + 2) - 2))) + 2 * (hmpprt::s32 ) (*(pBufL_cp_3 + w_5)) - (hmpprt::s32 ) (*(pBufL_cp_3 + (2 * width + (w_5 + 2) - 2)));
   # 117 "laplacian_fuse.cpp"
   *(pSrc_padding2 + ((height_7 + 2) * (width + 4) + (w_5 + 2))) = 2 * (hmpprt::s32 ) (*(pBufL_cp_3 + ((height_7 - 2) * width + (w_5 + 2) - 2))) - (hmpprt::s32 ) (*(pBufL_cp_3 + ((height_7 - 4) * width + (w_5 + 2) - 2)));
   # 118 "laplacian_fuse.cpp"
   *(pSrc_padding2 + ((height_7 + 3) * (width + 4) + (w_5 + 2))) = 4 * (hmpprt::s32 ) (*(pBufL_cp_3 + ((height_7 - 4) * width + (w_5 + 2) - 2))) - 4 * (hmpprt::s32 ) (*(pBufL_cp_3 + ((height_7 - 2) * width + (w_5 + 2) - 2))) + 2 * (hmpprt::s32 ) (*(pBufL_cp_3 + ((height_7 - 1) * width + (w_5 + 2) - 2))) - (hmpprt::s32 ) (*(pBufL_cp_3 + ((height_7 - 3) * width + (w_5 + 2) - 2)));
  }
  # 114 "laplacian_fuse.cpp"
 }
}
#endif // __HIPCC__



# 114 "laplacian_fuse.cpp"

#ifndef __HIPCC__
void hmpp_acc_region_main_108_internal_1(hmpprt::s32 height, hmpprt::s32 width_19, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s32>  pSrc_padding2_6, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  pBufL_cp_6)
{
 # 114 "laplacian_fuse.cpp"
 if (1)
 {
  hmpprt::CUDAGridCall __hmppcg_call;
  __hmppcg_call.setSizeX(192);
  __hmppcg_call.setSizeY(1);
  __hmppcg_call.setBlockSizeX(1);
  __hmppcg_call.setBlockSizeY(256);
  __hmppcg_call.addLocalParameter((hmpprt::s32) (height), "height_7");
  __hmppcg_call.addLocalParameter(&pBufL_cp_6, 8, "pBufL_cp_3");
  __hmppcg_call.addLocalParameter(&pSrc_padding2_6, 8, "pSrc_padding2");
  __hmppcg_call.addLocalParameter((hmpprt::s32) (width_19), "width");
  __hmppcg_call.launch(hmpp_acc_region_main_108_parallel_region_1, hmpprt::Context::getInstance()->getCUDADevice());
 }
 ;
}
#endif // __HIPCC__



# 114 "laplacian_fuse.cpp"

#ifndef __HIPCC__
extern "C" CDLT_API  void hmpp_acc_region_main_108(hmpprt::s32 height_14, hmpprt::s32 width_10, hmpprt::s32* pSrc_padding2_3, hmpprt::u08* pBufL_cp)
{
 # 126 "laplacian_fuse.cpp"
 (hmpp_acc_region_main_108_internal_1(height_14, width_10, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s32> (pSrc_padding2_3), hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08> (pBufL_cp)));
}
#endif // __HIPCC__



# 126 "laplacian_fuse.cpp"

#ifdef __HIPCC__

extern "C" __global__ void hmpp_acc_region_main_130_parallel_region_1(hmpprt::u08* FilterDst, hmpprt::s32 height_13, hmpprt::s32* pSrc_padding2_2, hmpprt::s32 width_9)
{
 # 128 "laplacian_fuse.cpp"
 {
  # 133 "laplacian_fuse.cpp"
  hmpprt::s32 iter_per_gang_9;
  # 133 "laplacian_fuse.cpp"
  hmpprt::s32 first_gang_iter_9;
  # 133 "laplacian_fuse.cpp"
  hmpprt::s32 last_gang_iter_9;
  # 133 "laplacian_fuse.cpp"
  iter_per_gang_9 = ((1 + (width_9 * height_13 - 1) / 192) > 256 ? (1 + (width_9 * height_13 - 1) / 192) : 256);
  # 133 "laplacian_fuse.cpp"
  first_gang_iter_9 = (hmpprt::gr_gbidx()) * iter_per_gang_9;
  # 133 "laplacian_fuse.cpp"
  last_gang_iter_9 = ((first_gang_iter_9 + iter_per_gang_9 - 1) < (width_9 * height_13 - 1) ? (first_gang_iter_9 + iter_per_gang_9 - 1) : (width_9 * height_13 - 1));
  # 133 "laplacian_fuse.cpp"
  hmpprt::s32 w_10;
  # 133 "laplacian_fuse.cpp"
  # 136 "laplacian_fuse.cpp"
  for (w_10 = first_gang_iter_9 + (hmpprt::gr_btidy()) ; w_10 <= last_gang_iter_9 ; w_10 = w_10 + (hmpprt::gr_btnumy()))
  {
   # 133 "laplacian_fuse.cpp"
   hmpprt::s32 w_6;
   # 135 "laplacian_fuse.cpp"
   hmpprt::s32 h_4;
   # 137 "laplacian_fuse.cpp"
   h_4 = w_10 % height_13;
   # 137 "laplacian_fuse.cpp"
   w_6 = w_10 / height_13;
   # 137 "laplacian_fuse.cpp"
   *(FilterDst + (h_4 * width_9 + w_6)) = (hmpprt::u08 ) (*(pSrc_padding2_2 + (h_4 * (width_9 + 4) + (w_6 + 2))) + (*(pSrc_padding2_2 + ((h_4 + 1) * (width_9 + 4) + (w_6 + 2))) << 2) + 6 * *(pSrc_padding2_2 + ((h_4 + 2) * (width_9 + 4) + (w_6 + 2))) + (*(pSrc_padding2_2 + ((h_4 + 3) * (width_9 + 4) + (w_6 + 2))) << 2) + *(pSrc_padding2_2 + ((h_4 + 4) * (width_9 + 4) + (w_6 + 2))) + 8 >> 4);
  }
  # 126 "laplacian_fuse.cpp"
 }
}
#endif // __HIPCC__



# 126 "laplacian_fuse.cpp"

#ifndef __HIPCC__
void hmpp_acc_region_main_130_internal_1(hmpprt::s32 height_18, hmpprt::s32 width_27, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  FilterDst_5, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s32>  pSrc_padding2_5)
{
 # 126 "laplacian_fuse.cpp"
 if (1)
 {
  hmpprt::CUDAGridCall __hmppcg_call;
  __hmppcg_call.setSizeX(192);
  __hmppcg_call.setSizeY(1);
  __hmppcg_call.setBlockSizeX(1);
  __hmppcg_call.setBlockSizeY(256);
  __hmppcg_call.addLocalParameter(&FilterDst_5, 8, "FilterDst");
  __hmppcg_call.addLocalParameter((hmpprt::s32) (height_18), "height_13");
  __hmppcg_call.addLocalParameter(&pSrc_padding2_5, 8, "pSrc_padding2_2");
  __hmppcg_call.addLocalParameter((hmpprt::s32) (width_27), "width_9");
  __hmppcg_call.launch(hmpp_acc_region_main_130_parallel_region_1, hmpprt::Context::getInstance()->getCUDADevice());
 }
 ;
}
#endif // __HIPCC__



# 126 "laplacian_fuse.cpp"

#ifndef __HIPCC__
extern "C" CDLT_API  void hmpp_acc_region_main_130(hmpprt::s32 height_6, hmpprt::s32 width_18, hmpprt::u08* FilterDst_2, hmpprt::s32* pSrc_padding2_8)
{
 # 148 "laplacian_fuse.cpp"
 (hmpp_acc_region_main_130_internal_1(height_6, width_18, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08> (FilterDst_2), hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s32> (pSrc_padding2_8)));
}
#endif // __HIPCC__



# 148 "laplacian_fuse.cpp"

#ifdef __HIPCC__

extern "C" __global__ void hmpp_acc_region_main_153_parallel_region_1(hmpprt::u08* DownsampleDst, hmpprt::u08* FilterDst_3, hmpprt::s32 halfHeight, hmpprt::s32 halfWidth, hmpprt::s32 width_8)
{
 # 150 "laplacian_fuse.cpp"
 {
  # 156 "laplacian_fuse.cpp"
  hmpprt::s32 iter_per_gang_8;
  # 156 "laplacian_fuse.cpp"
  hmpprt::s32 first_gang_iter_8;
  # 156 "laplacian_fuse.cpp"
  hmpprt::s32 last_gang_iter_8;
  # 156 "laplacian_fuse.cpp"
  iter_per_gang_8 = ((1 + (halfHeight * halfWidth - 1) / 192) > 256 ? (1 + (halfHeight * halfWidth - 1) / 192) : 256);
  # 156 "laplacian_fuse.cpp"
  first_gang_iter_8 = (hmpprt::gr_gbidx()) * iter_per_gang_8;
  # 156 "laplacian_fuse.cpp"
  last_gang_iter_8 = ((first_gang_iter_8 + iter_per_gang_8 - 1) < (halfHeight * halfWidth - 1) ? (first_gang_iter_8 + iter_per_gang_8 - 1) : (halfHeight * halfWidth - 1));
  # 156 "laplacian_fuse.cpp"
  hmpprt::s32 y_6;
  # 156 "laplacian_fuse.cpp"
  # 160 "laplacian_fuse.cpp"
  for (y_6 = first_gang_iter_8 + (hmpprt::gr_btidy()) ; y_6 <= last_gang_iter_8 ; y_6 = y_6 + (hmpprt::gr_btnumy()))
  {
   # 156 "laplacian_fuse.cpp"
   hmpprt::s32 y_3;
   # 159 "laplacian_fuse.cpp"
   hmpprt::s32 x_5;
   # 161 "laplacian_fuse.cpp"
   x_5 = y_6 % halfWidth;
   # 161 "laplacian_fuse.cpp"
   y_3 = y_6 / halfWidth;
   # 161 "laplacian_fuse.cpp"
   *(DownsampleDst + (y_3 * halfWidth + x_5)) = *(FilterDst_3 + ((y_3 << 1) * width_8 + (x_5 << 1)));
  }
  # 148 "laplacian_fuse.cpp"
 }
}
#endif // __HIPCC__



# 148 "laplacian_fuse.cpp"

#ifndef __HIPCC__
void hmpp_acc_region_main_153_internal_1(hmpprt::s32 width_26, hmpprt::s32 halfWidth_8, hmpprt::s32 halfHeight_6, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  FilterDst_4, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  DownsampleDst_8)
{
 # 148 "laplacian_fuse.cpp"
 if (1)
 {
  hmpprt::CUDAGridCall __hmppcg_call;
  __hmppcg_call.setSizeX(192);
  __hmppcg_call.setSizeY(1);
  __hmppcg_call.setBlockSizeX(1);
  __hmppcg_call.setBlockSizeY(256);
  __hmppcg_call.addLocalParameter(&DownsampleDst_8, 8, "DownsampleDst");
  __hmppcg_call.addLocalParameter(&FilterDst_4, 8, "FilterDst_3");
  __hmppcg_call.addLocalParameter((hmpprt::s32) (halfHeight_6), "halfHeight");
  __hmppcg_call.addLocalParameter((hmpprt::s32) (halfWidth_8), "halfWidth");
  __hmppcg_call.addLocalParameter((hmpprt::s32) (width_26), "width_8");
  __hmppcg_call.launch(hmpp_acc_region_main_153_parallel_region_1, hmpprt::Context::getInstance()->getCUDADevice());
 }
 ;
}
#endif // __HIPCC__



# 148 "laplacian_fuse.cpp"

#ifndef __HIPCC__
extern "C" CDLT_API  void hmpp_acc_region_main_153(hmpprt::s32 width_17, hmpprt::s32 halfWidth_4, hmpprt::s32 halfHeight_3, hmpprt::u08* FilterDst_1, hmpprt::u08* DownsampleDst_4)
{
 # 169 "laplacian_fuse.cpp"
 (hmpp_acc_region_main_153_internal_1(width_17, halfWidth_4, halfHeight_3, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08> (FilterDst_1), hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08> (DownsampleDst_4)));
}
#endif // __HIPCC__



# 169 "laplacian_fuse.cpp"

#ifdef __HIPCC__

extern "C" __global__ void hmpp_acc_region_main_183_parallel_region_1(hmpprt::u08* DownsampleDst_3, hmpprt::u08* UpsampleDst, hmpprt::s32 halfHeight_2, hmpprt::s32 halfWidth_3, hmpprt::s32 height_5, hmpprt::s32 width_7)
{
 # 171 "laplacian_fuse.cpp"
 {
  # 186 "laplacian_fuse.cpp"
  hmpprt::s32 iter_per_gang_7;
  # 186 "laplacian_fuse.cpp"
  hmpprt::s32 first_gang_iter_7;
  # 186 "laplacian_fuse.cpp"
  hmpprt::s32 last_gang_iter_7;
  # 186 "laplacian_fuse.cpp"
  iter_per_gang_7 = ((1 + (halfWidth_3 - 2) / 192) > 256 ? (1 + (halfWidth_3 - 2) / 192) : 256);
  # 186 "laplacian_fuse.cpp"
  first_gang_iter_7 = (hmpprt::gr_gbidx()) * iter_per_gang_7;
  # 186 "laplacian_fuse.cpp"
  last_gang_iter_7 = ((first_gang_iter_7 + iter_per_gang_7 - 1) < (halfWidth_3 - 2) ? (first_gang_iter_7 + iter_per_gang_7 - 1) : (halfWidth_3 - 2));
  # 186 "laplacian_fuse.cpp"
  hmpprt::s32 x_6;
  # 186 "laplacian_fuse.cpp"
  # 187 "laplacian_fuse.cpp"
  for (x_6 = first_gang_iter_7 + (hmpprt::gr_btidy()) ; x_6 <= last_gang_iter_7 ; x_6 = x_6 + (hmpprt::gr_btnumy()))
  {
   # 188 "laplacian_fuse.cpp"
   *(UpsampleDst + ((height_5 - 1) * width_7 + 2 * (x_6 + 1) - 1)) = (hmpprt::u08 ) ((hmpprt::s32 ) (*(DownsampleDst_3 + ((halfHeight_2 - 1) * halfWidth_3 + (x_6 + 1) - 1))) + (hmpprt::s32 ) (*(DownsampleDst_3 + ((halfHeight_2 - 1) * halfWidth_3 + (x_6 + 1)))) + 1 >> 1);
   # 189 "laplacian_fuse.cpp"
   *(UpsampleDst + ((height_5 - 1) * width_7 + 2 * (x_6 + 1))) = *(DownsampleDst_3 + ((halfHeight_2 - 1) * halfWidth_3 + (x_6 + 1)));
  }
  # 169 "laplacian_fuse.cpp"
 }
}
#endif // __HIPCC__



# 169 "laplacian_fuse.cpp"

#ifndef __HIPCC__
void hmpp_acc_region_main_183_internal_1(hmpprt::s32 height_17, hmpprt::s32 width_25, hmpprt::s32 halfWidth_7, hmpprt::s32 halfHeight_8, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  DownsampleDst_11, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  UpsampleDst_8)
{
 # 169 "laplacian_fuse.cpp"
 if (1)
 {
  hmpprt::CUDAGridCall __hmppcg_call;
  __hmppcg_call.setSizeX(192);
  __hmppcg_call.setSizeY(1);
  __hmppcg_call.setBlockSizeX(1);
  __hmppcg_call.setBlockSizeY(256);
  __hmppcg_call.addLocalParameter(&DownsampleDst_11, 8, "DownsampleDst_3");
  __hmppcg_call.addLocalParameter(&UpsampleDst_8, 8, "UpsampleDst");
  __hmppcg_call.addLocalParameter((hmpprt::s32) (halfHeight_8), "halfHeight_2");
  __hmppcg_call.addLocalParameter((hmpprt::s32) (halfWidth_7), "halfWidth_3");
  __hmppcg_call.addLocalParameter((hmpprt::s32) (height_17), "height_5");
  __hmppcg_call.addLocalParameter((hmpprt::s32) (width_25), "width_7");
  __hmppcg_call.launch(hmpp_acc_region_main_183_parallel_region_1, hmpprt::Context::getInstance()->getCUDADevice());
 }
 ;
}
#endif // __HIPCC__



# 169 "laplacian_fuse.cpp"

#ifndef __HIPCC__
extern "C" CDLT_API  void hmpp_acc_region_main_183(hmpprt::s32 height_12, hmpprt::s32 width_16, hmpprt::s32 halfWidth_11, hmpprt::s32 halfHeight_5, hmpprt::u08* DownsampleDst_7, hmpprt::u08* UpsampleDst_4)
{
 # 196 "laplacian_fuse.cpp"
 (hmpp_acc_region_main_183_internal_1(height_12, width_16, halfWidth_11, halfHeight_5, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08> (DownsampleDst_7), hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08> (UpsampleDst_4)));
}
#endif // __HIPCC__



# 196 "laplacian_fuse.cpp"

#ifdef __HIPCC__

extern "C" __global__ void hmpp_acc_region_main_192_parallel_region_1(hmpprt::u08* DownsampleDst_2, hmpprt::u08* UpsampleDst_3, hmpprt::s32 halfWidth_2)
{
 # 198 "laplacian_fuse.cpp"
 {
  # 201 "laplacian_fuse.cpp"
  hmpprt::s32 iter_per_gang_6;
  # 201 "laplacian_fuse.cpp"
  hmpprt::s32 first_gang_iter_6;
  # 201 "laplacian_fuse.cpp"
  hmpprt::s32 last_gang_iter_6;
  # 201 "laplacian_fuse.cpp"
  iter_per_gang_6 = ((1 +  - (1 - (halfWidth_2 - 1)) / 192) > 256 ? (1 +  - (1 - (halfWidth_2 - 1)) / 192) : 256);
  # 201 "laplacian_fuse.cpp"
  first_gang_iter_6 = (hmpprt::gr_gbidx()) * iter_per_gang_6;
  # 201 "laplacian_fuse.cpp"
  last_gang_iter_6 = ((first_gang_iter_6 + iter_per_gang_6 - 1) < ( - (1 - (halfWidth_2 - 1))) ? (first_gang_iter_6 + iter_per_gang_6 - 1) : ( - (1 - (halfWidth_2 - 1))));
  # 201 "laplacian_fuse.cpp"
  hmpprt::s32 x_7;
  # 201 "laplacian_fuse.cpp"
  # 202 "laplacian_fuse.cpp"
  for (x_7 = first_gang_iter_6 + (hmpprt::gr_btidy()) ; x_7 <= last_gang_iter_6 ; x_7 = x_7 + (hmpprt::gr_btnumy()))
  {
   # 203 "laplacian_fuse.cpp"
   *(UpsampleDst_3 + 2 * (halfWidth_2 - 1 - x_7)) = *(DownsampleDst_2 + (halfWidth_2 - 1 - x_7));
   # 204 "laplacian_fuse.cpp"
   *(UpsampleDst_3 + (2 * (halfWidth_2 - 1 - x_7) - 1)) = (hmpprt::u08 ) (((hmpprt::s32 ) (*(DownsampleDst_2 + (halfWidth_2 - 1 - x_7 - 1))) + (hmpprt::s32 ) (*(DownsampleDst_2 + (halfWidth_2 - 1 - x_7))) + 1) / 2);
  }
  # 196 "laplacian_fuse.cpp"
 }
}
#endif // __HIPCC__



# 196 "laplacian_fuse.cpp"

#ifndef __HIPCC__
void hmpp_acc_region_main_192_internal_1(hmpprt::s32 width_28, hmpprt::s32 halfWidth_10, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  DownsampleDst_6, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  UpsampleDst_7)
{
 # 196 "laplacian_fuse.cpp"
 if (1)
 {
  hmpprt::CUDAGridCall __hmppcg_call;
  __hmppcg_call.setSizeX(192);
  __hmppcg_call.setSizeY(1);
  __hmppcg_call.setBlockSizeX(1);
  __hmppcg_call.setBlockSizeY(256);
  __hmppcg_call.addLocalParameter(&DownsampleDst_6, 8, "DownsampleDst_2");
  __hmppcg_call.addLocalParameter(&UpsampleDst_7, 8, "UpsampleDst_3");
  __hmppcg_call.addLocalParameter((hmpprt::s32) (halfWidth_10), "halfWidth_2");
  __hmppcg_call.launch(hmpp_acc_region_main_192_parallel_region_1, hmpprt::Context::getInstance()->getCUDADevice());
 }
 ;
}
#endif // __HIPCC__



# 196 "laplacian_fuse.cpp"

#ifndef __HIPCC__
extern "C" CDLT_API  void hmpp_acc_region_main_192(hmpprt::s32 width_6, hmpprt::s32 halfWidth_6, hmpprt::u08* DownsampleDst_10, hmpprt::u08* UpsampleDst_11)
{
 # 211 "laplacian_fuse.cpp"
 (hmpp_acc_region_main_192_internal_1(width_6, halfWidth_6, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08> (DownsampleDst_10), hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08> (UpsampleDst_11)));
}
#endif // __HIPCC__



# 211 "laplacian_fuse.cpp"

#ifdef __HIPCC__

extern "C" __global__ void hmpp_acc_region_main_206_parallel_region_1(hmpprt::u08* DownsampleDst_1, hmpprt::u08* UpsampleDst_2, hmpprt::s32 halfHeight_1, hmpprt::s32 halfWidth_1, hmpprt::s32 width_5)
{
 # 213 "laplacian_fuse.cpp"
 {
  # 209 "laplacian_fuse.cpp"
  hmpprt::s32 iter_per_gang_5;
  # 209 "laplacian_fuse.cpp"
  hmpprt::s32 first_gang_iter_5;
  # 209 "laplacian_fuse.cpp"
  hmpprt::s32 last_gang_iter_5;
  # 209 "laplacian_fuse.cpp"
  iter_per_gang_5 = 1 +  - (1 - (halfHeight_1 - 1)) / 192;
  # 209 "laplacian_fuse.cpp"
  first_gang_iter_5 = (hmpprt::gr_gbidx()) * iter_per_gang_5;
  # 209 "laplacian_fuse.cpp"
  last_gang_iter_5 = ((first_gang_iter_5 + iter_per_gang_5 - 1) < ( - (1 - (halfHeight_1 - 1))) ? (first_gang_iter_5 + iter_per_gang_5 - 1) : ( - (1 - (halfHeight_1 - 1))));
  # 209 "laplacian_fuse.cpp"
  hmpprt::s32 y_4;
  # 209 "laplacian_fuse.cpp"
  # 210 "laplacian_fuse.cpp"
  for (y_4 = first_gang_iter_5 ; y_4 <= last_gang_iter_5 ; y_4 = y_4 + 1)
  {
   # 213 "laplacian_fuse.cpp"
   hmpprt::u08 tmp_18;
   # 213 "laplacian_fuse.cpp"
   hmpprt::u08 tmp_19;
   # 213 "laplacian_fuse.cpp"
   hmpprt::u08 tmp_20;
   # 213 "laplacian_fuse.cpp"
   hmpprt::u08 tmp_21;
   # 213 "laplacian_fuse.cpp"
   hmpprt::s32 end_6;
   # 213 "laplacian_fuse.cpp"
   hmpprt::s32 x_8;
   # 213 "laplacian_fuse.cpp"
   # 214 "laplacian_fuse.cpp"
   # 214 "laplacian_fuse.cpp"
   for (x_8 = (hmpprt::gr_btidy()), end_6 = halfWidth_1 - 2 ; x_8 <= end_6 ; x_8 = x_8 + (hmpprt::gr_btnumy()))
   {
    # 215 "laplacian_fuse.cpp"
    *(UpsampleDst_2 + ((2 * (halfHeight_1 - 1 - y_4) - 1) * width_5 + 2 * (x_8 + 1) - 1)) = (hmpprt::u08 ) ((hmpprt::s32 ) (*(DownsampleDst_1 + ((halfHeight_1 - 1 - y_4) * halfWidth_1 + (x_8 + 1) - 1))) + (hmpprt::s32 ) (*(DownsampleDst_1 + ((halfHeight_1 - 1 - y_4) * halfWidth_1 + (x_8 + 1)))) + (hmpprt::s32 ) (*(DownsampleDst_1 + ((halfHeight_1 - 1 - y_4 - 1) * halfWidth_1 + (x_8 + 1) - 1))) + (hmpprt::s32 ) (*(DownsampleDst_1 + ((halfHeight_1 - 1 - y_4 - 1) * halfWidth_1 + (x_8 + 1)))) + 2 >> 2);
    # 216 "laplacian_fuse.cpp"
    *(UpsampleDst_2 + ((2 * (halfHeight_1 - 1 - y_4) - 1) * width_5 + 2 * (x_8 + 1))) = (hmpprt::u08 ) ((hmpprt::s32 ) (*(DownsampleDst_1 + ((halfHeight_1 - 1 - y_4) * halfWidth_1 + (x_8 + 1)))) + (hmpprt::s32 ) (*(DownsampleDst_1 + ((halfHeight_1 - 1 - y_4 - 1) * halfWidth_1 + (x_8 + 1)))) + 1 >> 1);
    # 218 "laplacian_fuse.cpp"
    *(UpsampleDst_2 + (2 * (halfHeight_1 - 1 - y_4) * width_5 + 2 * (x_8 + 1) - 1)) = (hmpprt::u08 ) ((hmpprt::s32 ) (*(DownsampleDst_1 + ((halfHeight_1 - 1 - y_4) * halfWidth_1 + (x_8 + 1) - 1))) + (hmpprt::s32 ) (*(DownsampleDst_1 + ((halfHeight_1 - 1 - y_4) * halfWidth_1 + (x_8 + 1)))) + 1 >> 1);
    # 219 "laplacian_fuse.cpp"
    *(UpsampleDst_2 + (2 * (halfHeight_1 - 1 - y_4) * width_5 + 2 * (x_8 + 1))) = *(DownsampleDst_1 + ((halfHeight_1 - 1 - y_4) * halfWidth_1 + (x_8 + 1)));
   }
   # 211 "laplacian_fuse.cpp"
   # 211 "laplacian_fuse.cpp"
   tmp_18 = (hmpprt::u08 ) ((hmpprt::s32 ) (*(DownsampleDst_1 + ((halfHeight_1 - 1 - y_4) * halfWidth_1 + halfWidth_1 - 1))) + (hmpprt::s32 ) (*(DownsampleDst_1 + ((halfHeight_1 - 1 - y_4 - 1) * halfWidth_1 + halfWidth_1 - 1))) + 1 >> 1);
   # 211 "laplacian_fuse.cpp"
   if ((hmpprt::gr_btidy()) == 0)
   {
    # 211 "laplacian_fuse.cpp"
    *(UpsampleDst_2 + ((2 * (halfHeight_1 - 1 - y_4) - 1) * width_5 + width_5 - 1)) = tmp_18;
   }
   # 211 "laplacian_fuse.cpp"
   (hmpprt::gr_barrier());
   # 211 "laplacian_fuse.cpp"
   tmp_19 = *(DownsampleDst_1 + ((halfHeight_1 - 1 - y_4) * halfWidth_1 + halfWidth_1 - 1));
   # 211 "laplacian_fuse.cpp"
   if ((hmpprt::gr_btidy()) == 0)
   {
    # 211 "laplacian_fuse.cpp"
    *(UpsampleDst_2 + (2 * (halfHeight_1 - 1 - y_4) * width_5 + width_5 - 1)) = tmp_19;
   }
   # 211 "laplacian_fuse.cpp"
   (hmpprt::gr_barrier());
   # 211 "laplacian_fuse.cpp"
   tmp_20 = (hmpprt::u08 ) ((hmpprt::s32 ) (*(DownsampleDst_1 + (halfHeight_1 - 1 - y_4) * halfWidth_1)) + (hmpprt::s32 ) (*(DownsampleDst_1 + (halfHeight_1 - 1 - y_4 - 1) * halfWidth_1)) + 1 >> 1);
   # 211 "laplacian_fuse.cpp"
   if ((hmpprt::gr_btidy()) == 0)
   {
    # 211 "laplacian_fuse.cpp"
    *(UpsampleDst_2 + (2 * (halfHeight_1 - 1 - y_4) - 1) * width_5) = tmp_20;
   }
   # 211 "laplacian_fuse.cpp"
   (hmpprt::gr_barrier());
   # 211 "laplacian_fuse.cpp"
   tmp_21 = *(DownsampleDst_1 + (halfHeight_1 - 1 - y_4) * halfWidth_1);
   # 211 "laplacian_fuse.cpp"
   if ((hmpprt::gr_btidy()) == 0)
   {
    # 211 "laplacian_fuse.cpp"
    *(UpsampleDst_2 + 2 * (halfHeight_1 - 1 - y_4) * width_5) = tmp_21;
   }
   # 211 "laplacian_fuse.cpp"
   (hmpprt::gr_barrier());
  }
  # 211 "laplacian_fuse.cpp"
 }
}
#endif // __HIPCC__



# 211 "laplacian_fuse.cpp"

#ifndef __HIPCC__
void hmpp_acc_region_main_206_internal_1(hmpprt::s32 width_24, hmpprt::s32 halfWidth_9, hmpprt::s32 halfHeight_4, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  DownsampleDst_5, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  UpsampleDst_10)
{
 # 211 "laplacian_fuse.cpp"
 if (1)
 {
  hmpprt::CUDAGridCall __hmppcg_call;
  __hmppcg_call.setSizeX(192);
  __hmppcg_call.setSizeY(1);
  __hmppcg_call.setBlockSizeX(1);
  __hmppcg_call.setBlockSizeY(256);
  __hmppcg_call.addLocalParameter(&DownsampleDst_5, 8, "DownsampleDst_1");
  __hmppcg_call.addLocalParameter(&UpsampleDst_10, 8, "UpsampleDst_2");
  __hmppcg_call.addLocalParameter((hmpprt::s32) (halfHeight_4), "halfHeight_1");
  __hmppcg_call.addLocalParameter((hmpprt::s32) (halfWidth_9), "halfWidth_1");
  __hmppcg_call.addLocalParameter((hmpprt::s32) (width_24), "width_5");
  __hmppcg_call.launch(hmpp_acc_region_main_206_parallel_region_1, hmpprt::Context::getInstance()->getCUDADevice());
 }
 ;
}
#endif // __HIPCC__



# 211 "laplacian_fuse.cpp"

#ifndef __HIPCC__
extern "C" CDLT_API  void hmpp_acc_region_main_206(hmpprt::s32 width_15, hmpprt::s32 halfWidth_5, hmpprt::s32 halfHeight_7, hmpprt::u08* DownsampleDst_9, hmpprt::u08* UpsampleDst_6)
{
 # 233 "laplacian_fuse.cpp"
 (hmpp_acc_region_main_206_internal_1(width_15, halfWidth_5, halfHeight_7, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08> (DownsampleDst_9), hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08> (UpsampleDst_6)));
}
#endif // __HIPCC__



# 233 "laplacian_fuse.cpp"

#ifdef __HIPCC__

extern "C" __global__ void hmpp_acc_region_main_239_parallel_region_1(hmpprt::s16* LaplacianLayer, hmpprt::u08* Source, hmpprt::u08* UpsampleDst_1, hmpprt::s32 height_4, hmpprt::s32 width_4)
{
 # 235 "laplacian_fuse.cpp"
 {
  # 242 "laplacian_fuse.cpp"
  hmpprt::s32 iter_per_gang_4;
  # 242 "laplacian_fuse.cpp"
  hmpprt::s32 first_gang_iter_4;
  # 242 "laplacian_fuse.cpp"
  hmpprt::s32 last_gang_iter_4;
  # 242 "laplacian_fuse.cpp"
  iter_per_gang_4 = ((1 + (height_4 * width_4 - 1) / 192) > 256 ? (1 + (height_4 * width_4 - 1) / 192) : 256);
  # 242 "laplacian_fuse.cpp"
  first_gang_iter_4 = (hmpprt::gr_gbidx()) * iter_per_gang_4;
  # 242 "laplacian_fuse.cpp"
  last_gang_iter_4 = ((first_gang_iter_4 + iter_per_gang_4 - 1) < (height_4 * width_4 - 1) ? (first_gang_iter_4 + iter_per_gang_4 - 1) : (height_4 * width_4 - 1));
  # 242 "laplacian_fuse.cpp"
  hmpprt::s32 y_7;
  # 242 "laplacian_fuse.cpp"
  # 246 "laplacian_fuse.cpp"
  for (y_7 = first_gang_iter_4 + (hmpprt::gr_btidy()) ; y_7 <= last_gang_iter_4 ; y_7 = y_7 + (hmpprt::gr_btnumy()))
  {
   # 242 "laplacian_fuse.cpp"
   hmpprt::s32 y_5;
   # 245 "laplacian_fuse.cpp"
   hmpprt::s32 x_9;
   # 247 "laplacian_fuse.cpp"
   x_9 = y_7 % width_4;
   # 247 "laplacian_fuse.cpp"
   y_5 = y_7 / width_4;
   # 247 "laplacian_fuse.cpp"
   *(LaplacianLayer + (y_5 * width_4 + x_9)) = (hmpprt::s16 ) ((hmpprt::s32 ) ((hmpprt::s16 ) (*(Source + (y_5 * width_4 + x_9)))) - (hmpprt::s32 ) ((hmpprt::s16 ) (*(UpsampleDst_1 + (y_5 * width_4 + x_9)))));
  }
  # 233 "laplacian_fuse.cpp"
 }
}
#endif // __HIPCC__



# 233 "laplacian_fuse.cpp"

#ifndef __HIPCC__
void hmpp_acc_region_main_239_internal_1(hmpprt::s32 height_16, hmpprt::s32 width_23, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  Source_4, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s16>  LaplacianLayer_2, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  UpsampleDst_9)
{
 # 233 "laplacian_fuse.cpp"
 if (1)
 {
  hmpprt::CUDAGridCall __hmppcg_call;
  __hmppcg_call.setSizeX(192);
  __hmppcg_call.setSizeY(1);
  __hmppcg_call.setBlockSizeX(1);
  __hmppcg_call.setBlockSizeY(256);
  __hmppcg_call.addLocalParameter(&LaplacianLayer_2, 8, "LaplacianLayer");
  __hmppcg_call.addLocalParameter(&Source_4, 8, "Source");
  __hmppcg_call.addLocalParameter(&UpsampleDst_9, 8, "UpsampleDst_1");
  __hmppcg_call.addLocalParameter((hmpprt::s32) (height_16), "height_4");
  __hmppcg_call.addLocalParameter((hmpprt::s32) (width_23), "width_4");
  __hmppcg_call.launch(hmpp_acc_region_main_239_parallel_region_1, hmpprt::Context::getInstance()->getCUDADevice());
 }
 ;
}
#endif // __HIPCC__



# 233 "laplacian_fuse.cpp"

#ifndef __HIPCC__
extern "C" CDLT_API  void hmpp_acc_region_main_239(hmpprt::s32 height_11, hmpprt::s32 width_14, hmpprt::u08* Source_2, hmpprt::s16* LaplacianLayer_1, hmpprt::u08* UpsampleDst_5)
{
 # 12 "laplacian_fuse.cpp"
 (hmpp_acc_region_main_239_internal_1(height_11, width_14, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08> (Source_2), hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s16> (LaplacianLayer_1), hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08> (UpsampleDst_5)));
}
#endif // __HIPCC__



# 12 "laplacian_fuse.cpp"

#ifdef __HIPCC__

extern "C" __global__ void hmpp_acc_region_main_55_parallel_region_1(hmpprt::u08* Source_1, hmpprt::s32 height_3, hmpprt::s32* pSrc_padding, hmpprt::s32 width_3)
{
 # 14 "laplacian_fuse.cpp"
 {
  # 58 "laplacian_fuse.cpp"
  hmpprt::s32 iter_per_gang_3;
  # 58 "laplacian_fuse.cpp"
  hmpprt::s32 first_gang_iter_3;
  # 58 "laplacian_fuse.cpp"
  hmpprt::s32 last_gang_iter_3;
  # 58 "laplacian_fuse.cpp"
  iter_per_gang_3 = 1 + (height_3 - 1) / 192;
  # 58 "laplacian_fuse.cpp"
  first_gang_iter_3 = (hmpprt::gr_gbidx()) * iter_per_gang_3;
  # 58 "laplacian_fuse.cpp"
  last_gang_iter_3 = ((first_gang_iter_3 + iter_per_gang_3 - 1) < (height_3 - 1) ? (first_gang_iter_3 + iter_per_gang_3 - 1) : (height_3 - 1));
  # 58 "laplacian_fuse.cpp"
  hmpprt::s32 h_5;
  # 58 "laplacian_fuse.cpp"
  # 59 "laplacian_fuse.cpp"
  for (h_5 = first_gang_iter_3 ; h_5 <= last_gang_iter_3 ; h_5 = h_5 + 1)
  {
   # 67 "laplacian_fuse.cpp"
   hmpprt::s32 tmp_22;
   # 67 "laplacian_fuse.cpp"
   hmpprt::s32 tmp_23;
   # 67 "laplacian_fuse.cpp"
   hmpprt::s32 tmp_24;
   # 67 "laplacian_fuse.cpp"
   hmpprt::s32 tmp_25;
   # 67 "laplacian_fuse.cpp"
   tmp_22 = 2 * (hmpprt::s32 ) (*(Source_1 + (h_5 * width_3 + 1))) - (hmpprt::s32 ) (*(Source_1 + (h_5 * width_3 + 3)));
   # 67 "laplacian_fuse.cpp"
   if ((hmpprt::gr_btidy()) == 0)
   {
    # 67 "laplacian_fuse.cpp"
    *(pSrc_padding + ((h_5 + 2) * (width_3 + 4) + 1)) = tmp_22;
   }
   # 67 "laplacian_fuse.cpp"
   (hmpprt::gr_barrier());
   # 67 "laplacian_fuse.cpp"
   tmp_23 = 4 * (hmpprt::s32 ) (*(Source_1 + (h_5 * width_3 + 3))) - 4 * (hmpprt::s32 ) (*(Source_1 + (h_5 * width_3 + 1))) + 2 * (hmpprt::s32 ) (*(Source_1 + h_5 * width_3)) - (hmpprt::s32 ) (*(Source_1 + (h_5 * width_3 + 2)));
   # 67 "laplacian_fuse.cpp"
   if ((hmpprt::gr_btidy()) == 0)
   {
    # 67 "laplacian_fuse.cpp"
    *(pSrc_padding + (h_5 + 2) * (width_3 + 4)) = tmp_23;
   }
   # 67 "laplacian_fuse.cpp"
   (hmpprt::gr_barrier());
   # 67 "laplacian_fuse.cpp"
   tmp_24 = 2 * (hmpprt::s32 ) (*(Source_1 + (h_5 * width_3 + width_3 - 2))) - (hmpprt::s32 ) (*(Source_1 + (h_5 * width_3 + width_3 - 4)));
   # 67 "laplacian_fuse.cpp"
   if ((hmpprt::gr_btidy()) == 0)
   {
    # 67 "laplacian_fuse.cpp"
    *(pSrc_padding + ((h_5 + 2) * (width_3 + 4) + width_3 + 2)) = tmp_24;
   }
   # 67 "laplacian_fuse.cpp"
   (hmpprt::gr_barrier());
   # 67 "laplacian_fuse.cpp"
   tmp_25 = 4 * (hmpprt::s32 ) (*(Source_1 + (h_5 * width_3 + width_3 - 4))) - 4 * (hmpprt::s32 ) (*(Source_1 + (h_5 * width_3 + width_3 - 2))) + 2 * (hmpprt::s32 ) (*(Source_1 + (h_5 * width_3 + width_3 - 1))) - (hmpprt::s32 ) (*(Source_1 + (h_5 * width_3 + width_3 - 3)));
   # 67 "laplacian_fuse.cpp"
   if ((hmpprt::gr_btidy()) == 0)
   {
    # 67 "laplacian_fuse.cpp"
    *(pSrc_padding + ((h_5 + 2) * (width_3 + 4) + width_3 + 3)) = tmp_25;
   }
   # 67 "laplacian_fuse.cpp"
   (hmpprt::gr_barrier());
   # 67 "laplacian_fuse.cpp"
   hmpprt::s32 end_9;
   # 67 "laplacian_fuse.cpp"
   hmpprt::s32 w_7;
   # 67 "laplacian_fuse.cpp"
   # 68 "laplacian_fuse.cpp"
   # 68 "laplacian_fuse.cpp"
   for (w_7 = (hmpprt::gr_btidy()), end_9 = width_3 - 1 ; w_7 <= end_9 ; w_7 = w_7 + (hmpprt::gr_btnumy()))
   {
    # 68 "laplacian_fuse.cpp"
    *(pSrc_padding + ((h_5 + 2) * (width_3 + 4) + w_7 + 2)) = (hmpprt::s32 ) (*(Source_1 + (h_5 * width_3 + w_7)));
   }
   # 12 "laplacian_fuse.cpp"
  }
  # 12 "laplacian_fuse.cpp"
 }
}
#endif // __HIPCC__



# 12 "laplacian_fuse.cpp"

#ifndef __HIPCC__
void hmpp_acc_region_main_55_internal_1(hmpprt::s32 height_15, hmpprt::s32 width_22, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  Source_5, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s32>  pSrc_padding_4)
{
 # 12 "laplacian_fuse.cpp"
 if (1)
 {
  hmpprt::CUDAGridCall __hmppcg_call;
  __hmppcg_call.setSizeX(192);
  __hmppcg_call.setSizeY(1);
  __hmppcg_call.setBlockSizeX(1);
  __hmppcg_call.setBlockSizeY(256);
  __hmppcg_call.addLocalParameter(&Source_5, 8, "Source_1");
  __hmppcg_call.addLocalParameter((hmpprt::s32) (height_15), "height_3");
  __hmppcg_call.addLocalParameter(&pSrc_padding_4, 8, "pSrc_padding");
  __hmppcg_call.addLocalParameter((hmpprt::s32) (width_22), "width_3");
  __hmppcg_call.launch(hmpp_acc_region_main_55_parallel_region_1, hmpprt::Context::getInstance()->getCUDADevice());
 }
 ;
}
#endif // __HIPCC__



# 12 "laplacian_fuse.cpp"

#ifndef __HIPCC__
extern "C" CDLT_API  void hmpp_acc_region_main_55(hmpprt::s32 height_10, hmpprt::s32 width_13, hmpprt::u08* Source_3, hmpprt::s32* pSrc_padding_2)
{
 # 75 "laplacian_fuse.cpp"
 (hmpp_acc_region_main_55_internal_1(height_10, width_13, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08> (Source_3), hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s32> (pSrc_padding_2)));
}
#endif // __HIPCC__



# 75 "laplacian_fuse.cpp"

#ifdef __HIPCC__

extern "C" __global__ void hmpp_acc_region_main_80_parallel_region_1(hmpprt::s32 height_2, hmpprt::u08* pBufL_cp_2, hmpprt::s32* pSrc_padding_1, hmpprt::s32 width_2)
{
 # 77 "laplacian_fuse.cpp"
 {
  # 83 "laplacian_fuse.cpp"
  hmpprt::s32 iter_per_gang_2;
  # 83 "laplacian_fuse.cpp"
  hmpprt::s32 first_gang_iter_2;
  # 83 "laplacian_fuse.cpp"
  hmpprt::s32 last_gang_iter_2;
  # 83 "laplacian_fuse.cpp"
  iter_per_gang_2 = ((1 + (height_2 * width_2 - 1) / 192) > 256 ? (1 + (height_2 * width_2 - 1) / 192) : 256);
  # 83 "laplacian_fuse.cpp"
  first_gang_iter_2 = (hmpprt::gr_gbidx()) * iter_per_gang_2;
  # 83 "laplacian_fuse.cpp"
  last_gang_iter_2 = ((first_gang_iter_2 + iter_per_gang_2 - 1) < (height_2 * width_2 - 1) ? (first_gang_iter_2 + iter_per_gang_2 - 1) : (height_2 * width_2 - 1));
  # 83 "laplacian_fuse.cpp"
  hmpprt::s32 h_8;
  # 83 "laplacian_fuse.cpp"
  # 86 "laplacian_fuse.cpp"
  for (h_8 = first_gang_iter_2 + (hmpprt::gr_btidy()) ; h_8 <= last_gang_iter_2 ; h_8 = h_8 + (hmpprt::gr_btnumy()))
  {
   # 83 "laplacian_fuse.cpp"
   hmpprt::s32 h_6;
   # 85 "laplacian_fuse.cpp"
   hmpprt::s32 w_8;
   # 87 "laplacian_fuse.cpp"
   w_8 = h_8 % width_2;
   # 87 "laplacian_fuse.cpp"
   h_6 = h_8 / width_2;
   # 87 "laplacian_fuse.cpp"
   *(pBufL_cp_2 + (h_6 * width_2 + w_8)) = (hmpprt::u08 ) (*(pSrc_padding_1 + ((h_6 + 2) * (width_2 + 4) + w_8)) + (*(pSrc_padding_1 + ((h_6 + 2) * (width_2 + 4) + w_8 + 1)) << 2) + 6 * *(pSrc_padding_1 + ((h_6 + 2) * (width_2 + 4) + (w_8 + 2))) + (*(pSrc_padding_1 + ((h_6 + 2) * (width_2 + 4) + (w_8 + 3))) << 2) + *(pSrc_padding_1 + ((h_6 + 2) * (width_2 + 4) + (w_8 + 4))) + 8 >> 4);
  }
  # 75 "laplacian_fuse.cpp"
 }
}
#endif // __HIPCC__



# 75 "laplacian_fuse.cpp"

#ifndef __HIPCC__
void hmpp_acc_region_main_80_internal_1(hmpprt::s32 height_20, hmpprt::s32 width_21, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s32>  pSrc_padding_5, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  pBufL_cp_8)
{
 # 75 "laplacian_fuse.cpp"
 if (1)
 {
  hmpprt::CUDAGridCall __hmppcg_call;
  __hmppcg_call.setSizeX(192);
  __hmppcg_call.setSizeY(1);
  __hmppcg_call.setBlockSizeX(1);
  __hmppcg_call.setBlockSizeY(256);
  __hmppcg_call.addLocalParameter((hmpprt::s32) (height_20), "height_2");
  __hmppcg_call.addLocalParameter(&pBufL_cp_8, 8, "pBufL_cp_2");
  __hmppcg_call.addLocalParameter(&pSrc_padding_5, 8, "pSrc_padding_1");
  __hmppcg_call.addLocalParameter((hmpprt::s32) (width_21), "width_2");
  __hmppcg_call.launch(hmpp_acc_region_main_80_parallel_region_1, hmpprt::Context::getInstance()->getCUDADevice());
 }
 ;
}
#endif // __HIPCC__



# 75 "laplacian_fuse.cpp"

#ifndef __HIPCC__
extern "C" CDLT_API  void hmpp_acc_region_main_80(hmpprt::s32 height_9, hmpprt::s32 width_12, hmpprt::s32* pSrc_padding_3, hmpprt::u08* pBufL_cp_5)
{
 # 98 "laplacian_fuse.cpp"
 (hmpp_acc_region_main_80_internal_1(height_9, width_12, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s32> (pSrc_padding_3), hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08> (pBufL_cp_5)));
}
#endif // __HIPCC__



# 98 "laplacian_fuse.cpp"

#ifdef __HIPCC__

extern "C" __global__ void hmpp_acc_region_main_94_parallel_region_1(hmpprt::s32 height_1, hmpprt::u08* pBufL_cp_1, hmpprt::s32* pSrc_padding2_1, hmpprt::s32 width_1)
{
 # 100 "laplacian_fuse.cpp"
 {
  # 103 "laplacian_fuse.cpp"
  hmpprt::s32 iter_per_gang_1;
  # 103 "laplacian_fuse.cpp"
  hmpprt::s32 first_gang_iter_1;
  # 103 "laplacian_fuse.cpp"
  hmpprt::s32 last_gang_iter_1;
  # 103 "laplacian_fuse.cpp"
  iter_per_gang_1 = ((1 + (height_1 * width_1 - 1) / 192) > 256 ? (1 + (height_1 * width_1 - 1) / 192) : 256);
  # 103 "laplacian_fuse.cpp"
  first_gang_iter_1 = (hmpprt::gr_gbidx()) * iter_per_gang_1;
  # 103 "laplacian_fuse.cpp"
  last_gang_iter_1 = ((first_gang_iter_1 + iter_per_gang_1 - 1) < (height_1 * width_1 - 1) ? (first_gang_iter_1 + iter_per_gang_1 - 1) : (height_1 * width_1 - 1));
  # 103 "laplacian_fuse.cpp"
  hmpprt::s32 h_9;
  # 103 "laplacian_fuse.cpp"
  # 107 "laplacian_fuse.cpp"
  for (h_9 = first_gang_iter_1 + (hmpprt::gr_btidy()) ; h_9 <= last_gang_iter_1 ; h_9 = h_9 + (hmpprt::gr_btnumy()))
  {
   # 103 "laplacian_fuse.cpp"
   hmpprt::s32 h_7;
   # 106 "laplacian_fuse.cpp"
   hmpprt::s32 w_9;
   # 107 "laplacian_fuse.cpp"
   w_9 = h_9 % width_1;
   # 107 "laplacian_fuse.cpp"
   h_7 = h_9 / width_1;
   # 107 "laplacian_fuse.cpp"
   *(pSrc_padding2_1 + ((h_7 + 2) * (width_1 + 4) + w_9 + 2)) = (hmpprt::s32 ) (*(pBufL_cp_1 + (h_7 * width_1 + w_9)));
  }
  # 98 "laplacian_fuse.cpp"
 }
}
#endif // __HIPCC__



# 98 "laplacian_fuse.cpp"

#ifndef __HIPCC__
void hmpp_acc_region_main_94_internal_1(hmpprt::s32 height_19, hmpprt::s32 width_20, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s32>  pSrc_padding2_7, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  pBufL_cp_7)
{
 # 98 "laplacian_fuse.cpp"
 if (1)
 {
  hmpprt::CUDAGridCall __hmppcg_call;
  __hmppcg_call.setSizeX(192);
  __hmppcg_call.setSizeY(1);
  __hmppcg_call.setBlockSizeX(1);
  __hmppcg_call.setBlockSizeY(256);
  __hmppcg_call.addLocalParameter((hmpprt::s32) (height_19), "height_1");
  __hmppcg_call.addLocalParameter(&pBufL_cp_7, 8, "pBufL_cp_1");
  __hmppcg_call.addLocalParameter(&pSrc_padding2_7, 8, "pSrc_padding2_1");
  __hmppcg_call.addLocalParameter((hmpprt::s32) (width_20), "width_1");
  __hmppcg_call.launch(hmpp_acc_region_main_94_parallel_region_1, hmpprt::Context::getInstance()->getCUDADevice());
 }
 ;
}
#endif // __HIPCC__



# 98 "laplacian_fuse.cpp"

#ifndef __HIPCC__
extern "C" CDLT_API  void hmpp_acc_region_main_94(hmpprt::s32 height_8, hmpprt::s32 width_11, hmpprt::s32* pSrc_padding2_4, hmpprt::u08* pBufL_cp_4)
{
 # 1 "<preprocessor>"
 (hmpp_acc_region_main_94_internal_1(height_8, width_11, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s32> (pSrc_padding2_4), hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08> (pBufL_cp_4)));
}
#endif // __HIPCC__




#ifndef __HIPCC__
extern "C" const char * hmpprt_cuda_get_gpu_code();

static hmpprt::CUDAModule * hmpprt_module = 0;
static int hmpprt_uses = 0;

extern "C" CDLT_API void * hmpprt_init()
{
  try
  {
    if (hmpprt_uses++ == 0)
    {
      hmpprt_module = new hmpprt::CUDAModule(hmpprt_cuda_get_gpu_code());
      hmpp_acc_region_main_108_parallel_region_1 = new hmpprt::CUDAGrid(hmpprt_module, "hmpp_acc_region_main_108_parallel_region_1");
      hmpp_acc_region_main_130_parallel_region_1 = new hmpprt::CUDAGrid(hmpprt_module, "hmpp_acc_region_main_130_parallel_region_1");
      hmpp_acc_region_main_153_parallel_region_1 = new hmpprt::CUDAGrid(hmpprt_module, "hmpp_acc_region_main_153_parallel_region_1");
      hmpp_acc_region_main_183_parallel_region_1 = new hmpprt::CUDAGrid(hmpprt_module, "hmpp_acc_region_main_183_parallel_region_1");
      hmpp_acc_region_main_192_parallel_region_1 = new hmpprt::CUDAGrid(hmpprt_module, "hmpp_acc_region_main_192_parallel_region_1");
      hmpp_acc_region_main_206_parallel_region_1 = new hmpprt::CUDAGrid(hmpprt_module, "hmpp_acc_region_main_206_parallel_region_1");
      hmpp_acc_region_main_239_parallel_region_1 = new hmpprt::CUDAGrid(hmpprt_module, "hmpp_acc_region_main_239_parallel_region_1");
      hmpp_acc_region_main_55_parallel_region_1 = new hmpprt::CUDAGrid(hmpprt_module, "hmpp_acc_region_main_55_parallel_region_1");
      hmpp_acc_region_main_80_parallel_region_1 = new hmpprt::CUDAGrid(hmpprt_module, "hmpp_acc_region_main_80_parallel_region_1");
      hmpp_acc_region_main_94_parallel_region_1 = new hmpprt::CUDAGrid(hmpprt_module, "hmpp_acc_region_main_94_parallel_region_1");

    }
    hmpprt::Context::getInstance()->getGrouplet()->setTarget(hmpprt::CUDA);
    hmpprt::Context::getInstance()->getGrouplet()->addSignature("hmpp_acc_region_main_108", "prototype hmpp_acc_region_main_108(height: s32, width: s32, pSrc_padding2: ^cudaglob s32, pBufL_cp: ^cudaglob u8)");
    hmpprt::Context::getInstance()->getGrouplet()->addSignature("hmpp_acc_region_main_130", "prototype hmpp_acc_region_main_130(height: s32, width: s32, FilterDst: ^cudaglob u8, pSrc_padding2: ^cudaglob s32)");
    hmpprt::Context::getInstance()->getGrouplet()->addSignature("hmpp_acc_region_main_153", "prototype hmpp_acc_region_main_153(width: s32, halfWidth: s32, halfHeight: s32, FilterDst: ^cudaglob u8, DownsampleDst: ^cudaglob u8)");
    hmpprt::Context::getInstance()->getGrouplet()->addSignature("hmpp_acc_region_main_183", "prototype hmpp_acc_region_main_183(height: s32, width: s32, halfWidth: s32, halfHeight: s32, DownsampleDst: ^cudaglob u8, UpsampleDst: ^cudaglob u8)");
    hmpprt::Context::getInstance()->getGrouplet()->addSignature("hmpp_acc_region_main_192", "prototype hmpp_acc_region_main_192(width: s32, halfWidth: s32, DownsampleDst: ^cudaglob u8, UpsampleDst: ^cudaglob u8)");
    hmpprt::Context::getInstance()->getGrouplet()->addSignature("hmpp_acc_region_main_206", "prototype hmpp_acc_region_main_206(width: s32, halfWidth: s32, halfHeight: s32, DownsampleDst: ^cudaglob u8, UpsampleDst: ^cudaglob u8)");
    hmpprt::Context::getInstance()->getGrouplet()->addSignature("hmpp_acc_region_main_239", "prototype hmpp_acc_region_main_239(height: s32, width: s32, Source: ^cudaglob u8, LaplacianLayer: ^cudaglob s16, UpsampleDst: ^cudaglob u8)");
    hmpprt::Context::getInstance()->getGrouplet()->addSignature("hmpp_acc_region_main_55", "prototype hmpp_acc_region_main_55(height: s32, width: s32, Source: ^cudaglob u8, pSrc_padding: ^cudaglob s32)");
    hmpprt::Context::getInstance()->getGrouplet()->addSignature("hmpp_acc_region_main_80", "prototype hmpp_acc_region_main_80(height: s32, width: s32, pSrc_padding: ^cudaglob s32, pBufL_cp: ^cudaglob u8)");
    hmpprt::Context::getInstance()->getGrouplet()->addSignature("hmpp_acc_region_main_94", "prototype hmpp_acc_region_main_94(height: s32, width: s32, pSrc_padding2: ^cudaglob s32, pBufL_cp: ^cudaglob u8)");

  }
  catch (hmpperr::Error & e)
  {
    return e.clone();
  }
  catch(...)
  {
    fprintf(stderr,"Unexpected error in hmpprt_init()\n");
    abort();
  }
  return 0;
}
#endif // __HIPCC__

#ifndef __HIPCC__
extern "C" CDLT_API void * hmpprt_fini()
{
  try
  {
    if (--hmpprt_uses == 0)
    {
      delete hmpp_acc_region_main_108_parallel_region_1;
      delete hmpp_acc_region_main_130_parallel_region_1;
      delete hmpp_acc_region_main_153_parallel_region_1;
      delete hmpp_acc_region_main_183_parallel_region_1;
      delete hmpp_acc_region_main_192_parallel_region_1;
      delete hmpp_acc_region_main_206_parallel_region_1;
      delete hmpp_acc_region_main_239_parallel_region_1;
      delete hmpp_acc_region_main_55_parallel_region_1;
      delete hmpp_acc_region_main_80_parallel_region_1;
      delete hmpp_acc_region_main_94_parallel_region_1;

      delete hmpprt_module;
      hmpprt_module = 0;
    }
  }
  catch (hmpperr::Error & e)
  {
    return e.clone();
  }
  catch(...)
  {
    fprintf(stderr,"Unexpected error in hmpprt_fini()\n");
    abort();
  }
  return 0;
}
#endif // __HIPCC__

// footer
