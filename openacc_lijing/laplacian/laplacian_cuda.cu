#include "hip/hip_runtime.h"

#include <stdio.h>

#ifndef __HIPCC__
#include <stdlib.h>
#include <math.h>

#include <hmpprt/Grouplet.h>
#include <hmpprt/HostTypes.h>
#include <hmpprt/Context.h>
#include <hmpprt/CUDAGrid.h>
#include <hmpprt/CUDAModule.h>
#include <hmpprt/DeviceManager.h>
#include <hmpperr/hmpperr.h>

#include <openacci/openacci_c.h>

#ifdef _WIN32
#  define CDLT_API __declspec(dllexport)
#else /* ! _WIN32 */
#  define CDLT_API
#endif /* _WIN32 */



#else // ! __HIPCC__

#include <hmpprt/HostTypes.h>
#include <hmpprt/CUDAIntrinsics.h>

extern __shared__ int64_t hmpp_sharedmem[];
#endif // __HIPCC__



#ifndef __HIPCC__

#else


#endif

#define HMPPCG_SIMD_LENGTH 32

# 75 "laplacian.cpp"

#ifndef __HIPCC__
extern "C" CDLT_API  void hmpp_acc_region_main_79(hmpprt::s32 height_8, hmpprt::s32 width_11, hmpprt::s32* pSrc_padding_3, hmpprt::u08* pBufL_cp_4)
;
#endif // __HIPCC__



# 75 "laplacian.cpp"

#ifndef __HIPCC__
void hmpp_acc_region_main_79_internal_1(hmpprt::s32 height_19, hmpprt::s32 width_20, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s32>  pSrc_padding_5, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  pBufL_cp_8)
;
#endif // __HIPCC__



# 12 "laplacian.cpp"

#ifndef __HIPCC__
static hmpprt::CUDAGrid * hmpp_acc_region_main_79_parallel_region_1 = 0;
#else

extern "C" __global__ void hmpp_acc_region_main_79_parallel_region_1(hmpprt::s32 height_1, hmpprt::u08* pBufL_cp_1, hmpprt::s32* pSrc_padding_1, hmpprt::s32 width_1);
#endif // __HIPCC__




# 12 "laplacian.cpp"

#ifndef __HIPCC__
extern "C" CDLT_API  void hmpp_acc_region_main_55(hmpprt::s32 height_9, hmpprt::s32 width_12, hmpprt::u08* Source_3, hmpprt::s32* pSrc_padding_2)
;
#endif // __HIPCC__



# 12 "laplacian.cpp"

#ifndef __HIPCC__
void hmpp_acc_region_main_55_internal_1(hmpprt::s32 height_20, hmpprt::s32 width_21, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  Source_5, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s32>  pSrc_padding_4)
;
#endif // __HIPCC__



# 243 "laplacian.cpp"

#ifndef __HIPCC__
static hmpprt::CUDAGrid * hmpp_acc_region_main_55_parallel_region_1 = 0;
#else

extern "C" __global__ void hmpp_acc_region_main_55_parallel_region_1(hmpprt::u08* Source_1, hmpprt::s32 height_2, hmpprt::s32* pSrc_padding, hmpprt::s32 width_2);
#endif // __HIPCC__




# 243 "laplacian.cpp"

#ifndef __HIPCC__
extern "C" CDLT_API  void hmpp_acc_region_main_247(hmpprt::s32 height_10, hmpprt::s32 width_13, hmpprt::u08* Source_2, hmpprt::s16* LaplacianLayer_1, hmpprt::u08* UpsampleDst_5)
;
#endif // __HIPCC__



# 243 "laplacian.cpp"

#ifndef __HIPCC__
void hmpp_acc_region_main_247_internal_1(hmpprt::s32 height_15, hmpprt::s32 width_22, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  Source_4, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s16>  LaplacianLayer_2, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  UpsampleDst_9)
;
#endif // __HIPCC__



# 229 "laplacian.cpp"

#ifndef __HIPCC__
static hmpprt::CUDAGrid * hmpp_acc_region_main_247_parallel_region_1 = 0;
#else

extern "C" __global__ void hmpp_acc_region_main_247_parallel_region_1(hmpprt::s16* LaplacianLayer, hmpprt::u08* Source, hmpprt::u08* UpsampleDst_1, hmpprt::s32 height_3, hmpprt::s32 width_3);
#endif // __HIPCC__




# 229 "laplacian.cpp"

#ifndef __HIPCC__
extern "C" CDLT_API  void hmpp_acc_region_main_230(hmpprt::s32 width_4, hmpprt::s32 halfWidth_5, hmpprt::u08* DownsampleDst_5, hmpprt::u08* UpsampleDst_6)
;
#endif // __HIPCC__



# 229 "laplacian.cpp"

#ifndef __HIPCC__
void hmpp_acc_region_main_230_internal_1(hmpprt::s32 width_27, hmpprt::s32 halfWidth_9, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  DownsampleDst_9, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  UpsampleDst_10)
;
#endif // __HIPCC__



# 201 "laplacian.cpp"

#ifndef __HIPCC__
static hmpprt::CUDAGrid * hmpp_acc_region_main_230_parallel_region_1 = 0;
#else

extern "C" __global__ void hmpp_acc_region_main_230_parallel_region_1(hmpprt::u08* DownsampleDst_1, hmpprt::u08* UpsampleDst_2, hmpprt::s32 halfWidth_1);
#endif // __HIPCC__




# 201 "laplacian.cpp"

#ifndef __HIPCC__
extern "C" CDLT_API  void hmpp_acc_region_main_202(hmpprt::s32 width_14, hmpprt::s32 halfWidth_10, hmpprt::s32 halfHeight_8, hmpprt::u08* DownsampleDst_10, hmpprt::u08* UpsampleDst_11)
;
#endif // __HIPCC__



# 201 "laplacian.cpp"

#ifndef __HIPCC__
void hmpp_acc_region_main_202_internal_1(hmpprt::s32 width_23, hmpprt::s32 halfWidth_6, hmpprt::s32 halfHeight_4, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  DownsampleDst_6, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  UpsampleDst_7)
;
#endif // __HIPCC__



# 175 "laplacian.cpp"

#ifndef __HIPCC__
static hmpprt::CUDAGrid * hmpp_acc_region_main_202_parallel_region_1 = 0;
#else

extern "C" __global__ void hmpp_acc_region_main_202_parallel_region_1(hmpprt::u08* DownsampleDst_2, hmpprt::u08* UpsampleDst_3, hmpprt::s32 halfHeight_1, hmpprt::s32 halfWidth_2, hmpprt::s32 width_5);
#endif // __HIPCC__




# 175 "laplacian.cpp"

#ifndef __HIPCC__
extern "C" CDLT_API  void hmpp_acc_region_main_188(hmpprt::s32 height_11, hmpprt::s32 width_15, hmpprt::s32 halfWidth_7, hmpprt::s32 halfHeight_2, hmpprt::u08* DownsampleDst_11, hmpprt::u08* UpsampleDst_4)
;
#endif // __HIPCC__



# 175 "laplacian.cpp"

#ifndef __HIPCC__
void hmpp_acc_region_main_188_internal_1(hmpprt::s32 height_16, hmpprt::s32 width_24, hmpprt::s32 halfWidth_11, hmpprt::s32 halfHeight_6, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  DownsampleDst_7, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  UpsampleDst_8)
;
#endif // __HIPCC__



# 154 "laplacian.cpp"

#ifndef __HIPCC__
static hmpprt::CUDAGrid * hmpp_acc_region_main_188_parallel_region_1 = 0;
#else

extern "C" __global__ void hmpp_acc_region_main_188_parallel_region_1(hmpprt::u08* DownsampleDst_3, hmpprt::u08* UpsampleDst, hmpprt::s32 halfHeight_5, hmpprt::s32 halfWidth_3, hmpprt::s32 height_4, hmpprt::s32 width_6);
#endif // __HIPCC__




# 154 "laplacian.cpp"

#ifndef __HIPCC__
extern "C" CDLT_API  void hmpp_acc_region_main_159(hmpprt::s32 width_16, hmpprt::s32 halfWidth_4, hmpprt::s32 halfHeight_3, hmpprt::u08* FilterDst_3, hmpprt::u08* DownsampleDst_4)
;
#endif // __HIPCC__



# 154 "laplacian.cpp"

#ifndef __HIPCC__
void hmpp_acc_region_main_159_internal_1(hmpprt::s32 width_25, hmpprt::s32 halfWidth_8, hmpprt::s32 halfHeight_7, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  FilterDst_5, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  DownsampleDst_8)
;
#endif // __HIPCC__



# 132 "laplacian.cpp"

#ifndef __HIPCC__
static hmpprt::CUDAGrid * hmpp_acc_region_main_159_parallel_region_1 = 0;
#else

extern "C" __global__ void hmpp_acc_region_main_159_parallel_region_1(hmpprt::u08* DownsampleDst, hmpprt::u08* FilterDst_1, hmpprt::s32 halfHeight, hmpprt::s32 halfWidth, hmpprt::s32 width_7);
#endif // __HIPCC__




# 132 "laplacian.cpp"

#ifndef __HIPCC__
extern "C" CDLT_API  void hmpp_acc_region_main_136(hmpprt::s32 height_5, hmpprt::s32 width_8, hmpprt::u08* FilterDst_2, hmpprt::s32* pSrc_padding2_6)
;
#endif // __HIPCC__



# 132 "laplacian.cpp"

#ifndef __HIPCC__
void hmpp_acc_region_main_136_internal_1(hmpprt::s32 height_18, hmpprt::s32 width_28, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  FilterDst_4, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s32>  pSrc_padding2_1)
;
#endif // __HIPCC__



# 114 "laplacian.cpp"

#ifndef __HIPCC__
static hmpprt::CUDAGrid * hmpp_acc_region_main_136_parallel_region_1 = 0;
#else

extern "C" __global__ void hmpp_acc_region_main_136_parallel_region_1(hmpprt::u08* FilterDst, hmpprt::s32 height_12, hmpprt::s32* pSrc_padding2_4, hmpprt::s32 width_17);
#endif // __HIPCC__




# 114 "laplacian.cpp"

#ifndef __HIPCC__
extern "C" CDLT_API  void hmpp_acc_region_main_114(hmpprt::s32 height_17, hmpprt::s32 width_18, hmpprt::s32* pSrc_padding2_7, hmpprt::u08* pBufL_cp_2)
;
#endif // __HIPCC__



# 114 "laplacian.cpp"

#ifndef __HIPCC__
void hmpp_acc_region_main_114_internal_1(hmpprt::s32 height_13, hmpprt::s32 width_26, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s32>  pSrc_padding2_2, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  pBufL_cp_6)
;
#endif // __HIPCC__



# 97 "laplacian.cpp"

#ifndef __HIPCC__
static hmpprt::CUDAGrid * hmpp_acc_region_main_114_parallel_region_1 = 0;
#else

extern "C" __global__ void hmpp_acc_region_main_114_parallel_region_1(hmpprt::s32 height_6, hmpprt::u08* pBufL_cp_5, hmpprt::s32* pSrc_padding2_5, hmpprt::s32 width_9);
#endif // __HIPCC__




# 97 "laplacian.cpp"

#ifndef __HIPCC__
extern "C" CDLT_API  void hmpp_acc_region_main_100(hmpprt::s32 height_14, hmpprt::s32 width_10, hmpprt::s32* pSrc_padding2_3, hmpprt::u08* pBufL_cp_7)
;
#endif // __HIPCC__



# 97 "laplacian.cpp"

#ifndef __HIPCC__
void hmpp_acc_region_main_100_internal_1(hmpprt::s32 height, hmpprt::s32 width_19, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s32>  pSrc_padding2_8, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  pBufL_cp_3)
;
#endif // __HIPCC__



# 97 "laplacian.cpp"

#ifndef __HIPCC__
static hmpprt::CUDAGrid * hmpp_acc_region_main_100_parallel_region_1 = 0;
#else

extern "C" __global__ void hmpp_acc_region_main_100_parallel_region_1(hmpprt::s32 height_7, hmpprt::u08* pBufL_cp, hmpprt::s32* pSrc_padding2, hmpprt::s32 width);
#endif // __HIPCC__




# 97 "laplacian.cpp"

#ifdef __HIPCC__

extern "C" __global__ void hmpp_acc_region_main_100_parallel_region_1(hmpprt::s32 height_7, hmpprt::u08* pBufL_cp, hmpprt::s32* pSrc_padding2, hmpprt::s32 width)
{
 # 99 "laplacian.cpp"
 {
  # 103 "laplacian.cpp"
  hmpprt::s32 iter_per_gang_10;
  # 103 "laplacian.cpp"
  hmpprt::s32 first_gang_iter_10;
  # 103 "laplacian.cpp"
  hmpprt::s32 last_gang_iter_10;
  # 103 "laplacian.cpp"
  iter_per_gang_10 = ((1 + (height_7 * width - 1) / 192) > 256 ? (1 + (height_7 * width - 1) / 192) : 256);
  # 103 "laplacian.cpp"
  first_gang_iter_10 = (hmpprt::gr_gbidx()) * iter_per_gang_10;
  # 103 "laplacian.cpp"
  last_gang_iter_10 = ((first_gang_iter_10 + iter_per_gang_10 - 1) < (height_7 * width - 1) ? (first_gang_iter_10 + iter_per_gang_10 - 1) : (height_7 * width - 1));
  # 103 "laplacian.cpp"
  hmpprt::s32 h_9;
  # 103 "laplacian.cpp"
  # 107 "laplacian.cpp"
  for (h_9 = first_gang_iter_10 + (hmpprt::gr_btidy()) ; h_9 <= last_gang_iter_10 ; h_9 = h_9 + (hmpprt::gr_btnumy()))
  {
   # 103 "laplacian.cpp"
   hmpprt::s32 h_4;
   # 106 "laplacian.cpp"
   hmpprt::s32 w_5;
   # 107 "laplacian.cpp"
   w_5 = h_9 % width;
   # 107 "laplacian.cpp"
   h_4 = h_9 / width;
   # 107 "laplacian.cpp"
   *(pSrc_padding2 + ((h_4 + 2) * (width + 4) + w_5 + 2)) = (hmpprt::s32 ) (*(pBufL_cp + (h_4 * width + w_5)));
  }
  # 97 "laplacian.cpp"
 }
}
#endif // __HIPCC__



# 97 "laplacian.cpp"

#ifndef __HIPCC__
void hmpp_acc_region_main_100_internal_1(hmpprt::s32 height, hmpprt::s32 width_19, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s32>  pSrc_padding2_8, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  pBufL_cp_3)
{
 # 97 "laplacian.cpp"
 if (1)
 {
  hmpprt::CUDAGridCall __hmppcg_call;
  __hmppcg_call.setSizeX(192);
  __hmppcg_call.setSizeY(1);
  __hmppcg_call.setBlockSizeX(1);
  __hmppcg_call.setBlockSizeY(256);
  __hmppcg_call.addLocalParameter((hmpprt::s32) (height), "height_7");
  __hmppcg_call.addLocalParameter(&pBufL_cp_3, 8, "pBufL_cp");
  __hmppcg_call.addLocalParameter(&pSrc_padding2_8, 8, "pSrc_padding2");
  __hmppcg_call.addLocalParameter((hmpprt::s32) (width_19), "width");
  __hmppcg_call.launch(hmpp_acc_region_main_100_parallel_region_1, hmpprt::Context::getInstance()->getCUDADevice());
 }
 ;
}
#endif // __HIPCC__



# 97 "laplacian.cpp"

#ifndef __HIPCC__
extern "C" CDLT_API  void hmpp_acc_region_main_100(hmpprt::s32 height_14, hmpprt::s32 width_10, hmpprt::s32* pSrc_padding2_3, hmpprt::u08* pBufL_cp_7)
{
 # 114 "laplacian.cpp"
 (hmpp_acc_region_main_100_internal_1(height_14, width_10, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s32> (pSrc_padding2_3), hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08> (pBufL_cp_7)));
}
#endif // __HIPCC__



# 114 "laplacian.cpp"

#ifdef __HIPCC__

extern "C" __global__ void hmpp_acc_region_main_114_parallel_region_1(hmpprt::s32 height_6, hmpprt::u08* pBufL_cp_5, hmpprt::s32* pSrc_padding2_5, hmpprt::s32 width_9)
{
 # 116 "laplacian.cpp"
 {
  # 117 "laplacian.cpp"
  hmpprt::s32 iter_per_gang_9;
  # 117 "laplacian.cpp"
  hmpprt::s32 first_gang_iter_9;
  # 117 "laplacian.cpp"
  hmpprt::s32 last_gang_iter_9;
  # 117 "laplacian.cpp"
  iter_per_gang_9 = ((1 + (width_9 - 1) / 192) > 256 ? (1 + (width_9 - 1) / 192) : 256);
  # 117 "laplacian.cpp"
  first_gang_iter_9 = (hmpprt::gr_gbidx()) * iter_per_gang_9;
  # 117 "laplacian.cpp"
  last_gang_iter_9 = ((first_gang_iter_9 + iter_per_gang_9 - 1) < (width_9 - 1) ? (first_gang_iter_9 + iter_per_gang_9 - 1) : (width_9 - 1));
  # 117 "laplacian.cpp"
  hmpprt::s32 w_6;
  # 117 "laplacian.cpp"
  # 118 "laplacian.cpp"
  for (w_6 = first_gang_iter_9 + (hmpprt::gr_btidy()) ; w_6 <= last_gang_iter_9 ; w_6 = w_6 + (hmpprt::gr_btnumy()))
  {
   # 119 "laplacian.cpp"
   *(pSrc_padding2_5 + (width_9 + 4 + (w_6 + 2))) = 2 * (hmpprt::s32 ) (*(pBufL_cp_5 + (width_9 + (w_6 + 2) - 2))) - (hmpprt::s32 ) (*(pBufL_cp_5 + (3 * width_9 + (w_6 + 2) - 2)));
   # 120 "laplacian.cpp"
   *(pSrc_padding2_5 + (w_6 + 2)) = 4 * (hmpprt::s32 ) (*(pBufL_cp_5 + (3 * width_9 + (w_6 + 2) - 2))) - 4 * (hmpprt::s32 ) (*(pBufL_cp_5 + (width_9 + (w_6 + 2) - 2))) + 2 * (hmpprt::s32 ) (*(pBufL_cp_5 + w_6)) - (hmpprt::s32 ) (*(pBufL_cp_5 + (2 * width_9 + (w_6 + 2) - 2)));
   # 123 "laplacian.cpp"
   *(pSrc_padding2_5 + ((height_6 + 2) * (width_9 + 4) + (w_6 + 2))) = 2 * (hmpprt::s32 ) (*(pBufL_cp_5 + ((height_6 - 2) * width_9 + (w_6 + 2) - 2))) - (hmpprt::s32 ) (*(pBufL_cp_5 + ((height_6 - 4) * width_9 + (w_6 + 2) - 2)));
   # 124 "laplacian.cpp"
   *(pSrc_padding2_5 + ((height_6 + 3) * (width_9 + 4) + (w_6 + 2))) = 4 * (hmpprt::s32 ) (*(pBufL_cp_5 + ((height_6 - 4) * width_9 + (w_6 + 2) - 2))) - 4 * (hmpprt::s32 ) (*(pBufL_cp_5 + ((height_6 - 2) * width_9 + (w_6 + 2) - 2))) + 2 * (hmpprt::s32 ) (*(pBufL_cp_5 + ((height_6 - 1) * width_9 + (w_6 + 2) - 2))) - (hmpprt::s32 ) (*(pBufL_cp_5 + ((height_6 - 3) * width_9 + (w_6 + 2) - 2)));
  }
  # 114 "laplacian.cpp"
 }
}
#endif // __HIPCC__



# 114 "laplacian.cpp"

#ifndef __HIPCC__
void hmpp_acc_region_main_114_internal_1(hmpprt::s32 height_13, hmpprt::s32 width_26, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s32>  pSrc_padding2_2, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  pBufL_cp_6)
{
 # 114 "laplacian.cpp"
 if (1)
 {
  hmpprt::CUDAGridCall __hmppcg_call;
  __hmppcg_call.setSizeX(192);
  __hmppcg_call.setSizeY(1);
  __hmppcg_call.setBlockSizeX(1);
  __hmppcg_call.setBlockSizeY(256);
  __hmppcg_call.addLocalParameter((hmpprt::s32) (height_13), "height_6");
  __hmppcg_call.addLocalParameter(&pBufL_cp_6, 8, "pBufL_cp_5");
  __hmppcg_call.addLocalParameter(&pSrc_padding2_2, 8, "pSrc_padding2_5");
  __hmppcg_call.addLocalParameter((hmpprt::s32) (width_26), "width_9");
  __hmppcg_call.launch(hmpp_acc_region_main_114_parallel_region_1, hmpprt::Context::getInstance()->getCUDADevice());
 }
 ;
}
#endif // __HIPCC__



# 114 "laplacian.cpp"

#ifndef __HIPCC__
extern "C" CDLT_API  void hmpp_acc_region_main_114(hmpprt::s32 height_17, hmpprt::s32 width_18, hmpprt::s32* pSrc_padding2_7, hmpprt::u08* pBufL_cp_2)
{
 # 132 "laplacian.cpp"
 (hmpp_acc_region_main_114_internal_1(height_17, width_18, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s32> (pSrc_padding2_7), hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08> (pBufL_cp_2)));
}
#endif // __HIPCC__



# 132 "laplacian.cpp"

#ifdef __HIPCC__

extern "C" __global__ void hmpp_acc_region_main_136_parallel_region_1(hmpprt::u08* FilterDst, hmpprt::s32 height_12, hmpprt::s32* pSrc_padding2_4, hmpprt::s32 width_17)
{
 # 134 "laplacian.cpp"
 {
  # 139 "laplacian.cpp"
  hmpprt::s32 iter_per_gang_8;
  # 139 "laplacian.cpp"
  hmpprt::s32 first_gang_iter_8;
  # 139 "laplacian.cpp"
  hmpprt::s32 last_gang_iter_8;
  # 139 "laplacian.cpp"
  iter_per_gang_8 = ((1 + (width_17 * height_12 - 1) / 192) > 256 ? (1 + (width_17 * height_12 - 1) / 192) : 256);
  # 139 "laplacian.cpp"
  first_gang_iter_8 = (hmpprt::gr_gbidx()) * iter_per_gang_8;
  # 139 "laplacian.cpp"
  last_gang_iter_8 = ((first_gang_iter_8 + iter_per_gang_8 - 1) < (width_17 * height_12 - 1) ? (first_gang_iter_8 + iter_per_gang_8 - 1) : (width_17 * height_12 - 1));
  # 139 "laplacian.cpp"
  hmpprt::s32 w_10;
  # 139 "laplacian.cpp"
  # 142 "laplacian.cpp"
  for (w_10 = first_gang_iter_8 + (hmpprt::gr_btidy()) ; w_10 <= last_gang_iter_8 ; w_10 = w_10 + (hmpprt::gr_btnumy()))
  {
   # 139 "laplacian.cpp"
   hmpprt::s32 w_7;
   # 141 "laplacian.cpp"
   hmpprt::s32 h_5;
   # 143 "laplacian.cpp"
   h_5 = w_10 % height_12;
   # 143 "laplacian.cpp"
   w_7 = w_10 / height_12;
   # 143 "laplacian.cpp"
   *(FilterDst + (h_5 * width_17 + w_7)) = (hmpprt::u08 ) (*(pSrc_padding2_4 + (h_5 * (width_17 + 4) + (w_7 + 2))) + (*(pSrc_padding2_4 + ((h_5 + 1) * (width_17 + 4) + (w_7 + 2))) << 2) + 6 * *(pSrc_padding2_4 + ((h_5 + 2) * (width_17 + 4) + (w_7 + 2))) + (*(pSrc_padding2_4 + ((h_5 + 3) * (width_17 + 4) + (w_7 + 2))) << 2) + *(pSrc_padding2_4 + ((h_5 + 4) * (width_17 + 4) + (w_7 + 2))) + 8 >> 4);
  }
  # 132 "laplacian.cpp"
 }
}
#endif // __HIPCC__



# 132 "laplacian.cpp"

#ifndef __HIPCC__
void hmpp_acc_region_main_136_internal_1(hmpprt::s32 height_18, hmpprt::s32 width_28, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  FilterDst_4, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s32>  pSrc_padding2_1)
{
 # 132 "laplacian.cpp"
 if (1)
 {
  hmpprt::CUDAGridCall __hmppcg_call;
  __hmppcg_call.setSizeX(192);
  __hmppcg_call.setSizeY(1);
  __hmppcg_call.setBlockSizeX(1);
  __hmppcg_call.setBlockSizeY(256);
  __hmppcg_call.addLocalParameter(&FilterDst_4, 8, "FilterDst");
  __hmppcg_call.addLocalParameter((hmpprt::s32) (height_18), "height_12");
  __hmppcg_call.addLocalParameter(&pSrc_padding2_1, 8, "pSrc_padding2_4");
  __hmppcg_call.addLocalParameter((hmpprt::s32) (width_28), "width_17");
  __hmppcg_call.launch(hmpp_acc_region_main_136_parallel_region_1, hmpprt::Context::getInstance()->getCUDADevice());
 }
 ;
}
#endif // __HIPCC__



# 132 "laplacian.cpp"

#ifndef __HIPCC__
extern "C" CDLT_API  void hmpp_acc_region_main_136(hmpprt::s32 height_5, hmpprt::s32 width_8, hmpprt::u08* FilterDst_2, hmpprt::s32* pSrc_padding2_6)
{
 # 154 "laplacian.cpp"
 (hmpp_acc_region_main_136_internal_1(height_5, width_8, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08> (FilterDst_2), hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s32> (pSrc_padding2_6)));
}
#endif // __HIPCC__



# 154 "laplacian.cpp"

#ifdef __HIPCC__

extern "C" __global__ void hmpp_acc_region_main_159_parallel_region_1(hmpprt::u08* DownsampleDst, hmpprt::u08* FilterDst_1, hmpprt::s32 halfHeight, hmpprt::s32 halfWidth, hmpprt::s32 width_7)
{
 # 156 "laplacian.cpp"
 {
  # 162 "laplacian.cpp"
  hmpprt::s32 iter_per_gang_7;
  # 162 "laplacian.cpp"
  hmpprt::s32 first_gang_iter_7;
  # 162 "laplacian.cpp"
  hmpprt::s32 last_gang_iter_7;
  # 162 "laplacian.cpp"
  iter_per_gang_7 = ((1 + (halfHeight * halfWidth - 1) / 192) > 256 ? (1 + (halfHeight * halfWidth - 1) / 192) : 256);
  # 162 "laplacian.cpp"
  first_gang_iter_7 = (hmpprt::gr_gbidx()) * iter_per_gang_7;
  # 162 "laplacian.cpp"
  last_gang_iter_7 = ((first_gang_iter_7 + iter_per_gang_7 - 1) < (halfHeight * halfWidth - 1) ? (first_gang_iter_7 + iter_per_gang_7 - 1) : (halfHeight * halfWidth - 1));
  # 162 "laplacian.cpp"
  hmpprt::s32 y_7;
  # 162 "laplacian.cpp"
  # 166 "laplacian.cpp"
  for (y_7 = first_gang_iter_7 + (hmpprt::gr_btidy()) ; y_7 <= last_gang_iter_7 ; y_7 = y_7 + (hmpprt::gr_btnumy()))
  {
   # 162 "laplacian.cpp"
   hmpprt::s32 y_3;
   # 165 "laplacian.cpp"
   hmpprt::s32 x_5;
   # 167 "laplacian.cpp"
   x_5 = y_7 % halfWidth;
   # 167 "laplacian.cpp"
   y_3 = y_7 / halfWidth;
   # 167 "laplacian.cpp"
   *(DownsampleDst + (y_3 * halfWidth + x_5)) = *(FilterDst_1 + ((y_3 << 1) * width_7 + (x_5 << 1)));
  }
  # 154 "laplacian.cpp"
 }
}
#endif // __HIPCC__



# 154 "laplacian.cpp"

#ifndef __HIPCC__
void hmpp_acc_region_main_159_internal_1(hmpprt::s32 width_25, hmpprt::s32 halfWidth_8, hmpprt::s32 halfHeight_7, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  FilterDst_5, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  DownsampleDst_8)
{
 # 154 "laplacian.cpp"
 if (1)
 {
  hmpprt::CUDAGridCall __hmppcg_call;
  __hmppcg_call.setSizeX(192);
  __hmppcg_call.setSizeY(1);
  __hmppcg_call.setBlockSizeX(1);
  __hmppcg_call.setBlockSizeY(256);
  __hmppcg_call.addLocalParameter(&DownsampleDst_8, 8, "DownsampleDst");
  __hmppcg_call.addLocalParameter(&FilterDst_5, 8, "FilterDst_1");
  __hmppcg_call.addLocalParameter((hmpprt::s32) (halfHeight_7), "halfHeight");
  __hmppcg_call.addLocalParameter((hmpprt::s32) (halfWidth_8), "halfWidth");
  __hmppcg_call.addLocalParameter((hmpprt::s32) (width_25), "width_7");
  __hmppcg_call.launch(hmpp_acc_region_main_159_parallel_region_1, hmpprt::Context::getInstance()->getCUDADevice());
 }
 ;
}
#endif // __HIPCC__



# 154 "laplacian.cpp"

#ifndef __HIPCC__
extern "C" CDLT_API  void hmpp_acc_region_main_159(hmpprt::s32 width_16, hmpprt::s32 halfWidth_4, hmpprt::s32 halfHeight_3, hmpprt::u08* FilterDst_3, hmpprt::u08* DownsampleDst_4)
{
 # 175 "laplacian.cpp"
 (hmpp_acc_region_main_159_internal_1(width_16, halfWidth_4, halfHeight_3, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08> (FilterDst_3), hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08> (DownsampleDst_4)));
}
#endif // __HIPCC__



# 175 "laplacian.cpp"

#ifdef __HIPCC__

extern "C" __global__ void hmpp_acc_region_main_188_parallel_region_1(hmpprt::u08* DownsampleDst_3, hmpprt::u08* UpsampleDst, hmpprt::s32 halfHeight_5, hmpprt::s32 halfWidth_3, hmpprt::s32 height_4, hmpprt::s32 width_6)
{
 # 177 "laplacian.cpp"
 {
  # 191 "laplacian.cpp"
  hmpprt::s32 iter_per_gang_6;
  # 191 "laplacian.cpp"
  hmpprt::s32 first_gang_iter_6;
  # 191 "laplacian.cpp"
  hmpprt::s32 last_gang_iter_6;
  # 191 "laplacian.cpp"
  iter_per_gang_6 = ((1 + (halfWidth_3 - 2) / 192) > 256 ? (1 + (halfWidth_3 - 2) / 192) : 256);
  # 191 "laplacian.cpp"
  first_gang_iter_6 = (hmpprt::gr_gbidx()) * iter_per_gang_6;
  # 191 "laplacian.cpp"
  last_gang_iter_6 = ((first_gang_iter_6 + iter_per_gang_6 - 1) < (halfWidth_3 - 2) ? (first_gang_iter_6 + iter_per_gang_6 - 1) : (halfWidth_3 - 2));
  # 191 "laplacian.cpp"
  hmpprt::s32 x_6;
  # 191 "laplacian.cpp"
  # 192 "laplacian.cpp"
  for (x_6 = first_gang_iter_6 + (hmpprt::gr_btidy()) ; x_6 <= last_gang_iter_6 ; x_6 = x_6 + (hmpprt::gr_btnumy()))
  {
   # 193 "laplacian.cpp"
   *(UpsampleDst + ((height_4 - 1) * width_6 + 2 * (x_6 + 1) - 1)) = (hmpprt::u08 ) ((hmpprt::s32 ) (*(DownsampleDst_3 + ((halfHeight_5 - 1) * halfWidth_3 + (x_6 + 1) - 1))) + (hmpprt::s32 ) (*(DownsampleDst_3 + ((halfHeight_5 - 1) * halfWidth_3 + (x_6 + 1)))) + 1 >> 1);
   # 194 "laplacian.cpp"
   *(UpsampleDst + ((height_4 - 1) * width_6 + 2 * (x_6 + 1))) = *(DownsampleDst_3 + ((halfHeight_5 - 1) * halfWidth_3 + (x_6 + 1)));
  }
  # 175 "laplacian.cpp"
 }
}
#endif // __HIPCC__



# 175 "laplacian.cpp"

#ifndef __HIPCC__
void hmpp_acc_region_main_188_internal_1(hmpprt::s32 height_16, hmpprt::s32 width_24, hmpprt::s32 halfWidth_11, hmpprt::s32 halfHeight_6, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  DownsampleDst_7, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  UpsampleDst_8)
{
 # 175 "laplacian.cpp"
 if (1)
 {
  hmpprt::CUDAGridCall __hmppcg_call;
  __hmppcg_call.setSizeX(192);
  __hmppcg_call.setSizeY(1);
  __hmppcg_call.setBlockSizeX(1);
  __hmppcg_call.setBlockSizeY(256);
  __hmppcg_call.addLocalParameter(&DownsampleDst_7, 8, "DownsampleDst_3");
  __hmppcg_call.addLocalParameter(&UpsampleDst_8, 8, "UpsampleDst");
  __hmppcg_call.addLocalParameter((hmpprt::s32) (halfHeight_6), "halfHeight_5");
  __hmppcg_call.addLocalParameter((hmpprt::s32) (halfWidth_11), "halfWidth_3");
  __hmppcg_call.addLocalParameter((hmpprt::s32) (height_16), "height_4");
  __hmppcg_call.addLocalParameter((hmpprt::s32) (width_24), "width_6");
  __hmppcg_call.launch(hmpp_acc_region_main_188_parallel_region_1, hmpprt::Context::getInstance()->getCUDADevice());
 }
 ;
}
#endif // __HIPCC__



# 175 "laplacian.cpp"

#ifndef __HIPCC__
extern "C" CDLT_API  void hmpp_acc_region_main_188(hmpprt::s32 height_11, hmpprt::s32 width_15, hmpprt::s32 halfWidth_7, hmpprt::s32 halfHeight_2, hmpprt::u08* DownsampleDst_11, hmpprt::u08* UpsampleDst_4)
{
 # 201 "laplacian.cpp"
 (hmpp_acc_region_main_188_internal_1(height_11, width_15, halfWidth_7, halfHeight_2, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08> (DownsampleDst_11), hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08> (UpsampleDst_4)));
}
#endif // __HIPCC__



# 201 "laplacian.cpp"

#ifdef __HIPCC__

extern "C" __global__ void hmpp_acc_region_main_202_parallel_region_1(hmpprt::u08* DownsampleDst_2, hmpprt::u08* UpsampleDst_3, hmpprt::s32 halfHeight_1, hmpprt::s32 halfWidth_2, hmpprt::s32 width_5)
{
 # 203 "laplacian.cpp"
 {
  # 205 "laplacian.cpp"
  hmpprt::s32 iter_per_gang_5;
  # 205 "laplacian.cpp"
  hmpprt::s32 first_gang_iter_5;
  # 205 "laplacian.cpp"
  hmpprt::s32 last_gang_iter_5;
  # 205 "laplacian.cpp"
  iter_per_gang_5 = 1 +  - (1 - (halfHeight_1 - 1)) / 192;
  # 205 "laplacian.cpp"
  first_gang_iter_5 = (hmpprt::gr_gbidx()) * iter_per_gang_5;
  # 205 "laplacian.cpp"
  last_gang_iter_5 = ((first_gang_iter_5 + iter_per_gang_5 - 1) < ( - (1 - (halfHeight_1 - 1))) ? (first_gang_iter_5 + iter_per_gang_5 - 1) : ( - (1 - (halfHeight_1 - 1))));
  # 205 "laplacian.cpp"
  hmpprt::s32 y_4;
  # 205 "laplacian.cpp"
  # 206 "laplacian.cpp"
  for (y_4 = first_gang_iter_5 ; y_4 <= last_gang_iter_5 ; y_4 = y_4 + 1)
  {
   # 209 "laplacian.cpp"
   hmpprt::u08 tmp_18;
   # 209 "laplacian.cpp"
   hmpprt::u08 tmp_19;
   # 209 "laplacian.cpp"
   hmpprt::u08 tmp_20;
   # 209 "laplacian.cpp"
   hmpprt::u08 tmp_21;
   # 209 "laplacian.cpp"
   hmpprt::s32 end_6;
   # 209 "laplacian.cpp"
   hmpprt::s32 x_7;
   # 209 "laplacian.cpp"
   # 210 "laplacian.cpp"
   # 210 "laplacian.cpp"
   for (x_7 = (hmpprt::gr_btidy()), end_6 = halfWidth_2 - 2 ; x_7 <= end_6 ; x_7 = x_7 + (hmpprt::gr_btnumy()))
   {
    # 211 "laplacian.cpp"
    *(UpsampleDst_3 + ((2 * (halfHeight_1 - 1 - y_4) - 1) * width_5 + 2 * (x_7 + 1) - 1)) = (hmpprt::u08 ) ((hmpprt::s32 ) (*(DownsampleDst_2 + ((halfHeight_1 - 1 - y_4) * halfWidth_2 + (x_7 + 1) - 1))) + (hmpprt::s32 ) (*(DownsampleDst_2 + ((halfHeight_1 - 1 - y_4) * halfWidth_2 + (x_7 + 1)))) + (hmpprt::s32 ) (*(DownsampleDst_2 + ((halfHeight_1 - 1 - y_4 - 1) * halfWidth_2 + (x_7 + 1) - 1))) + (hmpprt::s32 ) (*(DownsampleDst_2 + ((halfHeight_1 - 1 - y_4 - 1) * halfWidth_2 + (x_7 + 1)))) + 2 >> 2);
    # 212 "laplacian.cpp"
    *(UpsampleDst_3 + ((2 * (halfHeight_1 - 1 - y_4) - 1) * width_5 + 2 * (x_7 + 1))) = (hmpprt::u08 ) ((hmpprt::s32 ) (*(DownsampleDst_2 + ((halfHeight_1 - 1 - y_4) * halfWidth_2 + (x_7 + 1)))) + (hmpprt::s32 ) (*(DownsampleDst_2 + ((halfHeight_1 - 1 - y_4 - 1) * halfWidth_2 + (x_7 + 1)))) + 1 >> 1);
    # 214 "laplacian.cpp"
    *(UpsampleDst_3 + (2 * (halfHeight_1 - 1 - y_4) * width_5 + 2 * (x_7 + 1) - 1)) = (hmpprt::u08 ) ((hmpprt::s32 ) (*(DownsampleDst_2 + ((halfHeight_1 - 1 - y_4) * halfWidth_2 + (x_7 + 1) - 1))) + (hmpprt::s32 ) (*(DownsampleDst_2 + ((halfHeight_1 - 1 - y_4) * halfWidth_2 + (x_7 + 1)))) + 1 >> 1);
    # 215 "laplacian.cpp"
    *(UpsampleDst_3 + (2 * (halfHeight_1 - 1 - y_4) * width_5 + 2 * (x_7 + 1))) = *(DownsampleDst_2 + ((halfHeight_1 - 1 - y_4) * halfWidth_2 + (x_7 + 1)));
   }
   # 201 "laplacian.cpp"
   # 201 "laplacian.cpp"
   tmp_18 = (hmpprt::u08 ) ((hmpprt::s32 ) (*(DownsampleDst_2 + ((halfHeight_1 - 1 - y_4) * halfWidth_2 + halfWidth_2 - 1))) + (hmpprt::s32 ) (*(DownsampleDst_2 + ((halfHeight_1 - 1 - y_4 - 1) * halfWidth_2 + halfWidth_2 - 1))) + 1 >> 1);
   # 201 "laplacian.cpp"
   if ((hmpprt::gr_btidy()) == 0)
   {
    # 201 "laplacian.cpp"
    *(UpsampleDst_3 + ((2 * (halfHeight_1 - 1 - y_4) - 1) * width_5 + width_5 - 1)) = tmp_18;
   }
   # 201 "laplacian.cpp"
   (hmpprt::gr_barrier());
   # 201 "laplacian.cpp"
   tmp_19 = *(DownsampleDst_2 + ((halfHeight_1 - 1 - y_4) * halfWidth_2 + halfWidth_2 - 1));
   # 201 "laplacian.cpp"
   if ((hmpprt::gr_btidy()) == 0)
   {
    # 201 "laplacian.cpp"
    *(UpsampleDst_3 + (2 * (halfHeight_1 - 1 - y_4) * width_5 + width_5 - 1)) = tmp_19;
   }
   # 201 "laplacian.cpp"
   (hmpprt::gr_barrier());
   # 201 "laplacian.cpp"
   tmp_20 = (hmpprt::u08 ) ((hmpprt::s32 ) (*(DownsampleDst_2 + (halfHeight_1 - 1 - y_4) * halfWidth_2)) + (hmpprt::s32 ) (*(DownsampleDst_2 + (halfHeight_1 - 1 - y_4 - 1) * halfWidth_2)) + 1 >> 1);
   # 201 "laplacian.cpp"
   if ((hmpprt::gr_btidy()) == 0)
   {
    # 201 "laplacian.cpp"
    *(UpsampleDst_3 + (2 * (halfHeight_1 - 1 - y_4) - 1) * width_5) = tmp_20;
   }
   # 201 "laplacian.cpp"
   (hmpprt::gr_barrier());
   # 201 "laplacian.cpp"
   tmp_21 = *(DownsampleDst_2 + (halfHeight_1 - 1 - y_4) * halfWidth_2);
   # 201 "laplacian.cpp"
   if ((hmpprt::gr_btidy()) == 0)
   {
    # 201 "laplacian.cpp"
    *(UpsampleDst_3 + 2 * (halfHeight_1 - 1 - y_4) * width_5) = tmp_21;
   }
   # 201 "laplacian.cpp"
   (hmpprt::gr_barrier());
  }
  # 201 "laplacian.cpp"
 }
}
#endif // __HIPCC__



# 201 "laplacian.cpp"

#ifndef __HIPCC__
void hmpp_acc_region_main_202_internal_1(hmpprt::s32 width_23, hmpprt::s32 halfWidth_6, hmpprt::s32 halfHeight_4, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  DownsampleDst_6, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  UpsampleDst_7)
{
 # 201 "laplacian.cpp"
 if (1)
 {
  hmpprt::CUDAGridCall __hmppcg_call;
  __hmppcg_call.setSizeX(192);
  __hmppcg_call.setSizeY(1);
  __hmppcg_call.setBlockSizeX(1);
  __hmppcg_call.setBlockSizeY(256);
  __hmppcg_call.addLocalParameter(&DownsampleDst_6, 8, "DownsampleDst_2");
  __hmppcg_call.addLocalParameter(&UpsampleDst_7, 8, "UpsampleDst_3");
  __hmppcg_call.addLocalParameter((hmpprt::s32) (halfHeight_4), "halfHeight_1");
  __hmppcg_call.addLocalParameter((hmpprt::s32) (halfWidth_6), "halfWidth_2");
  __hmppcg_call.addLocalParameter((hmpprt::s32) (width_23), "width_5");
  __hmppcg_call.launch(hmpp_acc_region_main_202_parallel_region_1, hmpprt::Context::getInstance()->getCUDADevice());
 }
 ;
}
#endif // __HIPCC__



# 201 "laplacian.cpp"

#ifndef __HIPCC__
extern "C" CDLT_API  void hmpp_acc_region_main_202(hmpprt::s32 width_14, hmpprt::s32 halfWidth_10, hmpprt::s32 halfHeight_8, hmpprt::u08* DownsampleDst_10, hmpprt::u08* UpsampleDst_11)
{
 # 229 "laplacian.cpp"
 (hmpp_acc_region_main_202_internal_1(width_14, halfWidth_10, halfHeight_8, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08> (DownsampleDst_10), hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08> (UpsampleDst_11)));
}
#endif // __HIPCC__



# 229 "laplacian.cpp"

#ifdef __HIPCC__

extern "C" __global__ void hmpp_acc_region_main_230_parallel_region_1(hmpprt::u08* DownsampleDst_1, hmpprt::u08* UpsampleDst_2, hmpprt::s32 halfWidth_1)
{
 # 231 "laplacian.cpp"
 {
  # 233 "laplacian.cpp"
  hmpprt::s32 iter_per_gang_4;
  # 233 "laplacian.cpp"
  hmpprt::s32 first_gang_iter_4;
  # 233 "laplacian.cpp"
  hmpprt::s32 last_gang_iter_4;
  # 233 "laplacian.cpp"
  iter_per_gang_4 = ((1 +  - (1 - (halfWidth_1 - 1)) / 192) > 256 ? (1 +  - (1 - (halfWidth_1 - 1)) / 192) : 256);
  # 233 "laplacian.cpp"
  first_gang_iter_4 = (hmpprt::gr_gbidx()) * iter_per_gang_4;
  # 233 "laplacian.cpp"
  last_gang_iter_4 = ((first_gang_iter_4 + iter_per_gang_4 - 1) < ( - (1 - (halfWidth_1 - 1))) ? (first_gang_iter_4 + iter_per_gang_4 - 1) : ( - (1 - (halfWidth_1 - 1))));
  # 233 "laplacian.cpp"
  hmpprt::s32 x_8;
  # 233 "laplacian.cpp"
  # 234 "laplacian.cpp"
  for (x_8 = first_gang_iter_4 + (hmpprt::gr_btidy()) ; x_8 <= last_gang_iter_4 ; x_8 = x_8 + (hmpprt::gr_btnumy()))
  {
   # 235 "laplacian.cpp"
   *(UpsampleDst_2 + 2 * (halfWidth_1 - 1 - x_8)) = *(DownsampleDst_1 + (halfWidth_1 - 1 - x_8));
   # 236 "laplacian.cpp"
   *(UpsampleDst_2 + (2 * (halfWidth_1 - 1 - x_8) - 1)) = (hmpprt::u08 ) (((hmpprt::s32 ) (*(DownsampleDst_1 + (halfWidth_1 - 1 - x_8 - 1))) + (hmpprt::s32 ) (*(DownsampleDst_1 + (halfWidth_1 - 1 - x_8))) + 1) / 2);
  }
  # 229 "laplacian.cpp"
 }
}
#endif // __HIPCC__



# 229 "laplacian.cpp"

#ifndef __HIPCC__
void hmpp_acc_region_main_230_internal_1(hmpprt::s32 width_27, hmpprt::s32 halfWidth_9, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  DownsampleDst_9, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  UpsampleDst_10)
{
 # 229 "laplacian.cpp"
 if (1)
 {
  hmpprt::CUDAGridCall __hmppcg_call;
  __hmppcg_call.setSizeX(192);
  __hmppcg_call.setSizeY(1);
  __hmppcg_call.setBlockSizeX(1);
  __hmppcg_call.setBlockSizeY(256);
  __hmppcg_call.addLocalParameter(&DownsampleDst_9, 8, "DownsampleDst_1");
  __hmppcg_call.addLocalParameter(&UpsampleDst_10, 8, "UpsampleDst_2");
  __hmppcg_call.addLocalParameter((hmpprt::s32) (halfWidth_9), "halfWidth_1");
  __hmppcg_call.launch(hmpp_acc_region_main_230_parallel_region_1, hmpprt::Context::getInstance()->getCUDADevice());
 }
 ;
}
#endif // __HIPCC__



# 229 "laplacian.cpp"

#ifndef __HIPCC__
extern "C" CDLT_API  void hmpp_acc_region_main_230(hmpprt::s32 width_4, hmpprt::s32 halfWidth_5, hmpprt::u08* DownsampleDst_5, hmpprt::u08* UpsampleDst_6)
{
 # 243 "laplacian.cpp"
 (hmpp_acc_region_main_230_internal_1(width_4, halfWidth_5, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08> (DownsampleDst_5), hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08> (UpsampleDst_6)));
}
#endif // __HIPCC__



# 243 "laplacian.cpp"

#ifdef __HIPCC__

extern "C" __global__ void hmpp_acc_region_main_247_parallel_region_1(hmpprt::s16* LaplacianLayer, hmpprt::u08* Source, hmpprt::u08* UpsampleDst_1, hmpprt::s32 height_3, hmpprt::s32 width_3)
{
 # 245 "laplacian.cpp"
 {
  # 250 "laplacian.cpp"
  hmpprt::s32 iter_per_gang_3;
  # 250 "laplacian.cpp"
  hmpprt::s32 first_gang_iter_3;
  # 250 "laplacian.cpp"
  hmpprt::s32 last_gang_iter_3;
  # 250 "laplacian.cpp"
  iter_per_gang_3 = ((1 + (height_3 * width_3 - 1) / 192) > 256 ? (1 + (height_3 * width_3 - 1) / 192) : 256);
  # 250 "laplacian.cpp"
  first_gang_iter_3 = (hmpprt::gr_gbidx()) * iter_per_gang_3;
  # 250 "laplacian.cpp"
  last_gang_iter_3 = ((first_gang_iter_3 + iter_per_gang_3 - 1) < (height_3 * width_3 - 1) ? (first_gang_iter_3 + iter_per_gang_3 - 1) : (height_3 * width_3 - 1));
  # 250 "laplacian.cpp"
  hmpprt::s32 y_6;
  # 250 "laplacian.cpp"
  # 254 "laplacian.cpp"
  for (y_6 = first_gang_iter_3 + (hmpprt::gr_btidy()) ; y_6 <= last_gang_iter_3 ; y_6 = y_6 + (hmpprt::gr_btnumy()))
  {
   # 250 "laplacian.cpp"
   hmpprt::s32 y_5;
   # 253 "laplacian.cpp"
   hmpprt::s32 x_9;
   # 255 "laplacian.cpp"
   x_9 = y_6 % width_3;
   # 255 "laplacian.cpp"
   y_5 = y_6 / width_3;
   # 255 "laplacian.cpp"
   *(LaplacianLayer + (y_5 * width_3 + x_9)) = (hmpprt::s16 ) ((hmpprt::s32 ) ((hmpprt::s16 ) (*(Source + (y_5 * width_3 + x_9)))) - (hmpprt::s32 ) ((hmpprt::s16 ) (*(UpsampleDst_1 + (y_5 * width_3 + x_9)))));
  }
  # 243 "laplacian.cpp"
 }
}
#endif // __HIPCC__



# 243 "laplacian.cpp"

#ifndef __HIPCC__
void hmpp_acc_region_main_247_internal_1(hmpprt::s32 height_15, hmpprt::s32 width_22, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  Source_4, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s16>  LaplacianLayer_2, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  UpsampleDst_9)
{
 # 243 "laplacian.cpp"
 if (1)
 {
  hmpprt::CUDAGridCall __hmppcg_call;
  __hmppcg_call.setSizeX(192);
  __hmppcg_call.setSizeY(1);
  __hmppcg_call.setBlockSizeX(1);
  __hmppcg_call.setBlockSizeY(256);
  __hmppcg_call.addLocalParameter(&LaplacianLayer_2, 8, "LaplacianLayer");
  __hmppcg_call.addLocalParameter(&Source_4, 8, "Source");
  __hmppcg_call.addLocalParameter(&UpsampleDst_9, 8, "UpsampleDst_1");
  __hmppcg_call.addLocalParameter((hmpprt::s32) (height_15), "height_3");
  __hmppcg_call.addLocalParameter((hmpprt::s32) (width_22), "width_3");
  __hmppcg_call.launch(hmpp_acc_region_main_247_parallel_region_1, hmpprt::Context::getInstance()->getCUDADevice());
 }
 ;
}
#endif // __HIPCC__



# 243 "laplacian.cpp"

#ifndef __HIPCC__
extern "C" CDLT_API  void hmpp_acc_region_main_247(hmpprt::s32 height_10, hmpprt::s32 width_13, hmpprt::u08* Source_2, hmpprt::s16* LaplacianLayer_1, hmpprt::u08* UpsampleDst_5)
{
 # 12 "laplacian.cpp"
 (hmpp_acc_region_main_247_internal_1(height_10, width_13, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08> (Source_2), hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s16> (LaplacianLayer_1), hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08> (UpsampleDst_5)));
}
#endif // __HIPCC__



# 12 "laplacian.cpp"

#ifdef __HIPCC__

extern "C" __global__ void hmpp_acc_region_main_55_parallel_region_1(hmpprt::u08* Source_1, hmpprt::s32 height_2, hmpprt::s32* pSrc_padding, hmpprt::s32 width_2)
{
 # 14 "laplacian.cpp"
 {
  # 58 "laplacian.cpp"
  hmpprt::s32 iter_per_gang_2;
  # 58 "laplacian.cpp"
  hmpprt::s32 first_gang_iter_2;
  # 58 "laplacian.cpp"
  hmpprt::s32 last_gang_iter_2;
  # 58 "laplacian.cpp"
  iter_per_gang_2 = 1 + (height_2 - 1) / 192;
  # 58 "laplacian.cpp"
  first_gang_iter_2 = (hmpprt::gr_gbidx()) * iter_per_gang_2;
  # 58 "laplacian.cpp"
  last_gang_iter_2 = ((first_gang_iter_2 + iter_per_gang_2 - 1) < (height_2 - 1) ? (first_gang_iter_2 + iter_per_gang_2 - 1) : (height_2 - 1));
  # 58 "laplacian.cpp"
  hmpprt::s32 h_6;
  # 58 "laplacian.cpp"
  # 59 "laplacian.cpp"
  for (h_6 = first_gang_iter_2 ; h_6 <= last_gang_iter_2 ; h_6 = h_6 + 1)
  {
   # 67 "laplacian.cpp"
   hmpprt::s32 tmp_22;
   # 67 "laplacian.cpp"
   hmpprt::s32 tmp_23;
   # 67 "laplacian.cpp"
   hmpprt::s32 tmp_24;
   # 67 "laplacian.cpp"
   hmpprt::s32 tmp_25;
   # 67 "laplacian.cpp"
   tmp_22 = 2 * (hmpprt::s32 ) (*(Source_1 + (h_6 * width_2 + 1))) - (hmpprt::s32 ) (*(Source_1 + (h_6 * width_2 + 3)));
   # 67 "laplacian.cpp"
   if ((hmpprt::gr_btidy()) == 0)
   {
    # 67 "laplacian.cpp"
    *(pSrc_padding + ((h_6 + 2) * (width_2 + 4) + 1)) = tmp_22;
   }
   # 67 "laplacian.cpp"
   (hmpprt::gr_barrier());
   # 67 "laplacian.cpp"
   tmp_23 = 4 * (hmpprt::s32 ) (*(Source_1 + (h_6 * width_2 + 3))) - 4 * (hmpprt::s32 ) (*(Source_1 + (h_6 * width_2 + 1))) + 2 * (hmpprt::s32 ) (*(Source_1 + h_6 * width_2)) - (hmpprt::s32 ) (*(Source_1 + (h_6 * width_2 + 2)));
   # 67 "laplacian.cpp"
   if ((hmpprt::gr_btidy()) == 0)
   {
    # 67 "laplacian.cpp"
    *(pSrc_padding + (h_6 + 2) * (width_2 + 4)) = tmp_23;
   }
   # 67 "laplacian.cpp"
   (hmpprt::gr_barrier());
   # 67 "laplacian.cpp"
   tmp_24 = 2 * (hmpprt::s32 ) (*(Source_1 + (h_6 * width_2 + width_2 - 2))) - (hmpprt::s32 ) (*(Source_1 + (h_6 * width_2 + width_2 - 4)));
   # 67 "laplacian.cpp"
   if ((hmpprt::gr_btidy()) == 0)
   {
    # 67 "laplacian.cpp"
    *(pSrc_padding + ((h_6 + 2) * (width_2 + 4) + width_2 + 2)) = tmp_24;
   }
   # 67 "laplacian.cpp"
   (hmpprt::gr_barrier());
   # 67 "laplacian.cpp"
   tmp_25 = 4 * (hmpprt::s32 ) (*(Source_1 + (h_6 * width_2 + width_2 - 4))) - 4 * (hmpprt::s32 ) (*(Source_1 + (h_6 * width_2 + width_2 - 2))) + 2 * (hmpprt::s32 ) (*(Source_1 + (h_6 * width_2 + width_2 - 1))) - (hmpprt::s32 ) (*(Source_1 + (h_6 * width_2 + width_2 - 3)));
   # 67 "laplacian.cpp"
   if ((hmpprt::gr_btidy()) == 0)
   {
    # 67 "laplacian.cpp"
    *(pSrc_padding + ((h_6 + 2) * (width_2 + 4) + width_2 + 3)) = tmp_25;
   }
   # 67 "laplacian.cpp"
   (hmpprt::gr_barrier());
   # 67 "laplacian.cpp"
   hmpprt::s32 end_10;
   # 67 "laplacian.cpp"
   hmpprt::s32 w_8;
   # 67 "laplacian.cpp"
   # 68 "laplacian.cpp"
   # 68 "laplacian.cpp"
   for (w_8 = (hmpprt::gr_btidy()), end_10 = width_2 - 1 ; w_8 <= end_10 ; w_8 = w_8 + (hmpprt::gr_btnumy()))
   {
    # 68 "laplacian.cpp"
    *(pSrc_padding + ((h_6 + 2) * (width_2 + 4) + w_8 + 2)) = (hmpprt::s32 ) (*(Source_1 + (h_6 * width_2 + w_8)));
   }
   # 12 "laplacian.cpp"
  }
  # 12 "laplacian.cpp"
 }
}
#endif // __HIPCC__



# 12 "laplacian.cpp"

#ifndef __HIPCC__
void hmpp_acc_region_main_55_internal_1(hmpprt::s32 height_20, hmpprt::s32 width_21, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  Source_5, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s32>  pSrc_padding_4)
{
 # 12 "laplacian.cpp"
 if (1)
 {
  hmpprt::CUDAGridCall __hmppcg_call;
  __hmppcg_call.setSizeX(192);
  __hmppcg_call.setSizeY(1);
  __hmppcg_call.setBlockSizeX(1);
  __hmppcg_call.setBlockSizeY(256);
  __hmppcg_call.addLocalParameter(&Source_5, 8, "Source_1");
  __hmppcg_call.addLocalParameter((hmpprt::s32) (height_20), "height_2");
  __hmppcg_call.addLocalParameter(&pSrc_padding_4, 8, "pSrc_padding");
  __hmppcg_call.addLocalParameter((hmpprt::s32) (width_21), "width_2");
  __hmppcg_call.launch(hmpp_acc_region_main_55_parallel_region_1, hmpprt::Context::getInstance()->getCUDADevice());
 }
 ;
}
#endif // __HIPCC__



# 12 "laplacian.cpp"

#ifndef __HIPCC__
extern "C" CDLT_API  void hmpp_acc_region_main_55(hmpprt::s32 height_9, hmpprt::s32 width_12, hmpprt::u08* Source_3, hmpprt::s32* pSrc_padding_2)
{
 # 75 "laplacian.cpp"
 (hmpp_acc_region_main_55_internal_1(height_9, width_12, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08> (Source_3), hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s32> (pSrc_padding_2)));
}
#endif // __HIPCC__



# 75 "laplacian.cpp"

#ifdef __HIPCC__

extern "C" __global__ void hmpp_acc_region_main_79_parallel_region_1(hmpprt::s32 height_1, hmpprt::u08* pBufL_cp_1, hmpprt::s32* pSrc_padding_1, hmpprt::s32 width_1)
{
 # 77 "laplacian.cpp"
 {
  # 82 "laplacian.cpp"
  hmpprt::s32 iter_per_gang_1;
  # 82 "laplacian.cpp"
  hmpprt::s32 first_gang_iter_1;
  # 82 "laplacian.cpp"
  hmpprt::s32 last_gang_iter_1;
  # 82 "laplacian.cpp"
  iter_per_gang_1 = ((1 + (height_1 * width_1 - 1) / 192) > 256 ? (1 + (height_1 * width_1 - 1) / 192) : 256);
  # 82 "laplacian.cpp"
  first_gang_iter_1 = (hmpprt::gr_gbidx()) * iter_per_gang_1;
  # 82 "laplacian.cpp"
  last_gang_iter_1 = ((first_gang_iter_1 + iter_per_gang_1 - 1) < (height_1 * width_1 - 1) ? (first_gang_iter_1 + iter_per_gang_1 - 1) : (height_1 * width_1 - 1));
  # 82 "laplacian.cpp"
  hmpprt::s32 h_8;
  # 82 "laplacian.cpp"
  # 85 "laplacian.cpp"
  for (h_8 = first_gang_iter_1 + (hmpprt::gr_btidy()) ; h_8 <= last_gang_iter_1 ; h_8 = h_8 + (hmpprt::gr_btnumy()))
  {
   # 82 "laplacian.cpp"
   hmpprt::s32 h_7;
   # 84 "laplacian.cpp"
   hmpprt::s32 w_9;
   # 86 "laplacian.cpp"
   w_9 = h_8 % width_1;
   # 86 "laplacian.cpp"
   h_7 = h_8 / width_1;
   # 86 "laplacian.cpp"
   *(pBufL_cp_1 + (h_7 * width_1 + w_9)) = (hmpprt::u08 ) (*(pSrc_padding_1 + ((h_7 + 2) * (width_1 + 4) + w_9)) + (*(pSrc_padding_1 + ((h_7 + 2) * (width_1 + 4) + w_9 + 1)) << 2) + 6 * *(pSrc_padding_1 + ((h_7 + 2) * (width_1 + 4) + (w_9 + 2))) + (*(pSrc_padding_1 + ((h_7 + 2) * (width_1 + 4) + (w_9 + 3))) << 2) + *(pSrc_padding_1 + ((h_7 + 2) * (width_1 + 4) + (w_9 + 4))) + 8 >> 4);
  }
  # 75 "laplacian.cpp"
 }
}
#endif // __HIPCC__



# 75 "laplacian.cpp"

#ifndef __HIPCC__
void hmpp_acc_region_main_79_internal_1(hmpprt::s32 height_19, hmpprt::s32 width_20, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s32>  pSrc_padding_5, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  pBufL_cp_8)
{
 # 75 "laplacian.cpp"
 if (1)
 {
  hmpprt::CUDAGridCall __hmppcg_call;
  __hmppcg_call.setSizeX(192);
  __hmppcg_call.setSizeY(1);
  __hmppcg_call.setBlockSizeX(1);
  __hmppcg_call.setBlockSizeY(256);
  __hmppcg_call.addLocalParameter((hmpprt::s32) (height_19), "height_1");
  __hmppcg_call.addLocalParameter(&pBufL_cp_8, 8, "pBufL_cp_1");
  __hmppcg_call.addLocalParameter(&pSrc_padding_5, 8, "pSrc_padding_1");
  __hmppcg_call.addLocalParameter((hmpprt::s32) (width_20), "width_1");
  __hmppcg_call.launch(hmpp_acc_region_main_79_parallel_region_1, hmpprt::Context::getInstance()->getCUDADevice());
 }
 ;
}
#endif // __HIPCC__



# 75 "laplacian.cpp"

#ifndef __HIPCC__
extern "C" CDLT_API  void hmpp_acc_region_main_79(hmpprt::s32 height_8, hmpprt::s32 width_11, hmpprt::s32* pSrc_padding_3, hmpprt::u08* pBufL_cp_4)
{
 # 1 "<preprocessor>"
 (hmpp_acc_region_main_79_internal_1(height_8, width_11, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s32> (pSrc_padding_3), hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08> (pBufL_cp_4)));
}
#endif // __HIPCC__




#ifndef __HIPCC__
extern "C" const char * hmpprt_cuda_get_gpu_code();

static hmpprt::CUDAModule * hmpprt_module = 0;
static int hmpprt_uses = 0;

extern "C" CDLT_API void * hmpprt_init()
{
  try
  {
    if (hmpprt_uses++ == 0)
    {
      hmpprt_module = new hmpprt::CUDAModule(hmpprt_cuda_get_gpu_code());
      hmpp_acc_region_main_100_parallel_region_1 = new hmpprt::CUDAGrid(hmpprt_module, "hmpp_acc_region_main_100_parallel_region_1");
      hmpp_acc_region_main_114_parallel_region_1 = new hmpprt::CUDAGrid(hmpprt_module, "hmpp_acc_region_main_114_parallel_region_1");
      hmpp_acc_region_main_136_parallel_region_1 = new hmpprt::CUDAGrid(hmpprt_module, "hmpp_acc_region_main_136_parallel_region_1");
      hmpp_acc_region_main_159_parallel_region_1 = new hmpprt::CUDAGrid(hmpprt_module, "hmpp_acc_region_main_159_parallel_region_1");
      hmpp_acc_region_main_188_parallel_region_1 = new hmpprt::CUDAGrid(hmpprt_module, "hmpp_acc_region_main_188_parallel_region_1");
      hmpp_acc_region_main_202_parallel_region_1 = new hmpprt::CUDAGrid(hmpprt_module, "hmpp_acc_region_main_202_parallel_region_1");
      hmpp_acc_region_main_230_parallel_region_1 = new hmpprt::CUDAGrid(hmpprt_module, "hmpp_acc_region_main_230_parallel_region_1");
      hmpp_acc_region_main_247_parallel_region_1 = new hmpprt::CUDAGrid(hmpprt_module, "hmpp_acc_region_main_247_parallel_region_1");
      hmpp_acc_region_main_55_parallel_region_1 = new hmpprt::CUDAGrid(hmpprt_module, "hmpp_acc_region_main_55_parallel_region_1");
      hmpp_acc_region_main_79_parallel_region_1 = new hmpprt::CUDAGrid(hmpprt_module, "hmpp_acc_region_main_79_parallel_region_1");

    }
    hmpprt::Context::getInstance()->getGrouplet()->setTarget(hmpprt::CUDA);
    hmpprt::Context::getInstance()->getGrouplet()->addSignature("hmpp_acc_region_main_100", "prototype hmpp_acc_region_main_100(height: s32, width: s32, pSrc_padding2: ^cudaglob s32, pBufL_cp: ^cudaglob u8)");
    hmpprt::Context::getInstance()->getGrouplet()->addSignature("hmpp_acc_region_main_114", "prototype hmpp_acc_region_main_114(height: s32, width: s32, pSrc_padding2: ^cudaglob s32, pBufL_cp: ^cudaglob u8)");
    hmpprt::Context::getInstance()->getGrouplet()->addSignature("hmpp_acc_region_main_136", "prototype hmpp_acc_region_main_136(height: s32, width: s32, FilterDst: ^cudaglob u8, pSrc_padding2: ^cudaglob s32)");
    hmpprt::Context::getInstance()->getGrouplet()->addSignature("hmpp_acc_region_main_159", "prototype hmpp_acc_region_main_159(width: s32, halfWidth: s32, halfHeight: s32, FilterDst: ^cudaglob u8, DownsampleDst: ^cudaglob u8)");
    hmpprt::Context::getInstance()->getGrouplet()->addSignature("hmpp_acc_region_main_188", "prototype hmpp_acc_region_main_188(height: s32, width: s32, halfWidth: s32, halfHeight: s32, DownsampleDst: ^cudaglob u8, UpsampleDst: ^cudaglob u8)");
    hmpprt::Context::getInstance()->getGrouplet()->addSignature("hmpp_acc_region_main_202", "prototype hmpp_acc_region_main_202(width: s32, halfWidth: s32, halfHeight: s32, DownsampleDst: ^cudaglob u8, UpsampleDst: ^cudaglob u8)");
    hmpprt::Context::getInstance()->getGrouplet()->addSignature("hmpp_acc_region_main_230", "prototype hmpp_acc_region_main_230(width: s32, halfWidth: s32, DownsampleDst: ^cudaglob u8, UpsampleDst: ^cudaglob u8)");
    hmpprt::Context::getInstance()->getGrouplet()->addSignature("hmpp_acc_region_main_247", "prototype hmpp_acc_region_main_247(height: s32, width: s32, Source: ^cudaglob u8, LaplacianLayer: ^cudaglob s16, UpsampleDst: ^cudaglob u8)");
    hmpprt::Context::getInstance()->getGrouplet()->addSignature("hmpp_acc_region_main_55", "prototype hmpp_acc_region_main_55(height: s32, width: s32, Source: ^cudaglob u8, pSrc_padding: ^cudaglob s32)");
    hmpprt::Context::getInstance()->getGrouplet()->addSignature("hmpp_acc_region_main_79", "prototype hmpp_acc_region_main_79(height: s32, width: s32, pSrc_padding: ^cudaglob s32, pBufL_cp: ^cudaglob u8)");

  }
  catch (hmpperr::Error & e)
  {
    return e.clone();
  }
  catch(...)
  {
    fprintf(stderr,"Unexpected error in hmpprt_init()\n");
    abort();
  }
  return 0;
}
#endif // __HIPCC__

#ifndef __HIPCC__
extern "C" CDLT_API void * hmpprt_fini()
{
  try
  {
    if (--hmpprt_uses == 0)
    {
      delete hmpp_acc_region_main_100_parallel_region_1;
      delete hmpp_acc_region_main_114_parallel_region_1;
      delete hmpp_acc_region_main_136_parallel_region_1;
      delete hmpp_acc_region_main_159_parallel_region_1;
      delete hmpp_acc_region_main_188_parallel_region_1;
      delete hmpp_acc_region_main_202_parallel_region_1;
      delete hmpp_acc_region_main_230_parallel_region_1;
      delete hmpp_acc_region_main_247_parallel_region_1;
      delete hmpp_acc_region_main_55_parallel_region_1;
      delete hmpp_acc_region_main_79_parallel_region_1;

      delete hmpprt_module;
      hmpprt_module = 0;
    }
  }
  catch (hmpperr::Error & e)
  {
    return e.clone();
  }
  catch(...)
  {
    fprintf(stderr,"Unexpected error in hmpprt_fini()\n");
    abort();
  }
  return 0;
}
#endif // __HIPCC__

// footer
