#include "hip/hip_runtime.h"

#include <stdio.h>

#ifndef __HIPCC__
#include <stdlib.h>
#include <math.h>

#include <hmpprt/Grouplet.h>
#include <hmpprt/HostTypes.h>
#include <hmpprt/Context.h>
#include <hmpprt/CUDAGrid.h>
#include <hmpprt/CUDAModule.h>
#include <hmpprt/DeviceManager.h>
#include <hmpperr/hmpperr.h>

#include <openacci/openacci_c.h>

#ifdef _WIN32
#  define CDLT_API __declspec(dllexport)
#else /* ! _WIN32 */
#  define CDLT_API
#endif /* _WIN32 */



#else // ! __HIPCC__

#include <hmpprt/HostTypes.h>
#include <hmpprt/CUDAIntrinsics.h>

extern __shared__ int64_t hmpp_sharedmem[];
#endif // __HIPCC__



#ifndef __HIPCC__

#else


#endif

#define HMPPCG_SIMD_LENGTH 32

# 15 "histogram.cpp"

#ifndef __HIPCC__
extern "C" CDLT_API  void hmpp_acc_region_main_103(hmpprt::s32 w_2, hmpprt::s32 h_2, hmpprt::s32* dstData_2, hmpprt::s32* mapPixel_2)
;
#endif // __HIPCC__



# 15 "histogram.cpp"

#ifndef __HIPCC__
void hmpp_acc_region_main_103_internal_1(hmpprt::s32 w, hmpprt::s32 h, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s32>  dstData_1, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s32>  mapPixel)
;
#endif // __HIPCC__



# 15 "histogram.cpp"

#ifndef __HIPCC__
static hmpprt::CUDAGrid * hmpp_acc_region_main_103_parallel_region_1 = 0;
#else

extern "C" __global__ void hmpp_acc_region_main_103_parallel_region_1(hmpprt::s32* dstData, hmpprt::s32 h_1, hmpprt::s32* mapPixel_1, hmpprt::s32 w_1);
#endif // __HIPCC__




# 15 "histogram.cpp"

#ifdef __HIPCC__

extern "C" __global__ void hmpp_acc_region_main_103_parallel_region_1(hmpprt::s32* dstData, hmpprt::s32 h_1, hmpprt::s32* mapPixel_1, hmpprt::s32 w_1)
{
 # 17 "histogram.cpp"
 {
  # 107 "histogram.cpp"
  hmpprt::s32 iter_per_gang_1;
  # 107 "histogram.cpp"
  hmpprt::s32 first_gang_iter_1;
  # 107 "histogram.cpp"
  hmpprt::s32 last_gang_iter_1;
  # 107 "histogram.cpp"
  iter_per_gang_1 = ((1 + (h_1 - 1) / 192) > 8LL ? (1 + (h_1 - 1) / 192) : 8LL);
  # 107 "histogram.cpp"
  first_gang_iter_1 = (hmpprt::gr_gbidx()) * iter_per_gang_1;
  # 107 "histogram.cpp"
  last_gang_iter_1 = ((first_gang_iter_1 + iter_per_gang_1 - 1) < (h_1 - 1) ? (first_gang_iter_1 + iter_per_gang_1 - 1) : (h_1 - 1));
  # 107 "histogram.cpp"
  hmpprt::s32 i_1;
  # 107 "histogram.cpp"
  # 108 "histogram.cpp"
  for (i_1 = first_gang_iter_1 + (hmpprt::gr_btidy()) ; i_1 <= last_gang_iter_1 ; i_1 = i_1 + (hmpprt::gr_btnumy()))
  {
   # 111 "histogram.cpp"
   hmpprt::s32 end_1;
   # 111 "histogram.cpp"
   hmpprt::s32 j_1;
   # 111 "histogram.cpp"
   # 112 "histogram.cpp"
   # 112 "histogram.cpp"
   for (j_1 = (hmpprt::gr_btidx()), end_1 = w_1 - 1 ; j_1 <= end_1 ; j_1 = j_1 + (hmpprt::gr_btnumx()))
   {
    # 113 "histogram.cpp"
    *(dstData + (i_1 * w_1 + j_1)) = *(mapPixel_1 + *(dstData + (i_1 * w_1 + j_1)));
   }
   # 15 "histogram.cpp"
  }
  # 15 "histogram.cpp"
 }
}
#endif // __HIPCC__



# 15 "histogram.cpp"

#ifndef __HIPCC__
void hmpp_acc_region_main_103_internal_1(hmpprt::s32 w, hmpprt::s32 h, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s32>  dstData_1, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s32>  mapPixel)
{
 # 15 "histogram.cpp"
 if (1)
 {
  hmpprt::CUDAGridCall __hmppcg_call;
  __hmppcg_call.setSizeX(192);
  __hmppcg_call.setSizeY(1);
  __hmppcg_call.setBlockSizeX(32);
  __hmppcg_call.setBlockSizeY(8LL);
  __hmppcg_call.addLocalParameter(&dstData_1, 8, "dstData");
  __hmppcg_call.addLocalParameter((hmpprt::s32) (h), "h_1");
  __hmppcg_call.addLocalParameter(&mapPixel, 8, "mapPixel_1");
  __hmppcg_call.addLocalParameter((hmpprt::s32) (w), "w_1");
  __hmppcg_call.launch(hmpp_acc_region_main_103_parallel_region_1, hmpprt::Context::getInstance()->getCUDADevice());
 }
 ;
}
#endif // __HIPCC__



# 15 "histogram.cpp"

#ifndef __HIPCC__
extern "C" CDLT_API  void hmpp_acc_region_main_103(hmpprt::s32 w_2, hmpprt::s32 h_2, hmpprt::s32* dstData_2, hmpprt::s32* mapPixel_2)
{
 # 1 "<preprocessor>"
 (hmpp_acc_region_main_103_internal_1(w_2, h_2, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s32> (dstData_2), hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s32> (mapPixel_2)));
}
#endif // __HIPCC__




#ifndef __HIPCC__
extern "C" const char * hmpprt_cuda_get_gpu_code();

static hmpprt::CUDAModule * hmpprt_module = 0;
static int hmpprt_uses = 0;

extern "C" CDLT_API void * hmpprt_init()
{
  try
  {
    if (hmpprt_uses++ == 0)
    {
      hmpprt_module = new hmpprt::CUDAModule(hmpprt_cuda_get_gpu_code());
      hmpp_acc_region_main_103_parallel_region_1 = new hmpprt::CUDAGrid(hmpprt_module, "hmpp_acc_region_main_103_parallel_region_1");

    }
    hmpprt::Context::getInstance()->getGrouplet()->setTarget(hmpprt::CUDA);
    hmpprt::Context::getInstance()->getGrouplet()->addSignature("hmpp_acc_region_main_103", "prototype hmpp_acc_region_main_103(w: s32, h: s32, dstData: ^cudaglob s32, mapPixel: ^cudaglob s32)");

  }
  catch (hmpperr::Error & e)
  {
    return e.clone();
  }
  catch(...)
  {
    fprintf(stderr,"Unexpected error in hmpprt_init()\n");
    abort();
  }
  return 0;
}
#endif // __HIPCC__

#ifndef __HIPCC__
extern "C" CDLT_API void * hmpprt_fini()
{
  try
  {
    if (--hmpprt_uses == 0)
    {
      delete hmpp_acc_region_main_103_parallel_region_1;

      delete hmpprt_module;
      hmpprt_module = 0;
    }
  }
  catch (hmpperr::Error & e)
  {
    return e.clone();
  }
  catch(...)
  {
    fprintf(stderr,"Unexpected error in hmpprt_fini()\n");
    abort();
  }
  return 0;
}
#endif // __HIPCC__

// footer
