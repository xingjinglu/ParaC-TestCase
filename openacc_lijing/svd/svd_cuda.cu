#include "hip/hip_runtime.h"

#include <stdio.h>

#ifndef __HIPCC__
#include <stdlib.h>
#include <math.h>

#include <hmpprt/Grouplet.h>
#include <hmpprt/HostTypes.h>
#include <hmpprt/Context.h>
#include <hmpprt/CUDAGrid.h>
#include <hmpprt/CUDAModule.h>
#include <hmpprt/DeviceManager.h>
#include <hmpperr/hmpperr.h>

#include <openacci/openacci_c.h>

#ifdef _WIN32
#  define CDLT_API __declspec(dllexport)
#else /* ! _WIN32 */
#  define CDLT_API
#endif /* _WIN32 */



#else // ! __HIPCC__

#include <hmpprt/HostTypes.h>
#include <hmpprt/CUDAIntrinsics.h>

extern __shared__ int64_t hmpp_sharedmem[];
#endif // __HIPCC__



#ifndef __HIPCC__

#else


#endif

#define HMPPCG_SIMD_LENGTH 32

# 363 "svd.cpp"

#ifdef __HIPCC__
__device__ double reduce_double_add_x(double priv, double* tmpa, double orig)
;
#endif // __HIPCC__



# 363 "svd.cpp"

#ifndef __HIPCC__
extern "C" CDLT_API  void hmpp_acc_region_svd_363(double* A_H_1, double* B_H_2, double* S_H_3)
;
#endif // __HIPCC__



# 363 "svd.cpp"

#ifndef __HIPCC__
void hmpp_acc_region_svd_363_internal_1(hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  A_H_7, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  B_H_1, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  S_H_5)
;
#endif // __HIPCC__



# 325 "svd.cpp"

#ifndef __HIPCC__
static hmpprt::CUDAGrid * hmpp_acc_region_svd_363_parallel_region_1 = 0;
#else

extern "C" __global__ void hmpp_acc_region_svd_363_parallel_region_1(double* A_H_4, double* B_H, double* S_H_1);
#endif // __HIPCC__




# 325 "svd.cpp"

#ifndef __HIPCC__
extern "C" CDLT_API  void hmpp_acc_region_svd_332(double* A_H_5, double* S_H_4, double* ci_3)
;
#endif // __HIPCC__



# 325 "svd.cpp"

#ifndef __HIPCC__
void hmpp_acc_region_svd_332_internal_1(hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  A_H_8, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  S_H_2, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  ci_5)
;
#endif // __HIPCC__



# 60 "svd.cpp"

#ifndef __HIPCC__
static hmpprt::CUDAGrid * hmpp_acc_region_svd_332_parallel_region_1 = 0;
#else

extern "C" __global__ void hmpp_acc_region_svd_332_parallel_region_1(double* A_H_2, double* S_H, double* ci_1);
#endif // __HIPCC__




# 60 "svd.cpp"

#ifndef __HIPCC__
extern "C" CDLT_API  void hmpp_acc_region_svd_124(hmpprt::s32* pass_3, double* A_H, double* V_H_2, hmpprt::s32* rotate_col_idx_1, double* ci_4, double* cj_2, double* vi_2, double* vj_2, hmpprt::s32 start_index)
;
#endif // __HIPCC__



# 60 "svd.cpp"

#ifndef __HIPCC__
void hmpp_acc_region_svd_124_internal_1(hmpprt::s32* pass, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  A_H_6, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  V_H_1, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s32>  rotate_col_idx_2, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  ci_2, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  cj_1, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  vi_1, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  vj_1, hmpprt::s32 start_index_2)
;
#endif // __HIPCC__



# 60 "svd.cpp"

#ifndef __HIPCC__
static hmpprt::CUDAGrid * hmpp_acc_region_svd_124_parallel_region_1 = 0;
#else

extern "C" __global__ void hmpp_acc_region_svd_124_parallel_region_1(double* A_H_3, double* V_H, double* ci, double* cj, hmpprt::s32* pass_1, hmpprt::s32* rotate_col_idx, hmpprt::s32 start_index_1, double* value_1, double* vi, double* vj);
#endif // __HIPCC__




# 60 "svd.cpp"

#ifdef __HIPCC__

extern "C" __global__ void hmpp_acc_region_svd_124_parallel_region_1(double* A_H_3, double* V_H, double* ci, double* cj, hmpprt::s32* pass_1, hmpprt::s32* rotate_col_idx, hmpprt::s32 start_index_1, double* value_1, double* vi, double* vj)
{
 
 double * tmpr_2 = (double *)(((char *)hmpp_sharedmem + 0));
 
 # 62 "svd.cpp"
 {
  # 127 "svd.cpp"
  hmpprt::s32 iter_per_gang_3;
  # 127 "svd.cpp"
  hmpprt::s32 first_gang_iter_3;
  # 127 "svd.cpp"
  hmpprt::s32 last_gang_iter_3;
  # 127 "svd.cpp"
  iter_per_gang_3 = ((1 + ((256 - start_index_1) / 2 - 1) / 192) > 8LL ? (1 + ((256 - start_index_1) / 2 - 1) / 192) : 8LL);
  # 127 "svd.cpp"
  first_gang_iter_3 = (hmpprt::gr_gbidx()) * iter_per_gang_3;
  # 127 "svd.cpp"
  last_gang_iter_3 = ((first_gang_iter_3 + iter_per_gang_3 - 1) < ((256 - start_index_1) / 2 - 1) ? (first_gang_iter_3 + iter_per_gang_3 - 1) : ((256 - start_index_1) / 2 - 1));
  # 127 "svd.cpp"
  hmpprt::s32 j_2;
  # 127 "svd.cpp"
  # 128 "svd.cpp"
  for (j_2 = first_gang_iter_3 + (hmpprt::gr_btidy()) ; j_2 <= last_gang_iter_3 ; j_2 = j_2 + (hmpprt::gr_btnumy()))
  {
   # 133 "svd.cpp"
   hmpprt::s32 col_p;
   # 133 "svd.cpp"
   col_p = *(rotate_col_idx + (j_2 * 2 + (start_index_1 + 1) - 1));
   # 134 "svd.cpp"
   hmpprt::s32 col_q;
   # 134 "svd.cpp"
   col_q = *(rotate_col_idx + (j_2 * 2 + (start_index_1 + 1)));
   # 135 "svd.cpp"
   double real1;
   # 136 "svd.cpp"
   double imag1;
   # 142 "svd.cpp"
   hmpprt::s32 k_18;
   # 142 "svd.cpp"
   # 143 "svd.cpp"
   for (k_18 = (hmpprt::gr_btidx()) ; k_18 <= 255 ; k_18 = k_18 + (hmpprt::gr_btnumx()))
   {
    # 144 "svd.cpp"
    *(ci + (j_2 * 2 + (start_index_1 + 1)) / 2 * 256 * 2 + k_18) = *(A_H_3 + (col_p * 256 + k_18));
    # 145 "svd.cpp"
    *(ci + (j_2 * 2 + (start_index_1 + 1)) / 2 * 256 * 2 + (256 + k_18)) = *(A_H_3 + (65536 + col_p * 256 + k_18));
   }
   # 150 "svd.cpp"
   # 150 "svd.cpp"
   hmpprt::s32 k_19;
   # 150 "svd.cpp"
   # 151 "svd.cpp"
   for (k_19 = (hmpprt::gr_btidx()) ; k_19 <= 255 ; k_19 = k_19 + (hmpprt::gr_btnumx()))
   {
    # 152 "svd.cpp"
    *(cj + (j_2 * 2 + (start_index_1 + 1)) / 2 * 256 * 2 + k_19) = *(A_H_3 + (col_q * 256 + k_19));
    # 153 "svd.cpp"
    *(cj + (j_2 * 2 + (start_index_1 + 1)) / 2 * 256 * 2 + (256 + k_19)) = *(A_H_3 + (65536 + col_q * 256 + k_19));
   }
   # 158 "svd.cpp"
   # 158 "svd.cpp"
   hmpprt::s32 k_20;
   # 158 "svd.cpp"
   # 159 "svd.cpp"
   for (k_20 = (hmpprt::gr_btidx()) ; k_20 <= 255 ; k_20 = k_20 + (hmpprt::gr_btnumx()))
   {
    # 160 "svd.cpp"
    *(vi + (j_2 * 2 + (start_index_1 + 1)) / 2 * 256 * 2 + k_20) = *(V_H + (col_p * 256 + k_20));
    # 161 "svd.cpp"
    *(vi + (j_2 * 2 + (start_index_1 + 1)) / 2 * 256 * 2 + (256 + k_20)) = *(V_H + (65536 + col_p * 256 + k_20));
   }
   # 166 "svd.cpp"
   # 166 "svd.cpp"
   hmpprt::s32 k_21;
   # 166 "svd.cpp"
   # 167 "svd.cpp"
   for (k_21 = (hmpprt::gr_btidx()) ; k_21 <= 255 ; k_21 = k_21 + (hmpprt::gr_btnumx()))
   {
    # 168 "svd.cpp"
    *(vj + (j_2 * 2 + (start_index_1 + 1)) / 2 * 256 * 2 + k_21) = *(V_H + (col_q * 256 + k_21));
    # 169 "svd.cpp"
    *(vj + (j_2 * 2 + (start_index_1 + 1)) / 2 * 256 * 2 + (256 + k_21)) = *(V_H + (65536 + col_q * 256 + k_21));
   }
   # 172 "svd.cpp"
   # 172 "svd.cpp"
   hmpprt::s32 skip_trans;
   # 172 "svd.cpp"
   skip_trans = 0;
   # 136 "svd.cpp"
   double imag1_1;
   # 135 "svd.cpp"
   imag1_1 = 0;
   # 135 "svd.cpp"
   double real1_1;
   # 176 "svd.cpp"
   real1_1 = 0;
   # 176 "svd.cpp"
   hmpprt::s32 k_22;
   # 176 "svd.cpp"
   # 177 "svd.cpp"
   for (k_22 = (hmpprt::gr_btidx()) ; k_22 <= 255 ; k_22 = k_22 + (hmpprt::gr_btnumx()))
   {
    # 178 "svd.cpp"
    real1_1 = real1_1 + (*(ci + (j_2 * 2 + (start_index_1 + 1)) / 2 * 256 * 2 + k_22) * *(cj + (j_2 * 2 + (start_index_1 + 1)) / 2 * 256 * 2 + k_22) + *(ci + (j_2 * 2 + (start_index_1 + 1)) / 2 * 256 * 2 + (256 + k_22)) * *(cj + (j_2 * 2 + (start_index_1 + 1)) / 2 * 256 * 2 + (256 + k_22)));
    # 179 "svd.cpp"
    imag1_1 = imag1_1 + (*(ci + (j_2 * 2 + (start_index_1 + 1)) / 2 * 256 * 2 + k_22) * *(cj + (j_2 * 2 + (start_index_1 + 1)) / 2 * 256 * 2 + (256 + k_22)) - *(ci + (j_2 * 2 + (start_index_1 + 1)) / 2 * 256 * 2 + (256 + k_22)) * *(cj + (j_2 * 2 + (start_index_1 + 1)) / 2 * 256 * 2 + k_22));
   }
   # 181 "svd.cpp"
   # 181 "svd.cpp"
   real1 = (reduce_double_add_x(real1_1, tmpr_2, (double) 0.));
   # 181 "svd.cpp"
   imag1 = (reduce_double_add_x(imag1_1, tmpr_2, (double) 0.));
   # 181 "svd.cpp"
   *(value_1 + (j_2 * 2 + (start_index_1 + 1)) / 2) = sqrt(real1 * real1 + imag1 * imag1);
   # 183 "svd.cpp"
   if (*(value_1 + (j_2 * 2 + (start_index_1 + 1)) / 2) < (double) 1.0000000000000000818e-05)
   {
    # 184 "svd.cpp"
    skip_trans = 1;
   }
   # 186 "svd.cpp"
   if (skip_trans == 0)
   {
    # 188 "svd.cpp"
    *pass_1 = 0;
    # 192 "svd.cpp"
    double result1;
    # 193 "svd.cpp"
    double result2;
    # 194 "svd.cpp"
    double result3;
    # 195 "svd.cpp"
    double result4;
    # 196 "svd.cpp"
    double result5;
    # 197 "svd.cpp"
    double result6;
    # 198 "svd.cpp"
    double result7;
    # 199 "svd.cpp"
    double result8;
    # 192 "svd.cpp"
    double result1_1;
    # 202 "svd.cpp"
    result1_1 = 0;
    # 202 "svd.cpp"
    hmpprt::s32 k_23;
    # 202 "svd.cpp"
    # 203 "svd.cpp"
    for (k_23 = (hmpprt::gr_btidx()) ; k_23 <= 255 ; k_23 = k_23 + (hmpprt::gr_btnumx()))
    {
     # 203 "svd.cpp"
     result1_1 = result1_1 + *(ci + (j_2 * 2 + (start_index_1 + 1)) / 2 * 256 * 2 + k_23) * *(cj + (j_2 * 2 + (start_index_1 + 1)) / 2 * 256 * 2 + (k_23 + 256));
    }
    # 193 "svd.cpp"
    # 193 "svd.cpp"
    result1 = (reduce_double_add_x(result1_1, tmpr_2, (double) 0.));
    # 193 "svd.cpp"
    double result2_1;
    # 208 "svd.cpp"
    result2_1 = 0;
    # 208 "svd.cpp"
    hmpprt::s32 k_24;
    # 208 "svd.cpp"
    # 209 "svd.cpp"
    for (k_24 = (hmpprt::gr_btidx()) ; k_24 <= 255 ; k_24 = k_24 + (hmpprt::gr_btnumx()))
    {
     # 209 "svd.cpp"
     result2_1 = result2_1 + *(ci + (j_2 * 2 + (start_index_1 + 1)) / 2 * 256 * 2 + (k_24 + 256)) * *(cj + (j_2 * 2 + (start_index_1 + 1)) / 2 * 256 * 2 + k_24);
    }
    # 194 "svd.cpp"
    # 194 "svd.cpp"
    result2 = (reduce_double_add_x(result2_1, tmpr_2, (double) 0.));
    # 194 "svd.cpp"
    double result3_1;
    # 214 "svd.cpp"
    result3_1 = 0;
    # 214 "svd.cpp"
    hmpprt::s32 k_25;
    # 214 "svd.cpp"
    # 215 "svd.cpp"
    for (k_25 = (hmpprt::gr_btidx()) ; k_25 <= 255 ; k_25 = k_25 + (hmpprt::gr_btnumx()))
    {
     # 215 "svd.cpp"
     result3_1 = result3_1 + *(ci + (j_2 * 2 + (start_index_1 + 1)) / 2 * 256 * 2 + k_25) * *(cj + (j_2 * 2 + (start_index_1 + 1)) / 2 * 256 * 2 + k_25);
    }
    # 195 "svd.cpp"
    # 195 "svd.cpp"
    result3 = (reduce_double_add_x(result3_1, tmpr_2, (double) 0.));
    # 195 "svd.cpp"
    double result4_1;
    # 220 "svd.cpp"
    result4_1 = 0;
    # 220 "svd.cpp"
    hmpprt::s32 k_26;
    # 220 "svd.cpp"
    # 221 "svd.cpp"
    for (k_26 = (hmpprt::gr_btidx()) ; k_26 <= 255 ; k_26 = k_26 + (hmpprt::gr_btnumx()))
    {
     # 221 "svd.cpp"
     result4_1 = result4_1 + *(ci + (j_2 * 2 + (start_index_1 + 1)) / 2 * 256 * 2 + (k_26 + 256)) * *(cj + (j_2 * 2 + (start_index_1 + 1)) / 2 * 256 * 2 + (k_26 + 256));
    }
    # 224 "svd.cpp"
    # 224 "svd.cpp"
    result4 = (reduce_double_add_x(result4_1, tmpr_2, (double) 0.));
    # 224 "svd.cpp"
    double a;
    # 224 "svd.cpp"
    a = result1 - result2;
    # 225 "svd.cpp"
    double b;
    # 225 "svd.cpp"
    b = result3 + result4;
    # 227 "svd.cpp"
    double s_a2b2;
    # 50 "svd.cpp"
    hmpprt::s32 rvalue_1;
    # 50 "svd.cpp"
    if (b > (double) 0.)
    {
     # 53 "svd.cpp"
     rvalue_1 = 1;
     # 53 "svd.cpp"
     goto endf_2;
    }
    # 53 "svd.cpp"
    if (b == (double) 0.)
    {
     # 227 "svd.cpp"
     rvalue_1 = 0;
     # 227 "svd.cpp"
     goto endf_2;
    }
    # 227 "svd.cpp"
    rvalue_1 =  -1;
    # 227 "svd.cpp"
    endf_2:;
    # 227 "svd.cpp"
    s_a2b2 = (double ) (rvalue_1) * sqrt(a * a + b * b);
    # 228 "svd.cpp"
    double sin_alpha;
    # 228 "svd.cpp"
    sin_alpha = a / s_a2b2;
    # 229 "svd.cpp"
    double cos_alpha;
    # 229 "svd.cpp"
    cos_alpha = b / s_a2b2;
    # 230 "svd.cpp"
    double m;
    # 230 "svd.cpp"
    m =  - (b * cos_alpha + a * sin_alpha);
    # 196 "svd.cpp"
    double result5_1;
    # 235 "svd.cpp"
    result5_1 = 0;
    # 235 "svd.cpp"
    hmpprt::s32 k_27;
    # 235 "svd.cpp"
    # 236 "svd.cpp"
    for (k_27 = (hmpprt::gr_btidx()) ; k_27 <= 255 ; k_27 = k_27 + (hmpprt::gr_btnumx()))
    {
     # 236 "svd.cpp"
     result5_1 = result5_1 + *(cj + (j_2 * 2 + (start_index_1 + 1)) / 2 * 256 * 2 + k_27) * *(cj + (j_2 * 2 + (start_index_1 + 1)) / 2 * 256 * 2 + k_27);
    }
    # 197 "svd.cpp"
    # 197 "svd.cpp"
    result5 = (reduce_double_add_x(result5_1, tmpr_2, (double) 0.));
    # 197 "svd.cpp"
    double result6_1;
    # 241 "svd.cpp"
    result6_1 = 0;
    # 241 "svd.cpp"
    hmpprt::s32 k_28;
    # 241 "svd.cpp"
    # 242 "svd.cpp"
    for (k_28 = (hmpprt::gr_btidx()) ; k_28 <= 255 ; k_28 = k_28 + (hmpprt::gr_btnumx()))
    {
     # 242 "svd.cpp"
     result6_1 = result6_1 + *(cj + (j_2 * 2 + (start_index_1 + 1)) / 2 * 256 * 2 + (k_28 + 256)) * *(cj + (j_2 * 2 + (start_index_1 + 1)) / 2 * 256 * 2 + (k_28 + 256));
    }
    # 198 "svd.cpp"
    # 198 "svd.cpp"
    result6 = (reduce_double_add_x(result6_1, tmpr_2, (double) 0.));
    # 198 "svd.cpp"
    double result7_1;
    # 247 "svd.cpp"
    result7_1 = 0;
    # 247 "svd.cpp"
    hmpprt::s32 k_29;
    # 247 "svd.cpp"
    # 248 "svd.cpp"
    for (k_29 = (hmpprt::gr_btidx()) ; k_29 <= 255 ; k_29 = k_29 + (hmpprt::gr_btnumx()))
    {
     # 248 "svd.cpp"
     result7_1 = result7_1 + *(ci + (j_2 * 2 + (start_index_1 + 1)) / 2 * 256 * 2 + k_29) * *(ci + (j_2 * 2 + (start_index_1 + 1)) / 2 * 256 * 2 + k_29);
    }
    # 199 "svd.cpp"
    # 199 "svd.cpp"
    result7 = (reduce_double_add_x(result7_1, tmpr_2, (double) 0.));
    # 199 "svd.cpp"
    double result8_1;
    # 253 "svd.cpp"
    result8_1 = 0;
    # 253 "svd.cpp"
    hmpprt::s32 k_30;
    # 253 "svd.cpp"
    # 254 "svd.cpp"
    for (k_30 = (hmpprt::gr_btidx()) ; k_30 <= 255 ; k_30 = k_30 + (hmpprt::gr_btnumx()))
    {
     # 254 "svd.cpp"
     result8_1 = result8_1 + *(ci + (j_2 * 2 + (start_index_1 + 1)) / 2 * 256 * 2 + (k_30 + 256)) * *(ci + (j_2 * 2 + (start_index_1 + 1)) / 2 * 256 * 2 + (k_30 + 256));
    }
    # 257 "svd.cpp"
    # 257 "svd.cpp"
    result8 = (reduce_double_add_x(result8_1, tmpr_2, (double) 0.));
    # 257 "svd.cpp"
    double n_1;
    # 257 "svd.cpp"
    n_1 = (double) 0.5 * (result5 + result6 - result7 - result8);
    # 258 "svd.cpp"
    double w;
    # 50 "svd.cpp"
    hmpprt::s32 rvalue_2;
    # 50 "svd.cpp"
    if (n_1 > (double) 0.)
    {
     # 53 "svd.cpp"
     rvalue_2 = 1;
     # 53 "svd.cpp"
     goto endf_3;
    }
    # 53 "svd.cpp"
    if (n_1 == (double) 0.)
    {
     # 258 "svd.cpp"
     rvalue_2 = 0;
     # 258 "svd.cpp"
     goto endf_3;
    }
    # 258 "svd.cpp"
    rvalue_2 =  -1;
    # 258 "svd.cpp"
    endf_3:;
    # 258 "svd.cpp"
    w = (double ) (rvalue_2) * m / sqrt(m * m + n_1 * n_1);
    # 260 "svd.cpp"
    double sin_x;
    # 260 "svd.cpp"
    sin_x = w / sqrt((double) 2. * ((double) 1. + sqrt((double) 1. - w * w)));
    # 261 "svd.cpp"
    double cos_x;
    # 261 "svd.cpp"
    cos_x = sqrt((double) 1. - sin_x * sin_x);
    # 264 "svd.cpp"
    double real2;
    # 265 "svd.cpp"
    double imag2;
    # 266 "svd.cpp"
    double real3;
    # 267 "svd.cpp"
    double imag3;
    # 268 "svd.cpp"
    double real4;
    # 269 "svd.cpp"
    double imag4;
    # 270 "svd.cpp"
    double real5;
    # 271 "svd.cpp"
    double imag5;
    # 274 "svd.cpp"
    hmpprt::s32 k_31;
    # 274 "svd.cpp"
    # 275 "svd.cpp"
    for (k_31 = (hmpprt::gr_btidx()) ; k_31 <= 255 ; k_31 = k_31 + (hmpprt::gr_btnumx()))
    {
     # 276 "svd.cpp"
     real2 = *(ci + (j_2 * 2 + (start_index_1 + 1)) / 2 * 256 * 2 + k_31) * cos_x + *(cj + (j_2 * 2 + (start_index_1 + 1)) / 2 * 256 * 2 + k_31) * cos_alpha * sin_x + *(cj + (j_2 * 2 + (start_index_1 + 1)) / 2 * 256 * 2 + (k_31 + 256)) * sin_alpha * sin_x;
     # 277 "svd.cpp"
     imag2 = *(ci + (j_2 * 2 + (start_index_1 + 1)) / 2 * 256 * 2 + (k_31 + 256)) * cos_x + *(cj + (j_2 * 2 + (start_index_1 + 1)) / 2 * 256 * 2 + (k_31 + 256)) * cos_alpha * sin_x - *(cj + (j_2 * 2 + (start_index_1 + 1)) / 2 * 256 * 2 + k_31) * sin_alpha * sin_x;
     # 279 "svd.cpp"
     *(A_H_3 + (col_p * 256 + k_31)) = real2;
     # 280 "svd.cpp"
     *(A_H_3 + (65536 + col_p * 256 + k_31)) = imag2;
    }
    # 285 "svd.cpp"
    # 285 "svd.cpp"
    hmpprt::s32 k_32;
    # 285 "svd.cpp"
    # 286 "svd.cpp"
    for (k_32 = (hmpprt::gr_btidx()) ; k_32 <= 255 ; k_32 = k_32 + (hmpprt::gr_btnumx()))
    {
     # 287 "svd.cpp"
     real3 = *(cj + (j_2 * 2 + (start_index_1 + 1)) / 2 * 256 * 2 + k_32) * cos_x +  - *(ci + (j_2 * 2 + (start_index_1 + 1)) / 2 * 256 * 2 + k_32) * cos_alpha * sin_x + *(ci + (j_2 * 2 + (start_index_1 + 1)) / 2 * 256 * 2 + (k_32 + 256)) * sin_alpha * sin_x;
     # 288 "svd.cpp"
     imag3 = *(cj + (j_2 * 2 + (start_index_1 + 1)) / 2 * 256 * 2 + (k_32 + 256)) * cos_x +  - *(ci + (j_2 * 2 + (start_index_1 + 1)) / 2 * 256 * 2 + (k_32 + 256)) * cos_alpha * sin_x - *(ci + (j_2 * 2 + (start_index_1 + 1)) / 2 * 256 * 2 + k_32) * sin_alpha * sin_x;
     # 290 "svd.cpp"
     *(A_H_3 + (col_q * 256 + k_32)) = real3;
     # 291 "svd.cpp"
     *(A_H_3 + (65536 + col_q * 256 + k_32)) = imag3;
    }
    # 296 "svd.cpp"
    # 296 "svd.cpp"
    hmpprt::s32 k_33;
    # 296 "svd.cpp"
    # 297 "svd.cpp"
    for (k_33 = (hmpprt::gr_btidx()) ; k_33 <= 255 ; k_33 = k_33 + (hmpprt::gr_btnumx()))
    {
     # 298 "svd.cpp"
     real4 = *(vi + (j_2 * 2 + (start_index_1 + 1)) / 2 * 256 * 2 + k_33) * cos_x + *(vj + (j_2 * 2 + (start_index_1 + 1)) / 2 * 256 * 2 + k_33) * cos_alpha * sin_x + *(vj + (j_2 * 2 + (start_index_1 + 1)) / 2 * 256 * 2 + (k_33 + 256)) * sin_alpha * sin_x;
     # 299 "svd.cpp"
     imag4 = *(vi + (j_2 * 2 + (start_index_1 + 1)) / 2 * 256 * 2 + (k_33 + 256)) * cos_x + *(vj + (j_2 * 2 + (start_index_1 + 1)) / 2 * 256 * 2 + (k_33 + 256)) * cos_alpha * sin_x - *(vj + (j_2 * 2 + (start_index_1 + 1)) / 2 * 256 * 2 + k_33) * sin_alpha * sin_x;
     # 301 "svd.cpp"
     *(V_H + (col_p * 256 + k_33)) = real4;
     # 302 "svd.cpp"
     *(V_H + (65536 + col_p * 256 + k_33)) = imag4;
    }
    # 307 "svd.cpp"
    # 307 "svd.cpp"
    hmpprt::s32 k_34;
    # 307 "svd.cpp"
    # 308 "svd.cpp"
    for (k_34 = (hmpprt::gr_btidx()) ; k_34 <= 255 ; k_34 = k_34 + (hmpprt::gr_btnumx()))
    {
     # 309 "svd.cpp"
     real5 = *(vj + (j_2 * 2 + (start_index_1 + 1)) / 2 * 256 * 2 + k_34) * cos_x +  - *(vi + (j_2 * 2 + (start_index_1 + 1)) / 2 * 256 * 2 + k_34) * cos_alpha * sin_x + *(vi + (j_2 * 2 + (start_index_1 + 1)) / 2 * 256 * 2 + (k_34 + 256)) * sin_alpha * sin_x;
     # 310 "svd.cpp"
     imag5 = *(vj + (j_2 * 2 + (start_index_1 + 1)) / 2 * 256 * 2 + (k_34 + 256)) * cos_x +  - *(vi + (j_2 * 2 + (start_index_1 + 1)) / 2 * 256 * 2 + (k_34 + 256)) * cos_alpha * sin_x - *(vi + (j_2 * 2 + (start_index_1 + 1)) / 2 * 256 * 2 + k_34) * sin_alpha * sin_x;
     # 312 "svd.cpp"
     *(V_H + (col_q * 256 + k_34)) = real5;
     # 313 "svd.cpp"
     *(V_H + (65536 + col_q * 256 + k_34)) = imag5;
    }
    # 317 "svd.cpp"
   }
   # 317 "svd.cpp"
   *(rotate_col_idx + (j_2 * 2 + (start_index_1 + 1) - 1)) = col_q;
   # 318 "svd.cpp"
   *(rotate_col_idx + (j_2 * 2 + (start_index_1 + 1))) = col_p;
  }
  # 60 "svd.cpp"
 }
}
#endif // __HIPCC__



# 60 "svd.cpp"

#ifndef __HIPCC__
void hmpp_acc_region_svd_124_internal_1(hmpprt::s32* pass, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  A_H_6, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  V_H_1, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s32>  rotate_col_idx_2, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  ci_2, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  cj_1, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  vi_1, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  vj_1, hmpprt::s32 start_index_2)
{
 # 60 "svd.cpp"
 hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s32>  pass_2;
 # 60 "svd.cpp"
 hmpprt::Context::getInstance()->allocate((void **) (&pass_2), hmpprt::MS_CUDA_GLOB, 4);
 # 60 "svd.cpp"
 *pass_2 = *pass;
 # 60 "svd.cpp"
 hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  value_2;
 # 60 "svd.cpp"
 hmpprt::Context::getInstance()->allocate((void **) (&value_2), hmpprt::MS_CUDA_GLOB, 1024uLL);
 # 60 "svd.cpp"
 hmpprt::DevicePtr<hmpprt::MS_CUDA_SHARED,double>  tmpr_4;
 # 60 "svd.cpp"
 hmpprt::Context::getInstance()->allocate((void **) (&tmpr_4), hmpprt::MS_CUDA_SHARED, 2048LL);
 # 60 "svd.cpp"
 if (1)
 {
  hmpprt::CUDAGridCall __hmppcg_call;
  __hmppcg_call.setSizeX(192);
  __hmppcg_call.setSizeY(1);
  __hmppcg_call.setBlockSizeX(32);
  __hmppcg_call.setBlockSizeY(8LL);
  __hmppcg_call.addLocalParameter(&A_H_6, 8, "A_H_3");
  __hmppcg_call.addLocalParameter(&V_H_1, 8, "V_H");
  __hmppcg_call.addLocalParameter(&ci_2, 8, "ci");
  __hmppcg_call.addLocalParameter(&cj_1, 8, "cj");
  __hmppcg_call.addLocalParameter(&pass_2, 8, "pass_1");
  __hmppcg_call.addLocalParameter(&rotate_col_idx_2, 8, "rotate_col_idx");
  __hmppcg_call.addLocalParameter((hmpprt::s32) (start_index_2), "start_index_1");
  __hmppcg_call.addSharedParameter(tmpr_4, "tmpr_2");
  __hmppcg_call.addLocalParameter(&value_2, 8, "value_1");
  __hmppcg_call.addLocalParameter(&vi_1, 8, "vi");
  __hmppcg_call.addLocalParameter(&vj_1, 8, "vj");
  __hmppcg_call.launch(hmpp_acc_region_svd_124_parallel_region_1, hmpprt::Context::getInstance()->getCUDADevice());
 }
 ;
 # 60 "svd.cpp"
 *pass = *pass_2;
 # 60 "svd.cpp"
 hmpprt::Context::getInstance()->free((void **) (&pass_2));
 # 60 "svd.cpp"
 hmpprt::Context::getInstance()->free((void **) (&value_2));
 # 60 "svd.cpp"
 hmpprt::Context::getInstance()->free((void **) (&tmpr_4));
}
#endif // __HIPCC__



# 60 "svd.cpp"

#ifndef __HIPCC__
extern "C" CDLT_API  void hmpp_acc_region_svd_124(hmpprt::s32* pass_3, double* A_H, double* V_H_2, hmpprt::s32* rotate_col_idx_1, double* ci_4, double* cj_2, double* vi_2, double* vj_2, hmpprt::s32 start_index)
{
 # 325 "svd.cpp"
 (hmpp_acc_region_svd_124_internal_1(pass_3, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double> (A_H), hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double> (V_H_2), hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s32> (rotate_col_idx_1), hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double> (ci_4), hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double> (cj_2), hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double> (vi_2), hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double> (vj_2), start_index));
}
#endif // __HIPCC__



# 325 "svd.cpp"

#ifdef __HIPCC__

extern "C" __global__ void hmpp_acc_region_svd_332_parallel_region_1(double* A_H_2, double* S_H, double* ci_1)
{
 
 double * tmpr_1 = (double *)(((char *)hmpp_sharedmem + 0));
 
 # 327 "svd.cpp"
 {
  # 335 "svd.cpp"
  hmpprt::s32 first_gang_iter_2;
  # 335 "svd.cpp"
  hmpprt::s32 last_gang_iter_2;
  # 335 "svd.cpp"
  first_gang_iter_2 = (hmpprt::gr_gbidx()) * 8;
  # 335 "svd.cpp"
  last_gang_iter_2 = ((first_gang_iter_2 + 7) < 255 ? (first_gang_iter_2 + 7) : 255);
  # 335 "svd.cpp"
  hmpprt::s32 i_2;
  # 335 "svd.cpp"
  # 336 "svd.cpp"
  for (i_2 = first_gang_iter_2 + (hmpprt::gr_btidy()) ; i_2 <= last_gang_iter_2 ; i_2 = i_2 + (hmpprt::gr_btnumy()))
  {
   # 337 "svd.cpp"
   double real;
   # 338 "svd.cpp"
   double imag;
   # 341 "svd.cpp"
   hmpprt::s32 k_35;
   # 341 "svd.cpp"
   # 342 "svd.cpp"
   for (k_35 = (hmpprt::gr_btidx()) ; k_35 <= 255 ; k_35 = k_35 + (hmpprt::gr_btnumx()))
   {
    # 343 "svd.cpp"
    *(ci_1 + i_2 * 256 * 2 + k_35) = *(A_H_2 + (i_2 * 256 + k_35));
    # 344 "svd.cpp"
    *(ci_1 + i_2 * 256 * 2 + (256 + k_35)) = *(A_H_2 + (65536 + i_2 * 256 + k_35));
   }
   # 338 "svd.cpp"
   # 338 "svd.cpp"
   double imag_1;
   # 337 "svd.cpp"
   imag_1 = 0;
   # 337 "svd.cpp"
   double real_1;
   # 349 "svd.cpp"
   real_1 = 0;
   # 349 "svd.cpp"
   hmpprt::s32 l_1;
   # 349 "svd.cpp"
   # 350 "svd.cpp"
   for (l_1 = (hmpprt::gr_btidx()) ; l_1 <= 255 ; l_1 = l_1 + (hmpprt::gr_btnumx()))
   {
    # 351 "svd.cpp"
    real_1 = real_1 + (*(ci_1 + i_2 * 256 * 2 + l_1) * *(ci_1 + i_2 * 256 * 2 + l_1) + *(ci_1 + i_2 * 256 * 2 + (256 + l_1)) * *(ci_1 + i_2 * 256 * 2 + (256 + l_1)));
    # 352 "svd.cpp"
    imag_1 = imag_1 + (*(ci_1 + i_2 * 256 * 2 + l_1) * *(ci_1 + i_2 * 256 * 2 + (256 + l_1)) - *(ci_1 + i_2 * 256 * 2 + (256 + l_1)) * *(ci_1 + i_2 * 256 * 2 + l_1));
   }
   # 355 "svd.cpp"
   # 355 "svd.cpp"
   real = (reduce_double_add_x(real_1, tmpr_1, (double) 0.));
   # 355 "svd.cpp"
   imag = (reduce_double_add_x(imag_1, tmpr_1, (double) 0.));
   # 355 "svd.cpp"
   double norm_A;
   # 355 "svd.cpp"
   norm_A = sqrt(sqrt(real * real + imag * imag));
   # 356 "svd.cpp"
   *(S_H + i_2) = norm_A;
  }
  # 325 "svd.cpp"
 }
}
#endif // __HIPCC__



# 325 "svd.cpp"

#ifndef __HIPCC__
void hmpp_acc_region_svd_332_internal_1(hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  A_H_8, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  S_H_2, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  ci_5)
{
 # 325 "svd.cpp"
 hmpprt::DevicePtr<hmpprt::MS_CUDA_SHARED,double>  tmpr_3;
 # 325 "svd.cpp"
 hmpprt::Context::getInstance()->allocate((void **) (&tmpr_3), hmpprt::MS_CUDA_SHARED, 2048LL);
 # 325 "svd.cpp"
 if (1)
 {
  hmpprt::CUDAGridCall __hmppcg_call;
  __hmppcg_call.setSizeX(192);
  __hmppcg_call.setSizeY(1);
  __hmppcg_call.setBlockSizeX(32);
  __hmppcg_call.setBlockSizeY(8LL);
  __hmppcg_call.addLocalParameter(&A_H_8, 8, "A_H_2");
  __hmppcg_call.addLocalParameter(&S_H_2, 8, "S_H");
  __hmppcg_call.addLocalParameter(&ci_5, 8, "ci_1");
  __hmppcg_call.addSharedParameter(tmpr_3, "tmpr_1");
  __hmppcg_call.launch(hmpp_acc_region_svd_332_parallel_region_1, hmpprt::Context::getInstance()->getCUDADevice());
 }
 ;
 # 325 "svd.cpp"
 hmpprt::Context::getInstance()->free((void **) (&tmpr_3));
}
#endif // __HIPCC__



# 325 "svd.cpp"

#ifndef __HIPCC__
extern "C" CDLT_API  void hmpp_acc_region_svd_332(double* A_H_5, double* S_H_4, double* ci_3)
{
 # 363 "svd.cpp"
 (hmpp_acc_region_svd_332_internal_1(hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double> (A_H_5), hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double> (S_H_4), hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double> (ci_3)));
}
#endif // __HIPCC__



# 363 "svd.cpp"

#ifdef __HIPCC__

extern "C" __global__ void hmpp_acc_region_svd_363_parallel_region_1(double* A_H_4, double* B_H, double* S_H_1)
{
 # 365 "svd.cpp"
 {
  # 366 "svd.cpp"
  hmpprt::s32 first_gang_iter_1;
  # 366 "svd.cpp"
  hmpprt::s32 last_gang_iter_1;
  # 366 "svd.cpp"
  first_gang_iter_1 = (hmpprt::gr_gbidx()) * 342;
  # 366 "svd.cpp"
  last_gang_iter_1 = ((first_gang_iter_1 + 341) < 65535 ? (first_gang_iter_1 + 341) : 65535);
  # 366 "svd.cpp"
  hmpprt::s32 j_4;
  # 366 "svd.cpp"
  # 369 "svd.cpp"
  for (j_4 = first_gang_iter_1 + (hmpprt::gr_btidy()) ; j_4 <= last_gang_iter_1 ; j_4 = j_4 + (hmpprt::gr_btnumy()))
  {
   # 366 "svd.cpp"
   hmpprt::s32 j_3;
   # 368 "svd.cpp"
   hmpprt::s32 i_3;
   # 370 "svd.cpp"
   i_3 = j_4 % 256;
   # 370 "svd.cpp"
   j_3 = j_4 / 256;
   # 370 "svd.cpp"
   *(B_H + (j_3 * 256 + i_3)) = *(A_H_4 + (j_3 * 256 + i_3)) / *(S_H_1 + j_3);
   # 371 "svd.cpp"
   *(B_H + (65536 + j_3 * 256 + i_3)) = *(A_H_4 + (65536 + j_3 * 256 + i_3)) / *(S_H_1 + j_3);
  }
  # 363 "svd.cpp"
 }
}
#endif // __HIPCC__



# 363 "svd.cpp"

#ifndef __HIPCC__
void hmpp_acc_region_svd_363_internal_1(hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  A_H_7, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  B_H_1, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  S_H_5)
{
 # 363 "svd.cpp"
 if (1)
 {
  hmpprt::CUDAGridCall __hmppcg_call;
  __hmppcg_call.setSizeX(192);
  __hmppcg_call.setSizeY(1);
  __hmppcg_call.setBlockSizeX(1);
  __hmppcg_call.setBlockSizeY(256);
  __hmppcg_call.addLocalParameter(&A_H_7, 8, "A_H_4");
  __hmppcg_call.addLocalParameter(&B_H_1, 8, "B_H");
  __hmppcg_call.addLocalParameter(&S_H_5, 8, "S_H_1");
  __hmppcg_call.launch(hmpp_acc_region_svd_363_parallel_region_1, hmpprt::Context::getInstance()->getCUDADevice());
 }
 ;
}
#endif // __HIPCC__



# 363 "svd.cpp"

#ifndef __HIPCC__
extern "C" CDLT_API  void hmpp_acc_region_svd_363(double* A_H_1, double* B_H_2, double* S_H_3)
{
 # 1 "<preprocessor>"
 (hmpp_acc_region_svd_363_internal_1(hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double> (A_H_1), hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double> (B_H_2), hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double> (S_H_3)));
}
#endif // __HIPCC__



# 1 "<preprocessor>"

#ifdef __HIPCC__
__device__ double reduce_double_add_x(double priv, double* tmpa, double orig)
{
 # 1 "<preprocessor>"
 *(tmpa + (hmpprt::gr_btidy()) + (hmpprt::gr_btnumy()) * (hmpprt::gr_btidx())) = priv;
 # 1 "<preprocessor>"
 hmpprt::s32 n;
 # 1 "<preprocessor>"
 # 1 "<preprocessor>"
 for (n = ((hmpprt::gr_btnumx()) - 1 >> 1) + 1 ; n > 0 ; n = (n >> 1))
 {
  # 1 "<preprocessor>"
  hmpprt::s32 next;
  # 1 "<preprocessor>"
  next = (hmpprt::gr_btidx()) + n;
  # 1 "<preprocessor>"
  if ((hmpprt::gr_btidx()) < n && next < (hmpprt::gr_btnumx()))
  {
   # 1 "<preprocessor>"
   *(tmpa + (hmpprt::gr_btidy()) + (hmpprt::gr_btnumy()) * (hmpprt::gr_btidx())) = *(tmpa + (hmpprt::gr_btidy()) + (hmpprt::gr_btnumy()) * (hmpprt::gr_btidx())) + *(tmpa + (hmpprt::gr_btidy()) + (hmpprt::gr_btnumy()) * next);
  }
 }
 # 1 "<preprocessor>"
 # 1 "<preprocessor>"
 return *(tmpa + (hmpprt::gr_btidy())) + orig;
}
#endif // __HIPCC__




#ifndef __HIPCC__
extern "C" const char * hmpprt_cuda_get_gpu_code();

static hmpprt::CUDAModule * hmpprt_module = 0;
static int hmpprt_uses = 0;

extern "C" CDLT_API void * hmpprt_init()
{
  try
  {
    if (hmpprt_uses++ == 0)
    {
      hmpprt_module = new hmpprt::CUDAModule(hmpprt_cuda_get_gpu_code());
      hmpp_acc_region_svd_124_parallel_region_1 = new hmpprt::CUDAGrid(hmpprt_module, "hmpp_acc_region_svd_124_parallel_region_1");
      hmpp_acc_region_svd_332_parallel_region_1 = new hmpprt::CUDAGrid(hmpprt_module, "hmpp_acc_region_svd_332_parallel_region_1");
      hmpp_acc_region_svd_363_parallel_region_1 = new hmpprt::CUDAGrid(hmpprt_module, "hmpp_acc_region_svd_363_parallel_region_1");

    }
    hmpprt::Context::getInstance()->getGrouplet()->setTarget(hmpprt::CUDA);
    hmpprt::Context::getInstance()->getGrouplet()->addSignature("hmpp_acc_region_svd_124", "prototype hmpp_acc_region_svd_124(pass: ^host s32, A_H: ^cudaglob double, V_H: ^cudaglob double, rotate_col_idx: ^cudaglob s32, ci: ^cudaglob double, cj: ^cudaglob double, vi: ^cudaglob double, vj: ^cudaglob double, start_index: s32)");
    hmpprt::Context::getInstance()->getGrouplet()->addSignature("hmpp_acc_region_svd_332", "prototype hmpp_acc_region_svd_332(A_H: ^cudaglob double, S_H: ^cudaglob double, ci: ^cudaglob double)");
    hmpprt::Context::getInstance()->getGrouplet()->addSignature("hmpp_acc_region_svd_363", "prototype hmpp_acc_region_svd_363(A_H: ^cudaglob double, B_H: ^cudaglob double, S_H: ^cudaglob double)");

  }
  catch (hmpperr::Error & e)
  {
    return e.clone();
  }
  catch(...)
  {
    fprintf(stderr,"Unexpected error in hmpprt_init()\n");
    abort();
  }
  return 0;
}
#endif // __HIPCC__

#ifndef __HIPCC__
extern "C" CDLT_API void * hmpprt_fini()
{
  try
  {
    if (--hmpprt_uses == 0)
    {
      delete hmpp_acc_region_svd_124_parallel_region_1;
      delete hmpp_acc_region_svd_332_parallel_region_1;
      delete hmpp_acc_region_svd_363_parallel_region_1;

      delete hmpprt_module;
      hmpprt_module = 0;
    }
  }
  catch (hmpperr::Error & e)
  {
    return e.clone();
  }
  catch(...)
  {
    fprintf(stderr,"Unexpected error in hmpprt_fini()\n");
    abort();
  }
  return 0;
}
#endif // __HIPCC__

// footer
