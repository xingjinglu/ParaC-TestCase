#include "hip/hip_runtime.h"

#include <stdio.h>

#ifndef __HIPCC__
#include <stdlib.h>
#include <math.h>

#include <hmpprt/Grouplet.h>
#include <hmpprt/HostTypes.h>
#include <hmpprt/Context.h>
#include <hmpprt/CUDAGrid.h>
#include <hmpprt/CUDAModule.h>
#include <hmpprt/DeviceManager.h>
#include <hmpperr/hmpperr.h>

#include <openacci/openacci_c.h>

#ifdef _WIN32
#  define CDLT_API __declspec(dllexport)
#else /* ! _WIN32 */
#  define CDLT_API
#endif /* _WIN32 */



#else // ! __HIPCC__

#include <hmpprt/HostTypes.h>
#include <hmpprt/CUDAIntrinsics.h>

extern __shared__ int64_t hmpp_sharedmem[];
#endif // __HIPCC__



#ifndef __HIPCC__

#else


#endif

#define HMPPCG_SIMD_LENGTH 32

# 12 "mt.cpp"

#ifndef __HIPCC__
extern "C" CDLT_API  void hmpp_acc_region_main_44(hmpprt::s32 n_2, float* A_2, float* B_2)
;
#endif // __HIPCC__



# 12 "mt.cpp"

#ifndef __HIPCC__
void hmpp_acc_region_main_44_internal_1(hmpprt::s32 n, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,float>  A, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,float>  B)
;
#endif // __HIPCC__



# 12 "mt.cpp"

#ifndef __HIPCC__
static hmpprt::CUDAGrid * hmpp_acc_region_main_44_parallel_region_1 = 0;
#else

extern "C" __global__ void hmpp_acc_region_main_44_parallel_region_1(float* A_1, float* B_1, hmpprt::s32 n_1);
#endif // __HIPCC__




# 12 "mt.cpp"

#ifdef __HIPCC__

extern "C" __global__ void hmpp_acc_region_main_44_parallel_region_1(float* A_1, float* B_1, hmpprt::s32 n_1)
{
 # 14 "mt.cpp"
 {
  # 45 "mt.cpp"
  hmpprt::s32 iter_per_gang_1;
  # 45 "mt.cpp"
  hmpprt::s32 first_gang_iter_1;
  # 45 "mt.cpp"
  hmpprt::s32 last_gang_iter_1;
  # 45 "mt.cpp"
  iter_per_gang_1 = ((1 + (n_1 * n_1 - 1) / 192) > 256 ? (1 + (n_1 * n_1 - 1) / 192) : 256);
  # 45 "mt.cpp"
  first_gang_iter_1 = (hmpprt::gr_gbidx()) * iter_per_gang_1;
  # 45 "mt.cpp"
  last_gang_iter_1 = ((first_gang_iter_1 + iter_per_gang_1 - 1) < (n_1 * n_1 - 1) ? (first_gang_iter_1 + iter_per_gang_1 - 1) : (n_1 * n_1 - 1));
  # 45 "mt.cpp"
  hmpprt::s32 i_2;
  # 45 "mt.cpp"
  # 49 "mt.cpp"
  for (i_2 = first_gang_iter_1 + (hmpprt::gr_btidy()) ; i_2 <= last_gang_iter_1 ; i_2 = i_2 + (hmpprt::gr_btnumy()))
  {
   # 45 "mt.cpp"
   hmpprt::s32 i_1;
   # 48 "mt.cpp"
   hmpprt::s32 j_1;
   # 50 "mt.cpp"
   j_1 = i_2 % n_1;
   # 50 "mt.cpp"
   i_1 = i_2 / n_1;
   # 50 "mt.cpp"
   *(B_1 + (j_1 * n_1 + i_1)) = *(A_1 + (i_1 * n_1 + j_1));
  }
  # 12 "mt.cpp"
 }
}
#endif // __HIPCC__



# 12 "mt.cpp"

#ifndef __HIPCC__
void hmpp_acc_region_main_44_internal_1(hmpprt::s32 n, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,float>  A, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,float>  B)
{
 # 12 "mt.cpp"
 if (1)
 {
  hmpprt::CUDAGridCall __hmppcg_call;
  __hmppcg_call.setSizeX(192);
  __hmppcg_call.setSizeY(1);
  __hmppcg_call.setBlockSizeX(1);
  __hmppcg_call.setBlockSizeY(256);
  __hmppcg_call.addLocalParameter(&A, 8, "A_1");
  __hmppcg_call.addLocalParameter(&B, 8, "B_1");
  __hmppcg_call.addLocalParameter((hmpprt::s32) (n), "n_1");
  __hmppcg_call.launch(hmpp_acc_region_main_44_parallel_region_1, hmpprt::Context::getInstance()->getCUDADevice());
 }
 ;
}
#endif // __HIPCC__



# 12 "mt.cpp"

#ifndef __HIPCC__
extern "C" CDLT_API  void hmpp_acc_region_main_44(hmpprt::s32 n_2, float* A_2, float* B_2)
{
 # 1 "<preprocessor>"
 (hmpp_acc_region_main_44_internal_1(n_2, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,float> (A_2), hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,float> (B_2)));
}
#endif // __HIPCC__




#ifndef __HIPCC__
extern "C" const char * hmpprt_cuda_get_gpu_code();

static hmpprt::CUDAModule * hmpprt_module = 0;
static int hmpprt_uses = 0;

extern "C" CDLT_API void * hmpprt_init()
{
  try
  {
    if (hmpprt_uses++ == 0)
    {
      hmpprt_module = new hmpprt::CUDAModule(hmpprt_cuda_get_gpu_code());
      hmpp_acc_region_main_44_parallel_region_1 = new hmpprt::CUDAGrid(hmpprt_module, "hmpp_acc_region_main_44_parallel_region_1");

    }
    hmpprt::Context::getInstance()->getGrouplet()->setTarget(hmpprt::CUDA);
    hmpprt::Context::getInstance()->getGrouplet()->addSignature("hmpp_acc_region_main_44", "prototype hmpp_acc_region_main_44(n: s32, A: ^cudaglob float, B: ^cudaglob float)");

  }
  catch (hmpperr::Error & e)
  {
    return e.clone();
  }
  catch(...)
  {
    fprintf(stderr,"Unexpected error in hmpprt_init()\n");
    abort();
  }
  return 0;
}
#endif // __HIPCC__

#ifndef __HIPCC__
extern "C" CDLT_API void * hmpprt_fini()
{
  try
  {
    if (--hmpprt_uses == 0)
    {
      delete hmpp_acc_region_main_44_parallel_region_1;

      delete hmpprt_module;
      hmpprt_module = 0;
    }
  }
  catch (hmpperr::Error & e)
  {
    return e.clone();
  }
  catch(...)
  {
    fprintf(stderr,"Unexpected error in hmpprt_fini()\n");
    abort();
  }
  return 0;
}
#endif // __HIPCC__

// footer
