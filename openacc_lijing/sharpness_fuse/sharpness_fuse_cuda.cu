#include "hip/hip_runtime.h"

#include <stdio.h>

#ifndef __HIPCC__
#include <stdlib.h>
#include <math.h>

#include <hmpprt/Grouplet.h>
#include <hmpprt/HostTypes.h>
#include <hmpprt/Context.h>
#include <hmpprt/CUDAGrid.h>
#include <hmpprt/CUDAModule.h>
#include <hmpprt/DeviceManager.h>
#include <hmpperr/hmpperr.h>

#include <openacci/openacci_c.h>

#ifdef _WIN32
#  define CDLT_API __declspec(dllexport)
#else /* ! _WIN32 */
#  define CDLT_API
#endif /* _WIN32 */



#else // ! __HIPCC__

#include <hmpprt/HostTypes.h>
#include <hmpprt/CUDAIntrinsics.h>

extern __shared__ int64_t hmpp_sharedmem[];
#endif // __HIPCC__



#ifndef __HIPCC__

#else


#endif

#define HMPPCG_SIMD_LENGTH 1

# 23 "sharpness_fuse.cpp"

#ifdef __HIPCC__
__device__ hmpprt::s64 reduce_s64_add_y(hmpprt::s64 priv, hmpprt::s64* tmpa_1, hmpprt::s64 orig)
;
#endif // __HIPCC__



# 23 "sharpness_fuse.cpp"

#ifndef __HIPCC__
extern "C" CDLT_API  void hmpp_acc_region_main_82(hmpprt::s32 width_32, hmpprt::s32 height_16, hmpprt::s32 quarterWidth_9, hmpprt::u08* yPlane_6, hmpprt::u08* yPlaneDown_7)
;
#endif // __HIPCC__



# 23 "sharpness_fuse.cpp"

#ifndef __HIPCC__
void hmpp_acc_region_main_82_internal_1(hmpprt::s32 width_60, hmpprt::s32 height_30, hmpprt::s32 quarterWidth_23, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  yPlane_11, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  yPlaneDown_15)
;
#endif // __HIPCC__



# 458 "sharpness_fuse.cpp"

#ifndef __HIPCC__
static hmpprt::CUDAGrid * hmpp_acc_region_main_82_parallel_region_1 = 0;
#else

extern "C" __global__ void hmpp_acc_region_main_82_parallel_region_1(hmpprt::s32 height_1, hmpprt::s32 quarterWidth_1, hmpprt::s32 width_1, hmpprt::u08* yPlaneDown_1, hmpprt::u08* yPlane_1);
#endif // __HIPCC__




# 458 "sharpness_fuse.cpp"

#ifndef __HIPCC__
extern "C" CDLT_API  void hmpp_acc_region_main_462(hmpprt::s32 width_34, hmpprt::s32 height_20, hmpprt::u08* yPlane_7, hmpprt::s16* pSharpness_4)
;
#endif // __HIPCC__



# 458 "sharpness_fuse.cpp"

#ifndef __HIPCC__
void hmpp_acc_region_main_462_internal_1(hmpprt::s32 width_61, hmpprt::s32 height_31, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  yPlane_12, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s16>  pSharpness_8)
;
#endif // __HIPCC__



# 417 "sharpness_fuse.cpp"

#ifndef __HIPCC__
static hmpprt::CUDAGrid * hmpp_acc_region_main_462_parallel_region_1 = 0;
#else

extern "C" __global__ void hmpp_acc_region_main_462_parallel_region_1(hmpprt::s32 height_2, hmpprt::s16* pSharpness_1, hmpprt::s32 width_2, hmpprt::u08* yPlane_2);
#endif // __HIPCC__




# 417 "sharpness_fuse.cpp"

#ifndef __HIPCC__
extern "C" CDLT_API  void hmpp_acc_region_main_408(hmpprt::s32 width_43, hmpprt::s32 height_21, hmpprt::u08* yPlane_8, hmpprt::s16* pEdge_5, hmpprt::s16* pSharpness_2, float cutoff3_1, float order3_2, float peak3, float low3_1)
;
#endif // __HIPCC__



# 417 "sharpness_fuse.cpp"

#ifndef __HIPCC__
void hmpp_acc_region_main_408_internal_1(hmpprt::s32 width_62, hmpprt::s32 height_32, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  yPlane_13, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s16>  pEdge_9, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s16>  pSharpness_6, float cutoff3_2, float order3_1, float peak3_2, float low3_2)
;
#endif // __HIPCC__



# 392 "sharpness_fuse.cpp"

#ifndef __HIPCC__
static hmpprt::CUDAGrid * hmpp_acc_region_main_408_parallel_region_1 = 0;
#else

extern "C" __global__ void hmpp_acc_region_main_408_parallel_region_1(float cutoff3, hmpprt::s32 height_3, float low3, float order3, hmpprt::s16* pEdge_1, hmpprt::s16* pSharpness_5, float peak3_1, hmpprt::s32 width_3, hmpprt::u08* yPlane_3);
#endif // __HIPCC__




# 392 "sharpness_fuse.cpp"

#ifndef __HIPCC__
extern "C" CDLT_API  void hmpp_acc_region_main_389(hmpprt::s32 width_45, hmpprt::s32 height_22, hmpprt::u08* yPlaneCSER_16, hmpprt::s16* pError_3, hmpprt::s16* pEdge_6, hmpprt::s16* pSharpness_3, double cutoff_1, double peak_1, double cutoff2_1, double order_1, double order2_1, double low_1)
;
#endif // __HIPCC__



# 392 "sharpness_fuse.cpp"

#ifndef __HIPCC__
void hmpp_acc_region_main_389_internal_1(hmpprt::s32 width_63, hmpprt::s32 height_33, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  yPlaneCSER_35, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s16>  pError_5, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s16>  pEdge_10, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s16>  pSharpness_7, double cutoff_2, double peak_2, double cutoff2_2, double order_2, double order2_2, double low_2)
;
#endif // __HIPCC__



# 361 "sharpness_fuse.cpp"

#ifndef __HIPCC__
static hmpprt::CUDAGrid * hmpp_acc_region_main_389_parallel_region_1 = 0;
#else

extern "C" __global__ void hmpp_acc_region_main_389_parallel_region_1(double cutoff, double cutoff2, hmpprt::s32 height_4, double low, double order, double order2, hmpprt::s16* pEdge_2, hmpprt::s16* pError_1, hmpprt::s16* pSharpness, double peak, hmpprt::s32 width_4, hmpprt::u08* yPlaneCSER_1);
#endif // __HIPCC__




# 361 "sharpness_fuse.cpp"

#ifndef __HIPCC__
extern "C" CDLT_API  void hmpp_acc_region_main_376(hmpprt::s32 width_46, hmpprt::s32 height_23, hmpprt::u08* yPlane_9, hmpprt::u08* yPlaneCSER_22, hmpprt::s16* pError_2)
;
#endif // __HIPCC__



# 361 "sharpness_fuse.cpp"

#ifndef __HIPCC__
void hmpp_acc_region_main_376_internal_1(hmpprt::s32 width_64, hmpprt::s32 height_34, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  yPlane_14, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  yPlaneCSER_37, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s16>  pError_4)
;
#endif // __HIPCC__



# 342 "sharpness_fuse.cpp"

#ifndef __HIPCC__
static hmpprt::CUDAGrid * hmpp_acc_region_main_376_parallel_region_1 = 0;
#else

extern "C" __global__ void hmpp_acc_region_main_376_parallel_region_1(hmpprt::s32 height_5, hmpprt::s16* pError, hmpprt::s32 width_5, hmpprt::u08* yPlaneCSER_2, hmpprt::u08* yPlane_4);
#endif // __HIPCC__




# 342 "sharpness_fuse.cpp"

#ifndef __HIPCC__
extern "C" CDLT_API  void hmpp_acc_region_main_339(hmpprt::s32 width_47, hmpprt::s32 height_24, hmpprt::s16* pEdge_7, hmpprt::s64* sum)
;
#endif // __HIPCC__



# 342 "sharpness_fuse.cpp"

#ifndef __HIPCC__
void hmpp_acc_region_main_339_internal_1(hmpprt::s32 width_65, hmpprt::s32 height_35, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s16>  pEdge_11, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s64>  sum_4)
;
#endif // __HIPCC__



# 313 "sharpness_fuse.cpp"

#ifndef __HIPCC__
static hmpprt::CUDAGrid * hmpp_acc_region_main_339_parallel_region_1 = 0;
#else

extern "C" __global__ void hmpp_acc_region_main_339_parallel_region_1(hmpprt::s64* gang_prv_red_sum_1, hmpprt::s32 height_6, hmpprt::s16* pEdge_3, hmpprt::s32 width_6);
#endif // __HIPCC__




# 313 "sharpness_fuse.cpp"

#ifndef __HIPCC__
static hmpprt::CUDAGrid * reduce_s64_add_xy = 0;
#else

extern "C" __global__ void reduce_s64_add_xy(hmpprt::s64 neutral, hmpprt::s64* scalar, hmpprt::s32 size, hmpprt::s64* array);
#endif // __HIPCC__




# 313 "sharpness_fuse.cpp"

#ifndef __HIPCC__
extern "C" CDLT_API  void hmpp_acc_region_main_316(hmpprt::s32 width_48, hmpprt::s32 height_25, hmpprt::u08* yPlane_5, hmpprt::s16* pEdge_4)
;
#endif // __HIPCC__



# 313 "sharpness_fuse.cpp"

#ifndef __HIPCC__
void hmpp_acc_region_main_316_internal_1(hmpprt::s32 width_66, hmpprt::s32 height_36, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  yPlane_10, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s16>  pEdge_8)
;
#endif // __HIPCC__



# 279 "sharpness_fuse.cpp"

#ifndef __HIPCC__
static hmpprt::CUDAGrid * hmpp_acc_region_main_316_parallel_region_1 = 0;
#else

extern "C" __global__ void hmpp_acc_region_main_316_parallel_region_1(hmpprt::s32 height_7, hmpprt::s16* pEdge, hmpprt::s32 width_7, hmpprt::u08* yPlane);
#endif // __HIPCC__




# 279 "sharpness_fuse.cpp"

#ifndef __HIPCC__
extern "C" CDLT_API  void hmpp_acc_region_main_279(hmpprt::s32* width_8, hmpprt::s32 quarterWidth_17, hmpprt::s32 quarterHeight_7, hmpprt::u08* yPlaneCSER_23, hmpprt::u08* yPlaneDown_8)
;
#endif // __HIPCC__



# 279 "sharpness_fuse.cpp"

#ifndef __HIPCC__
void hmpp_acc_region_main_279_internal_1(hmpprt::s32* width_67, hmpprt::s32 quarterWidth_10, hmpprt::s32 quarterHeight_15, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  yPlaneCSER_38, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  yPlaneDown_16)
;
#endif // __HIPCC__



# 267 "sharpness_fuse.cpp"

#ifndef __HIPCC__
static hmpprt::CUDAGrid * hmpp_acc_region_main_279_loop1D_1 = 0;
#else

extern "C" __global__ void hmpp_acc_region_main_279_loop1D_1(hmpprt::s32 quarterWidth_2, hmpprt::s32 quarterHeight_1, hmpprt::u08* yPlaneCSER_3, hmpprt::u08* yPlaneDown_2, hmpprt::s32 width_30);
#endif // __HIPCC__




# 267 "sharpness_fuse.cpp"

#ifndef __HIPCC__
extern "C" CDLT_API  void hmpp_acc_region_main_267(hmpprt::s32* width_9, hmpprt::s32 height_8, hmpprt::u08* yPlaneCSER_4)
;
#endif // __HIPCC__



# 267 "sharpness_fuse.cpp"

#ifndef __HIPCC__
void hmpp_acc_region_main_267_internal_1(hmpprt::s32* width_68, hmpprt::s32 height_37, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  yPlaneCSER_44)
;
#endif // __HIPCC__



# 244 "sharpness_fuse.cpp"

#ifndef __HIPCC__
static hmpprt::CUDAGrid * hmpp_acc_region_main_267_loop1D_1 = 0;
#else

extern "C" __global__ void hmpp_acc_region_main_267_loop1D_1(hmpprt::s32 width_42, hmpprt::s32 height_26, hmpprt::u08* yPlaneCSER_24);
#endif // __HIPCC__




# 244 "sharpness_fuse.cpp"

#ifndef __HIPCC__
extern "C" CDLT_API  void hmpp_acc_region_main_244(hmpprt::s32* width_10, hmpprt::s32 quarterHeight_16, hmpprt::u08* yPlaneCSER_39)
;
#endif // __HIPCC__



# 244 "sharpness_fuse.cpp"

#ifndef __HIPCC__
void hmpp_acc_region_main_244_internal_1(hmpprt::s32* width_50, hmpprt::s32 quarterHeight_8, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  yPlaneCSER_25)
;
#endif // __HIPCC__



# 238 "sharpness_fuse.cpp"

#ifndef __HIPCC__
static hmpprt::CUDAGrid * hmpp_acc_region_main_244_loop1D_1 = 0;
#else

extern "C" __global__ void hmpp_acc_region_main_244_loop1D_1(hmpprt::s32 quarterHeight_2, hmpprt::u08* yPlaneCSER_5, hmpprt::s32 width_28);
#endif // __HIPCC__




# 238 "sharpness_fuse.cpp"

#ifndef __HIPCC__
extern "C" CDLT_API  void hmpp_acc_region_main_232(hmpprt::s32* width_11, hmpprt::s32 quarterWidth_3, hmpprt::s32 quarterHeight_9, hmpprt::u08* yPlaneCSER_26, hmpprt::u08* yPlaneDown_9)
;
#endif // __HIPCC__



# 238 "sharpness_fuse.cpp"

#ifndef __HIPCC__
void hmpp_acc_region_main_232_internal_1(hmpprt::s32* width_51, hmpprt::s32 quarterWidth_21, hmpprt::s32 quarterHeight_17, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  yPlaneCSER_40, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  yPlaneDown_17)
;
#endif // __HIPCC__



# 223 "sharpness_fuse.cpp"

#ifndef __HIPCC__
static hmpprt::CUDAGrid * hmpp_acc_region_main_232_loop1D_1 = 0;
#else

extern "C" __global__ void hmpp_acc_region_main_232_loop1D_1(hmpprt::s32 quarterWidth_11, hmpprt::s32 quarterHeight_3, hmpprt::u08* yPlaneCSER_6, hmpprt::u08* yPlaneDown_3, hmpprt::s32 width_27);
#endif // __HIPCC__




# 223 "sharpness_fuse.cpp"

#ifndef __HIPCC__
extern "C" CDLT_API  void hmpp_acc_region_main_220(hmpprt::s32* width_12, hmpprt::s32* height_38, hmpprt::u08* yPlaneCSER_27)
;
#endif // __HIPCC__



# 223 "sharpness_fuse.cpp"

#ifndef __HIPCC__
void hmpp_acc_region_main_220_internal_1(hmpprt::s32* width_52, hmpprt::s32* height_9, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  yPlaneCSER_41)
;
#endif // __HIPCC__



# 216 "sharpness_fuse.cpp"

#ifndef __HIPCC__
static hmpprt::CUDAGrid * hmpp_acc_region_main_220_loop1D_1 = 0;
#else

extern "C" __global__ void hmpp_acc_region_main_220_loop1D_1(hmpprt::u08* yPlaneCSER_7, hmpprt::s32 width_26, hmpprt::s32 height_14);
#endif // __HIPCC__




# 216 "sharpness_fuse.cpp"

#ifndef __HIPCC__
extern "C" CDLT_API  void hmpp_acc_region_main_211(hmpprt::s32* width_13, hmpprt::u08* yPlaneCSER_28)
;
#endif // __HIPCC__



# 216 "sharpness_fuse.cpp"

#ifndef __HIPCC__
void hmpp_acc_region_main_211_internal_1(hmpprt::s32* width_53, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  yPlaneCSER_42)
;
#endif // __HIPCC__



# 193 "sharpness_fuse.cpp"

#ifndef __HIPCC__
static hmpprt::CUDAGrid * hmpp_acc_region_main_211_loop1D_1 = 0;
#else

extern "C" __global__ void hmpp_acc_region_main_211_loop1D_1(hmpprt::u08* yPlaneCSER_8, hmpprt::s32 width_25);
#endif // __HIPCC__




# 193 "sharpness_fuse.cpp"

#ifndef __HIPCC__
extern "C" CDLT_API  void hmpp_acc_region_main_189(hmpprt::s32* width_14, hmpprt::s32* height_10, hmpprt::s32 quarterWidth_12, hmpprt::u08* yPlaneCSER_29)
;
#endif // __HIPCC__



# 193 "sharpness_fuse.cpp"

#ifndef __HIPCC__
void hmpp_acc_region_main_189_internal_1(hmpprt::s32* width_54, hmpprt::s32* height_28, hmpprt::s32 quarterWidth_18, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  yPlaneCSER_43)
;
#endif // __HIPCC__



# 177 "sharpness_fuse.cpp"

#ifndef __HIPCC__
static hmpprt::CUDAGrid * hmpp_acc_region_main_189_loop1D_1 = 0;
#else

extern "C" __global__ void hmpp_acc_region_main_189_loop1D_1(hmpprt::s32 quarterWidth_4, hmpprt::u08* yPlaneCSER_9, hmpprt::s32 width_24, hmpprt::s32 height_13);
#endif // __HIPCC__




# 177 "sharpness_fuse.cpp"

#ifndef __HIPCC__
extern "C" CDLT_API  void hmpp_acc_region_main_172(hmpprt::s32* width_55, hmpprt::s32 quarterWidth_13, hmpprt::u08* yPlaneCSER_31)
;
#endif // __HIPCC__



# 177 "sharpness_fuse.cpp"

#ifndef __HIPCC__
void hmpp_acc_region_main_172_internal_1(hmpprt::s32* width_15, hmpprt::s32 quarterWidth_19, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  yPlaneCSER_10)
;
#endif // __HIPCC__



# 163 "sharpness_fuse.cpp"

#ifndef __HIPCC__
static hmpprt::CUDAGrid * hmpp_acc_region_main_172_loop1D_1 = 0;
#else

extern "C" __global__ void hmpp_acc_region_main_172_loop1D_1(hmpprt::s32 quarterWidth_5, hmpprt::u08* yPlaneCSER_17);
#endif // __HIPCC__




# 163 "sharpness_fuse.cpp"

#ifndef __HIPCC__
extern "C" CDLT_API  void hmpp_acc_region_main_159(hmpprt::s32* width_56, hmpprt::s32* height_29, hmpprt::s32 quarterWidth_14, hmpprt::s32 quarterHeight_13, hmpprt::u08* yPlaneCSER_36, hmpprt::u08* yPlaneDown_13)
;
#endif // __HIPCC__



# 163 "sharpness_fuse.cpp"

#ifndef __HIPCC__
void hmpp_acc_region_main_159_internal_1(hmpprt::s32* width_16, hmpprt::s32* height_11, hmpprt::s32 quarterWidth_20, hmpprt::s32 quarterHeight_10, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  yPlaneCSER_18, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  yPlaneDown_4)
;
#endif // __HIPCC__



# 150 "sharpness_fuse.cpp"

#ifndef __HIPCC__
static hmpprt::CUDAGrid * hmpp_acc_region_main_159_loop1D_1 = 0;
#else

extern "C" __global__ void hmpp_acc_region_main_159_loop1D_1(hmpprt::s32 width_22, hmpprt::s32 height_12, hmpprt::s32 quarterWidth_6, hmpprt::s32 quarterHeight_4, hmpprt::u08* yPlaneCSER_11, hmpprt::u08* yPlaneDown_10);
#endif // __HIPCC__




# 150 "sharpness_fuse.cpp"

#ifndef __HIPCC__
extern "C" CDLT_API  void hmpp_acc_region_main_151(hmpprt::s32* width_57, hmpprt::s32 quarterWidth_22, hmpprt::u08* yPlaneCSER_32, hmpprt::u08* yPlaneDown_14)
;
#endif // __HIPCC__



# 150 "sharpness_fuse.cpp"

#ifndef __HIPCC__
void hmpp_acc_region_main_151_internal_1(hmpprt::s32* width_17, hmpprt::s32 quarterWidth_7, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  yPlaneCSER_12, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  yPlaneDown_5)
;
#endif // __HIPCC__



# 138 "sharpness_fuse.cpp"

#ifndef __HIPCC__
static hmpprt::CUDAGrid * hmpp_acc_region_main_151_loop1D_1 = 0;
#else

extern "C" __global__ void hmpp_acc_region_main_151_loop1D_1(hmpprt::s32 quarterWidth_15, hmpprt::u08* yPlaneCSER_19, hmpprt::u08* yPlaneDown_11);
#endif // __HIPCC__




# 138 "sharpness_fuse.cpp"

#ifndef __HIPCC__
extern "C" CDLT_API  void hmpp_acc_region_main_138(hmpprt::s32* width_18, hmpprt::s32 height_27, hmpprt::u08* yPlaneCSER_33)
;
#endif // __HIPCC__



# 138 "sharpness_fuse.cpp"

#ifndef __HIPCC__
void hmpp_acc_region_main_138_internal_1(hmpprt::s32* width_58, hmpprt::s32 height, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  yPlaneCSER_13)
;
#endif // __HIPCC__



# 117 "sharpness_fuse.cpp"

#ifndef __HIPCC__
static hmpprt::CUDAGrid * hmpp_acc_region_main_138_loop1D_1 = 0;
#else

extern "C" __global__ void hmpp_acc_region_main_138_loop1D_1(hmpprt::s32 width_35, hmpprt::s32 height_15, hmpprt::u08* yPlaneCSER_20);
#endif // __HIPCC__




# 117 "sharpness_fuse.cpp"

#ifndef __HIPCC__
extern "C" CDLT_API  void hmpp_acc_region_main_117(hmpprt::s32 width_19, hmpprt::s32 quarterHeight_5, hmpprt::u08* yPlaneCSER_34)
;
#endif // __HIPCC__



# 117 "sharpness_fuse.cpp"

#ifndef __HIPCC__
void hmpp_acc_region_main_117_internal_1(hmpprt::s32 width_59, hmpprt::s32 quarterHeight_14, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  yPlaneCSER_14)
;
#endif // __HIPCC__



# 101 "sharpness_fuse.cpp"

#ifndef __HIPCC__
static hmpprt::CUDAGrid * hmpp_acc_region_main_117_parallel_region_1 = 0;
#else

extern "C" __global__ void hmpp_acc_region_main_117_parallel_region_1(hmpprt::s32 quarterHeight_11, hmpprt::s32 width_33, hmpprt::u08* yPlaneCSER_21);
#endif // __HIPCC__




# 101 "sharpness_fuse.cpp"

#ifndef __HIPCC__
extern "C" CDLT_API  void hmpp_acc_region_main_105(hmpprt::s32 width, hmpprt::s32 quarterWidth, hmpprt::s32 quarterHeight, hmpprt::u08* yPlaneCSER_30, hmpprt::u08* yPlaneDown)
;
#endif // __HIPCC__



# 101 "sharpness_fuse.cpp"

#ifndef __HIPCC__
void hmpp_acc_region_main_105_internal_1(hmpprt::s32 width_49, hmpprt::s32 quarterWidth_16, hmpprt::s32 quarterHeight_12, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  yPlaneCSER, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  yPlaneDown_12)
;
#endif // __HIPCC__



# 101 "sharpness_fuse.cpp"

#ifndef __HIPCC__
static hmpprt::CUDAGrid * hmpp_acc_region_main_105_parallel_region_1 = 0;
#else

extern "C" __global__ void hmpp_acc_region_main_105_parallel_region_1(hmpprt::s32 quarterHeight_6, hmpprt::s32 quarterWidth_8, hmpprt::s32 width_31, hmpprt::u08* yPlaneCSER_15, hmpprt::u08* yPlaneDown_6);
#endif // __HIPCC__




# 101 "sharpness_fuse.cpp"

#ifdef __HIPCC__

extern "C" __global__ void hmpp_acc_region_main_105_parallel_region_1(hmpprt::s32 quarterHeight_6, hmpprt::s32 quarterWidth_8, hmpprt::s32 width_31, hmpprt::u08* yPlaneCSER_15, hmpprt::u08* yPlaneDown_6)
{
 # 103 "sharpness_fuse.cpp"
 {
  # 108 "sharpness_fuse.cpp"
  hmpprt::s32 iter_per_gang_9;
  # 108 "sharpness_fuse.cpp"
  hmpprt::s32 first_gang_iter_9;
  # 108 "sharpness_fuse.cpp"
  hmpprt::s32 last_gang_iter_9;
  # 108 "sharpness_fuse.cpp"
  iter_per_gang_9 = ((1 + (quarterHeight_6 - 1) / 192) > 256 ? (1 + (quarterHeight_6 - 1) / 192) : 256);
  # 108 "sharpness_fuse.cpp"
  first_gang_iter_9 = (hmpprt::gr_gbidx()) * iter_per_gang_9;
  # 108 "sharpness_fuse.cpp"
  last_gang_iter_9 = ((first_gang_iter_9 + iter_per_gang_9 - 1) < (quarterHeight_6 - 1) ? (first_gang_iter_9 + iter_per_gang_9 - 1) : (quarterHeight_6 - 1));
  # 108 "sharpness_fuse.cpp"
  hmpprt::s32 h_11;
  # 108 "sharpness_fuse.cpp"
  # 109 "sharpness_fuse.cpp"
  for (h_11 = first_gang_iter_9 + (hmpprt::gr_btidy()) ; h_11 <= last_gang_iter_9 ; h_11 = h_11 + (hmpprt::gr_btnumy()))
  {
   # 110 "sharpness_fuse.cpp"
   *(yPlaneCSER_15 + h_11 * 4 * width_31) = *(yPlaneDown_6 + h_11 * quarterWidth_8);
  }
  # 101 "sharpness_fuse.cpp"
 }
}
#endif // __HIPCC__



# 101 "sharpness_fuse.cpp"

#ifndef __HIPCC__
void hmpp_acc_region_main_105_internal_1(hmpprt::s32 width_49, hmpprt::s32 quarterWidth_16, hmpprt::s32 quarterHeight_12, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  yPlaneCSER, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  yPlaneDown_12)
{
 # 101 "sharpness_fuse.cpp"
 if (1)
 {
  hmpprt::CUDAGridCall __hmppcg_call;
  __hmppcg_call.setSizeX(192);
  __hmppcg_call.setSizeY(1);
  __hmppcg_call.setBlockSizeX(1);
  __hmppcg_call.setBlockSizeY(256);
  __hmppcg_call.addLocalParameter((hmpprt::s32) (quarterHeight_12), "quarterHeight_6");
  __hmppcg_call.addLocalParameter((hmpprt::s32) (quarterWidth_16), "quarterWidth_8");
  __hmppcg_call.addLocalParameter((hmpprt::s32) (width_49), "width_31");
  __hmppcg_call.addLocalParameter(&yPlaneCSER, 8, "yPlaneCSER_15");
  __hmppcg_call.addLocalParameter(&yPlaneDown_12, 8, "yPlaneDown_6");
  __hmppcg_call.launch(hmpp_acc_region_main_105_parallel_region_1, hmpprt::Context::getInstance()->getCUDADevice());
 }
 ;
}
#endif // __HIPCC__



# 101 "sharpness_fuse.cpp"

#ifndef __HIPCC__
extern "C" CDLT_API  void hmpp_acc_region_main_105(hmpprt::s32 width, hmpprt::s32 quarterWidth, hmpprt::s32 quarterHeight, hmpprt::u08* yPlaneCSER_30, hmpprt::u08* yPlaneDown)
{
 # 117 "sharpness_fuse.cpp"
 (hmpp_acc_region_main_105_internal_1(width, quarterWidth, quarterHeight, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08> (yPlaneCSER_30), hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08> (yPlaneDown)));
}
#endif // __HIPCC__



# 117 "sharpness_fuse.cpp"

#ifdef __HIPCC__

extern "C" __global__ void hmpp_acc_region_main_117_parallel_region_1(hmpprt::s32 quarterHeight_11, hmpprt::s32 width_33, hmpprt::u08* yPlaneCSER_21)
{
 # 119 "sharpness_fuse.cpp"
 {
  # 120 "sharpness_fuse.cpp"
  hmpprt::s32 iter_per_gang_8;
  # 120 "sharpness_fuse.cpp"
  hmpprt::s32 first_gang_iter_8;
  # 120 "sharpness_fuse.cpp"
  hmpprt::s32 last_gang_iter_8;
  # 120 "sharpness_fuse.cpp"
  iter_per_gang_8 = ((1 + (quarterHeight_11 - 1) / 192) > 256 ? (1 + (quarterHeight_11 - 1) / 192) : 256);
  # 120 "sharpness_fuse.cpp"
  first_gang_iter_8 = (hmpprt::gr_gbidx()) * iter_per_gang_8;
  # 120 "sharpness_fuse.cpp"
  last_gang_iter_8 = ((first_gang_iter_8 + iter_per_gang_8 - 1) < (quarterHeight_11 - 1) ? (first_gang_iter_8 + iter_per_gang_8 - 1) : (quarterHeight_11 - 1));
  # 120 "sharpness_fuse.cpp"
  hmpprt::s32 h_12;
  # 120 "sharpness_fuse.cpp"
  # 121 "sharpness_fuse.cpp"
  for (h_12 = first_gang_iter_8 + (hmpprt::gr_btidy()) ; h_12 <= last_gang_iter_8 ; h_12 = h_12 + (hmpprt::gr_btnumy()))
  {
   # 122 "sharpness_fuse.cpp"
   if (h_12 < quarterHeight_11 - 1)
   {
    # 124 "sharpness_fuse.cpp"
    *(yPlaneCSER_21 + (h_12 * 4 + 1) * width_33) = (hmpprt::u08 ) (((hmpprt::s32 ) (*(yPlaneCSER_21 + h_12 * 4 * width_33)) * 3 + (hmpprt::s32 ) (*(yPlaneCSER_21 + (h_12 * 4 + 4) * width_33))) / 4);
    # 125 "sharpness_fuse.cpp"
    *(yPlaneCSER_21 + (h_12 * 4 + 2) * width_33) = (hmpprt::u08 ) (((hmpprt::s32 ) (*(yPlaneCSER_21 + h_12 * 4 * width_33)) * 2 + (hmpprt::s32 ) (*(yPlaneCSER_21 + (h_12 * 4 + 4) * width_33)) * 2) / 4);
    # 126 "sharpness_fuse.cpp"
    *(yPlaneCSER_21 + (h_12 * 4 + 3) * width_33) = (hmpprt::u08 ) (((hmpprt::s32 ) (*(yPlaneCSER_21 + h_12 * 4 * width_33)) + (hmpprt::s32 ) (*(yPlaneCSER_21 + (h_12 * 4 + 4) * width_33)) * 3) / 4);
   }
   else
   {
    # 130 "sharpness_fuse.cpp"
    *(yPlaneCSER_21 + (h_12 * 4 + 1) * width_33) = *(yPlaneCSER_21 + h_12 * 4 * width_33);
   }
  }
  # 117 "sharpness_fuse.cpp"
 }
}
#endif // __HIPCC__



# 117 "sharpness_fuse.cpp"

#ifndef __HIPCC__
void hmpp_acc_region_main_117_internal_1(hmpprt::s32 width_59, hmpprt::s32 quarterHeight_14, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  yPlaneCSER_14)
{
 # 117 "sharpness_fuse.cpp"
 if (1)
 {
  hmpprt::CUDAGridCall __hmppcg_call;
  __hmppcg_call.setSizeX(192);
  __hmppcg_call.setSizeY(1);
  __hmppcg_call.setBlockSizeX(1);
  __hmppcg_call.setBlockSizeY(256);
  __hmppcg_call.addLocalParameter((hmpprt::s32) (quarterHeight_14), "quarterHeight_11");
  __hmppcg_call.addLocalParameter((hmpprt::s32) (width_59), "width_33");
  __hmppcg_call.addLocalParameter(&yPlaneCSER_14, 8, "yPlaneCSER_21");
  __hmppcg_call.launch(hmpp_acc_region_main_117_parallel_region_1, hmpprt::Context::getInstance()->getCUDADevice());
 }
 ;
}
#endif // __HIPCC__



# 117 "sharpness_fuse.cpp"

#ifndef __HIPCC__
extern "C" CDLT_API  void hmpp_acc_region_main_117(hmpprt::s32 width_19, hmpprt::s32 quarterHeight_5, hmpprt::u08* yPlaneCSER_34)
{
 # 138 "sharpness_fuse.cpp"
 (hmpp_acc_region_main_117_internal_1(width_19, quarterHeight_5, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08> (yPlaneCSER_34)));
}
#endif // __HIPCC__



# 138 "sharpness_fuse.cpp"

#ifdef __HIPCC__

extern "C" __global__ void hmpp_acc_region_main_138_loop1D_1(hmpprt::s32 width_35, hmpprt::s32 height_15, hmpprt::u08* yPlaneCSER_20)
{
 # 141 "sharpness_fuse.cpp"
 hmpprt::s32 h_13;
 # 143 "sharpness_fuse.cpp"
 h_13 = (hmpprt::gr_atidf());
 # 143 "sharpness_fuse.cpp"
 if (h_13 > height_15 - 1)
 {
  # 143 "sharpness_fuse.cpp"
  goto __hmppcg_label_1;
 }
 # 143 "sharpness_fuse.cpp"
 *(yPlaneCSER_20 + (h_13 * width_35 + 1)) = *(yPlaneCSER_20 + h_13 * width_35);
 # 138 "sharpness_fuse.cpp"
 __hmppcg_label_1:;
}
#endif // __HIPCC__



# 138 "sharpness_fuse.cpp"

#ifndef __HIPCC__
void hmpp_acc_region_main_138_internal_1(hmpprt::s32* width_58, hmpprt::s32 height, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  yPlaneCSER_13)
{
 # 140 "sharpness_fuse.cpp"
 hmpprt::s32 width_20;
 # 140 "sharpness_fuse.cpp"
 width_20 = *width_58;
 # 140 "sharpness_fuse.cpp"
 {
  # 138 "sharpness_fuse.cpp"
  if (height - 1 >= 0)
  {
   hmpprt::CUDAGridCall __hmppcg_call;
   __hmppcg_call.setSizeX((height - 1) / 128 + 1);
   __hmppcg_call.setSizeY(1);
   __hmppcg_call.setBlockSizeX(32);
   __hmppcg_call.setBlockSizeY(4);
   __hmppcg_call.addLocalParameter((hmpprt::s32) (width_20), "width_35");
   __hmppcg_call.addLocalParameter((hmpprt::s32) (height), "height_15");
   __hmppcg_call.addLocalParameter(&yPlaneCSER_13, 8, "yPlaneCSER_20");
   __hmppcg_call.launch(hmpp_acc_region_main_138_loop1D_1, hmpprt::Context::getInstance()->getCUDADevice());
  }
  ;
 }
 # 138 "sharpness_fuse.cpp"
 *width_58 = width_20;
}
#endif // __HIPCC__



# 138 "sharpness_fuse.cpp"

#ifndef __HIPCC__
extern "C" CDLT_API  void hmpp_acc_region_main_138(hmpprt::s32* width_18, hmpprt::s32 height_27, hmpprt::u08* yPlaneCSER_33)
{
 # 150 "sharpness_fuse.cpp"
 (hmpp_acc_region_main_138_internal_1(width_18, height_27, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08> (yPlaneCSER_33)));
}
#endif // __HIPCC__



# 150 "sharpness_fuse.cpp"

#ifdef __HIPCC__

extern "C" __global__ void hmpp_acc_region_main_151_loop1D_1(hmpprt::s32 quarterWidth_15, hmpprt::u08* yPlaneCSER_19, hmpprt::u08* yPlaneDown_11)
{
 # 154 "sharpness_fuse.cpp"
 hmpprt::s32 w_11;
 # 156 "sharpness_fuse.cpp"
 w_11 = (hmpprt::gr_atidf());
 # 156 "sharpness_fuse.cpp"
 if (w_11 > quarterWidth_15 - 1)
 {
  # 156 "sharpness_fuse.cpp"
  goto __hmppcg_label_2;
 }
 # 156 "sharpness_fuse.cpp"
 *(yPlaneCSER_19 + w_11 * 4) = *(yPlaneDown_11 + w_11);
 # 150 "sharpness_fuse.cpp"
 __hmppcg_label_2:;
}
#endif // __HIPCC__



# 150 "sharpness_fuse.cpp"

#ifndef __HIPCC__
void hmpp_acc_region_main_151_internal_1(hmpprt::s32* width_17, hmpprt::s32 quarterWidth_7, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  yPlaneCSER_12, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  yPlaneDown_5)
{
 # 152 "sharpness_fuse.cpp"
 hmpprt::s32 width_21;
 # 152 "sharpness_fuse.cpp"
 width_21 = *width_17;
 # 152 "sharpness_fuse.cpp"
 {
  # 150 "sharpness_fuse.cpp"
  if (quarterWidth_7 - 1 >= 0)
  {
   hmpprt::CUDAGridCall __hmppcg_call;
   __hmppcg_call.setSizeX((quarterWidth_7 - 1) / 128 + 1);
   __hmppcg_call.setSizeY(1);
   __hmppcg_call.setBlockSizeX(32);
   __hmppcg_call.setBlockSizeY(4);
   __hmppcg_call.addLocalParameter((hmpprt::s32) (quarterWidth_7), "quarterWidth_15");
   __hmppcg_call.addLocalParameter(&yPlaneCSER_12, 8, "yPlaneCSER_19");
   __hmppcg_call.addLocalParameter(&yPlaneDown_5, 8, "yPlaneDown_11");
   __hmppcg_call.launch(hmpp_acc_region_main_151_loop1D_1, hmpprt::Context::getInstance()->getCUDADevice());
  }
  ;
 }
 # 150 "sharpness_fuse.cpp"
 *width_17 = width_21;
}
#endif // __HIPCC__



# 150 "sharpness_fuse.cpp"

#ifndef __HIPCC__
extern "C" CDLT_API  void hmpp_acc_region_main_151(hmpprt::s32* width_57, hmpprt::s32 quarterWidth_22, hmpprt::u08* yPlaneCSER_32, hmpprt::u08* yPlaneDown_14)
{
 # 163 "sharpness_fuse.cpp"
 (hmpp_acc_region_main_151_internal_1(width_57, quarterWidth_22, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08> (yPlaneCSER_32), hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08> (yPlaneDown_14)));
}
#endif // __HIPCC__



# 163 "sharpness_fuse.cpp"

#ifdef __HIPCC__

extern "C" __global__ void hmpp_acc_region_main_159_loop1D_1(hmpprt::s32 width_22, hmpprt::s32 height_12, hmpprt::s32 quarterWidth_6, hmpprt::s32 quarterHeight_4, hmpprt::u08* yPlaneCSER_11, hmpprt::u08* yPlaneDown_10)
{
 # 168 "sharpness_fuse.cpp"
 hmpprt::s32 w_12;
 # 170 "sharpness_fuse.cpp"
 w_12 = (hmpprt::gr_atidf());
 # 170 "sharpness_fuse.cpp"
 if (w_12 > quarterWidth_6 - 1)
 {
  # 170 "sharpness_fuse.cpp"
  goto __hmppcg_label_3;
 }
 # 170 "sharpness_fuse.cpp"
 *(yPlaneCSER_11 + ((height_12 - 2) * width_22 + w_12 * 4)) = *(yPlaneDown_10 + ((quarterHeight_4 - 1) * quarterWidth_6 + w_12));
 # 163 "sharpness_fuse.cpp"
 __hmppcg_label_3:;
}
#endif // __HIPCC__



# 163 "sharpness_fuse.cpp"

#ifndef __HIPCC__
void hmpp_acc_region_main_159_internal_1(hmpprt::s32* width_16, hmpprt::s32* height_11, hmpprt::s32 quarterWidth_20, hmpprt::s32 quarterHeight_10, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  yPlaneCSER_18, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  yPlaneDown_4)
{
 # 165 "sharpness_fuse.cpp"
 hmpprt::s32 height_17;
 # 165 "sharpness_fuse.cpp"
 height_17 = *height_11;
 # 165 "sharpness_fuse.cpp"
 hmpprt::s32 width_36;
 # 165 "sharpness_fuse.cpp"
 width_36 = *width_16;
 # 165 "sharpness_fuse.cpp"
 {
  # 163 "sharpness_fuse.cpp"
  if (quarterWidth_20 - 1 >= 0)
  {
   hmpprt::CUDAGridCall __hmppcg_call;
   __hmppcg_call.setSizeX((quarterWidth_20 - 1) / 128 + 1);
   __hmppcg_call.setSizeY(1);
   __hmppcg_call.setBlockSizeX(32);
   __hmppcg_call.setBlockSizeY(4);
   __hmppcg_call.addLocalParameter((hmpprt::s32) (width_36), "width_22");
   __hmppcg_call.addLocalParameter((hmpprt::s32) (height_17), "height_12");
   __hmppcg_call.addLocalParameter((hmpprt::s32) (quarterWidth_20), "quarterWidth_6");
   __hmppcg_call.addLocalParameter((hmpprt::s32) (quarterHeight_10), "quarterHeight_4");
   __hmppcg_call.addLocalParameter(&yPlaneCSER_18, 8, "yPlaneCSER_11");
   __hmppcg_call.addLocalParameter(&yPlaneDown_4, 8, "yPlaneDown_10");
   __hmppcg_call.launch(hmpp_acc_region_main_159_loop1D_1, hmpprt::Context::getInstance()->getCUDADevice());
  }
  ;
 }
 # 163 "sharpness_fuse.cpp"
 *width_16 = width_36;
 # 163 "sharpness_fuse.cpp"
 *height_11 = height_17;
}
#endif // __HIPCC__



# 163 "sharpness_fuse.cpp"

#ifndef __HIPCC__
extern "C" CDLT_API  void hmpp_acc_region_main_159(hmpprt::s32* width_56, hmpprt::s32* height_29, hmpprt::s32 quarterWidth_14, hmpprt::s32 quarterHeight_13, hmpprt::u08* yPlaneCSER_36, hmpprt::u08* yPlaneDown_13)
{
 # 177 "sharpness_fuse.cpp"
 (hmpp_acc_region_main_159_internal_1(width_56, height_29, quarterWidth_14, quarterHeight_13, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08> (yPlaneCSER_36), hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08> (yPlaneDown_13)));
}
#endif // __HIPCC__



# 177 "sharpness_fuse.cpp"

#ifdef __HIPCC__

extern "C" __global__ void hmpp_acc_region_main_172_loop1D_1(hmpprt::s32 quarterWidth_5, hmpprt::u08* yPlaneCSER_17)
{
 # 175 "sharpness_fuse.cpp"
 hmpprt::s32 w_13;
 # 177 "sharpness_fuse.cpp"
 w_13 = (hmpprt::gr_atidf());
 # 177 "sharpness_fuse.cpp"
 if (w_13 > quarterWidth_5 - 1)
 {
  # 177 "sharpness_fuse.cpp"
  goto __hmppcg_label_4;
 }
 # 177 "sharpness_fuse.cpp"
 if (w_13 < quarterWidth_5 - 1)
 {
  # 179 "sharpness_fuse.cpp"
  *(yPlaneCSER_17 + (w_13 * 4 + 1)) = (hmpprt::u08 ) (((hmpprt::s32 ) (*(yPlaneCSER_17 + w_13 * 4)) * 3 + (hmpprt::s32 ) (*(yPlaneCSER_17 + (w_13 * 4 + 4)))) / 4);
  # 180 "sharpness_fuse.cpp"
  *(yPlaneCSER_17 + (w_13 * 4 + 2)) = (hmpprt::u08 ) (((hmpprt::s32 ) (*(yPlaneCSER_17 + w_13 * 4)) * 2 + (hmpprt::s32 ) (*(yPlaneCSER_17 + (w_13 * 4 + 4))) * 2) / 4);
  # 181 "sharpness_fuse.cpp"
  *(yPlaneCSER_17 + (w_13 * 4 + 3)) = (hmpprt::u08 ) (((hmpprt::s32 ) (*(yPlaneCSER_17 + w_13 * 4)) + (hmpprt::s32 ) (*(yPlaneCSER_17 + (w_13 * 4 + 4))) * 3) / 4);
 }
 else
 {
  # 185 "sharpness_fuse.cpp"
  *(yPlaneCSER_17 + (w_13 * 4 + 1)) = *(yPlaneCSER_17 + w_13 * 4);
 }
 # 177 "sharpness_fuse.cpp"
 __hmppcg_label_4:;
}
#endif // __HIPCC__



# 177 "sharpness_fuse.cpp"

#ifndef __HIPCC__
void hmpp_acc_region_main_172_internal_1(hmpprt::s32* width_15, hmpprt::s32 quarterWidth_19, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  yPlaneCSER_10)
{
 # 179 "sharpness_fuse.cpp"
 hmpprt::s32 width_23;
 # 179 "sharpness_fuse.cpp"
 width_23 = *width_15;
 # 179 "sharpness_fuse.cpp"
 {
  # 177 "sharpness_fuse.cpp"
  if (quarterWidth_19 - 1 >= 0)
  {
   hmpprt::CUDAGridCall __hmppcg_call;
   __hmppcg_call.setSizeX((quarterWidth_19 - 1) / 128 + 1);
   __hmppcg_call.setSizeY(1);
   __hmppcg_call.setBlockSizeX(32);
   __hmppcg_call.setBlockSizeY(4);
   __hmppcg_call.addLocalParameter((hmpprt::s32) (quarterWidth_19), "quarterWidth_5");
   __hmppcg_call.addLocalParameter(&yPlaneCSER_10, 8, "yPlaneCSER_17");
   __hmppcg_call.launch(hmpp_acc_region_main_172_loop1D_1, hmpprt::Context::getInstance()->getCUDADevice());
  }
  ;
 }
 # 177 "sharpness_fuse.cpp"
 *width_15 = width_23;
}
#endif // __HIPCC__



# 177 "sharpness_fuse.cpp"

#ifndef __HIPCC__
extern "C" CDLT_API  void hmpp_acc_region_main_172(hmpprt::s32* width_55, hmpprt::s32 quarterWidth_13, hmpprt::u08* yPlaneCSER_31)
{
 # 193 "sharpness_fuse.cpp"
 (hmpp_acc_region_main_172_internal_1(width_55, quarterWidth_13, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08> (yPlaneCSER_31)));
}
#endif // __HIPCC__



# 193 "sharpness_fuse.cpp"

#ifdef __HIPCC__

extern "C" __global__ void hmpp_acc_region_main_189_loop1D_1(hmpprt::s32 quarterWidth_4, hmpprt::u08* yPlaneCSER_9, hmpprt::s32 width_24, hmpprt::s32 height_13)
{
 # 198 "sharpness_fuse.cpp"
 hmpprt::s32 w_14;
 # 200 "sharpness_fuse.cpp"
 w_14 = (hmpprt::gr_atidf());
 # 200 "sharpness_fuse.cpp"
 if (w_14 > quarterWidth_4 - 1)
 {
  # 200 "sharpness_fuse.cpp"
  goto __hmppcg_label_5;
 }
 # 200 "sharpness_fuse.cpp"
 if (w_14 < quarterWidth_4 - 1)
 {
  # 202 "sharpness_fuse.cpp"
  *(yPlaneCSER_9 + ((height_13 - 2) * width_24 + w_14 * 4 + 1)) = (hmpprt::u08 ) (((hmpprt::s32 ) (*(yPlaneCSER_9 + ((height_13 - 2) * width_24 + w_14 * 4))) * 3 + (hmpprt::s32 ) (*(yPlaneCSER_9 + ((height_13 - 2) * width_24 + w_14 * 4 + 4)))) / 4);
  # 203 "sharpness_fuse.cpp"
  *(yPlaneCSER_9 + ((height_13 - 2) * width_24 + w_14 * 4 + 2)) = (hmpprt::u08 ) (((hmpprt::s32 ) (*(yPlaneCSER_9 + ((height_13 - 2) * width_24 + w_14 * 4))) * 2 + (hmpprt::s32 ) (*(yPlaneCSER_9 + ((height_13 - 2) * width_24 + w_14 * 4 + 4))) * 2) / 4);
  # 204 "sharpness_fuse.cpp"
  *(yPlaneCSER_9 + ((height_13 - 2) * width_24 + w_14 * 4 + 3)) = (hmpprt::u08 ) (((hmpprt::s32 ) (*(yPlaneCSER_9 + ((height_13 - 2) * width_24 + w_14 * 4))) + (hmpprt::s32 ) (*(yPlaneCSER_9 + ((height_13 - 2) * width_24 + w_14 * 4 + 4))) * 3) / 4);
 }
 else
 {
  # 208 "sharpness_fuse.cpp"
  *(yPlaneCSER_9 + ((height_13 - 2) * width_24 + w_14 * 4 + 1)) = *(yPlaneCSER_9 + ((height_13 - 2) * width_24 + w_14 * 4));
 }
 # 193 "sharpness_fuse.cpp"
 __hmppcg_label_5:;
}
#endif // __HIPCC__



# 193 "sharpness_fuse.cpp"

#ifndef __HIPCC__
void hmpp_acc_region_main_189_internal_1(hmpprt::s32* width_54, hmpprt::s32* height_28, hmpprt::s32 quarterWidth_18, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  yPlaneCSER_43)
{
 # 195 "sharpness_fuse.cpp"
 hmpprt::s32 height_18;
 # 195 "sharpness_fuse.cpp"
 height_18 = *height_28;
 # 195 "sharpness_fuse.cpp"
 hmpprt::s32 width_37;
 # 195 "sharpness_fuse.cpp"
 width_37 = *width_54;
 # 195 "sharpness_fuse.cpp"
 {
  # 193 "sharpness_fuse.cpp"
  if (quarterWidth_18 - 1 >= 0)
  {
   hmpprt::CUDAGridCall __hmppcg_call;
   __hmppcg_call.setSizeX((quarterWidth_18 - 1) / 128 + 1);
   __hmppcg_call.setSizeY(1);
   __hmppcg_call.setBlockSizeX(32);
   __hmppcg_call.setBlockSizeY(4);
   __hmppcg_call.addLocalParameter((hmpprt::s32) (quarterWidth_18), "quarterWidth_4");
   __hmppcg_call.addLocalParameter(&yPlaneCSER_43, 8, "yPlaneCSER_9");
   __hmppcg_call.addLocalParameter((hmpprt::s32) (width_37), "width_24");
   __hmppcg_call.addLocalParameter((hmpprt::s32) (height_18), "height_13");
   __hmppcg_call.launch(hmpp_acc_region_main_189_loop1D_1, hmpprt::Context::getInstance()->getCUDADevice());
  }
  ;
 }
 # 193 "sharpness_fuse.cpp"
 *width_54 = width_37;
 # 193 "sharpness_fuse.cpp"
 *height_28 = height_18;
}
#endif // __HIPCC__



# 193 "sharpness_fuse.cpp"

#ifndef __HIPCC__
extern "C" CDLT_API  void hmpp_acc_region_main_189(hmpprt::s32* width_14, hmpprt::s32* height_10, hmpprt::s32 quarterWidth_12, hmpprt::u08* yPlaneCSER_29)
{
 # 216 "sharpness_fuse.cpp"
 (hmpp_acc_region_main_189_internal_1(width_14, height_10, quarterWidth_12, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08> (yPlaneCSER_29)));
}
#endif // __HIPCC__



# 216 "sharpness_fuse.cpp"

#ifdef __HIPCC__

extern "C" __global__ void hmpp_acc_region_main_211_loop1D_1(hmpprt::u08* yPlaneCSER_8, hmpprt::s32 width_25)
{
 # 214 "sharpness_fuse.cpp"
 hmpprt::s32 w_15;
 # 216 "sharpness_fuse.cpp"
 w_15 = (hmpprt::gr_atidf());
 # 216 "sharpness_fuse.cpp"
 if (w_15 > width_25 - 1)
 {
  # 216 "sharpness_fuse.cpp"
  goto __hmppcg_label_6;
 }
 # 216 "sharpness_fuse.cpp"
 *(yPlaneCSER_8 + (width_25 + w_15)) = *(yPlaneCSER_8 + w_15);
 # 216 "sharpness_fuse.cpp"
 __hmppcg_label_6:;
}
#endif // __HIPCC__



# 216 "sharpness_fuse.cpp"

#ifndef __HIPCC__
void hmpp_acc_region_main_211_internal_1(hmpprt::s32* width_53, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  yPlaneCSER_42)
{
 # 218 "sharpness_fuse.cpp"
 hmpprt::s32 width_38;
 # 218 "sharpness_fuse.cpp"
 width_38 = *width_53;
 # 218 "sharpness_fuse.cpp"
 {
  # 216 "sharpness_fuse.cpp"
  if (width_38 - 1 >= 0)
  {
   hmpprt::CUDAGridCall __hmppcg_call;
   __hmppcg_call.setSizeX((width_38 - 1) / 128 + 1);
   __hmppcg_call.setSizeY(1);
   __hmppcg_call.setBlockSizeX(32);
   __hmppcg_call.setBlockSizeY(4);
   __hmppcg_call.addLocalParameter(&yPlaneCSER_42, 8, "yPlaneCSER_8");
   __hmppcg_call.addLocalParameter((hmpprt::s32) (width_38), "width_25");
   __hmppcg_call.launch(hmpp_acc_region_main_211_loop1D_1, hmpprt::Context::getInstance()->getCUDADevice());
  }
  ;
 }
 # 216 "sharpness_fuse.cpp"
 *width_53 = width_38;
}
#endif // __HIPCC__



# 216 "sharpness_fuse.cpp"

#ifndef __HIPCC__
extern "C" CDLT_API  void hmpp_acc_region_main_211(hmpprt::s32* width_13, hmpprt::u08* yPlaneCSER_28)
{
 # 223 "sharpness_fuse.cpp"
 (hmpp_acc_region_main_211_internal_1(width_13, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08> (yPlaneCSER_28)));
}
#endif // __HIPCC__



# 223 "sharpness_fuse.cpp"

#ifdef __HIPCC__

extern "C" __global__ void hmpp_acc_region_main_220_loop1D_1(hmpprt::u08* yPlaneCSER_7, hmpprt::s32 width_26, hmpprt::s32 height_14)
{
 # 229 "sharpness_fuse.cpp"
 hmpprt::s32 w_16;
 # 231 "sharpness_fuse.cpp"
 w_16 = (hmpprt::gr_atidf());
 # 231 "sharpness_fuse.cpp"
 if (w_16 > width_26 - 1)
 {
  # 231 "sharpness_fuse.cpp"
  goto __hmppcg_label_7;
 }
 # 231 "sharpness_fuse.cpp"
 *(yPlaneCSER_7 + ((height_14 - 1) * width_26 + w_16)) = *(yPlaneCSER_7 + ((height_14 - 2) * width_26 + w_16));
 # 223 "sharpness_fuse.cpp"
 __hmppcg_label_7:;
}
#endif // __HIPCC__



# 223 "sharpness_fuse.cpp"

#ifndef __HIPCC__
void hmpp_acc_region_main_220_internal_1(hmpprt::s32* width_52, hmpprt::s32* height_9, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  yPlaneCSER_41)
{
 # 225 "sharpness_fuse.cpp"
 hmpprt::s32 height_19;
 # 225 "sharpness_fuse.cpp"
 height_19 = *height_9;
 # 225 "sharpness_fuse.cpp"
 hmpprt::s32 width_39;
 # 225 "sharpness_fuse.cpp"
 width_39 = *width_52;
 # 225 "sharpness_fuse.cpp"
 {
  # 223 "sharpness_fuse.cpp"
  if (width_39 - 1 >= 0)
  {
   hmpprt::CUDAGridCall __hmppcg_call;
   __hmppcg_call.setSizeX((width_39 - 1) / 128 + 1);
   __hmppcg_call.setSizeY(1);
   __hmppcg_call.setBlockSizeX(32);
   __hmppcg_call.setBlockSizeY(4);
   __hmppcg_call.addLocalParameter(&yPlaneCSER_41, 8, "yPlaneCSER_7");
   __hmppcg_call.addLocalParameter((hmpprt::s32) (width_39), "width_26");
   __hmppcg_call.addLocalParameter((hmpprt::s32) (height_19), "height_14");
   __hmppcg_call.launch(hmpp_acc_region_main_220_loop1D_1, hmpprt::Context::getInstance()->getCUDADevice());
  }
  ;
 }
 # 223 "sharpness_fuse.cpp"
 *width_52 = width_39;
 # 223 "sharpness_fuse.cpp"
 *height_9 = height_19;
}
#endif // __HIPCC__



# 223 "sharpness_fuse.cpp"

#ifndef __HIPCC__
extern "C" CDLT_API  void hmpp_acc_region_main_220(hmpprt::s32* width_12, hmpprt::s32* height_38, hmpprt::u08* yPlaneCSER_27)
{
 # 238 "sharpness_fuse.cpp"
 (hmpp_acc_region_main_220_internal_1(width_12, height_38, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08> (yPlaneCSER_27)));
}
#endif // __HIPCC__



# 238 "sharpness_fuse.cpp"

#ifdef __HIPCC__

extern "C" __global__ void hmpp_acc_region_main_232_loop1D_1(hmpprt::s32 quarterWidth_11, hmpprt::s32 quarterHeight_3, hmpprt::u08* yPlaneCSER_6, hmpprt::u08* yPlaneDown_3, hmpprt::s32 width_27)
{
 # 235 "sharpness_fuse.cpp"
 hmpprt::s32 h_14;
 # 237 "sharpness_fuse.cpp"
 h_14 = (hmpprt::gr_atidf());
 # 237 "sharpness_fuse.cpp"
 if (h_14 > quarterHeight_3 - 1)
 {
  # 237 "sharpness_fuse.cpp"
  goto __hmppcg_label_8;
 }
 # 237 "sharpness_fuse.cpp"
 *(yPlaneCSER_6 + (h_14 * 4 * width_27 + width_27 - 2)) = *(yPlaneDown_3 + (h_14 * quarterWidth_11 + quarterWidth_11 - 1));
 # 238 "sharpness_fuse.cpp"
 __hmppcg_label_8:;
}
#endif // __HIPCC__



# 238 "sharpness_fuse.cpp"

#ifndef __HIPCC__
void hmpp_acc_region_main_232_internal_1(hmpprt::s32* width_51, hmpprt::s32 quarterWidth_21, hmpprt::s32 quarterHeight_17, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  yPlaneCSER_40, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  yPlaneDown_17)
{
 # 240 "sharpness_fuse.cpp"
 hmpprt::s32 width_40;
 # 240 "sharpness_fuse.cpp"
 width_40 = *width_51;
 # 240 "sharpness_fuse.cpp"
 {
  # 238 "sharpness_fuse.cpp"
  if (quarterHeight_17 - 1 >= 0)
  {
   hmpprt::CUDAGridCall __hmppcg_call;
   __hmppcg_call.setSizeX((quarterHeight_17 - 1) / 128 + 1);
   __hmppcg_call.setSizeY(1);
   __hmppcg_call.setBlockSizeX(32);
   __hmppcg_call.setBlockSizeY(4);
   __hmppcg_call.addLocalParameter((hmpprt::s32) (quarterWidth_21), "quarterWidth_11");
   __hmppcg_call.addLocalParameter((hmpprt::s32) (quarterHeight_17), "quarterHeight_3");
   __hmppcg_call.addLocalParameter(&yPlaneCSER_40, 8, "yPlaneCSER_6");
   __hmppcg_call.addLocalParameter(&yPlaneDown_17, 8, "yPlaneDown_3");
   __hmppcg_call.addLocalParameter((hmpprt::s32) (width_40), "width_27");
   __hmppcg_call.launch(hmpp_acc_region_main_232_loop1D_1, hmpprt::Context::getInstance()->getCUDADevice());
  }
  ;
 }
 # 238 "sharpness_fuse.cpp"
 *width_51 = width_40;
}
#endif // __HIPCC__



# 238 "sharpness_fuse.cpp"

#ifndef __HIPCC__
extern "C" CDLT_API  void hmpp_acc_region_main_232(hmpprt::s32* width_11, hmpprt::s32 quarterWidth_3, hmpprt::s32 quarterHeight_9, hmpprt::u08* yPlaneCSER_26, hmpprt::u08* yPlaneDown_9)
{
 # 244 "sharpness_fuse.cpp"
 (hmpp_acc_region_main_232_internal_1(width_11, quarterWidth_3, quarterHeight_9, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08> (yPlaneCSER_26), hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08> (yPlaneDown_9)));
}
#endif // __HIPCC__



# 244 "sharpness_fuse.cpp"

#ifdef __HIPCC__

extern "C" __global__ void hmpp_acc_region_main_244_loop1D_1(hmpprt::s32 quarterHeight_2, hmpprt::u08* yPlaneCSER_5, hmpprt::s32 width_28)
{
 # 247 "sharpness_fuse.cpp"
 hmpprt::s32 h_15;
 # 249 "sharpness_fuse.cpp"
 h_15 = (hmpprt::gr_atidf());
 # 249 "sharpness_fuse.cpp"
 if (h_15 > quarterHeight_2 - 1)
 {
  # 249 "sharpness_fuse.cpp"
  goto __hmppcg_label_9;
 }
 # 249 "sharpness_fuse.cpp"
 if (h_15 < quarterHeight_2 - 1)
 {
  # 251 "sharpness_fuse.cpp"
  *(yPlaneCSER_5 + ((h_15 * 4 + 1) * width_28 + width_28 - 2)) = (hmpprt::u08 ) (((hmpprt::s32 ) (*(yPlaneCSER_5 + (h_15 * 4 * width_28 + width_28 - 2))) * 3 + (hmpprt::s32 ) (*(yPlaneCSER_5 + ((h_15 * 4 + 4) * width_28 + width_28 - 2)))) / 4);
  # 252 "sharpness_fuse.cpp"
  *(yPlaneCSER_5 + ((h_15 * 4 + 2) * width_28 + width_28 - 2)) = (hmpprt::u08 ) (((hmpprt::s32 ) (*(yPlaneCSER_5 + (h_15 * 4 * width_28 + width_28 - 2))) * 2 + (hmpprt::s32 ) (*(yPlaneCSER_5 + ((h_15 * 4 + 4) * width_28 + width_28 - 2))) * 2) / 4);
  # 253 "sharpness_fuse.cpp"
  *(yPlaneCSER_5 + ((h_15 * 4 + 3) * width_28 + width_28 - 2)) = (hmpprt::u08 ) (((hmpprt::s32 ) (*(yPlaneCSER_5 + (h_15 * 4 * width_28 + width_28 - 2))) + (hmpprt::s32 ) (*(yPlaneCSER_5 + ((h_15 * 4 + 4) * width_28 + width_28 - 2))) * 3) / 4);
 }
 else
 {
  # 257 "sharpness_fuse.cpp"
  *(yPlaneCSER_5 + ((h_15 * 4 + 1) * width_28 + width_28 - 2)) = *(yPlaneCSER_5 + (h_15 * 4 * width_28 + width_28 - 2));
  # 258 "sharpness_fuse.cpp"
  *(yPlaneCSER_5 + ((h_15 * 4 + 2) * width_28 + width_28 - 2)) = *(yPlaneCSER_5 + (h_15 * 4 * width_28 + width_28 - 2));
  # 259 "sharpness_fuse.cpp"
  *(yPlaneCSER_5 + ((h_15 * 4 + 3) * width_28 + width_28 - 2)) = *(yPlaneCSER_5 + (h_15 * 4 * width_28 + width_28 - 2));
 }
 # 244 "sharpness_fuse.cpp"
 __hmppcg_label_9:;
}
#endif // __HIPCC__



# 244 "sharpness_fuse.cpp"

#ifndef __HIPCC__
void hmpp_acc_region_main_244_internal_1(hmpprt::s32* width_50, hmpprt::s32 quarterHeight_8, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  yPlaneCSER_25)
{
 # 246 "sharpness_fuse.cpp"
 hmpprt::s32 width_41;
 # 246 "sharpness_fuse.cpp"
 width_41 = *width_50;
 # 246 "sharpness_fuse.cpp"
 {
  # 244 "sharpness_fuse.cpp"
  if (quarterHeight_8 - 1 >= 0)
  {
   hmpprt::CUDAGridCall __hmppcg_call;
   __hmppcg_call.setSizeX((quarterHeight_8 - 1) / 128 + 1);
   __hmppcg_call.setSizeY(1);
   __hmppcg_call.setBlockSizeX(32);
   __hmppcg_call.setBlockSizeY(4);
   __hmppcg_call.addLocalParameter((hmpprt::s32) (quarterHeight_8), "quarterHeight_2");
   __hmppcg_call.addLocalParameter(&yPlaneCSER_25, 8, "yPlaneCSER_5");
   __hmppcg_call.addLocalParameter((hmpprt::s32) (width_41), "width_28");
   __hmppcg_call.launch(hmpp_acc_region_main_244_loop1D_1, hmpprt::Context::getInstance()->getCUDADevice());
  }
  ;
 }
 # 244 "sharpness_fuse.cpp"
 *width_50 = width_41;
}
#endif // __HIPCC__



# 244 "sharpness_fuse.cpp"

#ifndef __HIPCC__
extern "C" CDLT_API  void hmpp_acc_region_main_244(hmpprt::s32* width_10, hmpprt::s32 quarterHeight_16, hmpprt::u08* yPlaneCSER_39)
{
 # 267 "sharpness_fuse.cpp"
 (hmpp_acc_region_main_244_internal_1(width_10, quarterHeight_16, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08> (yPlaneCSER_39)));
}
#endif // __HIPCC__



# 267 "sharpness_fuse.cpp"

#ifdef __HIPCC__

extern "C" __global__ void hmpp_acc_region_main_267_loop1D_1(hmpprt::s32 width_42, hmpprt::s32 height_26, hmpprt::u08* yPlaneCSER_24)
{
 # 270 "sharpness_fuse.cpp"
 hmpprt::s32 h_16;
 # 272 "sharpness_fuse.cpp"
 h_16 = (hmpprt::gr_atidf());
 # 272 "sharpness_fuse.cpp"
 if (h_16 > height_26 - 1)
 {
  # 272 "sharpness_fuse.cpp"
  goto __hmppcg_label_10;
 }
 # 272 "sharpness_fuse.cpp"
 *(yPlaneCSER_24 + (h_16 * width_42 + width_42 - 1)) = *(yPlaneCSER_24 + (h_16 * width_42 + width_42 - 2));
 # 267 "sharpness_fuse.cpp"
 __hmppcg_label_10:;
}
#endif // __HIPCC__



# 267 "sharpness_fuse.cpp"

#ifndef __HIPCC__
void hmpp_acc_region_main_267_internal_1(hmpprt::s32* width_68, hmpprt::s32 height_37, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  yPlaneCSER_44)
{
 # 269 "sharpness_fuse.cpp"
 hmpprt::s32 width_29;
 # 269 "sharpness_fuse.cpp"
 width_29 = *width_68;
 # 269 "sharpness_fuse.cpp"
 {
  # 267 "sharpness_fuse.cpp"
  if (height_37 - 1 >= 0)
  {
   hmpprt::CUDAGridCall __hmppcg_call;
   __hmppcg_call.setSizeX((height_37 - 1) / 128 + 1);
   __hmppcg_call.setSizeY(1);
   __hmppcg_call.setBlockSizeX(32);
   __hmppcg_call.setBlockSizeY(4);
   __hmppcg_call.addLocalParameter((hmpprt::s32) (width_29), "width_42");
   __hmppcg_call.addLocalParameter((hmpprt::s32) (height_37), "height_26");
   __hmppcg_call.addLocalParameter(&yPlaneCSER_44, 8, "yPlaneCSER_24");
   __hmppcg_call.launch(hmpp_acc_region_main_267_loop1D_1, hmpprt::Context::getInstance()->getCUDADevice());
  }
  ;
 }
 # 267 "sharpness_fuse.cpp"
 *width_68 = width_29;
}
#endif // __HIPCC__



# 267 "sharpness_fuse.cpp"

#ifndef __HIPCC__
extern "C" CDLT_API  void hmpp_acc_region_main_267(hmpprt::s32* width_9, hmpprt::s32 height_8, hmpprt::u08* yPlaneCSER_4)
{
 # 279 "sharpness_fuse.cpp"
 (hmpp_acc_region_main_267_internal_1(width_9, height_8, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08> (yPlaneCSER_4)));
}
#endif // __HIPCC__



# 279 "sharpness_fuse.cpp"

#ifdef __HIPCC__

extern "C" __global__ void hmpp_acc_region_main_279_loop1D_1(hmpprt::s32 quarterWidth_2, hmpprt::s32 quarterHeight_1, hmpprt::u08* yPlaneCSER_3, hmpprt::u08* yPlaneDown_2, hmpprt::s32 width_30)
{
 # 282 "sharpness_fuse.cpp"
 hmpprt::s32 h_17;
 # 285 "sharpness_fuse.cpp"
 hmpprt::s32 w_17;
 # 282 "sharpness_fuse.cpp"
 hmpprt::s32 h_22;
 # 287 "sharpness_fuse.cpp"
 h_22 = (hmpprt::gr_atidf());
 # 287 "sharpness_fuse.cpp"
 if (h_22 > (quarterHeight_1 +  -1) * (quarterWidth_2 - 1) - 1)
 {
  # 287 "sharpness_fuse.cpp"
  goto __hmppcg_label_11;
 }
 # 287 "sharpness_fuse.cpp"
 w_17 = h_22 % (quarterWidth_2 +  -1);
 # 287 "sharpness_fuse.cpp"
 h_17 = h_22 / (quarterWidth_2 +  -1);
 # 287 "sharpness_fuse.cpp"
 *(yPlaneCSER_3 + ((h_17 * 4 + 2) * width_30 + w_17 * 4 + 2)) = (hmpprt::u08 ) (((hmpprt::s32 ) (*(yPlaneDown_2 + (h_17 * quarterWidth_2 + w_17))) * 7 + (hmpprt::s32 ) (*(yPlaneDown_2 + (h_17 * quarterWidth_2 + w_17 + 1)))) * 7 / 8 / 8 + ((hmpprt::s32 ) (*(yPlaneDown_2 + ((h_17 + 1) * quarterWidth_2 + w_17))) * 7 + (hmpprt::s32 ) (*(yPlaneDown_2 + ((h_17 + 1) * quarterWidth_2 + w_17 + 1)))) / 8 / 8);
 # 288 "sharpness_fuse.cpp"
 *(yPlaneCSER_3 + ((h_17 * 4 + 2) * width_30 + w_17 * 4 + 3)) = (hmpprt::u08 ) (((hmpprt::s32 ) (*(yPlaneDown_2 + (h_17 * quarterWidth_2 + w_17))) * 5 + (hmpprt::s32 ) (*(yPlaneDown_2 + (h_17 * quarterWidth_2 + w_17 + 1))) * 3) * 7 / 8 / 8 + ((hmpprt::s32 ) (*(yPlaneDown_2 + ((h_17 + 1) * quarterWidth_2 + w_17))) * 5 + (hmpprt::s32 ) (*(yPlaneDown_2 + ((h_17 + 1) * quarterWidth_2 + w_17 + 1))) * 3) / 8 / 8);
 # 289 "sharpness_fuse.cpp"
 *(yPlaneCSER_3 + ((h_17 * 4 + 2) * width_30 + w_17 * 4 + 4)) = (hmpprt::u08 ) (((hmpprt::s32 ) (*(yPlaneDown_2 + (h_17 * quarterWidth_2 + w_17))) * 3 + (hmpprt::s32 ) (*(yPlaneDown_2 + (h_17 * quarterWidth_2 + w_17 + 1))) * 5) * 7 / 8 / 8 + ((hmpprt::s32 ) (*(yPlaneDown_2 + ((h_17 + 1) * quarterWidth_2 + w_17))) * 3 + (hmpprt::s32 ) (*(yPlaneDown_2 + ((h_17 + 1) * quarterWidth_2 + w_17 + 1))) * 5) / 8 / 8);
 # 290 "sharpness_fuse.cpp"
 *(yPlaneCSER_3 + ((h_17 * 4 + 2) * width_30 + w_17 * 4 + 5)) = (hmpprt::u08 ) (((hmpprt::s32 ) (*(yPlaneDown_2 + (h_17 * quarterWidth_2 + w_17))) + (hmpprt::s32 ) (*(yPlaneDown_2 + (h_17 * quarterWidth_2 + w_17 + 1))) * 7) * 7 / 8 / 8 + ((hmpprt::s32 ) (*(yPlaneDown_2 + ((h_17 + 1) * quarterWidth_2 + w_17))) + (hmpprt::s32 ) (*(yPlaneDown_2 + ((h_17 + 1) * quarterWidth_2 + w_17 + 1))) * 7) / 8 / 8);
 # 292 "sharpness_fuse.cpp"
 *(yPlaneCSER_3 + ((h_17 * 4 + 3) * width_30 + w_17 * 4 + 2)) = (hmpprt::u08 ) (((hmpprt::s32 ) (*(yPlaneDown_2 + (h_17 * quarterWidth_2 + w_17))) * 7 + (hmpprt::s32 ) (*(yPlaneDown_2 + (h_17 * quarterWidth_2 + w_17 + 1)))) * 5 / 8 / 8 + ((hmpprt::s32 ) (*(yPlaneDown_2 + ((h_17 + 1) * quarterWidth_2 + w_17))) * 7 + (hmpprt::s32 ) (*(yPlaneDown_2 + ((h_17 + 1) * quarterWidth_2 + w_17 + 1)))) * 3 / 8 / 8);
 # 293 "sharpness_fuse.cpp"
 *(yPlaneCSER_3 + ((h_17 * 4 + 3) * width_30 + w_17 * 4 + 3)) = (hmpprt::u08 ) (((hmpprt::s32 ) (*(yPlaneDown_2 + (h_17 * quarterWidth_2 + w_17))) * 5 + (hmpprt::s32 ) (*(yPlaneDown_2 + (h_17 * quarterWidth_2 + w_17 + 1))) * 3) * 5 / 8 / 8 + ((hmpprt::s32 ) (*(yPlaneDown_2 + ((h_17 + 1) * quarterWidth_2 + w_17))) * 5 + (hmpprt::s32 ) (*(yPlaneDown_2 + ((h_17 + 1) * quarterWidth_2 + w_17 + 1))) * 3) * 3 / 8 / 8);
 # 294 "sharpness_fuse.cpp"
 *(yPlaneCSER_3 + ((h_17 * 4 + 3) * width_30 + w_17 * 4 + 4)) = (hmpprt::u08 ) (((hmpprt::s32 ) (*(yPlaneDown_2 + (h_17 * quarterWidth_2 + w_17))) * 3 + (hmpprt::s32 ) (*(yPlaneDown_2 + (h_17 * quarterWidth_2 + w_17 + 1))) * 5) * 5 / 8 / 8 + ((hmpprt::s32 ) (*(yPlaneDown_2 + ((h_17 + 1) * quarterWidth_2 + w_17))) * 3 + (hmpprt::s32 ) (*(yPlaneDown_2 + ((h_17 + 1) * quarterWidth_2 + w_17 + 1))) * 5) * 3 / 8 / 8);
 # 295 "sharpness_fuse.cpp"
 *(yPlaneCSER_3 + ((h_17 * 4 + 3) * width_30 + w_17 * 4 + 5)) = (hmpprt::u08 ) (((hmpprt::s32 ) (*(yPlaneDown_2 + (h_17 * quarterWidth_2 + w_17))) + (hmpprt::s32 ) (*(yPlaneDown_2 + (h_17 * quarterWidth_2 + w_17 + 1))) * 7) * 5 / 8 / 8 + ((hmpprt::s32 ) (*(yPlaneDown_2 + ((h_17 + 1) * quarterWidth_2 + w_17))) + (hmpprt::s32 ) (*(yPlaneDown_2 + ((h_17 + 1) * quarterWidth_2 + w_17 + 1))) * 7) * 3 / 8 / 8);
 # 297 "sharpness_fuse.cpp"
 *(yPlaneCSER_3 + ((h_17 * 4 + 4) * width_30 + w_17 * 4 + 2)) = (hmpprt::u08 ) (((hmpprt::s32 ) (*(yPlaneDown_2 + (h_17 * quarterWidth_2 + w_17))) * 7 + (hmpprt::s32 ) (*(yPlaneDown_2 + (h_17 * quarterWidth_2 + w_17 + 1)))) * 3 / 8 / 8 + ((hmpprt::s32 ) (*(yPlaneDown_2 + ((h_17 + 1) * quarterWidth_2 + w_17))) * 7 + (hmpprt::s32 ) (*(yPlaneDown_2 + ((h_17 + 1) * quarterWidth_2 + w_17 + 1)))) * 5 / 8 / 8);
 # 298 "sharpness_fuse.cpp"
 *(yPlaneCSER_3 + ((h_17 * 4 + 4) * width_30 + w_17 * 4 + 3)) = (hmpprt::u08 ) (((hmpprt::s32 ) (*(yPlaneDown_2 + (h_17 * quarterWidth_2 + w_17))) * 5 + (hmpprt::s32 ) (*(yPlaneDown_2 + (h_17 * quarterWidth_2 + w_17 + 1))) * 3) * 3 / 8 / 8 + ((hmpprt::s32 ) (*(yPlaneDown_2 + ((h_17 + 1) * quarterWidth_2 + w_17))) * 5 + (hmpprt::s32 ) (*(yPlaneDown_2 + ((h_17 + 1) * quarterWidth_2 + w_17 + 1))) * 3) * 5 / 8 / 8);
 # 299 "sharpness_fuse.cpp"
 *(yPlaneCSER_3 + ((h_17 * 4 + 4) * width_30 + w_17 * 4 + 4)) = (hmpprt::u08 ) (((hmpprt::s32 ) (*(yPlaneDown_2 + (h_17 * quarterWidth_2 + w_17))) * 3 + (hmpprt::s32 ) (*(yPlaneDown_2 + (h_17 * quarterWidth_2 + w_17 + 1))) * 5) * 3 / 8 / 8 + ((hmpprt::s32 ) (*(yPlaneDown_2 + ((h_17 + 1) * quarterWidth_2 + w_17))) * 3 + (hmpprt::s32 ) (*(yPlaneDown_2 + ((h_17 + 1) * quarterWidth_2 + w_17 + 1))) * 5) * 5 / 8 / 8);
 # 300 "sharpness_fuse.cpp"
 *(yPlaneCSER_3 + ((h_17 * 4 + 4) * width_30 + w_17 * 4 + 5)) = (hmpprt::u08 ) (((hmpprt::s32 ) (*(yPlaneDown_2 + (h_17 * quarterWidth_2 + w_17))) + (hmpprt::s32 ) (*(yPlaneDown_2 + (h_17 * quarterWidth_2 + w_17 + 1))) * 7) * 3 / 8 / 8 + ((hmpprt::s32 ) (*(yPlaneDown_2 + ((h_17 + 1) * quarterWidth_2 + w_17))) + (hmpprt::s32 ) (*(yPlaneDown_2 + ((h_17 + 1) * quarterWidth_2 + w_17 + 1))) * 7) * 5 / 8 / 8);
 # 302 "sharpness_fuse.cpp"
 *(yPlaneCSER_3 + ((h_17 * 4 + 5) * width_30 + w_17 * 4 + 2)) = (hmpprt::u08 ) (((hmpprt::s32 ) (*(yPlaneDown_2 + (h_17 * quarterWidth_2 + w_17))) * 7 + (hmpprt::s32 ) (*(yPlaneDown_2 + (h_17 * quarterWidth_2 + w_17 + 1)))) / 8 / 8 + ((hmpprt::s32 ) (*(yPlaneDown_2 + ((h_17 + 1) * quarterWidth_2 + w_17))) * 7 + (hmpprt::s32 ) (*(yPlaneDown_2 + ((h_17 + 1) * quarterWidth_2 + w_17 + 1)))) * 7 / 8 / 8);
 # 303 "sharpness_fuse.cpp"
 *(yPlaneCSER_3 + ((h_17 * 4 + 5) * width_30 + w_17 * 4 + 3)) = (hmpprt::u08 ) (((hmpprt::s32 ) (*(yPlaneDown_2 + (h_17 * quarterWidth_2 + w_17))) * 5 + (hmpprt::s32 ) (*(yPlaneDown_2 + (h_17 * quarterWidth_2 + w_17 + 1))) * 3) / 8 / 8 + ((hmpprt::s32 ) (*(yPlaneDown_2 + ((h_17 + 1) * quarterWidth_2 + w_17))) * 5 + (hmpprt::s32 ) (*(yPlaneDown_2 + ((h_17 + 1) * quarterWidth_2 + w_17 + 1))) * 3) * 7 / 8 / 8);
 # 304 "sharpness_fuse.cpp"
 *(yPlaneCSER_3 + ((h_17 * 4 + 5) * width_30 + w_17 * 4 + 4)) = (hmpprt::u08 ) (((hmpprt::s32 ) (*(yPlaneDown_2 + (h_17 * quarterWidth_2 + w_17))) * 3 + (hmpprt::s32 ) (*(yPlaneDown_2 + (h_17 * quarterWidth_2 + w_17 + 1))) * 5) / 8 / 8 + ((hmpprt::s32 ) (*(yPlaneDown_2 + ((h_17 + 1) * quarterWidth_2 + w_17))) * 3 + (hmpprt::s32 ) (*(yPlaneDown_2 + ((h_17 + 1) * quarterWidth_2 + w_17 + 1))) * 5) * 7 / 8 / 8);
 # 305 "sharpness_fuse.cpp"
 *(yPlaneCSER_3 + ((h_17 * 4 + 5) * width_30 + w_17 * 4 + 5)) = (hmpprt::u08 ) (((hmpprt::s32 ) (*(yPlaneDown_2 + (h_17 * quarterWidth_2 + w_17))) + (hmpprt::s32 ) (*(yPlaneDown_2 + (h_17 * quarterWidth_2 + w_17 + 1))) * 7) / 8 / 8 + ((hmpprt::s32 ) (*(yPlaneDown_2 + ((h_17 + 1) * quarterWidth_2 + w_17))) + (hmpprt::s32 ) (*(yPlaneDown_2 + ((h_17 + 1) * quarterWidth_2 + w_17 + 1))) * 7) * 7 / 8 / 8);
 # 279 "sharpness_fuse.cpp"
 __hmppcg_label_11:;
}
#endif // __HIPCC__



# 279 "sharpness_fuse.cpp"

#ifndef __HIPCC__
void hmpp_acc_region_main_279_internal_1(hmpprt::s32* width_67, hmpprt::s32 quarterWidth_10, hmpprt::s32 quarterHeight_15, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  yPlaneCSER_38, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  yPlaneDown_16)
{
 # 281 "sharpness_fuse.cpp"
 hmpprt::s32 width_44;
 # 281 "sharpness_fuse.cpp"
 width_44 = *width_67;
 # 281 "sharpness_fuse.cpp"
 {
  # 279 "sharpness_fuse.cpp"
  if ((quarterHeight_15 +  -1) * (quarterWidth_10 - 1) - 1 >= 0)
  {
   hmpprt::CUDAGridCall __hmppcg_call;
   __hmppcg_call.setSizeX(((quarterHeight_15 +  -1) * (quarterWidth_10 - 1) - 1) / 128 + 1);
   __hmppcg_call.setSizeY(1);
   __hmppcg_call.setBlockSizeX(32);
   __hmppcg_call.setBlockSizeY(4);
   __hmppcg_call.addLocalParameter((hmpprt::s32) (quarterWidth_10), "quarterWidth_2");
   __hmppcg_call.addLocalParameter((hmpprt::s32) (quarterHeight_15), "quarterHeight_1");
   __hmppcg_call.addLocalParameter(&yPlaneCSER_38, 8, "yPlaneCSER_3");
   __hmppcg_call.addLocalParameter(&yPlaneDown_16, 8, "yPlaneDown_2");
   __hmppcg_call.addLocalParameter((hmpprt::s32) (width_44), "width_30");
   __hmppcg_call.launch(hmpp_acc_region_main_279_loop1D_1, hmpprt::Context::getInstance()->getCUDADevice());
  }
  ;
 }
 # 279 "sharpness_fuse.cpp"
 *width_67 = width_44;
}
#endif // __HIPCC__



# 279 "sharpness_fuse.cpp"

#ifndef __HIPCC__
extern "C" CDLT_API  void hmpp_acc_region_main_279(hmpprt::s32* width_8, hmpprt::s32 quarterWidth_17, hmpprt::s32 quarterHeight_7, hmpprt::u08* yPlaneCSER_23, hmpprt::u08* yPlaneDown_8)
{
 # 313 "sharpness_fuse.cpp"
 (hmpp_acc_region_main_279_internal_1(width_8, quarterWidth_17, quarterHeight_7, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08> (yPlaneCSER_23), hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08> (yPlaneDown_8)));
}
#endif // __HIPCC__



# 313 "sharpness_fuse.cpp"

#ifdef __HIPCC__

extern "C" __global__ void hmpp_acc_region_main_316_parallel_region_1(hmpprt::s32 height_7, hmpprt::s16* pEdge, hmpprt::s32 width_7, hmpprt::u08* yPlane)
{
 # 315 "sharpness_fuse.cpp"
 {
  # 319 "sharpness_fuse.cpp"
  hmpprt::s32 iter_per_gang_7;
  # 319 "sharpness_fuse.cpp"
  hmpprt::s32 first_gang_iter_7;
  # 319 "sharpness_fuse.cpp"
  hmpprt::s32 last_gang_iter_7;
  # 319 "sharpness_fuse.cpp"
  iter_per_gang_7 = ((1 + (height_7 * width_7 - 1) / 192) > 256 ? (1 + (height_7 * width_7 - 1) / 192) : 256);
  # 319 "sharpness_fuse.cpp"
  first_gang_iter_7 = (hmpprt::gr_gbidx()) * iter_per_gang_7;
  # 319 "sharpness_fuse.cpp"
  last_gang_iter_7 = ((first_gang_iter_7 + iter_per_gang_7 - 1) < (height_7 * width_7 - 1) ? (first_gang_iter_7 + iter_per_gang_7 - 1) : (height_7 * width_7 - 1));
  # 319 "sharpness_fuse.cpp"
  hmpprt::s32 y_2;
  # 319 "sharpness_fuse.cpp"
  # 323 "sharpness_fuse.cpp"
  for (y_2 = first_gang_iter_7 + (hmpprt::gr_btidy()) ; y_2 <= last_gang_iter_7 ; y_2 = y_2 + (hmpprt::gr_btnumy()))
  {
   # 319 "sharpness_fuse.cpp"
   hmpprt::s32 y_1;
   # 322 "sharpness_fuse.cpp"
   hmpprt::s32 x_1;
   # 324 "sharpness_fuse.cpp"
   x_1 = y_2 % width_7;
   # 324 "sharpness_fuse.cpp"
   y_1 = y_2 / width_7;
   # 324 "sharpness_fuse.cpp"
   if (y_1 == 0 || y_1 == height_7 - 1 || x_1 == 0 || x_1 == width_7 - 1)
   {
    # 326 "sharpness_fuse.cpp"
    *(pEdge + (y_1 * width_7 + x_1)) = (hmpprt::s16) 0;
   }
   else
   {
    # 330 "sharpness_fuse.cpp"
    hmpprt::s32 gx;
    # 330 "sharpness_fuse.cpp"
    gx = (hmpprt::s32 ) (*(yPlane + ((y_1 - 1) * width_7 + x_1 + 1))) + ((hmpprt::s32 ) (*(yPlane + (y_1 * width_7 + x_1 + 1))) << 1) + (hmpprt::s32 ) (*(yPlane + ((y_1 + 1) * width_7 + x_1 + 1))) - ((hmpprt::s32 ) (*(yPlane + ((y_1 - 1) * width_7 + x_1 - 1))) + ((hmpprt::s32 ) (*(yPlane + (y_1 * width_7 + x_1 - 1))) << 1) + (hmpprt::s32 ) (*(yPlane + ((y_1 + 1) * width_7 + x_1 - 1))));
    # 331 "sharpness_fuse.cpp"
    hmpprt::s32 gy;
    # 331 "sharpness_fuse.cpp"
    gy = (hmpprt::s32 ) (*(yPlane + ((y_1 - 1) * width_7 + x_1 - 1))) + ((hmpprt::s32 ) (*(yPlane + ((y_1 - 1) * width_7 + x_1))) << 1) + (hmpprt::s32 ) (*(yPlane + ((y_1 - 1) * width_7 + x_1 + 1))) - ((hmpprt::s32 ) (*(yPlane + ((y_1 + 1) * width_7 + x_1 - 1))) + ((hmpprt::s32 ) (*(yPlane + ((y_1 + 1) * width_7 + x_1))) << 1) + (hmpprt::s32 ) (*(yPlane + ((y_1 + 1) * width_7 + x_1 + 1))));
    # 332 "sharpness_fuse.cpp"
    hmpprt::s32 edgeV;
    # 332 "sharpness_fuse.cpp"
    edgeV = ((gx) > 0 ? (gx) : -(gx)) + ((gy) > 0 ? (gy) : -(gy));
    # 333 "sharpness_fuse.cpp"
    *(pEdge + (y_1 * width_7 + x_1)) = (hmpprt::s16 ) (edgeV);
   }
  }
  # 313 "sharpness_fuse.cpp"
 }
}
#endif // __HIPCC__



# 313 "sharpness_fuse.cpp"

#ifndef __HIPCC__
void hmpp_acc_region_main_316_internal_1(hmpprt::s32 width_66, hmpprt::s32 height_36, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  yPlane_10, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s16>  pEdge_8)
{
 # 313 "sharpness_fuse.cpp"
 if (1)
 {
  hmpprt::CUDAGridCall __hmppcg_call;
  __hmppcg_call.setSizeX(192);
  __hmppcg_call.setSizeY(1);
  __hmppcg_call.setBlockSizeX(1);
  __hmppcg_call.setBlockSizeY(256);
  __hmppcg_call.addLocalParameter((hmpprt::s32) (height_36), "height_7");
  __hmppcg_call.addLocalParameter(&pEdge_8, 8, "pEdge");
  __hmppcg_call.addLocalParameter((hmpprt::s32) (width_66), "width_7");
  __hmppcg_call.addLocalParameter(&yPlane_10, 8, "yPlane");
  __hmppcg_call.launch(hmpp_acc_region_main_316_parallel_region_1, hmpprt::Context::getInstance()->getCUDADevice());
 }
 ;
}
#endif // __HIPCC__



# 313 "sharpness_fuse.cpp"

#ifndef __HIPCC__
extern "C" CDLT_API  void hmpp_acc_region_main_316(hmpprt::s32 width_48, hmpprt::s32 height_25, hmpprt::u08* yPlane_5, hmpprt::s16* pEdge_4)
{
 # 342 "sharpness_fuse.cpp"
 (hmpp_acc_region_main_316_internal_1(width_48, height_25, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08> (yPlane_5), hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s16> (pEdge_4)));
}
#endif // __HIPCC__



# 342 "sharpness_fuse.cpp"

#ifdef __HIPCC__

extern "C" __global__ void reduce_s64_add_xy(hmpprt::s64 neutral, hmpprt::s64* scalar, hmpprt::s32 size, hmpprt::s64* array)
{
 
 hmpprt::s64 * tmpa = (hmpprt::s64 *)(((char *)hmpp_sharedmem + 0));
 
 # 342 "sharpness_fuse.cpp"
 hmpprt::s64 var;
 # 342 "sharpness_fuse.cpp"
 var = neutral;
 # 342 "sharpness_fuse.cpp"
 hmpprt::s32 end_4;
 # 342 "sharpness_fuse.cpp"
 hmpprt::s32 i;
 # 342 "sharpness_fuse.cpp"
 # 342 "sharpness_fuse.cpp"
 # 342 "sharpness_fuse.cpp"
 for (i = (hmpprt::gr_btidf()), end_4 = size - 1 ; i <= end_4 ; i = i + 512)
 {
  # 342 "sharpness_fuse.cpp"
  var = var + *(array + i);
 }
 # 342 "sharpness_fuse.cpp"
 # 342 "sharpness_fuse.cpp"
 *(tmpa + (hmpprt::gr_btidf())) = var;
 # 342 "sharpness_fuse.cpp"
 hmpprt::s32 n_1;
 # 342 "sharpness_fuse.cpp"
 # 342 "sharpness_fuse.cpp"
 for (n_1 = (511 >> 1) + 1 ; n_1 > 0 ; n_1 = (n_1 >> 1))
 {
  # 342 "sharpness_fuse.cpp"
  hmpprt::s32 next_1;
  # 342 "sharpness_fuse.cpp"
  next_1 = (hmpprt::gr_btidf()) + n_1;
  # 342 "sharpness_fuse.cpp"
  (hmpprt::gr_barrier());
  # 342 "sharpness_fuse.cpp"
  if ((hmpprt::gr_btidf()) < n_1 && next_1 < 512)
  {
   # 342 "sharpness_fuse.cpp"
   *(tmpa + (hmpprt::gr_btidf())) = *(tmpa + (hmpprt::gr_btidf())) + *(tmpa + next_1);
  }
 }
 # 342 "sharpness_fuse.cpp"
 # 342 "sharpness_fuse.cpp"
 if ((hmpprt::gr_btidf()) == 0)
 {
  # 342 "sharpness_fuse.cpp"
  *scalar = *tmpa + *scalar;
 }
}
#endif // __HIPCC__



# 342 "sharpness_fuse.cpp"

#ifdef __HIPCC__

extern "C" __global__ void hmpp_acc_region_main_339_parallel_region_1(hmpprt::s64* gang_prv_red_sum_1, hmpprt::s32 height_6, hmpprt::s16* pEdge_3, hmpprt::s32 width_6)
{
 
 hmpprt::s64 * sum_1 = (hmpprt::s64 *)(((char *)hmpp_sharedmem + 0));
 
 hmpprt::s64 * tmpr_3 = (hmpprt::s64 *)(((char *)hmpp_sharedmem + 0+8));
 
 # 344 "sharpness_fuse.cpp"
 hmpprt::s64 tmp_27;
 # 344 "sharpness_fuse.cpp"
 if ((hmpprt::gr_btidf()) == 0)
 {
  # 344 "sharpness_fuse.cpp"
  *sum_1 = 0;
 }
 # 344 "sharpness_fuse.cpp"
 (hmpprt::gr_barrier());
 # 344 "sharpness_fuse.cpp"
 {
  # 342 "sharpness_fuse.cpp"
  hmpprt::s32 iter_per_gang_6;
  # 342 "sharpness_fuse.cpp"
  hmpprt::s32 first_gang_iter_6;
  # 342 "sharpness_fuse.cpp"
  hmpprt::s32 last_gang_iter_6;
  # 342 "sharpness_fuse.cpp"
  iter_per_gang_6 = ((1 + (height_6 * width_6 - 1) / 192) > 256 ? (1 + (height_6 * width_6 - 1) / 192) : 256);
  # 342 "sharpness_fuse.cpp"
  first_gang_iter_6 = (hmpprt::gr_gbidx()) * iter_per_gang_6;
  # 342 "sharpness_fuse.cpp"
  last_gang_iter_6 = ((first_gang_iter_6 + iter_per_gang_6 - 1) < (height_6 * width_6 - 1) ? (first_gang_iter_6 + iter_per_gang_6 - 1) : (height_6 * width_6 - 1));
  # 342 "sharpness_fuse.cpp"
  hmpprt::s64 sum_2;
  # 348 "sharpness_fuse.cpp"
  sum_2 = 0;
  # 348 "sharpness_fuse.cpp"
  hmpprt::s32 i_4;
  # 348 "sharpness_fuse.cpp"
  # 352 "sharpness_fuse.cpp"
  for (i_4 = first_gang_iter_6 + (hmpprt::gr_btidy()) ; i_4 <= last_gang_iter_6 ; i_4 = i_4 + (hmpprt::gr_btnumy()))
  {
   # 348 "sharpness_fuse.cpp"
   hmpprt::s32 i_2;
   # 351 "sharpness_fuse.cpp"
   hmpprt::s32 j_2;
   # 353 "sharpness_fuse.cpp"
   j_2 = i_4 % width_6;
   # 353 "sharpness_fuse.cpp"
   i_2 = i_4 / width_6;
   # 353 "sharpness_fuse.cpp"
   sum_2 = sum_2 + (hmpprt::s64 ) (*(pEdge_3 + (i_2 * width_6 + j_2)));
  }
  # 342 "sharpness_fuse.cpp"
  # 342 "sharpness_fuse.cpp"
  *sum_1 = (reduce_s64_add_y(sum_2, tmpr_3, *sum_1));
 }
 # 342 "sharpness_fuse.cpp"
 tmp_27 = *sum_1;
 # 342 "sharpness_fuse.cpp"
 if ((hmpprt::gr_btidy()) == 0)
 {
  # 342 "sharpness_fuse.cpp"
  *(gang_prv_red_sum_1 + (hmpprt::gr_gbidx())) = tmp_27;
 }
 # 342 "sharpness_fuse.cpp"
 (hmpprt::gr_barrier());
}
#endif // __HIPCC__



# 342 "sharpness_fuse.cpp"

#ifndef __HIPCC__
void hmpp_acc_region_main_339_internal_1(hmpprt::s32 width_65, hmpprt::s32 height_35, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s16>  pEdge_11, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s64>  sum_4)
{
 # 342 "sharpness_fuse.cpp"
 hmpprt::DevicePtr<hmpprt::MS_CUDA_SHARED,hmpprt::s64>  sum_3;
 # 342 "sharpness_fuse.cpp"
 hmpprt::s64 tmp_28;
 # 342 "sharpness_fuse.cpp"
 hmpprt::Context::getInstance()->allocate((void **) (&sum_3), hmpprt::MS_CUDA_SHARED, 8);
 # 342 "sharpness_fuse.cpp"
 hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s64>  gang_prv_red_sum_2;
 # 342 "sharpness_fuse.cpp"
 hmpprt::Context::getInstance()->allocate((void **) (&gang_prv_red_sum_2), hmpprt::MS_CUDA_GLOB, 1536);
 # 342 "sharpness_fuse.cpp"
 hmpprt::DevicePtr<hmpprt::MS_CUDA_SHARED,hmpprt::s64>  tmpr_2;
 # 342 "sharpness_fuse.cpp"
 hmpprt::Context::getInstance()->allocate((void **) (&tmpr_2), hmpprt::MS_CUDA_SHARED, 2048);
 # 342 "sharpness_fuse.cpp"
 if (1)
 {
  hmpprt::CUDAGridCall __hmppcg_call;
  __hmppcg_call.setSizeX(192);
  __hmppcg_call.setSizeY(1);
  __hmppcg_call.setBlockSizeX(1);
  __hmppcg_call.setBlockSizeY(256);
  __hmppcg_call.addLocalParameter(&gang_prv_red_sum_2, 8, "gang_prv_red_sum_1");
  __hmppcg_call.addLocalParameter((hmpprt::s32) (height_35), "height_6");
  __hmppcg_call.addLocalParameter(&pEdge_11, 8, "pEdge_3");
  __hmppcg_call.addSharedParameter(sum_3, "sum_1");
  __hmppcg_call.addSharedParameter(tmpr_2, "tmpr_3");
  __hmppcg_call.addLocalParameter((hmpprt::s32) (width_65), "width_6");
  __hmppcg_call.launch(hmpp_acc_region_main_339_parallel_region_1, hmpprt::Context::getInstance()->getCUDADevice());
 }
 ;
 # 342 "sharpness_fuse.cpp"
 hmpprt::DevicePtr<hmpprt::MS_CUDA_SHARED,hmpprt::s64>  tmpr_1;
 # 342 "sharpness_fuse.cpp"
 hmpprt::Context::getInstance()->allocate((void **) (&tmpr_1), hmpprt::MS_CUDA_SHARED, 4096);
 # 342 "sharpness_fuse.cpp"
 tmp_28 = 0LL;
 # 342 "sharpness_fuse.cpp"
 if (1)
 {
  hmpprt::CUDAGridCall __hmppcg_call;
  __hmppcg_call.setSizeX(1);
  __hmppcg_call.setSizeY(1);
  __hmppcg_call.setBlockSizeX(512);
  __hmppcg_call.setBlockSizeY(1);
  __hmppcg_call.addLocalParameter(&tmp_28, 8, "neutral");
  __hmppcg_call.addLocalParameter(&sum_4, 8, "scalar");
  __hmppcg_call.addSharedParameter(tmpr_1, "tmpa");
  __hmppcg_call.addLocalParameter((hmpprt::s32) (192), "size");
  __hmppcg_call.addLocalParameter(&gang_prv_red_sum_2, 8, "array");
  __hmppcg_call.launch(reduce_s64_add_xy, hmpprt::Context::getInstance()->getCUDADevice());
 }
 ;
 # 342 "sharpness_fuse.cpp"
 hmpprt::Context::getInstance()->free((void **) (&tmpr_2));
 # 342 "sharpness_fuse.cpp"
 hmpprt::Context::getInstance()->free((void **) (&tmpr_1));
 # 342 "sharpness_fuse.cpp"
 hmpprt::Context::getInstance()->free((void **) (&gang_prv_red_sum_2));
 # 342 "sharpness_fuse.cpp"
 hmpprt::Context::getInstance()->free((void **) (&sum_3));
}
#endif // __HIPCC__



# 342 "sharpness_fuse.cpp"

#ifndef __HIPCC__
extern "C" CDLT_API  void hmpp_acc_region_main_339(hmpprt::s32 width_47, hmpprt::s32 height_24, hmpprt::s16* pEdge_7, hmpprt::s64* sum)
{
 # 361 "sharpness_fuse.cpp"
 (hmpp_acc_region_main_339_internal_1(width_47, height_24, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s16> (pEdge_7), hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s64> (sum)));
}
#endif // __HIPCC__



# 361 "sharpness_fuse.cpp"

#ifdef __HIPCC__

extern "C" __global__ void hmpp_acc_region_main_376_parallel_region_1(hmpprt::s32 height_5, hmpprt::s16* pError, hmpprt::s32 width_5, hmpprt::u08* yPlaneCSER_2, hmpprt::u08* yPlane_4)
{
 # 363 "sharpness_fuse.cpp"
 {
  # 379 "sharpness_fuse.cpp"
  hmpprt::s32 iter_per_gang_5;
  # 379 "sharpness_fuse.cpp"
  hmpprt::s32 first_gang_iter_5;
  # 379 "sharpness_fuse.cpp"
  hmpprt::s32 last_gang_iter_5;
  # 379 "sharpness_fuse.cpp"
  iter_per_gang_5 = ((1 + (height_5 * width_5 - 1) / 192) > 256 ? (1 + (height_5 * width_5 - 1) / 192) : 256);
  # 379 "sharpness_fuse.cpp"
  first_gang_iter_5 = (hmpprt::gr_gbidx()) * iter_per_gang_5;
  # 379 "sharpness_fuse.cpp"
  last_gang_iter_5 = ((first_gang_iter_5 + iter_per_gang_5 - 1) < (height_5 * width_5 - 1) ? (first_gang_iter_5 + iter_per_gang_5 - 1) : (height_5 * width_5 - 1));
  # 379 "sharpness_fuse.cpp"
  hmpprt::s32 h_23;
  # 379 "sharpness_fuse.cpp"
  # 383 "sharpness_fuse.cpp"
  for (h_23 = first_gang_iter_5 + (hmpprt::gr_btidy()) ; h_23 <= last_gang_iter_5 ; h_23 = h_23 + (hmpprt::gr_btnumy()))
  {
   # 379 "sharpness_fuse.cpp"
   hmpprt::s32 h_18;
   # 382 "sharpness_fuse.cpp"
   hmpprt::s32 w_18;
   # 384 "sharpness_fuse.cpp"
   w_18 = h_23 % width_5;
   # 384 "sharpness_fuse.cpp"
   h_18 = h_23 / width_5;
   # 384 "sharpness_fuse.cpp"
   *(pError + (h_18 * width_5 + w_18)) = (hmpprt::s16 ) ((hmpprt::s32 ) (*(yPlane_4 + (h_18 * width_5 + w_18))) - (hmpprt::s32 ) (*(yPlaneCSER_2 + (h_18 * width_5 + w_18))));
  }
  # 361 "sharpness_fuse.cpp"
 }
}
#endif // __HIPCC__



# 361 "sharpness_fuse.cpp"

#ifndef __HIPCC__
void hmpp_acc_region_main_376_internal_1(hmpprt::s32 width_64, hmpprt::s32 height_34, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  yPlane_14, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  yPlaneCSER_37, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s16>  pError_4)
{
 # 361 "sharpness_fuse.cpp"
 if (1)
 {
  hmpprt::CUDAGridCall __hmppcg_call;
  __hmppcg_call.setSizeX(192);
  __hmppcg_call.setSizeY(1);
  __hmppcg_call.setBlockSizeX(1);
  __hmppcg_call.setBlockSizeY(256);
  __hmppcg_call.addLocalParameter((hmpprt::s32) (height_34), "height_5");
  __hmppcg_call.addLocalParameter(&pError_4, 8, "pError");
  __hmppcg_call.addLocalParameter((hmpprt::s32) (width_64), "width_5");
  __hmppcg_call.addLocalParameter(&yPlaneCSER_37, 8, "yPlaneCSER_2");
  __hmppcg_call.addLocalParameter(&yPlane_14, 8, "yPlane_4");
  __hmppcg_call.launch(hmpp_acc_region_main_376_parallel_region_1, hmpprt::Context::getInstance()->getCUDADevice());
 }
 ;
}
#endif // __HIPCC__



# 361 "sharpness_fuse.cpp"

#ifndef __HIPCC__
extern "C" CDLT_API  void hmpp_acc_region_main_376(hmpprt::s32 width_46, hmpprt::s32 height_23, hmpprt::u08* yPlane_9, hmpprt::u08* yPlaneCSER_22, hmpprt::s16* pError_2)
{
 # 392 "sharpness_fuse.cpp"
 (hmpp_acc_region_main_376_internal_1(width_46, height_23, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08> (yPlane_9), hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08> (yPlaneCSER_22), hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s16> (pError_2)));
}
#endif // __HIPCC__



# 392 "sharpness_fuse.cpp"

#ifdef __HIPCC__

extern "C" __global__ void hmpp_acc_region_main_389_parallel_region_1(double cutoff, double cutoff2, hmpprt::s32 height_4, double low, double order, double order2, hmpprt::s16* pEdge_2, hmpprt::s16* pError_1, hmpprt::s16* pSharpness, double peak, hmpprt::s32 width_4, hmpprt::u08* yPlaneCSER_1)
{
 # 394 "sharpness_fuse.cpp"
 {
  # 398 "sharpness_fuse.cpp"
  hmpprt::s32 iter_per_gang_4;
  # 398 "sharpness_fuse.cpp"
  hmpprt::s32 first_gang_iter_4;
  # 398 "sharpness_fuse.cpp"
  hmpprt::s32 last_gang_iter_4;
  # 398 "sharpness_fuse.cpp"
  iter_per_gang_4 = ((1 + (height_4 * width_4 - 1) / 192) > 256 ? (1 + (height_4 * width_4 - 1) / 192) : 256);
  # 398 "sharpness_fuse.cpp"
  first_gang_iter_4 = (hmpprt::gr_gbidx()) * iter_per_gang_4;
  # 398 "sharpness_fuse.cpp"
  last_gang_iter_4 = ((first_gang_iter_4 + iter_per_gang_4 - 1) < (height_4 * width_4 - 1) ? (first_gang_iter_4 + iter_per_gang_4 - 1) : (height_4 * width_4 - 1));
  # 398 "sharpness_fuse.cpp"
  hmpprt::s32 h_24;
  # 398 "sharpness_fuse.cpp"
  # 402 "sharpness_fuse.cpp"
  for (h_24 = first_gang_iter_4 + (hmpprt::gr_btidy()) ; h_24 <= last_gang_iter_4 ; h_24 = h_24 + (hmpprt::gr_btnumy()))
  {
   # 403 "sharpness_fuse.cpp"
   hmpprt::s16 edge;
   # 398 "sharpness_fuse.cpp"
   hmpprt::s32 h_19;
   # 401 "sharpness_fuse.cpp"
   hmpprt::s32 w_19;
   # 403 "sharpness_fuse.cpp"
   w_19 = h_24 % width_4;
   # 403 "sharpness_fuse.cpp"
   h_19 = h_24 / width_4;
   # 403 "sharpness_fuse.cpp"
   edge = *(pEdge_2 + (h_19 * width_4 + w_19));
   # 404 "sharpness_fuse.cpp"
   double weight;
   # 404 "sharpness_fuse.cpp"
   weight = (double) 1. / ((double) 1. + pow((double ) (edge) / cutoff2, (double) 2. * order2));
   # 405 "sharpness_fuse.cpp"
   double strength;
   # 405 "sharpness_fuse.cpp"
   strength = (peak - low) * ((double) 1. - (double) 1. / ((double) 1. + pow((double ) (edge) / cutoff, (double) 2. * order)));
   # 406 "sharpness_fuse.cpp"
   strength = strength * weight + low;
   # 408 "sharpness_fuse.cpp"
   *(pSharpness + (h_19 * width_4 + w_19)) = (hmpprt::s16 ) (*(yPlaneCSER_1 + (h_19 * width_4 + w_19)));
   # 409 "sharpness_fuse.cpp"
   *(pSharpness + (h_19 * width_4 + w_19)) = *(pSharpness + (h_19 * width_4 + w_19)) + (double ) (*(pError_1 + (h_19 * width_4 + w_19))) * strength;
  }
  # 392 "sharpness_fuse.cpp"
 }
}
#endif // __HIPCC__



# 392 "sharpness_fuse.cpp"

#ifndef __HIPCC__
void hmpp_acc_region_main_389_internal_1(hmpprt::s32 width_63, hmpprt::s32 height_33, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  yPlaneCSER_35, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s16>  pError_5, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s16>  pEdge_10, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s16>  pSharpness_7, double cutoff_2, double peak_2, double cutoff2_2, double order_2, double order2_2, double low_2)
{
 # 392 "sharpness_fuse.cpp"
 if (1)
 {
  hmpprt::CUDAGridCall __hmppcg_call;
  __hmppcg_call.setSizeX(192);
  __hmppcg_call.setSizeY(1);
  __hmppcg_call.setBlockSizeX(1);
  __hmppcg_call.setBlockSizeY(256);
  __hmppcg_call.addLocalParameter(&cutoff_2, 8, "cutoff");
  __hmppcg_call.addLocalParameter(&cutoff2_2, 8, "cutoff2");
  __hmppcg_call.addLocalParameter((hmpprt::s32) (height_33), "height_4");
  __hmppcg_call.addLocalParameter(&low_2, 8, "low");
  __hmppcg_call.addLocalParameter(&order_2, 8, "order");
  __hmppcg_call.addLocalParameter(&order2_2, 8, "order2");
  __hmppcg_call.addLocalParameter(&pEdge_10, 8, "pEdge_2");
  __hmppcg_call.addLocalParameter(&pError_5, 8, "pError_1");
  __hmppcg_call.addLocalParameter(&pSharpness_7, 8, "pSharpness");
  __hmppcg_call.addLocalParameter(&peak_2, 8, "peak");
  __hmppcg_call.addLocalParameter((hmpprt::s32) (width_63), "width_4");
  __hmppcg_call.addLocalParameter(&yPlaneCSER_35, 8, "yPlaneCSER_1");
  __hmppcg_call.launch(hmpp_acc_region_main_389_parallel_region_1, hmpprt::Context::getInstance()->getCUDADevice());
 }
 ;
}
#endif // __HIPCC__



# 392 "sharpness_fuse.cpp"

#ifndef __HIPCC__
extern "C" CDLT_API  void hmpp_acc_region_main_389(hmpprt::s32 width_45, hmpprt::s32 height_22, hmpprt::u08* yPlaneCSER_16, hmpprt::s16* pError_3, hmpprt::s16* pEdge_6, hmpprt::s16* pSharpness_3, double cutoff_1, double peak_1, double cutoff2_1, double order_1, double order2_1, double low_1)
{
 # 417 "sharpness_fuse.cpp"
 (hmpp_acc_region_main_389_internal_1(width_45, height_22, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08> (yPlaneCSER_16), hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s16> (pError_3), hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s16> (pEdge_6), hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s16> (pSharpness_3), cutoff_1, peak_1, cutoff2_1, order_1, order2_1, low_1));
}
#endif // __HIPCC__



# 417 "sharpness_fuse.cpp"

#ifdef __HIPCC__

extern "C" __global__ void hmpp_acc_region_main_408_parallel_region_1(float cutoff3, hmpprt::s32 height_3, float low3, float order3, hmpprt::s16* pEdge_1, hmpprt::s16* pSharpness_5, float peak3_1, hmpprt::s32 width_3, hmpprt::u08* yPlane_3)
{
 # 419 "sharpness_fuse.cpp"
 {
  # 423 "sharpness_fuse.cpp"
  hmpprt::s32 iter_per_gang_3;
  # 423 "sharpness_fuse.cpp"
  hmpprt::s32 first_gang_iter_3;
  # 423 "sharpness_fuse.cpp"
  hmpprt::s32 last_gang_iter_3;
  # 423 "sharpness_fuse.cpp"
  iter_per_gang_3 = ((1 + (height_3 * width_3 - 1) / 192) > 256 ? (1 + (height_3 * width_3 - 1) / 192) : 256);
  # 423 "sharpness_fuse.cpp"
  first_gang_iter_3 = (hmpprt::gr_gbidx()) * iter_per_gang_3;
  # 423 "sharpness_fuse.cpp"
  last_gang_iter_3 = ((first_gang_iter_3 + iter_per_gang_3 - 1) < (height_3 * width_3 - 1) ? (first_gang_iter_3 + iter_per_gang_3 - 1) : (height_3 * width_3 - 1));
  # 423 "sharpness_fuse.cpp"
  hmpprt::s32 h_25;
  # 423 "sharpness_fuse.cpp"
  # 427 "sharpness_fuse.cpp"
  for (h_25 = first_gang_iter_3 + (hmpprt::gr_btidy()) ; h_25 <= last_gang_iter_3 ; h_25 = h_25 + (hmpprt::gr_btnumy()))
  {
   # 423 "sharpness_fuse.cpp"
   hmpprt::s32 h_20;
   # 426 "sharpness_fuse.cpp"
   hmpprt::s32 w_20;
   # 428 "sharpness_fuse.cpp"
   w_20 = h_25 % width_3;
   # 428 "sharpness_fuse.cpp"
   h_20 = h_25 / width_3;
   # 428 "sharpness_fuse.cpp"
   if (!(h_20 == 0 || h_20 == height_3 - 1 || w_20 == 0 || w_20 == width_3 - 1))
   {
    # 430 "sharpness_fuse.cpp"
    hmpprt::s32 enhanced;
    # 430 "sharpness_fuse.cpp"
    hmpprt::s32 maxValue;
    # 430 "sharpness_fuse.cpp"
    hmpprt::s32 minValue;
    # 431 "sharpness_fuse.cpp"
    double oscStrength;
    # 432 "sharpness_fuse.cpp"
    enhanced = (hmpprt::s32 ) (*(pSharpness_5 + (h_20 * width_3 + w_20)));
    # 434 "sharpness_fuse.cpp"
    maxValue = (hmpprt::s32 ) ( ((hmpprt::s32 ) ( ((hmpprt::s32 ) (*(yPlane_3 + ((h_20 - 1) * width_3 + w_20 - 1))) > (hmpprt::s32 ) (*(yPlane_3 + ((h_20 - 1) * width_3 + w_20)))) ? (*(yPlane_3 + ((h_20 - 1) * width_3 + w_20 - 1))) : (*(yPlane_3 + ((h_20 - 1) * width_3 + w_20)))) > (hmpprt::s32 ) (*(yPlane_3 + ((h_20 - 1) * width_3 + w_20 + 1)))) ? ( ((hmpprt::s32 ) (*(yPlane_3 + ((h_20 - 1) * width_3 + w_20 - 1))) > (hmpprt::s32 ) (*(yPlane_3 + ((h_20 - 1) * width_3 + w_20)))) ? (*(yPlane_3 + ((h_20 - 1) * width_3 + w_20 - 1))) : (*(yPlane_3 + ((h_20 - 1) * width_3 + w_20)))) : (*(yPlane_3 + ((h_20 - 1) * width_3 + w_20 + 1))));
    # 423 "sharpness_fuse.cpp"
    maxValue = ( (maxValue > (hmpprt::s32 ) ( ((hmpprt::s32 ) ( ((hmpprt::s32 ) (*(yPlane_3 + (h_20 * width_3 + w_20 - 1))) > (hmpprt::s32 ) (*(yPlane_3 + (h_20 * width_3 + w_20)))) ? (*(yPlane_3 + (h_20 * width_3 + w_20 - 1))) : (*(yPlane_3 + (h_20 * width_3 + w_20)))) > (hmpprt::s32 ) (*(yPlane_3 + (h_20 * width_3 + w_20 + 1)))) ? ( ((hmpprt::s32 ) (*(yPlane_3 + (h_20 * width_3 + w_20 - 1))) > (hmpprt::s32 ) (*(yPlane_3 + (h_20 * width_3 + w_20)))) ? (*(yPlane_3 + (h_20 * width_3 + w_20 - 1))) : (*(yPlane_3 + (h_20 * width_3 + w_20)))) : (*(yPlane_3 + (h_20 * width_3 + w_20 + 1))))) ? maxValue : ((hmpprt::s32 ) ( ((hmpprt::s32 ) ( ((hmpprt::s32 ) (*(yPlane_3 + (h_20 * width_3 + w_20 - 1))) > (hmpprt::s32 ) (*(yPlane_3 + (h_20 * width_3 + w_20)))) ? (*(yPlane_3 + (h_20 * width_3 + w_20 - 1))) : (*(yPlane_3 + (h_20 * width_3 + w_20)))) > (hmpprt::s32 ) (*(yPlane_3 + (h_20 * width_3 + w_20 + 1)))) ? ( ((hmpprt::s32 ) (*(yPlane_3 + (h_20 * width_3 + w_20 - 1))) > (hmpprt::s32 ) (*(yPlane_3 + (h_20 * width_3 + w_20)))) ? (*(yPlane_3 + (h_20 * width_3 + w_20 - 1))) : (*(yPlane_3 + (h_20 * width_3 + w_20)))) : (*(yPlane_3 + (h_20 * width_3 + w_20 + 1))))));
    # 424 "sharpness_fuse.cpp"
    maxValue = ( (maxValue > (hmpprt::s32 ) ( ((hmpprt::s32 ) ( ((hmpprt::s32 ) (*(yPlane_3 + ((h_20 + 1) * width_3 + w_20 - 1))) > (hmpprt::s32 ) (*(yPlane_3 + ((h_20 + 1) * width_3 + w_20)))) ? (*(yPlane_3 + ((h_20 + 1) * width_3 + w_20 - 1))) : (*(yPlane_3 + ((h_20 + 1) * width_3 + w_20)))) > (hmpprt::s32 ) (*(yPlane_3 + ((h_20 + 1) * width_3 + w_20 + 1)))) ? ( ((hmpprt::s32 ) (*(yPlane_3 + ((h_20 + 1) * width_3 + w_20 - 1))) > (hmpprt::s32 ) (*(yPlane_3 + ((h_20 + 1) * width_3 + w_20)))) ? (*(yPlane_3 + ((h_20 + 1) * width_3 + w_20 - 1))) : (*(yPlane_3 + ((h_20 + 1) * width_3 + w_20)))) : (*(yPlane_3 + ((h_20 + 1) * width_3 + w_20 + 1))))) ? maxValue : ((hmpprt::s32 ) ( ((hmpprt::s32 ) ( ((hmpprt::s32 ) (*(yPlane_3 + ((h_20 + 1) * width_3 + w_20 - 1))) > (hmpprt::s32 ) (*(yPlane_3 + ((h_20 + 1) * width_3 + w_20)))) ? (*(yPlane_3 + ((h_20 + 1) * width_3 + w_20 - 1))) : (*(yPlane_3 + ((h_20 + 1) * width_3 + w_20)))) > (hmpprt::s32 ) (*(yPlane_3 + ((h_20 + 1) * width_3 + w_20 + 1)))) ? ( ((hmpprt::s32 ) (*(yPlane_3 + ((h_20 + 1) * width_3 + w_20 - 1))) > (hmpprt::s32 ) (*(yPlane_3 + ((h_20 + 1) * width_3 + w_20)))) ? (*(yPlane_3 + ((h_20 + 1) * width_3 + w_20 - 1))) : (*(yPlane_3 + ((h_20 + 1) * width_3 + w_20)))) : (*(yPlane_3 + ((h_20 + 1) * width_3 + w_20 + 1))))));
    # 426 "sharpness_fuse.cpp"
    minValue = (hmpprt::s32 ) ( ((hmpprt::s32 ) (*(yPlane_3 + ((h_20 - 1) * width_3 + w_20 - 1))) < (hmpprt::s32 ) ( ((hmpprt::s32 ) (*(yPlane_3 + ((h_20 - 1) * width_3 + w_20))) < (hmpprt::s32 ) (*(yPlane_3 + ((h_20 - 1) * width_3 + w_20 + 1)))) ? (*(yPlane_3 + ((h_20 - 1) * width_3 + w_20))) : (*(yPlane_3 + ((h_20 - 1) * width_3 + w_20 + 1))))) ? (*(yPlane_3 + ((h_20 - 1) * width_3 + w_20 - 1))) : ( ((hmpprt::s32 ) (*(yPlane_3 + ((h_20 - 1) * width_3 + w_20))) < (hmpprt::s32 ) (*(yPlane_3 + ((h_20 - 1) * width_3 + w_20 + 1)))) ? (*(yPlane_3 + ((h_20 - 1) * width_3 + w_20))) : (*(yPlane_3 + ((h_20 - 1) * width_3 + w_20 + 1)))));
    # 427 "sharpness_fuse.cpp"
    minValue = ( (minValue < (hmpprt::s32 ) ( ((hmpprt::s32 ) (*(yPlane_3 + (h_20 * width_3 + w_20 - 1))) < (hmpprt::s32 ) ( ((hmpprt::s32 ) (*(yPlane_3 + (h_20 * width_3 + w_20))) < (hmpprt::s32 ) (*(yPlane_3 + (h_20 * width_3 + w_20 + 1)))) ? (*(yPlane_3 + (h_20 * width_3 + w_20))) : (*(yPlane_3 + (h_20 * width_3 + w_20 + 1))))) ? (*(yPlane_3 + (h_20 * width_3 + w_20 - 1))) : ( ((hmpprt::s32 ) (*(yPlane_3 + (h_20 * width_3 + w_20))) < (hmpprt::s32 ) (*(yPlane_3 + (h_20 * width_3 + w_20 + 1)))) ? (*(yPlane_3 + (h_20 * width_3 + w_20))) : (*(yPlane_3 + (h_20 * width_3 + w_20 + 1)))))) ? minValue : ((hmpprt::s32 ) ( ((hmpprt::s32 ) (*(yPlane_3 + (h_20 * width_3 + w_20 - 1))) < (hmpprt::s32 ) ( ((hmpprt::s32 ) (*(yPlane_3 + (h_20 * width_3 + w_20))) < (hmpprt::s32 ) (*(yPlane_3 + (h_20 * width_3 + w_20 + 1)))) ? (*(yPlane_3 + (h_20 * width_3 + w_20))) : (*(yPlane_3 + (h_20 * width_3 + w_20 + 1))))) ? (*(yPlane_3 + (h_20 * width_3 + w_20 - 1))) : ( ((hmpprt::s32 ) (*(yPlane_3 + (h_20 * width_3 + w_20))) < (hmpprt::s32 ) (*(yPlane_3 + (h_20 * width_3 + w_20 + 1)))) ? (*(yPlane_3 + (h_20 * width_3 + w_20))) : (*(yPlane_3 + (h_20 * width_3 + w_20 + 1)))))));
    # 428 "sharpness_fuse.cpp"
    minValue = ( (minValue < (hmpprt::s32 ) ( ((hmpprt::s32 ) (*(yPlane_3 + ((h_20 + 1) * width_3 + w_20 - 1))) < (hmpprt::s32 ) ( ((hmpprt::s32 ) (*(yPlane_3 + ((h_20 + 1) * width_3 + w_20))) < (hmpprt::s32 ) (*(yPlane_3 + ((h_20 + 1) * width_3 + w_20 + 1)))) ? (*(yPlane_3 + ((h_20 + 1) * width_3 + w_20))) : (*(yPlane_3 + ((h_20 + 1) * width_3 + w_20 + 1))))) ? (*(yPlane_3 + ((h_20 + 1) * width_3 + w_20 - 1))) : ( ((hmpprt::s32 ) (*(yPlane_3 + ((h_20 + 1) * width_3 + w_20))) < (hmpprt::s32 ) (*(yPlane_3 + ((h_20 + 1) * width_3 + w_20 + 1)))) ? (*(yPlane_3 + ((h_20 + 1) * width_3 + w_20))) : (*(yPlane_3 + ((h_20 + 1) * width_3 + w_20 + 1)))))) ? minValue : ((hmpprt::s32 ) ( ((hmpprt::s32 ) (*(yPlane_3 + ((h_20 + 1) * width_3 + w_20 - 1))) < (hmpprt::s32 ) ( ((hmpprt::s32 ) (*(yPlane_3 + ((h_20 + 1) * width_3 + w_20))) < (hmpprt::s32 ) (*(yPlane_3 + ((h_20 + 1) * width_3 + w_20 + 1)))) ? (*(yPlane_3 + ((h_20 + 1) * width_3 + w_20))) : (*(yPlane_3 + ((h_20 + 1) * width_3 + w_20 + 1))))) ? (*(yPlane_3 + ((h_20 + 1) * width_3 + w_20 - 1))) : ( ((hmpprt::s32 ) (*(yPlane_3 + ((h_20 + 1) * width_3 + w_20))) < (hmpprt::s32 ) (*(yPlane_3 + ((h_20 + 1) * width_3 + w_20 + 1)))) ? (*(yPlane_3 + ((h_20 + 1) * width_3 + w_20))) : (*(yPlane_3 + ((h_20 + 1) * width_3 + w_20 + 1)))))));
    # 432 "sharpness_fuse.cpp"
    double cutoff2Order;
    # 432 "sharpness_fuse.cpp"
    cutoff2Order = pow((double ) (cutoff3), (double ) ((float) 2. * order3));
    # 433 "sharpness_fuse.cpp"
    double edge2Order;
    # 433 "sharpness_fuse.cpp"
    edge2Order = pow((double ) (*(pEdge_1 + (h_20 * width_3 + w_20))), (double) 2. * (double ) (order3));
    # 434 "sharpness_fuse.cpp"
    oscStrength = ((double ) (peak3_1) * cutoff2Order + (double ) (low3) * edge2Order) / (cutoff2Order + edge2Order);
    # 438 "sharpness_fuse.cpp"
    if (enhanced > maxValue)
    {
     # 440 "sharpness_fuse.cpp"
     *(pSharpness_5 + (h_20 * width_3 + w_20)) = (hmpprt::s16 ) ( ((double ) (maxValue) + oscStrength * (double ) (enhanced - maxValue) < (double) 255.) ? ((double ) (maxValue) + oscStrength * (double ) (enhanced - maxValue)) : ((double) 255.));
    }
    else
    {
     # 442 "sharpness_fuse.cpp"
     if (enhanced < minValue)
     {
      # 444 "sharpness_fuse.cpp"
      *(pSharpness_5 + (h_20 * width_3 + w_20)) = (hmpprt::s16 ) ( ((double) 0. > (double ) (minValue) - oscStrength * (double ) (minValue - enhanced)) ? ((double) 0.) : ((double ) (minValue) - oscStrength * (double ) (minValue - enhanced)));
     }
     else
     {
      # 448 "sharpness_fuse.cpp"
      *(pSharpness_5 + (h_20 * width_3 + w_20)) = (hmpprt::s16 ) ( (( (enhanced > 0) ? enhanced : 0) < 255) ? ( (enhanced > 0) ? enhanced : 0) : 255);
     }
    }
   }
  }
  # 417 "sharpness_fuse.cpp"
 }
}
#endif // __HIPCC__



# 417 "sharpness_fuse.cpp"

#ifndef __HIPCC__
void hmpp_acc_region_main_408_internal_1(hmpprt::s32 width_62, hmpprt::s32 height_32, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  yPlane_13, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s16>  pEdge_9, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s16>  pSharpness_6, float cutoff3_2, float order3_1, float peak3_2, float low3_2)
{
 # 417 "sharpness_fuse.cpp"
 if (1)
 {
  hmpprt::CUDAGridCall __hmppcg_call;
  __hmppcg_call.setSizeX(192);
  __hmppcg_call.setSizeY(1);
  __hmppcg_call.setBlockSizeX(1);
  __hmppcg_call.setBlockSizeY(256);
  __hmppcg_call.addLocalParameter((float) (cutoff3_2) , "cutoff3");
  __hmppcg_call.addLocalParameter((hmpprt::s32) (height_32), "height_3");
  __hmppcg_call.addLocalParameter((float) (low3_2) , "low3");
  __hmppcg_call.addLocalParameter((float) (order3_1) , "order3");
  __hmppcg_call.addLocalParameter(&pEdge_9, 8, "pEdge_1");
  __hmppcg_call.addLocalParameter(&pSharpness_6, 8, "pSharpness_5");
  __hmppcg_call.addLocalParameter((float) (peak3_2) , "peak3_1");
  __hmppcg_call.addLocalParameter((hmpprt::s32) (width_62), "width_3");
  __hmppcg_call.addLocalParameter(&yPlane_13, 8, "yPlane_3");
  __hmppcg_call.launch(hmpp_acc_region_main_408_parallel_region_1, hmpprt::Context::getInstance()->getCUDADevice());
 }
 ;
}
#endif // __HIPCC__



# 417 "sharpness_fuse.cpp"

#ifndef __HIPCC__
extern "C" CDLT_API  void hmpp_acc_region_main_408(hmpprt::s32 width_43, hmpprt::s32 height_21, hmpprt::u08* yPlane_8, hmpprt::s16* pEdge_5, hmpprt::s16* pSharpness_2, float cutoff3_1, float order3_2, float peak3, float low3_1)
{
 # 458 "sharpness_fuse.cpp"
 (hmpp_acc_region_main_408_internal_1(width_43, height_21, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08> (yPlane_8), hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s16> (pEdge_5), hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s16> (pSharpness_2), cutoff3_1, order3_2, peak3, low3_1));
}
#endif // __HIPCC__



# 458 "sharpness_fuse.cpp"

#ifdef __HIPCC__

extern "C" __global__ void hmpp_acc_region_main_462_parallel_region_1(hmpprt::s32 height_2, hmpprt::s16* pSharpness_1, hmpprt::s32 width_2, hmpprt::u08* yPlane_2)
{
 # 460 "sharpness_fuse.cpp"
 {
  # 465 "sharpness_fuse.cpp"
  hmpprt::s32 iter_per_gang_2;
  # 465 "sharpness_fuse.cpp"
  hmpprt::s32 first_gang_iter_2;
  # 465 "sharpness_fuse.cpp"
  hmpprt::s32 last_gang_iter_2;
  # 465 "sharpness_fuse.cpp"
  iter_per_gang_2 = ((1 + (height_2 * width_2 - 1) / 192) > 256 ? (1 + (height_2 * width_2 - 1) / 192) : 256);
  # 465 "sharpness_fuse.cpp"
  first_gang_iter_2 = (hmpprt::gr_gbidx()) * iter_per_gang_2;
  # 465 "sharpness_fuse.cpp"
  last_gang_iter_2 = ((first_gang_iter_2 + iter_per_gang_2 - 1) < (height_2 * width_2 - 1) ? (first_gang_iter_2 + iter_per_gang_2 - 1) : (height_2 * width_2 - 1));
  # 465 "sharpness_fuse.cpp"
  hmpprt::s32 h_26;
  # 465 "sharpness_fuse.cpp"
  # 469 "sharpness_fuse.cpp"
  for (h_26 = first_gang_iter_2 + (hmpprt::gr_btidy()) ; h_26 <= last_gang_iter_2 ; h_26 = h_26 + (hmpprt::gr_btnumy()))
  {
   # 465 "sharpness_fuse.cpp"
   hmpprt::s32 h_21;
   # 468 "sharpness_fuse.cpp"
   hmpprt::s32 w_21;
   # 470 "sharpness_fuse.cpp"
   w_21 = h_26 % width_2;
   # 470 "sharpness_fuse.cpp"
   h_21 = h_26 / width_2;
   # 470 "sharpness_fuse.cpp"
   *(yPlane_2 + (h_21 * width_2 + w_21)) = (hmpprt::u08 ) ( ((hmpprt::s32 ) (*(pSharpness_1 + (h_21 * width_2 + w_21))) > 255) ? 255 : ( ((hmpprt::s32 ) (*(pSharpness_1 + (h_21 * width_2 + w_21))) < 0) ? 0 : ((hmpprt::s32 ) (*(pSharpness_1 + (h_21 * width_2 + w_21))))));
  }
  # 458 "sharpness_fuse.cpp"
 }
}
#endif // __HIPCC__



# 458 "sharpness_fuse.cpp"

#ifndef __HIPCC__
void hmpp_acc_region_main_462_internal_1(hmpprt::s32 width_61, hmpprt::s32 height_31, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  yPlane_12, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s16>  pSharpness_8)
{
 # 458 "sharpness_fuse.cpp"
 if (1)
 {
  hmpprt::CUDAGridCall __hmppcg_call;
  __hmppcg_call.setSizeX(192);
  __hmppcg_call.setSizeY(1);
  __hmppcg_call.setBlockSizeX(1);
  __hmppcg_call.setBlockSizeY(256);
  __hmppcg_call.addLocalParameter((hmpprt::s32) (height_31), "height_2");
  __hmppcg_call.addLocalParameter(&pSharpness_8, 8, "pSharpness_1");
  __hmppcg_call.addLocalParameter((hmpprt::s32) (width_61), "width_2");
  __hmppcg_call.addLocalParameter(&yPlane_12, 8, "yPlane_2");
  __hmppcg_call.launch(hmpp_acc_region_main_462_parallel_region_1, hmpprt::Context::getInstance()->getCUDADevice());
 }
 ;
}
#endif // __HIPCC__



# 458 "sharpness_fuse.cpp"

#ifndef __HIPCC__
extern "C" CDLT_API  void hmpp_acc_region_main_462(hmpprt::s32 width_34, hmpprt::s32 height_20, hmpprt::u08* yPlane_7, hmpprt::s16* pSharpness_4)
{
 # 23 "sharpness_fuse.cpp"
 (hmpp_acc_region_main_462_internal_1(width_34, height_20, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08> (yPlane_7), hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s16> (pSharpness_4)));
}
#endif // __HIPCC__



# 23 "sharpness_fuse.cpp"

#ifdef __HIPCC__

extern "C" __global__ void hmpp_acc_region_main_82_parallel_region_1(hmpprt::s32 height_1, hmpprt::s32 quarterWidth_1, hmpprt::s32 width_1, hmpprt::u08* yPlaneDown_1, hmpprt::u08* yPlane_1)
{
 # 25 "sharpness_fuse.cpp"
 {
  # 85 "sharpness_fuse.cpp"
  hmpprt::s32 iter_per_gang_1;
  # 85 "sharpness_fuse.cpp"
  hmpprt::s32 first_gang_iter_1;
  # 85 "sharpness_fuse.cpp"
  hmpprt::s32 last_gang_iter_1;
  # 85 "sharpness_fuse.cpp"
  iter_per_gang_1 = ((1 + ((height_1 + 3) / 4 * ((width_1 + 3) / 4) - 1) / 192) > 256 ? (1 + ((height_1 + 3) / 4 * ((width_1 + 3) / 4) - 1) / 192) : 256);
  # 85 "sharpness_fuse.cpp"
  first_gang_iter_1 = (hmpprt::gr_gbidx()) * iter_per_gang_1;
  # 85 "sharpness_fuse.cpp"
  last_gang_iter_1 = ((first_gang_iter_1 + iter_per_gang_1 - 1) < ((height_1 + 3) / 4 * ((width_1 + 3) / 4) - 1) ? (first_gang_iter_1 + iter_per_gang_1 - 1) : ((height_1 + 3) / 4 * ((width_1 + 3) / 4) - 1));
  # 85 "sharpness_fuse.cpp"
  hmpprt::s32 i_5;
  # 85 "sharpness_fuse.cpp"
  # 89 "sharpness_fuse.cpp"
  for (i_5 = first_gang_iter_1 + (hmpprt::gr_btidy()) ; i_5 <= last_gang_iter_1 ; i_5 = i_5 + (hmpprt::gr_btnumy()))
  {
   # 85 "sharpness_fuse.cpp"
   hmpprt::u08 tbt_1;
   # 85 "sharpness_fuse.cpp"
   hmpprt::s32 tbt_2;
   # 85 "sharpness_fuse.cpp"
   hmpprt::s32 tbt_3;
   # 85 "sharpness_fuse.cpp"
   hmpprt::s32 i_3;
   # 88 "sharpness_fuse.cpp"
   hmpprt::s32 j_3;
   # 90 "sharpness_fuse.cpp"
   j_3 = i_5 % ((width_1 + 3) / 4);
   # 90 "sharpness_fuse.cpp"
   i_3 = i_5 / ((width_1 + 3) / 4);
   # 90 "sharpness_fuse.cpp"
   tbt_1 = *(yPlane_1 + ((i_3 * 4 + 1) * width_1 + j_3 * 4 + 3));
   # 90 "sharpness_fuse.cpp"
   tbt_2 = (hmpprt::s32 ) (*(yPlane_1 + ((i_3 * 4 + 1) * width_1 + j_3 * 4 + 2)));
   # 90 "sharpness_fuse.cpp"
   tbt_3 = (hmpprt::s32 ) (*(yPlane_1 + (i_3 * 4 * width_1 + j_3 * 4))) + (hmpprt::s32 ) (*(yPlane_1 + (i_3 * 4 * width_1 + j_3 * 4 + 1))) + (hmpprt::s32 ) (*(yPlane_1 + (i_3 * 4 * width_1 + j_3 * 4 + 2))) + (hmpprt::s32 ) (*(yPlane_1 + (i_3 * 4 * width_1 + j_3 * 4 + 3))) + (hmpprt::s32 ) (*(yPlane_1 + ((i_3 * 4 + 1) * width_1 + j_3 * 4))) + (hmpprt::s32 ) (*(yPlane_1 + ((i_3 * 4 + 1) * width_1 + j_3 * 4 + 1)));
   # 90 "sharpness_fuse.cpp"
   *(yPlaneDown_1 + (i_3 * 4 / 4 * quarterWidth_1 + j_3 * 4 / 4)) = (hmpprt::u08 ) (tbt_3 + tbt_2 + (hmpprt::s32 ) (tbt_1) + (hmpprt::s32 ) (*(yPlane_1 + ((i_3 * 4 + 2) * width_1 + j_3 * 4))) + (hmpprt::s32 ) (*(yPlane_1 + ((i_3 * 4 + 2) * width_1 + j_3 * 4 + 1))) + (hmpprt::s32 ) (*(yPlane_1 + ((i_3 * 4 + 2) * width_1 + j_3 * 4 + 2))) + (hmpprt::s32 ) (*(yPlane_1 + ((i_3 * 4 + 2) * width_1 + j_3 * 4 + 3))) + (hmpprt::s32 ) (*(yPlane_1 + ((i_3 * 4 + 3) * width_1 + j_3 * 4))) + (hmpprt::s32 ) (*(yPlane_1 + ((i_3 * 4 + 3) * width_1 + j_3 * 4 + 1))) + (hmpprt::s32 ) (*(yPlane_1 + ((i_3 * 4 + 3) * width_1 + j_3 * 4 + 2))) + (hmpprt::s32 ) (*(yPlane_1 + ((i_3 * 4 + 3) * width_1 + j_3 * 4 + 3))) + 8 >> 4);
  }
  # 23 "sharpness_fuse.cpp"
 }
}
#endif // __HIPCC__



# 23 "sharpness_fuse.cpp"

#ifndef __HIPCC__
void hmpp_acc_region_main_82_internal_1(hmpprt::s32 width_60, hmpprt::s32 height_30, hmpprt::s32 quarterWidth_23, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  yPlane_11, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  yPlaneDown_15)
{
 # 23 "sharpness_fuse.cpp"
 if (1)
 {
  hmpprt::CUDAGridCall __hmppcg_call;
  __hmppcg_call.setSizeX(192);
  __hmppcg_call.setSizeY(1);
  __hmppcg_call.setBlockSizeX(1);
  __hmppcg_call.setBlockSizeY(256);
  __hmppcg_call.addLocalParameter((hmpprt::s32) (height_30), "height_1");
  __hmppcg_call.addLocalParameter((hmpprt::s32) (quarterWidth_23), "quarterWidth_1");
  __hmppcg_call.addLocalParameter((hmpprt::s32) (width_60), "width_1");
  __hmppcg_call.addLocalParameter(&yPlaneDown_15, 8, "yPlaneDown_1");
  __hmppcg_call.addLocalParameter(&yPlane_11, 8, "yPlane_1");
  __hmppcg_call.launch(hmpp_acc_region_main_82_parallel_region_1, hmpprt::Context::getInstance()->getCUDADevice());
 }
 ;
}
#endif // __HIPCC__



# 23 "sharpness_fuse.cpp"

#ifndef __HIPCC__
extern "C" CDLT_API  void hmpp_acc_region_main_82(hmpprt::s32 width_32, hmpprt::s32 height_16, hmpprt::s32 quarterWidth_9, hmpprt::u08* yPlane_6, hmpprt::u08* yPlaneDown_7)
{
 # 1 "<preprocessor>"
 (hmpp_acc_region_main_82_internal_1(width_32, height_16, quarterWidth_9, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08> (yPlane_6), hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08> (yPlaneDown_7)));
}
#endif // __HIPCC__



# 1 "<preprocessor>"

#ifdef __HIPCC__
__device__ hmpprt::s64 reduce_s64_add_y(hmpprt::s64 priv, hmpprt::s64* tmpa_1, hmpprt::s64 orig)
{
 # 1 "<preprocessor>"
 (hmpprt::gr_barrier());
 # 1 "<preprocessor>"
 *(tmpa_1 + (hmpprt::gr_btidy()) + (hmpprt::gr_btnumy()) * (hmpprt::gr_btidx())) = priv;
 # 1 "<preprocessor>"
 hmpprt::s32 n;
 # 1 "<preprocessor>"
 # 1 "<preprocessor>"
 for (n = ((hmpprt::gr_btnumy()) - 1 >> 1) + 1 ; n > 0 ; n = (n >> 1))
 {
  # 1 "<preprocessor>"
  hmpprt::s32 next;
  # 1 "<preprocessor>"
  next = (hmpprt::gr_btidy()) + n;
  # 1 "<preprocessor>"
  (hmpprt::gr_barrier());
  # 1 "<preprocessor>"
  if ((hmpprt::gr_btidy()) < n && next < (hmpprt::gr_btnumy()))
  {
   # 1 "<preprocessor>"
   *(tmpa_1 + (hmpprt::gr_btidy()) + (hmpprt::gr_btnumy()) * (hmpprt::gr_btidx())) = *(tmpa_1 + (hmpprt::gr_btidy()) + (hmpprt::gr_btnumy()) * (hmpprt::gr_btidx())) + *(tmpa_1 + next + (hmpprt::gr_btnumy()) * (hmpprt::gr_btidx()));
  }
 }
 # 1 "<preprocessor>"
 # 1 "<preprocessor>"
 (hmpprt::gr_barrier());
 # 1 "<preprocessor>"
 return *(tmpa_1 + (hmpprt::gr_btnumy()) * (hmpprt::gr_btidx())) + orig;
}
#endif // __HIPCC__




#ifndef __HIPCC__
extern "C" const char * hmpprt_cuda_get_gpu_code();

static hmpprt::CUDAModule * hmpprt_module = 0;
static int hmpprt_uses = 0;

extern "C" CDLT_API void * hmpprt_init()
{
  try
  {
    if (hmpprt_uses++ == 0)
    {
      hmpprt_module = new hmpprt::CUDAModule(hmpprt_cuda_get_gpu_code());
      hmpp_acc_region_main_105_parallel_region_1 = new hmpprt::CUDAGrid(hmpprt_module, "hmpp_acc_region_main_105_parallel_region_1");
      hmpp_acc_region_main_117_parallel_region_1 = new hmpprt::CUDAGrid(hmpprt_module, "hmpp_acc_region_main_117_parallel_region_1");
      hmpp_acc_region_main_138_loop1D_1 = new hmpprt::CUDAGrid(hmpprt_module, "hmpp_acc_region_main_138_loop1D_1");
      hmpp_acc_region_main_151_loop1D_1 = new hmpprt::CUDAGrid(hmpprt_module, "hmpp_acc_region_main_151_loop1D_1");
      hmpp_acc_region_main_159_loop1D_1 = new hmpprt::CUDAGrid(hmpprt_module, "hmpp_acc_region_main_159_loop1D_1");
      hmpp_acc_region_main_172_loop1D_1 = new hmpprt::CUDAGrid(hmpprt_module, "hmpp_acc_region_main_172_loop1D_1");
      hmpp_acc_region_main_189_loop1D_1 = new hmpprt::CUDAGrid(hmpprt_module, "hmpp_acc_region_main_189_loop1D_1");
      hmpp_acc_region_main_211_loop1D_1 = new hmpprt::CUDAGrid(hmpprt_module, "hmpp_acc_region_main_211_loop1D_1");
      hmpp_acc_region_main_220_loop1D_1 = new hmpprt::CUDAGrid(hmpprt_module, "hmpp_acc_region_main_220_loop1D_1");
      hmpp_acc_region_main_232_loop1D_1 = new hmpprt::CUDAGrid(hmpprt_module, "hmpp_acc_region_main_232_loop1D_1");
      hmpp_acc_region_main_244_loop1D_1 = new hmpprt::CUDAGrid(hmpprt_module, "hmpp_acc_region_main_244_loop1D_1");
      hmpp_acc_region_main_267_loop1D_1 = new hmpprt::CUDAGrid(hmpprt_module, "hmpp_acc_region_main_267_loop1D_1");
      hmpp_acc_region_main_279_loop1D_1 = new hmpprt::CUDAGrid(hmpprt_module, "hmpp_acc_region_main_279_loop1D_1");
      hmpp_acc_region_main_316_parallel_region_1 = new hmpprt::CUDAGrid(hmpprt_module, "hmpp_acc_region_main_316_parallel_region_1");
      reduce_s64_add_xy = new hmpprt::CUDAGrid(hmpprt_module, "reduce_s64_add_xy");
      hmpp_acc_region_main_339_parallel_region_1 = new hmpprt::CUDAGrid(hmpprt_module, "hmpp_acc_region_main_339_parallel_region_1");
      hmpp_acc_region_main_376_parallel_region_1 = new hmpprt::CUDAGrid(hmpprt_module, "hmpp_acc_region_main_376_parallel_region_1");
      hmpp_acc_region_main_389_parallel_region_1 = new hmpprt::CUDAGrid(hmpprt_module, "hmpp_acc_region_main_389_parallel_region_1");
      hmpp_acc_region_main_408_parallel_region_1 = new hmpprt::CUDAGrid(hmpprt_module, "hmpp_acc_region_main_408_parallel_region_1");
      hmpp_acc_region_main_462_parallel_region_1 = new hmpprt::CUDAGrid(hmpprt_module, "hmpp_acc_region_main_462_parallel_region_1");
      hmpp_acc_region_main_82_parallel_region_1 = new hmpprt::CUDAGrid(hmpprt_module, "hmpp_acc_region_main_82_parallel_region_1");

    }
    hmpprt::Context::getInstance()->getGrouplet()->setTarget(hmpprt::CUDA);
    hmpprt::Context::getInstance()->getGrouplet()->addSignature("hmpp_acc_region_main_105", "prototype hmpp_acc_region_main_105(width: s32, quarterWidth: s32, quarterHeight: s32, yPlaneCSER: ^cudaglob u8, yPlaneDown: ^cudaglob u8)");
    hmpprt::Context::getInstance()->getGrouplet()->addSignature("hmpp_acc_region_main_117", "prototype hmpp_acc_region_main_117(width: s32, quarterHeight: s32, yPlaneCSER: ^cudaglob u8)");
    hmpprt::Context::getInstance()->getGrouplet()->addSignature("hmpp_acc_region_main_138", "prototype hmpp_acc_region_main_138(width: ^host s32, height: s32, yPlaneCSER: ^cudaglob u8)");
    hmpprt::Context::getInstance()->getGrouplet()->addSignature("hmpp_acc_region_main_151", "prototype hmpp_acc_region_main_151(width: ^host s32, quarterWidth: s32, yPlaneCSER: ^cudaglob u8, yPlaneDown: ^cudaglob u8)");
    hmpprt::Context::getInstance()->getGrouplet()->addSignature("hmpp_acc_region_main_159", "prototype hmpp_acc_region_main_159(width: ^host s32, height: ^host s32, quarterWidth: s32, quarterHeight: s32, yPlaneCSER: ^cudaglob u8, yPlaneDown: ^cudaglob u8)");
    hmpprt::Context::getInstance()->getGrouplet()->addSignature("hmpp_acc_region_main_172", "prototype hmpp_acc_region_main_172(width: ^host s32, quarterWidth: s32, yPlaneCSER: ^cudaglob u8)");
    hmpprt::Context::getInstance()->getGrouplet()->addSignature("hmpp_acc_region_main_189", "prototype hmpp_acc_region_main_189(width: ^host s32, height: ^host s32, quarterWidth: s32, yPlaneCSER: ^cudaglob u8)");
    hmpprt::Context::getInstance()->getGrouplet()->addSignature("hmpp_acc_region_main_211", "prototype hmpp_acc_region_main_211(width: ^host s32, yPlaneCSER: ^cudaglob u8)");
    hmpprt::Context::getInstance()->getGrouplet()->addSignature("hmpp_acc_region_main_220", "prototype hmpp_acc_region_main_220(width: ^host s32, height: ^host s32, yPlaneCSER: ^cudaglob u8)");
    hmpprt::Context::getInstance()->getGrouplet()->addSignature("hmpp_acc_region_main_232", "prototype hmpp_acc_region_main_232(width: ^host s32, quarterWidth: s32, quarterHeight: s32, yPlaneCSER: ^cudaglob u8, yPlaneDown: ^cudaglob u8)");
    hmpprt::Context::getInstance()->getGrouplet()->addSignature("hmpp_acc_region_main_244", "prototype hmpp_acc_region_main_244(width: ^host s32, quarterHeight: s32, yPlaneCSER: ^cudaglob u8)");
    hmpprt::Context::getInstance()->getGrouplet()->addSignature("hmpp_acc_region_main_267", "prototype hmpp_acc_region_main_267(width: ^host s32, height: s32, yPlaneCSER: ^cudaglob u8)");
    hmpprt::Context::getInstance()->getGrouplet()->addSignature("hmpp_acc_region_main_279", "prototype hmpp_acc_region_main_279(width: ^host s32, quarterWidth: s32, quarterHeight: s32, yPlaneCSER: ^cudaglob u8, yPlaneDown: ^cudaglob u8)");
    hmpprt::Context::getInstance()->getGrouplet()->addSignature("hmpp_acc_region_main_316", "prototype hmpp_acc_region_main_316(width: s32, height: s32, yPlane: ^cudaglob u8, pEdge: ^cudaglob s16)");
    hmpprt::Context::getInstance()->getGrouplet()->addSignature("hmpp_acc_region_main_339", "prototype hmpp_acc_region_main_339(width: s32, height: s32, pEdge: ^cudaglob s16, sum: ^cudaglob s64)");
    hmpprt::Context::getInstance()->getGrouplet()->addSignature("hmpp_acc_region_main_376", "prototype hmpp_acc_region_main_376(width: s32, height: s32, yPlane: ^cudaglob u8, yPlaneCSER: ^cudaglob u8, pError: ^cudaglob s16)");
    hmpprt::Context::getInstance()->getGrouplet()->addSignature("hmpp_acc_region_main_389", "prototype hmpp_acc_region_main_389(width: s32, height: s32, yPlaneCSER: ^cudaglob u8, pError: ^cudaglob s16, pEdge: ^cudaglob s16, pSharpness: ^cudaglob s16, cutoff: double, peak: double, cutoff2: double, order: double, order2: double, low: double)");
    hmpprt::Context::getInstance()->getGrouplet()->addSignature("hmpp_acc_region_main_408", "prototype hmpp_acc_region_main_408(width: s32, height: s32, yPlane: ^cudaglob u8, pEdge: ^cudaglob s16, pSharpness: ^cudaglob s16, cutoff3: float, order3: float, peak3: float, low3: float)");
    hmpprt::Context::getInstance()->getGrouplet()->addSignature("hmpp_acc_region_main_462", "prototype hmpp_acc_region_main_462(width: s32, height: s32, yPlane: ^cudaglob u8, pSharpness: ^cudaglob s16)");
    hmpprt::Context::getInstance()->getGrouplet()->addSignature("hmpp_acc_region_main_82", "prototype hmpp_acc_region_main_82(width: s32, height: s32, quarterWidth: s32, yPlane: ^cudaglob u8, yPlaneDown: ^cudaglob u8)");

  }
  catch (hmpperr::Error & e)
  {
    return e.clone();
  }
  catch(...)
  {
    fprintf(stderr,"Unexpected error in hmpprt_init()\n");
    abort();
  }
  return 0;
}
#endif // __HIPCC__

#ifndef __HIPCC__
extern "C" CDLT_API void * hmpprt_fini()
{
  try
  {
    if (--hmpprt_uses == 0)
    {
      delete hmpp_acc_region_main_105_parallel_region_1;
      delete hmpp_acc_region_main_117_parallel_region_1;
      delete hmpp_acc_region_main_138_loop1D_1;
      delete hmpp_acc_region_main_151_loop1D_1;
      delete hmpp_acc_region_main_159_loop1D_1;
      delete hmpp_acc_region_main_172_loop1D_1;
      delete hmpp_acc_region_main_189_loop1D_1;
      delete hmpp_acc_region_main_211_loop1D_1;
      delete hmpp_acc_region_main_220_loop1D_1;
      delete hmpp_acc_region_main_232_loop1D_1;
      delete hmpp_acc_region_main_244_loop1D_1;
      delete hmpp_acc_region_main_267_loop1D_1;
      delete hmpp_acc_region_main_279_loop1D_1;
      delete hmpp_acc_region_main_316_parallel_region_1;
      delete reduce_s64_add_xy;
      delete hmpp_acc_region_main_339_parallel_region_1;
      delete hmpp_acc_region_main_376_parallel_region_1;
      delete hmpp_acc_region_main_389_parallel_region_1;
      delete hmpp_acc_region_main_408_parallel_region_1;
      delete hmpp_acc_region_main_462_parallel_region_1;
      delete hmpp_acc_region_main_82_parallel_region_1;

      delete hmpprt_module;
      hmpprt_module = 0;
    }
  }
  catch (hmpperr::Error & e)
  {
    return e.clone();
  }
  catch(...)
  {
    fprintf(stderr,"Unexpected error in hmpprt_fini()\n");
    abort();
  }
  return 0;
}
#endif // __HIPCC__

// footer
