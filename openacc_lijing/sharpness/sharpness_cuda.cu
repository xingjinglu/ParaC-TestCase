#include "hip/hip_runtime.h"

#include <stdio.h>

#ifndef __HIPCC__
#include <stdlib.h>
#include <math.h>

#include <hmpprt/Grouplet.h>
#include <hmpprt/HostTypes.h>
#include <hmpprt/Context.h>
#include <hmpprt/CUDAGrid.h>
#include <hmpprt/CUDAModule.h>
#include <hmpprt/DeviceManager.h>
#include <hmpperr/hmpperr.h>

#include <openacci/openacci_c.h>

#ifdef _WIN32
#  define CDLT_API __declspec(dllexport)
#else /* ! _WIN32 */
#  define CDLT_API
#endif /* _WIN32 */



#else // ! __HIPCC__

#include <hmpprt/HostTypes.h>
#include <hmpprt/CUDAIntrinsics.h>

extern __shared__ int64_t hmpp_sharedmem[];
#endif // __HIPCC__



#ifndef __HIPCC__

#else


#endif

#define HMPPCG_SIMD_LENGTH 1

# 23 "sharpness.cpp"

#ifdef __HIPCC__
__device__ hmpprt::s64 reduce_s64_add_y(hmpprt::s64 priv, hmpprt::s64* tmpa_1, hmpprt::s64 orig)
;
#endif // __HIPCC__



# 23 "sharpness.cpp"

#ifndef __HIPCC__
extern "C" CDLT_API  void hmpp_acc_region_main_84(hmpprt::s32 width_32, hmpprt::s32 height_16, hmpprt::s32 quarterWidth_9, hmpprt::u08* yPlane_6, hmpprt::u08* yPlaneDown_7)
;
#endif // __HIPCC__



# 23 "sharpness.cpp"

#ifndef __HIPCC__
void hmpp_acc_region_main_84_internal_1(hmpprt::s32 width_60, hmpprt::s32 height_30, hmpprt::s32 quarterWidth_23, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  yPlane_11, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  yPlaneDown_15)
;
#endif // __HIPCC__



# 486 "sharpness.cpp"

#ifndef __HIPCC__
static hmpprt::CUDAGrid * hmpp_acc_region_main_84_parallel_region_1 = 0;
#else

extern "C" __global__ void hmpp_acc_region_main_84_parallel_region_1(hmpprt::s32 height_1, hmpprt::s32 quarterWidth_1, hmpprt::s32 width_1, hmpprt::u08* yPlaneDown_1, hmpprt::u08* yPlane_1);
#endif // __HIPCC__




# 486 "sharpness.cpp"

#ifndef __HIPCC__
extern "C" CDLT_API  void hmpp_acc_region_main_490(hmpprt::s32 width_34, hmpprt::s32 height_20, hmpprt::u08* yPlane_7, hmpprt::s16* pSharpness_4)
;
#endif // __HIPCC__



# 486 "sharpness.cpp"

#ifndef __HIPCC__
void hmpp_acc_region_main_490_internal_1(hmpprt::s32 width_61, hmpprt::s32 height_31, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  yPlane_12, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s16>  pSharpness_7)
;
#endif // __HIPCC__



# 433 "sharpness.cpp"

#ifndef __HIPCC__
static hmpprt::CUDAGrid * hmpp_acc_region_main_490_parallel_region_1 = 0;
#else

extern "C" __global__ void hmpp_acc_region_main_490_parallel_region_1(hmpprt::s32 height_2, hmpprt::s16* pSharpness_1, hmpprt::s32 width_2, hmpprt::u08* yPlane_2);
#endif // __HIPCC__




# 433 "sharpness.cpp"

#ifndef __HIPCC__
extern "C" CDLT_API  void hmpp_acc_region_main_436(hmpprt::s32 width_43, hmpprt::s32 height_21, hmpprt::u08* yPlane_8, hmpprt::s16* pEdge_5, hmpprt::s16* pSharpness_5, float cutoff3_1, float order3_1, float peak3_1, float low3_1)
;
#endif // __HIPCC__



# 433 "sharpness.cpp"

#ifndef __HIPCC__
void hmpp_acc_region_main_436_internal_1(hmpprt::s32 width_62, hmpprt::s32 height_32, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  yPlane_13, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s16>  pEdge_9, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s16>  pSharpness_8, float cutoff3_2, float order3_2, float peak3_2, float low3_2)
;
#endif // __HIPCC__



# 391 "sharpness.cpp"

#ifndef __HIPCC__
static hmpprt::CUDAGrid * hmpp_acc_region_main_436_parallel_region_1 = 0;
#else

extern "C" __global__ void hmpp_acc_region_main_436_parallel_region_1(float cutoff3, hmpprt::s32 height_3, float low3, float order3, hmpprt::s16* pEdge_1, hmpprt::s16* pSharpness_2, float peak3, hmpprt::s32 width_3, hmpprt::u08* yPlane_3);
#endif // __HIPCC__




# 391 "sharpness.cpp"

#ifndef __HIPCC__
extern "C" CDLT_API  void hmpp_acc_region_main_411(hmpprt::s32 width_45, hmpprt::s32 height_22, hmpprt::u08* yPlaneCSER_16, hmpprt::s16* pError_3, hmpprt::s16* pEdge_10, hmpprt::s16* pSharpness_3, double cutoff_1, double peak_1, double cutoff2_1, double order_1, double order2_1, double low_1)
;
#endif // __HIPCC__



# 391 "sharpness.cpp"

#ifndef __HIPCC__
void hmpp_acc_region_main_411_internal_1(hmpprt::s32 width_63, hmpprt::s32 height_33, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  yPlaneCSER_34, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s16>  pError_5, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s16>  pEdge_6, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s16>  pSharpness_6, double cutoff_2, double peak_2, double cutoff2_2, double order_2, double order2_2, double low_2)
;
#endif // __HIPCC__



# 372 "sharpness.cpp"

#ifndef __HIPCC__
static hmpprt::CUDAGrid * hmpp_acc_region_main_411_parallel_region_1 = 0;
#else

extern "C" __global__ void hmpp_acc_region_main_411_parallel_region_1(double cutoff, double cutoff2, hmpprt::s32 height_4, double low, double order, double order2, hmpprt::s16* pEdge_2, hmpprt::s16* pError_1, hmpprt::s16* pSharpness, double peak, hmpprt::s32 width_4, hmpprt::u08* yPlaneCSER_1);
#endif // __HIPCC__




# 372 "sharpness.cpp"

#ifndef __HIPCC__
extern "C" CDLT_API  void hmpp_acc_region_main_375(hmpprt::s32 width_46, hmpprt::s32 height_23, hmpprt::s16* pEdge_7, hmpprt::s64* sum)
;
#endif // __HIPCC__



# 372 "sharpness.cpp"

#ifndef __HIPCC__
void hmpp_acc_region_main_375_internal_1(hmpprt::s32 width_64, hmpprt::s32 height_34, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s16>  pEdge_11, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s64>  sum_4)
;
#endif // __HIPCC__



# 342 "sharpness.cpp"

#ifndef __HIPCC__
static hmpprt::CUDAGrid * hmpp_acc_region_main_375_parallel_region_1 = 0;
#else

extern "C" __global__ void hmpp_acc_region_main_375_parallel_region_1(hmpprt::s64* gang_prv_red_sum_2, hmpprt::s32 height_5, hmpprt::s16* pEdge_3, hmpprt::s32 width_5);
#endif // __HIPCC__




# 342 "sharpness.cpp"

#ifndef __HIPCC__
static hmpprt::CUDAGrid * reduce_s64_add_xy = 0;
#else

extern "C" __global__ void reduce_s64_add_xy(hmpprt::s64 neutral, hmpprt::s64* scalar, hmpprt::s32 size, hmpprt::s64* array);
#endif // __HIPCC__




# 342 "sharpness.cpp"

#ifndef __HIPCC__
extern "C" CDLT_API  void hmpp_acc_region_main_346(hmpprt::s32 width_47, hmpprt::s32 height_24, hmpprt::u08* yPlane_9, hmpprt::s16* pEdge_4)
;
#endif // __HIPCC__



# 342 "sharpness.cpp"

#ifndef __HIPCC__
void hmpp_acc_region_main_346_internal_1(hmpprt::s32 width_65, hmpprt::s32 height_35, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  yPlane_14, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s16>  pEdge_8)
;
#endif // __HIPCC__



# 323 "sharpness.cpp"

#ifndef __HIPCC__
static hmpprt::CUDAGrid * hmpp_acc_region_main_346_parallel_region_1 = 0;
#else

extern "C" __global__ void hmpp_acc_region_main_346_parallel_region_1(hmpprt::s32 height_6, hmpprt::s16* pEdge, hmpprt::s32 width_6, hmpprt::u08* yPlane_4);
#endif // __HIPCC__




# 323 "sharpness.cpp"

#ifndef __HIPCC__
extern "C" CDLT_API  void hmpp_acc_region_main_326(hmpprt::s32 width_48, hmpprt::s32 height_25, hmpprt::u08* yPlane_5, hmpprt::u08* yPlaneCSER_22, hmpprt::s16* pError_2)
;
#endif // __HIPCC__



# 323 "sharpness.cpp"

#ifndef __HIPCC__
void hmpp_acc_region_main_326_internal_1(hmpprt::s32 width_66, hmpprt::s32 height_36, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  yPlane_10, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  yPlaneCSER_37, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s16>  pError_4)
;
#endif // __HIPCC__



# 289 "sharpness.cpp"

#ifndef __HIPCC__
static hmpprt::CUDAGrid * hmpp_acc_region_main_326_parallel_region_1 = 0;
#else

extern "C" __global__ void hmpp_acc_region_main_326_parallel_region_1(hmpprt::s32 height_7, hmpprt::s16* pError, hmpprt::s32 width_7, hmpprt::u08* yPlane, hmpprt::u08* yPlaneCSER_2);
#endif // __HIPCC__




# 289 "sharpness.cpp"

#ifndef __HIPCC__
extern "C" CDLT_API  void hmpp_acc_region_main_289(hmpprt::s32* width_8, hmpprt::s32 quarterWidth_10, hmpprt::s32 quarterHeight_7, hmpprt::u08* yPlaneCSER_23, hmpprt::u08* yPlaneDown_8)
;
#endif // __HIPCC__



# 289 "sharpness.cpp"

#ifndef __HIPCC__
void hmpp_acc_region_main_289_internal_1(hmpprt::s32* width_67, hmpprt::s32 quarterWidth_16, hmpprt::s32 quarterHeight_16, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  yPlaneCSER_38, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  yPlaneDown_16)
;
#endif // __HIPCC__



# 277 "sharpness.cpp"

#ifndef __HIPCC__
static hmpprt::CUDAGrid * hmpp_acc_region_main_289_loop1D_1 = 0;
#else

extern "C" __global__ void hmpp_acc_region_main_289_loop1D_1(hmpprt::s32 width_30, hmpprt::s32 quarterWidth_2, hmpprt::s32 quarterHeight_1, hmpprt::u08* yPlaneCSER_3, hmpprt::u08* yPlaneDown_2);
#endif // __HIPCC__




# 277 "sharpness.cpp"

#ifndef __HIPCC__
extern "C" CDLT_API  void hmpp_acc_region_main_277(hmpprt::s32* width_68, hmpprt::s32 height_8, hmpprt::u08* yPlaneCSER_4)
;
#endif // __HIPCC__



# 277 "sharpness.cpp"

#ifndef __HIPCC__
void hmpp_acc_region_main_277_internal_1(hmpprt::s32* width_9, hmpprt::s32 height_37, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  yPlaneCSER_42)
;
#endif // __HIPCC__



# 254 "sharpness.cpp"

#ifndef __HIPCC__
static hmpprt::CUDAGrid * hmpp_acc_region_main_277_loop1D_1 = 0;
#else

extern "C" __global__ void hmpp_acc_region_main_277_loop1D_1(hmpprt::s32 height_26, hmpprt::u08* yPlaneCSER_24, hmpprt::s32 width_29);
#endif // __HIPCC__




# 254 "sharpness.cpp"

#ifndef __HIPCC__
extern "C" CDLT_API  void hmpp_acc_region_main_254(hmpprt::s32* width_10, hmpprt::s32 quarterHeight_2, hmpprt::u08* yPlaneCSER_5)
;
#endif // __HIPCC__



# 254 "sharpness.cpp"

#ifndef __HIPCC__
void hmpp_acc_region_main_254_internal_1(hmpprt::s32* width_49, hmpprt::s32 quarterHeight_13, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  yPlaneCSER_43)
;
#endif // __HIPCC__



# 242 "sharpness.cpp"

#ifndef __HIPCC__
static hmpprt::CUDAGrid * hmpp_acc_region_main_254_loop1D_1 = 0;
#else

extern "C" __global__ void hmpp_acc_region_main_254_loop1D_1(hmpprt::s32 quarterHeight_8, hmpprt::u08* yPlaneCSER_25, hmpprt::s32 width_28);
#endif // __HIPCC__




# 242 "sharpness.cpp"

#ifndef __HIPCC__
extern "C" CDLT_API  void hmpp_acc_region_main_242(hmpprt::s32* width_11, hmpprt::s32 quarterWidth_3, hmpprt::s32 quarterHeight_9, hmpprt::u08* yPlaneCSER_26, hmpprt::u08* yPlaneDown_9)
;
#endif // __HIPCC__



# 242 "sharpness.cpp"

#ifndef __HIPCC__
void hmpp_acc_region_main_242_internal_1(hmpprt::s32* width_50, hmpprt::s32 quarterWidth_20, hmpprt::s32 quarterHeight_17, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  yPlaneCSER_39, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  yPlaneDown_17)
;
#endif // __HIPCC__



# 230 "sharpness.cpp"

#ifndef __HIPCC__
static hmpprt::CUDAGrid * hmpp_acc_region_main_242_loop1D_1 = 0;
#else

extern "C" __global__ void hmpp_acc_region_main_242_loop1D_1(hmpprt::s32 quarterWidth_11, hmpprt::s32 quarterHeight_3, hmpprt::u08* yPlaneCSER_6, hmpprt::u08* yPlaneDown_3, hmpprt::s32 width_27);
#endif // __HIPCC__




# 230 "sharpness.cpp"

#ifndef __HIPCC__
extern "C" CDLT_API  void hmpp_acc_region_main_230(hmpprt::s32* width_12, hmpprt::s32* height_9, hmpprt::u08* yPlaneCSER_40)
;
#endif // __HIPCC__



# 230 "sharpness.cpp"

#ifndef __HIPCC__
void hmpp_acc_region_main_230_internal_1(hmpprt::s32* width_51, hmpprt::s32* height_38, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  yPlaneCSER_27)
;
#endif // __HIPCC__



# 209 "sharpness.cpp"

#ifndef __HIPCC__
static hmpprt::CUDAGrid * hmpp_acc_region_main_230_loop1D_1 = 0;
#else

extern "C" __global__ void hmpp_acc_region_main_230_loop1D_1(hmpprt::u08* yPlaneCSER_7, hmpprt::s32 width_26, hmpprt::s32 height_14);
#endif // __HIPCC__




# 209 "sharpness.cpp"

#ifndef __HIPCC__
extern "C" CDLT_API  void hmpp_acc_region_main_209(hmpprt::s32* width_13, hmpprt::s32* height_10, hmpprt::s32 quarterWidth_12, hmpprt::u08* yPlaneCSER_28)
;
#endif // __HIPCC__



# 209 "sharpness.cpp"

#ifndef __HIPCC__
void hmpp_acc_region_main_209_internal_1(hmpprt::s32* width_52, hmpprt::s32* height_27, hmpprt::s32 quarterWidth_17, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  yPlaneCSER_41)
;
#endif // __HIPCC__



# 197 "sharpness.cpp"

#ifndef __HIPCC__
static hmpprt::CUDAGrid * hmpp_acc_region_main_209_loop1D_1 = 0;
#else

extern "C" __global__ void hmpp_acc_region_main_209_loop1D_1(hmpprt::s32 quarterWidth_4, hmpprt::u08* yPlaneCSER_8, hmpprt::s32 width_25, hmpprt::s32 height_13);
#endif // __HIPCC__




# 197 "sharpness.cpp"

#ifndef __HIPCC__
extern "C" CDLT_API  void hmpp_acc_region_main_197(hmpprt::s32* width_53, hmpprt::s32* height_28, hmpprt::s32 quarterWidth_18, hmpprt::s32 quarterHeight_14, hmpprt::u08* yPlaneCSER_9, hmpprt::u08* yPlaneDown_12)
;
#endif // __HIPCC__



# 197 "sharpness.cpp"

#ifndef __HIPCC__
void hmpp_acc_region_main_197_internal_1(hmpprt::s32* width_14, hmpprt::s32* height_11, hmpprt::s32 quarterWidth_13, hmpprt::s32 quarterHeight_4, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  yPlaneCSER_44, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  yPlaneDown_4)
;
#endif // __HIPCC__



# 185 "sharpness.cpp"

#ifndef __HIPCC__
static hmpprt::CUDAGrid * hmpp_acc_region_main_197_loop1D_1 = 0;
#else

extern "C" __global__ void hmpp_acc_region_main_197_loop1D_1(hmpprt::s32 quarterWidth_5, hmpprt::s32 quarterHeight_10, hmpprt::u08* yPlaneCSER_29, hmpprt::u08* yPlaneDown_10, hmpprt::s32 width_24, hmpprt::s32 height_17);
#endif // __HIPCC__




# 185 "sharpness.cpp"

#ifndef __HIPCC__
extern "C" CDLT_API  void hmpp_acc_region_main_185(hmpprt::s32* width_54, hmpprt::u08* yPlaneCSER_35)
;
#endif // __HIPCC__



# 185 "sharpness.cpp"

#ifndef __HIPCC__
void hmpp_acc_region_main_185_internal_1(hmpprt::s32* width_15, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  yPlaneCSER_17)
;
#endif // __HIPCC__



# 164 "sharpness.cpp"

#ifndef __HIPCC__
static hmpprt::CUDAGrid * hmpp_acc_region_main_185_loop1D_1 = 0;
#else

extern "C" __global__ void hmpp_acc_region_main_185_loop1D_1(hmpprt::s32 width_36, hmpprt::u08* yPlaneCSER_10);
#endif // __HIPCC__




# 164 "sharpness.cpp"

#ifndef __HIPCC__
extern "C" CDLT_API  void hmpp_acc_region_main_164(hmpprt::s32* width_55, hmpprt::s32 quarterWidth_19, hmpprt::u08* yPlaneCSER_36)
;
#endif // __HIPCC__



# 164 "sharpness.cpp"

#ifndef __HIPCC__
void hmpp_acc_region_main_164_internal_1(hmpprt::s32* width_16, hmpprt::s32 quarterWidth_14, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  yPlaneCSER_18)
;
#endif // __HIPCC__



# 152 "sharpness.cpp"

#ifndef __HIPCC__
static hmpprt::CUDAGrid * hmpp_acc_region_main_164_loop1D_1 = 0;
#else

extern "C" __global__ void hmpp_acc_region_main_164_loop1D_1(hmpprt::s32 quarterWidth_6, hmpprt::u08* yPlaneCSER_11);
#endif // __HIPCC__




# 152 "sharpness.cpp"

#ifndef __HIPCC__
extern "C" CDLT_API  void hmpp_acc_region_main_152(hmpprt::s32* width_17, hmpprt::s32 quarterWidth_7, hmpprt::u08* yPlaneCSER_30, hmpprt::u08* yPlaneDown_5)
;
#endif // __HIPCC__



# 152 "sharpness.cpp"

#ifndef __HIPCC__
void hmpp_acc_region_main_152_internal_1(hmpprt::s32* width_56, hmpprt::s32 quarterWidth_21, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  yPlaneCSER_12, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  yPlaneDown_13)
;
#endif // __HIPCC__



# 140 "sharpness.cpp"

#ifndef __HIPCC__
static hmpprt::CUDAGrid * hmpp_acc_region_main_152_loop1D_1 = 0;
#else

extern "C" __global__ void hmpp_acc_region_main_152_loop1D_1(hmpprt::s32 quarterWidth_15, hmpprt::u08* yPlaneCSER_19, hmpprt::u08* yPlaneDown_11);
#endif // __HIPCC__




# 140 "sharpness.cpp"

#ifndef __HIPCC__
extern "C" CDLT_API  void hmpp_acc_region_main_140(hmpprt::s32* width_57, hmpprt::s32 height_29, hmpprt::u08* yPlaneCSER_31)
;
#endif // __HIPCC__



# 140 "sharpness.cpp"

#ifndef __HIPCC__
void hmpp_acc_region_main_140_internal_1(hmpprt::s32* width_18, hmpprt::s32 height_15, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  yPlaneCSER_13)
;
#endif // __HIPCC__



# 119 "sharpness.cpp"

#ifndef __HIPCC__
static hmpprt::CUDAGrid * hmpp_acc_region_main_140_loop1D_1 = 0;
#else

extern "C" __global__ void hmpp_acc_region_main_140_loop1D_1(hmpprt::s32 width_35, hmpprt::s32 height, hmpprt::u08* yPlaneCSER_20);
#endif // __HIPCC__




# 119 "sharpness.cpp"

#ifndef __HIPCC__
extern "C" CDLT_API  void hmpp_acc_region_main_119(hmpprt::s32 width_19, hmpprt::s32 quarterHeight_15, hmpprt::u08* yPlaneCSER_14)
;
#endif // __HIPCC__



# 119 "sharpness.cpp"

#ifndef __HIPCC__
void hmpp_acc_region_main_119_internal_1(hmpprt::s32 width_58, hmpprt::s32 quarterHeight_5, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  yPlaneCSER_32)
;
#endif // __HIPCC__



# 103 "sharpness.cpp"

#ifndef __HIPCC__
static hmpprt::CUDAGrid * hmpp_acc_region_main_119_parallel_region_1 = 0;
#else

extern "C" __global__ void hmpp_acc_region_main_119_parallel_region_1(hmpprt::s32 quarterHeight_11, hmpprt::s32 width_33, hmpprt::u08* yPlaneCSER_21);
#endif // __HIPCC__




# 103 "sharpness.cpp"

#ifndef __HIPCC__
extern "C" CDLT_API  void hmpp_acc_region_main_107(hmpprt::s32 width_31, hmpprt::s32 quarterWidth_22, hmpprt::s32 quarterHeight, hmpprt::u08* yPlaneCSER_33, hmpprt::u08* yPlaneDown_6)
;
#endif // __HIPCC__



# 103 "sharpness.cpp"

#ifndef __HIPCC__
void hmpp_acc_region_main_107_internal_1(hmpprt::s32 width_59, hmpprt::s32 quarterWidth_8, hmpprt::s32 quarterHeight_12, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  yPlaneCSER_15, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  yPlaneDown_14)
;
#endif // __HIPCC__



# 103 "sharpness.cpp"

#ifndef __HIPCC__
static hmpprt::CUDAGrid * hmpp_acc_region_main_107_parallel_region_1 = 0;
#else

extern "C" __global__ void hmpp_acc_region_main_107_parallel_region_1(hmpprt::s32 quarterHeight_6, hmpprt::s32 quarterWidth, hmpprt::s32 width, hmpprt::u08* yPlaneCSER, hmpprt::u08* yPlaneDown);
#endif // __HIPCC__




# 103 "sharpness.cpp"

#ifdef __HIPCC__

extern "C" __global__ void hmpp_acc_region_main_107_parallel_region_1(hmpprt::s32 quarterHeight_6, hmpprt::s32 quarterWidth, hmpprt::s32 width, hmpprt::u08* yPlaneCSER, hmpprt::u08* yPlaneDown)
{
 # 105 "sharpness.cpp"
 {
  # 110 "sharpness.cpp"
  hmpprt::s32 iter_per_gang_9;
  # 110 "sharpness.cpp"
  hmpprt::s32 first_gang_iter_9;
  # 110 "sharpness.cpp"
  hmpprt::s32 last_gang_iter_9;
  # 110 "sharpness.cpp"
  iter_per_gang_9 = ((1 + (quarterHeight_6 - 1) / 192) > 256 ? (1 + (quarterHeight_6 - 1) / 192) : 256);
  # 110 "sharpness.cpp"
  first_gang_iter_9 = (hmpprt::gr_gbidx()) * iter_per_gang_9;
  # 110 "sharpness.cpp"
  last_gang_iter_9 = ((first_gang_iter_9 + iter_per_gang_9 - 1) < (quarterHeight_6 - 1) ? (first_gang_iter_9 + iter_per_gang_9 - 1) : (quarterHeight_6 - 1));
  # 110 "sharpness.cpp"
  hmpprt::s32 h_11;
  # 110 "sharpness.cpp"
  # 111 "sharpness.cpp"
  for (h_11 = first_gang_iter_9 + (hmpprt::gr_btidy()) ; h_11 <= last_gang_iter_9 ; h_11 = h_11 + (hmpprt::gr_btnumy()))
  {
   # 112 "sharpness.cpp"
   *(yPlaneCSER + h_11 * 4 * width) = *(yPlaneDown + h_11 * quarterWidth);
  }
  # 103 "sharpness.cpp"
 }
}
#endif // __HIPCC__



# 103 "sharpness.cpp"

#ifndef __HIPCC__
void hmpp_acc_region_main_107_internal_1(hmpprt::s32 width_59, hmpprt::s32 quarterWidth_8, hmpprt::s32 quarterHeight_12, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  yPlaneCSER_15, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  yPlaneDown_14)
{
 # 103 "sharpness.cpp"
 if (1)
 {
  hmpprt::CUDAGridCall __hmppcg_call;
  __hmppcg_call.setSizeX(192);
  __hmppcg_call.setSizeY(1);
  __hmppcg_call.setBlockSizeX(1);
  __hmppcg_call.setBlockSizeY(256);
  __hmppcg_call.addLocalParameter((hmpprt::s32) (quarterHeight_12), "quarterHeight_6");
  __hmppcg_call.addLocalParameter((hmpprt::s32) (quarterWidth_8), "quarterWidth");
  __hmppcg_call.addLocalParameter((hmpprt::s32) (width_59), "width");
  __hmppcg_call.addLocalParameter(&yPlaneCSER_15, 8, "yPlaneCSER");
  __hmppcg_call.addLocalParameter(&yPlaneDown_14, 8, "yPlaneDown");
  __hmppcg_call.launch(hmpp_acc_region_main_107_parallel_region_1, hmpprt::Context::getInstance()->getCUDADevice());
 }
 ;
}
#endif // __HIPCC__



# 103 "sharpness.cpp"

#ifndef __HIPCC__
extern "C" CDLT_API  void hmpp_acc_region_main_107(hmpprt::s32 width_31, hmpprt::s32 quarterWidth_22, hmpprt::s32 quarterHeight, hmpprt::u08* yPlaneCSER_33, hmpprt::u08* yPlaneDown_6)
{
 # 119 "sharpness.cpp"
 (hmpp_acc_region_main_107_internal_1(width_31, quarterWidth_22, quarterHeight, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08> (yPlaneCSER_33), hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08> (yPlaneDown_6)));
}
#endif // __HIPCC__



# 119 "sharpness.cpp"

#ifdef __HIPCC__

extern "C" __global__ void hmpp_acc_region_main_119_parallel_region_1(hmpprt::s32 quarterHeight_11, hmpprt::s32 width_33, hmpprt::u08* yPlaneCSER_21)
{
 # 121 "sharpness.cpp"
 {
  # 122 "sharpness.cpp"
  hmpprt::s32 iter_per_gang_8;
  # 122 "sharpness.cpp"
  hmpprt::s32 first_gang_iter_8;
  # 122 "sharpness.cpp"
  hmpprt::s32 last_gang_iter_8;
  # 122 "sharpness.cpp"
  iter_per_gang_8 = ((1 + (quarterHeight_11 - 1) / 192) > 256 ? (1 + (quarterHeight_11 - 1) / 192) : 256);
  # 122 "sharpness.cpp"
  first_gang_iter_8 = (hmpprt::gr_gbidx()) * iter_per_gang_8;
  # 122 "sharpness.cpp"
  last_gang_iter_8 = ((first_gang_iter_8 + iter_per_gang_8 - 1) < (quarterHeight_11 - 1) ? (first_gang_iter_8 + iter_per_gang_8 - 1) : (quarterHeight_11 - 1));
  # 122 "sharpness.cpp"
  hmpprt::s32 h_12;
  # 122 "sharpness.cpp"
  # 123 "sharpness.cpp"
  for (h_12 = first_gang_iter_8 + (hmpprt::gr_btidy()) ; h_12 <= last_gang_iter_8 ; h_12 = h_12 + (hmpprt::gr_btnumy()))
  {
   # 124 "sharpness.cpp"
   if (h_12 < quarterHeight_11 - 1)
   {
    # 126 "sharpness.cpp"
    *(yPlaneCSER_21 + (h_12 * 4 + 1) * width_33) = (hmpprt::u08 ) (((hmpprt::s32 ) (*(yPlaneCSER_21 + h_12 * 4 * width_33)) * 3 + (hmpprt::s32 ) (*(yPlaneCSER_21 + (h_12 * 4 + 4) * width_33))) / 4);
    # 127 "sharpness.cpp"
    *(yPlaneCSER_21 + (h_12 * 4 + 2) * width_33) = (hmpprt::u08 ) (((hmpprt::s32 ) (*(yPlaneCSER_21 + h_12 * 4 * width_33)) * 2 + (hmpprt::s32 ) (*(yPlaneCSER_21 + (h_12 * 4 + 4) * width_33)) * 2) / 4);
    # 128 "sharpness.cpp"
    *(yPlaneCSER_21 + (h_12 * 4 + 3) * width_33) = (hmpprt::u08 ) (((hmpprt::s32 ) (*(yPlaneCSER_21 + h_12 * 4 * width_33)) + (hmpprt::s32 ) (*(yPlaneCSER_21 + (h_12 * 4 + 4) * width_33)) * 3) / 4);
   }
   else
   {
    # 132 "sharpness.cpp"
    *(yPlaneCSER_21 + (h_12 * 4 + 1) * width_33) = *(yPlaneCSER_21 + h_12 * 4 * width_33);
   }
  }
  # 119 "sharpness.cpp"
 }
}
#endif // __HIPCC__



# 119 "sharpness.cpp"

#ifndef __HIPCC__
void hmpp_acc_region_main_119_internal_1(hmpprt::s32 width_58, hmpprt::s32 quarterHeight_5, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  yPlaneCSER_32)
{
 # 119 "sharpness.cpp"
 if (1)
 {
  hmpprt::CUDAGridCall __hmppcg_call;
  __hmppcg_call.setSizeX(192);
  __hmppcg_call.setSizeY(1);
  __hmppcg_call.setBlockSizeX(1);
  __hmppcg_call.setBlockSizeY(256);
  __hmppcg_call.addLocalParameter((hmpprt::s32) (quarterHeight_5), "quarterHeight_11");
  __hmppcg_call.addLocalParameter((hmpprt::s32) (width_58), "width_33");
  __hmppcg_call.addLocalParameter(&yPlaneCSER_32, 8, "yPlaneCSER_21");
  __hmppcg_call.launch(hmpp_acc_region_main_119_parallel_region_1, hmpprt::Context::getInstance()->getCUDADevice());
 }
 ;
}
#endif // __HIPCC__



# 119 "sharpness.cpp"

#ifndef __HIPCC__
extern "C" CDLT_API  void hmpp_acc_region_main_119(hmpprt::s32 width_19, hmpprt::s32 quarterHeight_15, hmpprt::u08* yPlaneCSER_14)
{
 # 140 "sharpness.cpp"
 (hmpp_acc_region_main_119_internal_1(width_19, quarterHeight_15, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08> (yPlaneCSER_14)));
}
#endif // __HIPCC__



# 140 "sharpness.cpp"

#ifdef __HIPCC__

extern "C" __global__ void hmpp_acc_region_main_140_loop1D_1(hmpprt::s32 width_35, hmpprt::s32 height, hmpprt::u08* yPlaneCSER_20)
{
 # 143 "sharpness.cpp"
 hmpprt::s32 h_13;
 # 145 "sharpness.cpp"
 h_13 = (hmpprt::gr_atidf());
 # 145 "sharpness.cpp"
 if (h_13 > height - 1)
 {
  # 145 "sharpness.cpp"
  goto __hmppcg_label_1;
 }
 # 145 "sharpness.cpp"
 *(yPlaneCSER_20 + (h_13 * width_35 + 1)) = *(yPlaneCSER_20 + h_13 * width_35);
 # 140 "sharpness.cpp"
 __hmppcg_label_1:;
}
#endif // __HIPCC__



# 140 "sharpness.cpp"

#ifndef __HIPCC__
void hmpp_acc_region_main_140_internal_1(hmpprt::s32* width_18, hmpprt::s32 height_15, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  yPlaneCSER_13)
{
 # 142 "sharpness.cpp"
 hmpprt::s32 width_20;
 # 142 "sharpness.cpp"
 width_20 = *width_18;
 # 142 "sharpness.cpp"
 {
  # 140 "sharpness.cpp"
  if (height_15 - 1 >= 0)
  {
   hmpprt::CUDAGridCall __hmppcg_call;
   __hmppcg_call.setSizeX((height_15 - 1) / 128 + 1);
   __hmppcg_call.setSizeY(1);
   __hmppcg_call.setBlockSizeX(32);
   __hmppcg_call.setBlockSizeY(4);
   __hmppcg_call.addLocalParameter((hmpprt::s32) (width_20), "width_35");
   __hmppcg_call.addLocalParameter((hmpprt::s32) (height_15), "height");
   __hmppcg_call.addLocalParameter(&yPlaneCSER_13, 8, "yPlaneCSER_20");
   __hmppcg_call.launch(hmpp_acc_region_main_140_loop1D_1, hmpprt::Context::getInstance()->getCUDADevice());
  }
  ;
 }
 # 140 "sharpness.cpp"
 *width_18 = width_20;
}
#endif // __HIPCC__



# 140 "sharpness.cpp"

#ifndef __HIPCC__
extern "C" CDLT_API  void hmpp_acc_region_main_140(hmpprt::s32* width_57, hmpprt::s32 height_29, hmpprt::u08* yPlaneCSER_31)
{
 # 152 "sharpness.cpp"
 (hmpp_acc_region_main_140_internal_1(width_57, height_29, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08> (yPlaneCSER_31)));
}
#endif // __HIPCC__



# 152 "sharpness.cpp"

#ifdef __HIPCC__

extern "C" __global__ void hmpp_acc_region_main_152_loop1D_1(hmpprt::s32 quarterWidth_15, hmpprt::u08* yPlaneCSER_19, hmpprt::u08* yPlaneDown_11)
{
 # 155 "sharpness.cpp"
 hmpprt::s32 w_11;
 # 157 "sharpness.cpp"
 w_11 = (hmpprt::gr_atidf());
 # 157 "sharpness.cpp"
 if (w_11 > quarterWidth_15 - 1)
 {
  # 157 "sharpness.cpp"
  goto __hmppcg_label_2;
 }
 # 157 "sharpness.cpp"
 *(yPlaneCSER_19 + w_11 * 4) = *(yPlaneDown_11 + w_11);
 # 152 "sharpness.cpp"
 __hmppcg_label_2:;
}
#endif // __HIPCC__



# 152 "sharpness.cpp"

#ifndef __HIPCC__
void hmpp_acc_region_main_152_internal_1(hmpprt::s32* width_56, hmpprt::s32 quarterWidth_21, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  yPlaneCSER_12, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  yPlaneDown_13)
{
 # 154 "sharpness.cpp"
 hmpprt::s32 width_21;
 # 154 "sharpness.cpp"
 width_21 = *width_56;
 # 154 "sharpness.cpp"
 {
  # 152 "sharpness.cpp"
  if (quarterWidth_21 - 1 >= 0)
  {
   hmpprt::CUDAGridCall __hmppcg_call;
   __hmppcg_call.setSizeX((quarterWidth_21 - 1) / 128 + 1);
   __hmppcg_call.setSizeY(1);
   __hmppcg_call.setBlockSizeX(32);
   __hmppcg_call.setBlockSizeY(4);
   __hmppcg_call.addLocalParameter((hmpprt::s32) (quarterWidth_21), "quarterWidth_15");
   __hmppcg_call.addLocalParameter(&yPlaneCSER_12, 8, "yPlaneCSER_19");
   __hmppcg_call.addLocalParameter(&yPlaneDown_13, 8, "yPlaneDown_11");
   __hmppcg_call.launch(hmpp_acc_region_main_152_loop1D_1, hmpprt::Context::getInstance()->getCUDADevice());
  }
  ;
 }
 # 152 "sharpness.cpp"
 *width_56 = width_21;
}
#endif // __HIPCC__



# 152 "sharpness.cpp"

#ifndef __HIPCC__
extern "C" CDLT_API  void hmpp_acc_region_main_152(hmpprt::s32* width_17, hmpprt::s32 quarterWidth_7, hmpprt::u08* yPlaneCSER_30, hmpprt::u08* yPlaneDown_5)
{
 # 164 "sharpness.cpp"
 (hmpp_acc_region_main_152_internal_1(width_17, quarterWidth_7, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08> (yPlaneCSER_30), hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08> (yPlaneDown_5)));
}
#endif // __HIPCC__



# 164 "sharpness.cpp"

#ifdef __HIPCC__

extern "C" __global__ void hmpp_acc_region_main_164_loop1D_1(hmpprt::s32 quarterWidth_6, hmpprt::u08* yPlaneCSER_11)
{
 # 167 "sharpness.cpp"
 hmpprt::s32 w_12;
 # 169 "sharpness.cpp"
 w_12 = (hmpprt::gr_atidf());
 # 169 "sharpness.cpp"
 if (w_12 > quarterWidth_6 - 1)
 {
  # 169 "sharpness.cpp"
  goto __hmppcg_label_3;
 }
 # 169 "sharpness.cpp"
 if (w_12 < quarterWidth_6 - 1)
 {
  # 171 "sharpness.cpp"
  *(yPlaneCSER_11 + (w_12 * 4 + 1)) = (hmpprt::u08 ) (((hmpprt::s32 ) (*(yPlaneCSER_11 + w_12 * 4)) * 3 + (hmpprt::s32 ) (*(yPlaneCSER_11 + (w_12 * 4 + 4)))) / 4);
  # 172 "sharpness.cpp"
  *(yPlaneCSER_11 + (w_12 * 4 + 2)) = (hmpprt::u08 ) (((hmpprt::s32 ) (*(yPlaneCSER_11 + w_12 * 4)) * 2 + (hmpprt::s32 ) (*(yPlaneCSER_11 + (w_12 * 4 + 4))) * 2) / 4);
  # 173 "sharpness.cpp"
  *(yPlaneCSER_11 + (w_12 * 4 + 3)) = (hmpprt::u08 ) (((hmpprt::s32 ) (*(yPlaneCSER_11 + w_12 * 4)) + (hmpprt::s32 ) (*(yPlaneCSER_11 + (w_12 * 4 + 4))) * 3) / 4);
 }
 else
 {
  # 177 "sharpness.cpp"
  *(yPlaneCSER_11 + (w_12 * 4 + 1)) = *(yPlaneCSER_11 + w_12 * 4);
 }
 # 164 "sharpness.cpp"
 __hmppcg_label_3:;
}
#endif // __HIPCC__



# 164 "sharpness.cpp"

#ifndef __HIPCC__
void hmpp_acc_region_main_164_internal_1(hmpprt::s32* width_16, hmpprt::s32 quarterWidth_14, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  yPlaneCSER_18)
{
 # 166 "sharpness.cpp"
 hmpprt::s32 width_22;
 # 166 "sharpness.cpp"
 width_22 = *width_16;
 # 166 "sharpness.cpp"
 {
  # 164 "sharpness.cpp"
  if (quarterWidth_14 - 1 >= 0)
  {
   hmpprt::CUDAGridCall __hmppcg_call;
   __hmppcg_call.setSizeX((quarterWidth_14 - 1) / 128 + 1);
   __hmppcg_call.setSizeY(1);
   __hmppcg_call.setBlockSizeX(32);
   __hmppcg_call.setBlockSizeY(4);
   __hmppcg_call.addLocalParameter((hmpprt::s32) (quarterWidth_14), "quarterWidth_6");
   __hmppcg_call.addLocalParameter(&yPlaneCSER_18, 8, "yPlaneCSER_11");
   __hmppcg_call.launch(hmpp_acc_region_main_164_loop1D_1, hmpprt::Context::getInstance()->getCUDADevice());
  }
  ;
 }
 # 164 "sharpness.cpp"
 *width_16 = width_22;
}
#endif // __HIPCC__



# 164 "sharpness.cpp"

#ifndef __HIPCC__
extern "C" CDLT_API  void hmpp_acc_region_main_164(hmpprt::s32* width_55, hmpprt::s32 quarterWidth_19, hmpprt::u08* yPlaneCSER_36)
{
 # 185 "sharpness.cpp"
 (hmpp_acc_region_main_164_internal_1(width_55, quarterWidth_19, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08> (yPlaneCSER_36)));
}
#endif // __HIPCC__



# 185 "sharpness.cpp"

#ifdef __HIPCC__

extern "C" __global__ void hmpp_acc_region_main_185_loop1D_1(hmpprt::s32 width_36, hmpprt::u08* yPlaneCSER_10)
{
 # 188 "sharpness.cpp"
 hmpprt::s32 w_13;
 # 190 "sharpness.cpp"
 w_13 = (hmpprt::gr_atidf());
 # 190 "sharpness.cpp"
 if (w_13 > width_36 - 1)
 {
  # 190 "sharpness.cpp"
  goto __hmppcg_label_4;
 }
 # 190 "sharpness.cpp"
 *(yPlaneCSER_10 + (width_36 + w_13)) = *(yPlaneCSER_10 + w_13);
 # 185 "sharpness.cpp"
 __hmppcg_label_4:;
}
#endif // __HIPCC__



# 185 "sharpness.cpp"

#ifndef __HIPCC__
void hmpp_acc_region_main_185_internal_1(hmpprt::s32* width_15, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  yPlaneCSER_17)
{
 # 187 "sharpness.cpp"
 hmpprt::s32 width_23;
 # 187 "sharpness.cpp"
 width_23 = *width_15;
 # 187 "sharpness.cpp"
 {
  # 185 "sharpness.cpp"
  if (width_23 - 1 >= 0)
  {
   hmpprt::CUDAGridCall __hmppcg_call;
   __hmppcg_call.setSizeX((width_23 - 1) / 128 + 1);
   __hmppcg_call.setSizeY(1);
   __hmppcg_call.setBlockSizeX(32);
   __hmppcg_call.setBlockSizeY(4);
   __hmppcg_call.addLocalParameter((hmpprt::s32) (width_23), "width_36");
   __hmppcg_call.addLocalParameter(&yPlaneCSER_17, 8, "yPlaneCSER_10");
   __hmppcg_call.launch(hmpp_acc_region_main_185_loop1D_1, hmpprt::Context::getInstance()->getCUDADevice());
  }
  ;
 }
 # 185 "sharpness.cpp"
 *width_15 = width_23;
}
#endif // __HIPCC__



# 185 "sharpness.cpp"

#ifndef __HIPCC__
extern "C" CDLT_API  void hmpp_acc_region_main_185(hmpprt::s32* width_54, hmpprt::u08* yPlaneCSER_35)
{
 # 197 "sharpness.cpp"
 (hmpp_acc_region_main_185_internal_1(width_54, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08> (yPlaneCSER_35)));
}
#endif // __HIPCC__



# 197 "sharpness.cpp"

#ifdef __HIPCC__

extern "C" __global__ void hmpp_acc_region_main_197_loop1D_1(hmpprt::s32 quarterWidth_5, hmpprt::s32 quarterHeight_10, hmpprt::u08* yPlaneCSER_29, hmpprt::u08* yPlaneDown_10, hmpprt::s32 width_24, hmpprt::s32 height_17)
{
 # 200 "sharpness.cpp"
 hmpprt::s32 w_14;
 # 202 "sharpness.cpp"
 w_14 = (hmpprt::gr_atidf());
 # 202 "sharpness.cpp"
 if (w_14 > quarterWidth_5 - 1)
 {
  # 202 "sharpness.cpp"
  goto __hmppcg_label_5;
 }
 # 202 "sharpness.cpp"
 *(yPlaneCSER_29 + ((height_17 - 2) * width_24 + w_14 * 4)) = *(yPlaneDown_10 + ((quarterHeight_10 - 1) * quarterWidth_5 + w_14));
 # 197 "sharpness.cpp"
 __hmppcg_label_5:;
}
#endif // __HIPCC__



# 197 "sharpness.cpp"

#ifndef __HIPCC__
void hmpp_acc_region_main_197_internal_1(hmpprt::s32* width_14, hmpprt::s32* height_11, hmpprt::s32 quarterWidth_13, hmpprt::s32 quarterHeight_4, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  yPlaneCSER_44, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  yPlaneDown_4)
{
 # 199 "sharpness.cpp"
 hmpprt::s32 height_12;
 # 199 "sharpness.cpp"
 height_12 = *height_11;
 # 199 "sharpness.cpp"
 hmpprt::s32 width_37;
 # 199 "sharpness.cpp"
 width_37 = *width_14;
 # 199 "sharpness.cpp"
 {
  # 197 "sharpness.cpp"
  if (quarterWidth_13 - 1 >= 0)
  {
   hmpprt::CUDAGridCall __hmppcg_call;
   __hmppcg_call.setSizeX((quarterWidth_13 - 1) / 128 + 1);
   __hmppcg_call.setSizeY(1);
   __hmppcg_call.setBlockSizeX(32);
   __hmppcg_call.setBlockSizeY(4);
   __hmppcg_call.addLocalParameter((hmpprt::s32) (quarterWidth_13), "quarterWidth_5");
   __hmppcg_call.addLocalParameter((hmpprt::s32) (quarterHeight_4), "quarterHeight_10");
   __hmppcg_call.addLocalParameter(&yPlaneCSER_44, 8, "yPlaneCSER_29");
   __hmppcg_call.addLocalParameter(&yPlaneDown_4, 8, "yPlaneDown_10");
   __hmppcg_call.addLocalParameter((hmpprt::s32) (width_37), "width_24");
   __hmppcg_call.addLocalParameter((hmpprt::s32) (height_12), "height_17");
   __hmppcg_call.launch(hmpp_acc_region_main_197_loop1D_1, hmpprt::Context::getInstance()->getCUDADevice());
  }
  ;
 }
 # 197 "sharpness.cpp"
 *width_14 = width_37;
 # 197 "sharpness.cpp"
 *height_11 = height_12;
}
#endif // __HIPCC__



# 197 "sharpness.cpp"

#ifndef __HIPCC__
extern "C" CDLT_API  void hmpp_acc_region_main_197(hmpprt::s32* width_53, hmpprt::s32* height_28, hmpprt::s32 quarterWidth_18, hmpprt::s32 quarterHeight_14, hmpprt::u08* yPlaneCSER_9, hmpprt::u08* yPlaneDown_12)
{
 # 209 "sharpness.cpp"
 (hmpp_acc_region_main_197_internal_1(width_53, height_28, quarterWidth_18, quarterHeight_14, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08> (yPlaneCSER_9), hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08> (yPlaneDown_12)));
}
#endif // __HIPCC__



# 209 "sharpness.cpp"

#ifdef __HIPCC__

extern "C" __global__ void hmpp_acc_region_main_209_loop1D_1(hmpprt::s32 quarterWidth_4, hmpprt::u08* yPlaneCSER_8, hmpprt::s32 width_25, hmpprt::s32 height_13)
{
 # 212 "sharpness.cpp"
 hmpprt::s32 w_15;
 # 214 "sharpness.cpp"
 w_15 = (hmpprt::gr_atidf());
 # 214 "sharpness.cpp"
 if (w_15 > quarterWidth_4 - 1)
 {
  # 214 "sharpness.cpp"
  goto __hmppcg_label_6;
 }
 # 214 "sharpness.cpp"
 if (w_15 < quarterWidth_4 - 1)
 {
  # 216 "sharpness.cpp"
  *(yPlaneCSER_8 + ((height_13 - 2) * width_25 + w_15 * 4 + 1)) = (hmpprt::u08 ) (((hmpprt::s32 ) (*(yPlaneCSER_8 + ((height_13 - 2) * width_25 + w_15 * 4))) * 3 + (hmpprt::s32 ) (*(yPlaneCSER_8 + ((height_13 - 2) * width_25 + w_15 * 4 + 4)))) / 4);
  # 217 "sharpness.cpp"
  *(yPlaneCSER_8 + ((height_13 - 2) * width_25 + w_15 * 4 + 2)) = (hmpprt::u08 ) (((hmpprt::s32 ) (*(yPlaneCSER_8 + ((height_13 - 2) * width_25 + w_15 * 4))) * 2 + (hmpprt::s32 ) (*(yPlaneCSER_8 + ((height_13 - 2) * width_25 + w_15 * 4 + 4))) * 2) / 4);
  # 218 "sharpness.cpp"
  *(yPlaneCSER_8 + ((height_13 - 2) * width_25 + w_15 * 4 + 3)) = (hmpprt::u08 ) (((hmpprt::s32 ) (*(yPlaneCSER_8 + ((height_13 - 2) * width_25 + w_15 * 4))) + (hmpprt::s32 ) (*(yPlaneCSER_8 + ((height_13 - 2) * width_25 + w_15 * 4 + 4))) * 3) / 4);
 }
 else
 {
  # 222 "sharpness.cpp"
  *(yPlaneCSER_8 + ((height_13 - 2) * width_25 + w_15 * 4 + 1)) = *(yPlaneCSER_8 + ((height_13 - 2) * width_25 + w_15 * 4));
 }
 # 209 "sharpness.cpp"
 __hmppcg_label_6:;
}
#endif // __HIPCC__



# 209 "sharpness.cpp"

#ifndef __HIPCC__
void hmpp_acc_region_main_209_internal_1(hmpprt::s32* width_52, hmpprt::s32* height_27, hmpprt::s32 quarterWidth_17, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  yPlaneCSER_41)
{
 # 211 "sharpness.cpp"
 hmpprt::s32 height_18;
 # 211 "sharpness.cpp"
 height_18 = *height_27;
 # 211 "sharpness.cpp"
 hmpprt::s32 width_38;
 # 211 "sharpness.cpp"
 width_38 = *width_52;
 # 211 "sharpness.cpp"
 {
  # 209 "sharpness.cpp"
  if (quarterWidth_17 - 1 >= 0)
  {
   hmpprt::CUDAGridCall __hmppcg_call;
   __hmppcg_call.setSizeX((quarterWidth_17 - 1) / 128 + 1);
   __hmppcg_call.setSizeY(1);
   __hmppcg_call.setBlockSizeX(32);
   __hmppcg_call.setBlockSizeY(4);
   __hmppcg_call.addLocalParameter((hmpprt::s32) (quarterWidth_17), "quarterWidth_4");
   __hmppcg_call.addLocalParameter(&yPlaneCSER_41, 8, "yPlaneCSER_8");
   __hmppcg_call.addLocalParameter((hmpprt::s32) (width_38), "width_25");
   __hmppcg_call.addLocalParameter((hmpprt::s32) (height_18), "height_13");
   __hmppcg_call.launch(hmpp_acc_region_main_209_loop1D_1, hmpprt::Context::getInstance()->getCUDADevice());
  }
  ;
 }
 # 209 "sharpness.cpp"
 *width_52 = width_38;
 # 209 "sharpness.cpp"
 *height_27 = height_18;
}
#endif // __HIPCC__



# 209 "sharpness.cpp"

#ifndef __HIPCC__
extern "C" CDLT_API  void hmpp_acc_region_main_209(hmpprt::s32* width_13, hmpprt::s32* height_10, hmpprt::s32 quarterWidth_12, hmpprt::u08* yPlaneCSER_28)
{
 # 230 "sharpness.cpp"
 (hmpp_acc_region_main_209_internal_1(width_13, height_10, quarterWidth_12, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08> (yPlaneCSER_28)));
}
#endif // __HIPCC__



# 230 "sharpness.cpp"

#ifdef __HIPCC__

extern "C" __global__ void hmpp_acc_region_main_230_loop1D_1(hmpprt::u08* yPlaneCSER_7, hmpprt::s32 width_26, hmpprt::s32 height_14)
{
 # 233 "sharpness.cpp"
 hmpprt::s32 w_16;
 # 235 "sharpness.cpp"
 w_16 = (hmpprt::gr_atidf());
 # 235 "sharpness.cpp"
 if (w_16 > width_26 - 1)
 {
  # 235 "sharpness.cpp"
  goto __hmppcg_label_7;
 }
 # 235 "sharpness.cpp"
 *(yPlaneCSER_7 + ((height_14 - 1) * width_26 + w_16)) = *(yPlaneCSER_7 + ((height_14 - 2) * width_26 + w_16));
 # 230 "sharpness.cpp"
 __hmppcg_label_7:;
}
#endif // __HIPCC__



# 230 "sharpness.cpp"

#ifndef __HIPCC__
void hmpp_acc_region_main_230_internal_1(hmpprt::s32* width_51, hmpprt::s32* height_38, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  yPlaneCSER_27)
{
 # 232 "sharpness.cpp"
 hmpprt::s32 height_19;
 # 232 "sharpness.cpp"
 height_19 = *height_38;
 # 232 "sharpness.cpp"
 hmpprt::s32 width_39;
 # 232 "sharpness.cpp"
 width_39 = *width_51;
 # 232 "sharpness.cpp"
 {
  # 230 "sharpness.cpp"
  if (width_39 - 1 >= 0)
  {
   hmpprt::CUDAGridCall __hmppcg_call;
   __hmppcg_call.setSizeX((width_39 - 1) / 128 + 1);
   __hmppcg_call.setSizeY(1);
   __hmppcg_call.setBlockSizeX(32);
   __hmppcg_call.setBlockSizeY(4);
   __hmppcg_call.addLocalParameter(&yPlaneCSER_27, 8, "yPlaneCSER_7");
   __hmppcg_call.addLocalParameter((hmpprt::s32) (width_39), "width_26");
   __hmppcg_call.addLocalParameter((hmpprt::s32) (height_19), "height_14");
   __hmppcg_call.launch(hmpp_acc_region_main_230_loop1D_1, hmpprt::Context::getInstance()->getCUDADevice());
  }
  ;
 }
 # 230 "sharpness.cpp"
 *width_51 = width_39;
 # 230 "sharpness.cpp"
 *height_38 = height_19;
}
#endif // __HIPCC__



# 230 "sharpness.cpp"

#ifndef __HIPCC__
extern "C" CDLT_API  void hmpp_acc_region_main_230(hmpprt::s32* width_12, hmpprt::s32* height_9, hmpprt::u08* yPlaneCSER_40)
{
 # 242 "sharpness.cpp"
 (hmpp_acc_region_main_230_internal_1(width_12, height_9, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08> (yPlaneCSER_40)));
}
#endif // __HIPCC__



# 242 "sharpness.cpp"

#ifdef __HIPCC__

extern "C" __global__ void hmpp_acc_region_main_242_loop1D_1(hmpprt::s32 quarterWidth_11, hmpprt::s32 quarterHeight_3, hmpprt::u08* yPlaneCSER_6, hmpprt::u08* yPlaneDown_3, hmpprt::s32 width_27)
{
 # 245 "sharpness.cpp"
 hmpprt::s32 h_14;
 # 247 "sharpness.cpp"
 h_14 = (hmpprt::gr_atidf());
 # 247 "sharpness.cpp"
 if (h_14 > quarterHeight_3 - 1)
 {
  # 247 "sharpness.cpp"
  goto __hmppcg_label_8;
 }
 # 247 "sharpness.cpp"
 *(yPlaneCSER_6 + (h_14 * 4 * width_27 + width_27 - 2)) = *(yPlaneDown_3 + (h_14 * quarterWidth_11 + quarterWidth_11 - 1));
 # 242 "sharpness.cpp"
 __hmppcg_label_8:;
}
#endif // __HIPCC__



# 242 "sharpness.cpp"

#ifndef __HIPCC__
void hmpp_acc_region_main_242_internal_1(hmpprt::s32* width_50, hmpprt::s32 quarterWidth_20, hmpprt::s32 quarterHeight_17, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  yPlaneCSER_39, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  yPlaneDown_17)
{
 # 244 "sharpness.cpp"
 hmpprt::s32 width_40;
 # 244 "sharpness.cpp"
 width_40 = *width_50;
 # 244 "sharpness.cpp"
 {
  # 242 "sharpness.cpp"
  if (quarterHeight_17 - 1 >= 0)
  {
   hmpprt::CUDAGridCall __hmppcg_call;
   __hmppcg_call.setSizeX((quarterHeight_17 - 1) / 128 + 1);
   __hmppcg_call.setSizeY(1);
   __hmppcg_call.setBlockSizeX(32);
   __hmppcg_call.setBlockSizeY(4);
   __hmppcg_call.addLocalParameter((hmpprt::s32) (quarterWidth_20), "quarterWidth_11");
   __hmppcg_call.addLocalParameter((hmpprt::s32) (quarterHeight_17), "quarterHeight_3");
   __hmppcg_call.addLocalParameter(&yPlaneCSER_39, 8, "yPlaneCSER_6");
   __hmppcg_call.addLocalParameter(&yPlaneDown_17, 8, "yPlaneDown_3");
   __hmppcg_call.addLocalParameter((hmpprt::s32) (width_40), "width_27");
   __hmppcg_call.launch(hmpp_acc_region_main_242_loop1D_1, hmpprt::Context::getInstance()->getCUDADevice());
  }
  ;
 }
 # 242 "sharpness.cpp"
 *width_50 = width_40;
}
#endif // __HIPCC__



# 242 "sharpness.cpp"

#ifndef __HIPCC__
extern "C" CDLT_API  void hmpp_acc_region_main_242(hmpprt::s32* width_11, hmpprt::s32 quarterWidth_3, hmpprt::s32 quarterHeight_9, hmpprt::u08* yPlaneCSER_26, hmpprt::u08* yPlaneDown_9)
{
 # 254 "sharpness.cpp"
 (hmpp_acc_region_main_242_internal_1(width_11, quarterWidth_3, quarterHeight_9, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08> (yPlaneCSER_26), hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08> (yPlaneDown_9)));
}
#endif // __HIPCC__



# 254 "sharpness.cpp"

#ifdef __HIPCC__

extern "C" __global__ void hmpp_acc_region_main_254_loop1D_1(hmpprt::s32 quarterHeight_8, hmpprt::u08* yPlaneCSER_25, hmpprt::s32 width_28)
{
 # 257 "sharpness.cpp"
 hmpprt::s32 h_15;
 # 259 "sharpness.cpp"
 h_15 = (hmpprt::gr_atidf());
 # 259 "sharpness.cpp"
 if (h_15 > quarterHeight_8 - 1)
 {
  # 259 "sharpness.cpp"
  goto __hmppcg_label_9;
 }
 # 259 "sharpness.cpp"
 if (h_15 < quarterHeight_8 - 1)
 {
  # 261 "sharpness.cpp"
  *(yPlaneCSER_25 + ((h_15 * 4 + 1) * width_28 + width_28 - 2)) = (hmpprt::u08 ) (((hmpprt::s32 ) (*(yPlaneCSER_25 + (h_15 * 4 * width_28 + width_28 - 2))) * 3 + (hmpprt::s32 ) (*(yPlaneCSER_25 + ((h_15 * 4 + 4) * width_28 + width_28 - 2)))) / 4);
  # 262 "sharpness.cpp"
  *(yPlaneCSER_25 + ((h_15 * 4 + 2) * width_28 + width_28 - 2)) = (hmpprt::u08 ) (((hmpprt::s32 ) (*(yPlaneCSER_25 + (h_15 * 4 * width_28 + width_28 - 2))) * 2 + (hmpprt::s32 ) (*(yPlaneCSER_25 + ((h_15 * 4 + 4) * width_28 + width_28 - 2))) * 2) / 4);
  # 263 "sharpness.cpp"
  *(yPlaneCSER_25 + ((h_15 * 4 + 3) * width_28 + width_28 - 2)) = (hmpprt::u08 ) (((hmpprt::s32 ) (*(yPlaneCSER_25 + (h_15 * 4 * width_28 + width_28 - 2))) + (hmpprt::s32 ) (*(yPlaneCSER_25 + ((h_15 * 4 + 4) * width_28 + width_28 - 2))) * 3) / 4);
 }
 else
 {
  # 267 "sharpness.cpp"
  *(yPlaneCSER_25 + ((h_15 * 4 + 1) * width_28 + width_28 - 2)) = *(yPlaneCSER_25 + (h_15 * 4 * width_28 + width_28 - 2));
  # 268 "sharpness.cpp"
  *(yPlaneCSER_25 + ((h_15 * 4 + 2) * width_28 + width_28 - 2)) = *(yPlaneCSER_25 + (h_15 * 4 * width_28 + width_28 - 2));
  # 269 "sharpness.cpp"
  *(yPlaneCSER_25 + ((h_15 * 4 + 3) * width_28 + width_28 - 2)) = *(yPlaneCSER_25 + (h_15 * 4 * width_28 + width_28 - 2));
 }
 # 254 "sharpness.cpp"
 __hmppcg_label_9:;
}
#endif // __HIPCC__



# 254 "sharpness.cpp"

#ifndef __HIPCC__
void hmpp_acc_region_main_254_internal_1(hmpprt::s32* width_49, hmpprt::s32 quarterHeight_13, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  yPlaneCSER_43)
{
 # 256 "sharpness.cpp"
 hmpprt::s32 width_41;
 # 256 "sharpness.cpp"
 width_41 = *width_49;
 # 256 "sharpness.cpp"
 {
  # 254 "sharpness.cpp"
  if (quarterHeight_13 - 1 >= 0)
  {
   hmpprt::CUDAGridCall __hmppcg_call;
   __hmppcg_call.setSizeX((quarterHeight_13 - 1) / 128 + 1);
   __hmppcg_call.setSizeY(1);
   __hmppcg_call.setBlockSizeX(32);
   __hmppcg_call.setBlockSizeY(4);
   __hmppcg_call.addLocalParameter((hmpprt::s32) (quarterHeight_13), "quarterHeight_8");
   __hmppcg_call.addLocalParameter(&yPlaneCSER_43, 8, "yPlaneCSER_25");
   __hmppcg_call.addLocalParameter((hmpprt::s32) (width_41), "width_28");
   __hmppcg_call.launch(hmpp_acc_region_main_254_loop1D_1, hmpprt::Context::getInstance()->getCUDADevice());
  }
  ;
 }
 # 254 "sharpness.cpp"
 *width_49 = width_41;
}
#endif // __HIPCC__



# 254 "sharpness.cpp"

#ifndef __HIPCC__
extern "C" CDLT_API  void hmpp_acc_region_main_254(hmpprt::s32* width_10, hmpprt::s32 quarterHeight_2, hmpprt::u08* yPlaneCSER_5)
{
 # 277 "sharpness.cpp"
 (hmpp_acc_region_main_254_internal_1(width_10, quarterHeight_2, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08> (yPlaneCSER_5)));
}
#endif // __HIPCC__



# 277 "sharpness.cpp"

#ifdef __HIPCC__

extern "C" __global__ void hmpp_acc_region_main_277_loop1D_1(hmpprt::s32 height_26, hmpprt::u08* yPlaneCSER_24, hmpprt::s32 width_29)
{
 # 280 "sharpness.cpp"
 hmpprt::s32 h_16;
 # 282 "sharpness.cpp"
 h_16 = (hmpprt::gr_atidf());
 # 282 "sharpness.cpp"
 if (h_16 > height_26 - 1)
 {
  # 282 "sharpness.cpp"
  goto __hmppcg_label_10;
 }
 # 282 "sharpness.cpp"
 *(yPlaneCSER_24 + (h_16 * width_29 + width_29 - 1)) = *(yPlaneCSER_24 + (h_16 * width_29 + width_29 - 2));
 # 277 "sharpness.cpp"
 __hmppcg_label_10:;
}
#endif // __HIPCC__



# 277 "sharpness.cpp"

#ifndef __HIPCC__
void hmpp_acc_region_main_277_internal_1(hmpprt::s32* width_9, hmpprt::s32 height_37, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  yPlaneCSER_42)
{
 # 279 "sharpness.cpp"
 hmpprt::s32 width_42;
 # 279 "sharpness.cpp"
 width_42 = *width_9;
 # 279 "sharpness.cpp"
 {
  # 277 "sharpness.cpp"
  if (height_37 - 1 >= 0)
  {
   hmpprt::CUDAGridCall __hmppcg_call;
   __hmppcg_call.setSizeX((height_37 - 1) / 128 + 1);
   __hmppcg_call.setSizeY(1);
   __hmppcg_call.setBlockSizeX(32);
   __hmppcg_call.setBlockSizeY(4);
   __hmppcg_call.addLocalParameter((hmpprt::s32) (height_37), "height_26");
   __hmppcg_call.addLocalParameter(&yPlaneCSER_42, 8, "yPlaneCSER_24");
   __hmppcg_call.addLocalParameter((hmpprt::s32) (width_42), "width_29");
   __hmppcg_call.launch(hmpp_acc_region_main_277_loop1D_1, hmpprt::Context::getInstance()->getCUDADevice());
  }
  ;
 }
 # 277 "sharpness.cpp"
 *width_9 = width_42;
}
#endif // __HIPCC__



# 277 "sharpness.cpp"

#ifndef __HIPCC__
extern "C" CDLT_API  void hmpp_acc_region_main_277(hmpprt::s32* width_68, hmpprt::s32 height_8, hmpprt::u08* yPlaneCSER_4)
{
 # 289 "sharpness.cpp"
 (hmpp_acc_region_main_277_internal_1(width_68, height_8, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08> (yPlaneCSER_4)));
}
#endif // __HIPCC__



# 289 "sharpness.cpp"

#ifdef __HIPCC__

extern "C" __global__ void hmpp_acc_region_main_289_loop1D_1(hmpprt::s32 width_30, hmpprt::s32 quarterWidth_2, hmpprt::s32 quarterHeight_1, hmpprt::u08* yPlaneCSER_3, hmpprt::u08* yPlaneDown_2)
{
 # 292 "sharpness.cpp"
 hmpprt::s32 h_17;
 # 295 "sharpness.cpp"
 hmpprt::s32 w_17;
 # 292 "sharpness.cpp"
 hmpprt::s32 h_22;
 # 297 "sharpness.cpp"
 h_22 = (hmpprt::gr_atidf());
 # 297 "sharpness.cpp"
 if (h_22 > (quarterHeight_1 +  -1) * (quarterWidth_2 - 1) - 1)
 {
  # 297 "sharpness.cpp"
  goto __hmppcg_label_11;
 }
 # 297 "sharpness.cpp"
 w_17 = h_22 % (quarterWidth_2 +  -1);
 # 297 "sharpness.cpp"
 h_17 = h_22 / (quarterWidth_2 +  -1);
 # 297 "sharpness.cpp"
 *(yPlaneCSER_3 + ((h_17 * 4 + 2) * width_30 + w_17 * 4 + 2)) = (hmpprt::u08 ) (((hmpprt::s32 ) (*(yPlaneDown_2 + (h_17 * quarterWidth_2 + w_17))) * 7 + (hmpprt::s32 ) (*(yPlaneDown_2 + (h_17 * quarterWidth_2 + w_17 + 1)))) * 7 / 8 / 8 + ((hmpprt::s32 ) (*(yPlaneDown_2 + ((h_17 + 1) * quarterWidth_2 + w_17))) * 7 + (hmpprt::s32 ) (*(yPlaneDown_2 + ((h_17 + 1) * quarterWidth_2 + w_17 + 1)))) / 8 / 8);
 # 298 "sharpness.cpp"
 *(yPlaneCSER_3 + ((h_17 * 4 + 2) * width_30 + w_17 * 4 + 3)) = (hmpprt::u08 ) (((hmpprt::s32 ) (*(yPlaneDown_2 + (h_17 * quarterWidth_2 + w_17))) * 5 + (hmpprt::s32 ) (*(yPlaneDown_2 + (h_17 * quarterWidth_2 + w_17 + 1))) * 3) * 7 / 8 / 8 + ((hmpprt::s32 ) (*(yPlaneDown_2 + ((h_17 + 1) * quarterWidth_2 + w_17))) * 5 + (hmpprt::s32 ) (*(yPlaneDown_2 + ((h_17 + 1) * quarterWidth_2 + w_17 + 1))) * 3) / 8 / 8);
 # 299 "sharpness.cpp"
 *(yPlaneCSER_3 + ((h_17 * 4 + 2) * width_30 + w_17 * 4 + 4)) = (hmpprt::u08 ) (((hmpprt::s32 ) (*(yPlaneDown_2 + (h_17 * quarterWidth_2 + w_17))) * 3 + (hmpprt::s32 ) (*(yPlaneDown_2 + (h_17 * quarterWidth_2 + w_17 + 1))) * 5) * 7 / 8 / 8 + ((hmpprt::s32 ) (*(yPlaneDown_2 + ((h_17 + 1) * quarterWidth_2 + w_17))) * 3 + (hmpprt::s32 ) (*(yPlaneDown_2 + ((h_17 + 1) * quarterWidth_2 + w_17 + 1))) * 5) / 8 / 8);
 # 300 "sharpness.cpp"
 *(yPlaneCSER_3 + ((h_17 * 4 + 2) * width_30 + w_17 * 4 + 5)) = (hmpprt::u08 ) (((hmpprt::s32 ) (*(yPlaneDown_2 + (h_17 * quarterWidth_2 + w_17))) + (hmpprt::s32 ) (*(yPlaneDown_2 + (h_17 * quarterWidth_2 + w_17 + 1))) * 7) * 7 / 8 / 8 + ((hmpprt::s32 ) (*(yPlaneDown_2 + ((h_17 + 1) * quarterWidth_2 + w_17))) + (hmpprt::s32 ) (*(yPlaneDown_2 + ((h_17 + 1) * quarterWidth_2 + w_17 + 1))) * 7) / 8 / 8);
 # 302 "sharpness.cpp"
 *(yPlaneCSER_3 + ((h_17 * 4 + 3) * width_30 + w_17 * 4 + 2)) = (hmpprt::u08 ) (((hmpprt::s32 ) (*(yPlaneDown_2 + (h_17 * quarterWidth_2 + w_17))) * 7 + (hmpprt::s32 ) (*(yPlaneDown_2 + (h_17 * quarterWidth_2 + w_17 + 1)))) * 5 / 8 / 8 + ((hmpprt::s32 ) (*(yPlaneDown_2 + ((h_17 + 1) * quarterWidth_2 + w_17))) * 7 + (hmpprt::s32 ) (*(yPlaneDown_2 + ((h_17 + 1) * quarterWidth_2 + w_17 + 1)))) * 3 / 8 / 8);
 # 303 "sharpness.cpp"
 *(yPlaneCSER_3 + ((h_17 * 4 + 3) * width_30 + w_17 * 4 + 3)) = (hmpprt::u08 ) (((hmpprt::s32 ) (*(yPlaneDown_2 + (h_17 * quarterWidth_2 + w_17))) * 5 + (hmpprt::s32 ) (*(yPlaneDown_2 + (h_17 * quarterWidth_2 + w_17 + 1))) * 3) * 5 / 8 / 8 + ((hmpprt::s32 ) (*(yPlaneDown_2 + ((h_17 + 1) * quarterWidth_2 + w_17))) * 5 + (hmpprt::s32 ) (*(yPlaneDown_2 + ((h_17 + 1) * quarterWidth_2 + w_17 + 1))) * 3) * 3 / 8 / 8);
 # 304 "sharpness.cpp"
 *(yPlaneCSER_3 + ((h_17 * 4 + 3) * width_30 + w_17 * 4 + 4)) = (hmpprt::u08 ) (((hmpprt::s32 ) (*(yPlaneDown_2 + (h_17 * quarterWidth_2 + w_17))) * 3 + (hmpprt::s32 ) (*(yPlaneDown_2 + (h_17 * quarterWidth_2 + w_17 + 1))) * 5) * 5 / 8 / 8 + ((hmpprt::s32 ) (*(yPlaneDown_2 + ((h_17 + 1) * quarterWidth_2 + w_17))) * 3 + (hmpprt::s32 ) (*(yPlaneDown_2 + ((h_17 + 1) * quarterWidth_2 + w_17 + 1))) * 5) * 3 / 8 / 8);
 # 305 "sharpness.cpp"
 *(yPlaneCSER_3 + ((h_17 * 4 + 3) * width_30 + w_17 * 4 + 5)) = (hmpprt::u08 ) (((hmpprt::s32 ) (*(yPlaneDown_2 + (h_17 * quarterWidth_2 + w_17))) + (hmpprt::s32 ) (*(yPlaneDown_2 + (h_17 * quarterWidth_2 + w_17 + 1))) * 7) * 5 / 8 / 8 + ((hmpprt::s32 ) (*(yPlaneDown_2 + ((h_17 + 1) * quarterWidth_2 + w_17))) + (hmpprt::s32 ) (*(yPlaneDown_2 + ((h_17 + 1) * quarterWidth_2 + w_17 + 1))) * 7) * 3 / 8 / 8);
 # 307 "sharpness.cpp"
 *(yPlaneCSER_3 + ((h_17 * 4 + 4) * width_30 + w_17 * 4 + 2)) = (hmpprt::u08 ) (((hmpprt::s32 ) (*(yPlaneDown_2 + (h_17 * quarterWidth_2 + w_17))) * 7 + (hmpprt::s32 ) (*(yPlaneDown_2 + (h_17 * quarterWidth_2 + w_17 + 1)))) * 3 / 8 / 8 + ((hmpprt::s32 ) (*(yPlaneDown_2 + ((h_17 + 1) * quarterWidth_2 + w_17))) * 7 + (hmpprt::s32 ) (*(yPlaneDown_2 + ((h_17 + 1) * quarterWidth_2 + w_17 + 1)))) * 5 / 8 / 8);
 # 308 "sharpness.cpp"
 *(yPlaneCSER_3 + ((h_17 * 4 + 4) * width_30 + w_17 * 4 + 3)) = (hmpprt::u08 ) (((hmpprt::s32 ) (*(yPlaneDown_2 + (h_17 * quarterWidth_2 + w_17))) * 5 + (hmpprt::s32 ) (*(yPlaneDown_2 + (h_17 * quarterWidth_2 + w_17 + 1))) * 3) * 3 / 8 / 8 + ((hmpprt::s32 ) (*(yPlaneDown_2 + ((h_17 + 1) * quarterWidth_2 + w_17))) * 5 + (hmpprt::s32 ) (*(yPlaneDown_2 + ((h_17 + 1) * quarterWidth_2 + w_17 + 1))) * 3) * 5 / 8 / 8);
 # 309 "sharpness.cpp"
 *(yPlaneCSER_3 + ((h_17 * 4 + 4) * width_30 + w_17 * 4 + 4)) = (hmpprt::u08 ) (((hmpprt::s32 ) (*(yPlaneDown_2 + (h_17 * quarterWidth_2 + w_17))) * 3 + (hmpprt::s32 ) (*(yPlaneDown_2 + (h_17 * quarterWidth_2 + w_17 + 1))) * 5) * 3 / 8 / 8 + ((hmpprt::s32 ) (*(yPlaneDown_2 + ((h_17 + 1) * quarterWidth_2 + w_17))) * 3 + (hmpprt::s32 ) (*(yPlaneDown_2 + ((h_17 + 1) * quarterWidth_2 + w_17 + 1))) * 5) * 5 / 8 / 8);
 # 310 "sharpness.cpp"
 *(yPlaneCSER_3 + ((h_17 * 4 + 4) * width_30 + w_17 * 4 + 5)) = (hmpprt::u08 ) (((hmpprt::s32 ) (*(yPlaneDown_2 + (h_17 * quarterWidth_2 + w_17))) + (hmpprt::s32 ) (*(yPlaneDown_2 + (h_17 * quarterWidth_2 + w_17 + 1))) * 7) * 3 / 8 / 8 + ((hmpprt::s32 ) (*(yPlaneDown_2 + ((h_17 + 1) * quarterWidth_2 + w_17))) + (hmpprt::s32 ) (*(yPlaneDown_2 + ((h_17 + 1) * quarterWidth_2 + w_17 + 1))) * 7) * 5 / 8 / 8);
 # 312 "sharpness.cpp"
 *(yPlaneCSER_3 + ((h_17 * 4 + 5) * width_30 + w_17 * 4 + 2)) = (hmpprt::u08 ) (((hmpprt::s32 ) (*(yPlaneDown_2 + (h_17 * quarterWidth_2 + w_17))) * 7 + (hmpprt::s32 ) (*(yPlaneDown_2 + (h_17 * quarterWidth_2 + w_17 + 1)))) / 8 / 8 + ((hmpprt::s32 ) (*(yPlaneDown_2 + ((h_17 + 1) * quarterWidth_2 + w_17))) * 7 + (hmpprt::s32 ) (*(yPlaneDown_2 + ((h_17 + 1) * quarterWidth_2 + w_17 + 1)))) * 7 / 8 / 8);
 # 313 "sharpness.cpp"
 *(yPlaneCSER_3 + ((h_17 * 4 + 5) * width_30 + w_17 * 4 + 3)) = (hmpprt::u08 ) (((hmpprt::s32 ) (*(yPlaneDown_2 + (h_17 * quarterWidth_2 + w_17))) * 5 + (hmpprt::s32 ) (*(yPlaneDown_2 + (h_17 * quarterWidth_2 + w_17 + 1))) * 3) / 8 / 8 + ((hmpprt::s32 ) (*(yPlaneDown_2 + ((h_17 + 1) * quarterWidth_2 + w_17))) * 5 + (hmpprt::s32 ) (*(yPlaneDown_2 + ((h_17 + 1) * quarterWidth_2 + w_17 + 1))) * 3) * 7 / 8 / 8);
 # 314 "sharpness.cpp"
 *(yPlaneCSER_3 + ((h_17 * 4 + 5) * width_30 + w_17 * 4 + 4)) = (hmpprt::u08 ) (((hmpprt::s32 ) (*(yPlaneDown_2 + (h_17 * quarterWidth_2 + w_17))) * 3 + (hmpprt::s32 ) (*(yPlaneDown_2 + (h_17 * quarterWidth_2 + w_17 + 1))) * 5) / 8 / 8 + ((hmpprt::s32 ) (*(yPlaneDown_2 + ((h_17 + 1) * quarterWidth_2 + w_17))) * 3 + (hmpprt::s32 ) (*(yPlaneDown_2 + ((h_17 + 1) * quarterWidth_2 + w_17 + 1))) * 5) * 7 / 8 / 8);
 # 315 "sharpness.cpp"
 *(yPlaneCSER_3 + ((h_17 * 4 + 5) * width_30 + w_17 * 4 + 5)) = (hmpprt::u08 ) (((hmpprt::s32 ) (*(yPlaneDown_2 + (h_17 * quarterWidth_2 + w_17))) + (hmpprt::s32 ) (*(yPlaneDown_2 + (h_17 * quarterWidth_2 + w_17 + 1))) * 7) / 8 / 8 + ((hmpprt::s32 ) (*(yPlaneDown_2 + ((h_17 + 1) * quarterWidth_2 + w_17))) + (hmpprt::s32 ) (*(yPlaneDown_2 + ((h_17 + 1) * quarterWidth_2 + w_17 + 1))) * 7) * 7 / 8 / 8);
 # 289 "sharpness.cpp"
 __hmppcg_label_11:;
}
#endif // __HIPCC__



# 289 "sharpness.cpp"

#ifndef __HIPCC__
void hmpp_acc_region_main_289_internal_1(hmpprt::s32* width_67, hmpprt::s32 quarterWidth_16, hmpprt::s32 quarterHeight_16, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  yPlaneCSER_38, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  yPlaneDown_16)
{
 # 291 "sharpness.cpp"
 hmpprt::s32 width_44;
 # 291 "sharpness.cpp"
 width_44 = *width_67;
 # 291 "sharpness.cpp"
 {
  # 289 "sharpness.cpp"
  if ((quarterHeight_16 +  -1) * (quarterWidth_16 - 1) - 1 >= 0)
  {
   hmpprt::CUDAGridCall __hmppcg_call;
   __hmppcg_call.setSizeX(((quarterHeight_16 +  -1) * (quarterWidth_16 - 1) - 1) / 128 + 1);
   __hmppcg_call.setSizeY(1);
   __hmppcg_call.setBlockSizeX(32);
   __hmppcg_call.setBlockSizeY(4);
   __hmppcg_call.addLocalParameter((hmpprt::s32) (width_44), "width_30");
   __hmppcg_call.addLocalParameter((hmpprt::s32) (quarterWidth_16), "quarterWidth_2");
   __hmppcg_call.addLocalParameter((hmpprt::s32) (quarterHeight_16), "quarterHeight_1");
   __hmppcg_call.addLocalParameter(&yPlaneCSER_38, 8, "yPlaneCSER_3");
   __hmppcg_call.addLocalParameter(&yPlaneDown_16, 8, "yPlaneDown_2");
   __hmppcg_call.launch(hmpp_acc_region_main_289_loop1D_1, hmpprt::Context::getInstance()->getCUDADevice());
  }
  ;
 }
 # 289 "sharpness.cpp"
 *width_67 = width_44;
}
#endif // __HIPCC__



# 289 "sharpness.cpp"

#ifndef __HIPCC__
extern "C" CDLT_API  void hmpp_acc_region_main_289(hmpprt::s32* width_8, hmpprt::s32 quarterWidth_10, hmpprt::s32 quarterHeight_7, hmpprt::u08* yPlaneCSER_23, hmpprt::u08* yPlaneDown_8)
{
 # 323 "sharpness.cpp"
 (hmpp_acc_region_main_289_internal_1(width_8, quarterWidth_10, quarterHeight_7, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08> (yPlaneCSER_23), hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08> (yPlaneDown_8)));
}
#endif // __HIPCC__



# 323 "sharpness.cpp"

#ifdef __HIPCC__

extern "C" __global__ void hmpp_acc_region_main_326_parallel_region_1(hmpprt::s32 height_7, hmpprt::s16* pError, hmpprt::s32 width_7, hmpprt::u08* yPlane, hmpprt::u08* yPlaneCSER_2)
{
 # 325 "sharpness.cpp"
 {
  # 329 "sharpness.cpp"
  hmpprt::s32 iter_per_gang_7;
  # 329 "sharpness.cpp"
  hmpprt::s32 first_gang_iter_7;
  # 329 "sharpness.cpp"
  hmpprt::s32 last_gang_iter_7;
  # 329 "sharpness.cpp"
  iter_per_gang_7 = ((1 + (height_7 * width_7 - 1) / 192) > 256 ? (1 + (height_7 * width_7 - 1) / 192) : 256);
  # 329 "sharpness.cpp"
  first_gang_iter_7 = (hmpprt::gr_gbidx()) * iter_per_gang_7;
  # 329 "sharpness.cpp"
  last_gang_iter_7 = ((first_gang_iter_7 + iter_per_gang_7 - 1) < (height_7 * width_7 - 1) ? (first_gang_iter_7 + iter_per_gang_7 - 1) : (height_7 * width_7 - 1));
  # 329 "sharpness.cpp"
  hmpprt::s32 h_23;
  # 329 "sharpness.cpp"
  # 333 "sharpness.cpp"
  for (h_23 = first_gang_iter_7 + (hmpprt::gr_btidy()) ; h_23 <= last_gang_iter_7 ; h_23 = h_23 + (hmpprt::gr_btnumy()))
  {
   # 329 "sharpness.cpp"
   hmpprt::s32 h_18;
   # 332 "sharpness.cpp"
   hmpprt::s32 w_18;
   # 334 "sharpness.cpp"
   w_18 = h_23 % width_7;
   # 334 "sharpness.cpp"
   h_18 = h_23 / width_7;
   # 334 "sharpness.cpp"
   *(pError + (h_18 * width_7 + w_18)) = (hmpprt::s16 ) ((hmpprt::s32 ) (*(yPlane + (h_18 * width_7 + w_18))) - (hmpprt::s32 ) (*(yPlaneCSER_2 + (h_18 * width_7 + w_18))));
  }
  # 323 "sharpness.cpp"
 }
}
#endif // __HIPCC__



# 323 "sharpness.cpp"

#ifndef __HIPCC__
void hmpp_acc_region_main_326_internal_1(hmpprt::s32 width_66, hmpprt::s32 height_36, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  yPlane_10, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  yPlaneCSER_37, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s16>  pError_4)
{
 # 323 "sharpness.cpp"
 if (1)
 {
  hmpprt::CUDAGridCall __hmppcg_call;
  __hmppcg_call.setSizeX(192);
  __hmppcg_call.setSizeY(1);
  __hmppcg_call.setBlockSizeX(1);
  __hmppcg_call.setBlockSizeY(256);
  __hmppcg_call.addLocalParameter((hmpprt::s32) (height_36), "height_7");
  __hmppcg_call.addLocalParameter(&pError_4, 8, "pError");
  __hmppcg_call.addLocalParameter((hmpprt::s32) (width_66), "width_7");
  __hmppcg_call.addLocalParameter(&yPlane_10, 8, "yPlane");
  __hmppcg_call.addLocalParameter(&yPlaneCSER_37, 8, "yPlaneCSER_2");
  __hmppcg_call.launch(hmpp_acc_region_main_326_parallel_region_1, hmpprt::Context::getInstance()->getCUDADevice());
 }
 ;
}
#endif // __HIPCC__



# 323 "sharpness.cpp"

#ifndef __HIPCC__
extern "C" CDLT_API  void hmpp_acc_region_main_326(hmpprt::s32 width_48, hmpprt::s32 height_25, hmpprt::u08* yPlane_5, hmpprt::u08* yPlaneCSER_22, hmpprt::s16* pError_2)
{
 # 342 "sharpness.cpp"
 (hmpp_acc_region_main_326_internal_1(width_48, height_25, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08> (yPlane_5), hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08> (yPlaneCSER_22), hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s16> (pError_2)));
}
#endif // __HIPCC__



# 342 "sharpness.cpp"

#ifdef __HIPCC__

extern "C" __global__ void hmpp_acc_region_main_346_parallel_region_1(hmpprt::s32 height_6, hmpprt::s16* pEdge, hmpprt::s32 width_6, hmpprt::u08* yPlane_4)
{
 # 344 "sharpness.cpp"
 {
  # 349 "sharpness.cpp"
  hmpprt::s32 iter_per_gang_6;
  # 349 "sharpness.cpp"
  hmpprt::s32 first_gang_iter_6;
  # 349 "sharpness.cpp"
  hmpprt::s32 last_gang_iter_6;
  # 349 "sharpness.cpp"
  iter_per_gang_6 = ((1 + (height_6 * width_6 - 1) / 192) > 256 ? (1 + (height_6 * width_6 - 1) / 192) : 256);
  # 349 "sharpness.cpp"
  first_gang_iter_6 = (hmpprt::gr_gbidx()) * iter_per_gang_6;
  # 349 "sharpness.cpp"
  last_gang_iter_6 = ((first_gang_iter_6 + iter_per_gang_6 - 1) < (height_6 * width_6 - 1) ? (first_gang_iter_6 + iter_per_gang_6 - 1) : (height_6 * width_6 - 1));
  # 349 "sharpness.cpp"
  hmpprt::s32 y_2;
  # 349 "sharpness.cpp"
  # 353 "sharpness.cpp"
  for (y_2 = first_gang_iter_6 + (hmpprt::gr_btidy()) ; y_2 <= last_gang_iter_6 ; y_2 = y_2 + (hmpprt::gr_btnumy()))
  {
   # 349 "sharpness.cpp"
   hmpprt::s32 y_1;
   # 352 "sharpness.cpp"
   hmpprt::s32 x_1;
   # 354 "sharpness.cpp"
   x_1 = y_2 % width_6;
   # 354 "sharpness.cpp"
   y_1 = y_2 / width_6;
   # 354 "sharpness.cpp"
   if (y_1 == 0 || y_1 == height_6 - 1 || x_1 == 0 || x_1 == width_6 - 1)
   {
    # 356 "sharpness.cpp"
    *(pEdge + (y_1 * width_6 + x_1)) = (hmpprt::s16) 0;
   }
   else
   {
    # 360 "sharpness.cpp"
    hmpprt::s32 gx;
    # 360 "sharpness.cpp"
    gx = (hmpprt::s32 ) (*(yPlane_4 + ((y_1 - 1) * width_6 + x_1 + 1))) + ((hmpprt::s32 ) (*(yPlane_4 + (y_1 * width_6 + x_1 + 1))) << 1) + (hmpprt::s32 ) (*(yPlane_4 + ((y_1 + 1) * width_6 + x_1 + 1))) - ((hmpprt::s32 ) (*(yPlane_4 + ((y_1 - 1) * width_6 + x_1 - 1))) + ((hmpprt::s32 ) (*(yPlane_4 + (y_1 * width_6 + x_1 - 1))) << 1) + (hmpprt::s32 ) (*(yPlane_4 + ((y_1 + 1) * width_6 + x_1 - 1))));
    # 361 "sharpness.cpp"
    hmpprt::s32 gy;
    # 361 "sharpness.cpp"
    gy = (hmpprt::s32 ) (*(yPlane_4 + ((y_1 - 1) * width_6 + x_1 - 1))) + ((hmpprt::s32 ) (*(yPlane_4 + ((y_1 - 1) * width_6 + x_1))) << 1) + (hmpprt::s32 ) (*(yPlane_4 + ((y_1 - 1) * width_6 + x_1 + 1))) - ((hmpprt::s32 ) (*(yPlane_4 + ((y_1 + 1) * width_6 + x_1 - 1))) + ((hmpprt::s32 ) (*(yPlane_4 + ((y_1 + 1) * width_6 + x_1))) << 1) + (hmpprt::s32 ) (*(yPlane_4 + ((y_1 + 1) * width_6 + x_1 + 1))));
    # 362 "sharpness.cpp"
    hmpprt::s32 edgeV;
    # 362 "sharpness.cpp"
    edgeV = ((gx) > 0 ? (gx) : -(gx)) + ((gy) > 0 ? (gy) : -(gy));
    # 363 "sharpness.cpp"
    *(pEdge + (y_1 * width_6 + x_1)) = (hmpprt::s16 ) (edgeV);
   }
  }
  # 342 "sharpness.cpp"
 }
}
#endif // __HIPCC__



# 342 "sharpness.cpp"

#ifndef __HIPCC__
void hmpp_acc_region_main_346_internal_1(hmpprt::s32 width_65, hmpprt::s32 height_35, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  yPlane_14, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s16>  pEdge_8)
{
 # 342 "sharpness.cpp"
 if (1)
 {
  hmpprt::CUDAGridCall __hmppcg_call;
  __hmppcg_call.setSizeX(192);
  __hmppcg_call.setSizeY(1);
  __hmppcg_call.setBlockSizeX(1);
  __hmppcg_call.setBlockSizeY(256);
  __hmppcg_call.addLocalParameter((hmpprt::s32) (height_35), "height_6");
  __hmppcg_call.addLocalParameter(&pEdge_8, 8, "pEdge");
  __hmppcg_call.addLocalParameter((hmpprt::s32) (width_65), "width_6");
  __hmppcg_call.addLocalParameter(&yPlane_14, 8, "yPlane_4");
  __hmppcg_call.launch(hmpp_acc_region_main_346_parallel_region_1, hmpprt::Context::getInstance()->getCUDADevice());
 }
 ;
}
#endif // __HIPCC__



# 342 "sharpness.cpp"

#ifndef __HIPCC__
extern "C" CDLT_API  void hmpp_acc_region_main_346(hmpprt::s32 width_47, hmpprt::s32 height_24, hmpprt::u08* yPlane_9, hmpprt::s16* pEdge_4)
{
 # 372 "sharpness.cpp"
 (hmpp_acc_region_main_346_internal_1(width_47, height_24, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08> (yPlane_9), hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s16> (pEdge_4)));
}
#endif // __HIPCC__



# 372 "sharpness.cpp"

#ifdef __HIPCC__

extern "C" __global__ void reduce_s64_add_xy(hmpprt::s64 neutral, hmpprt::s64* scalar, hmpprt::s32 size, hmpprt::s64* array)
{
 
 hmpprt::s64 * tmpa = (hmpprt::s64 *)(((char *)hmpp_sharedmem + 0));
 
 # 372 "sharpness.cpp"
 hmpprt::s64 var;
 # 372 "sharpness.cpp"
 var = neutral;
 # 372 "sharpness.cpp"
 hmpprt::s32 end_5;
 # 372 "sharpness.cpp"
 hmpprt::s32 i;
 # 372 "sharpness.cpp"
 # 372 "sharpness.cpp"
 # 372 "sharpness.cpp"
 for (i = (hmpprt::gr_btidf()), end_5 = size - 1 ; i <= end_5 ; i = i + 512)
 {
  # 372 "sharpness.cpp"
  var = var + *(array + i);
 }
 # 372 "sharpness.cpp"
 # 372 "sharpness.cpp"
 *(tmpa + (hmpprt::gr_btidf())) = var;
 # 372 "sharpness.cpp"
 hmpprt::s32 n_1;
 # 372 "sharpness.cpp"
 # 372 "sharpness.cpp"
 for (n_1 = (511 >> 1) + 1 ; n_1 > 0 ; n_1 = (n_1 >> 1))
 {
  # 372 "sharpness.cpp"
  hmpprt::s32 next_1;
  # 372 "sharpness.cpp"
  next_1 = (hmpprt::gr_btidf()) + n_1;
  # 372 "sharpness.cpp"
  (hmpprt::gr_barrier());
  # 372 "sharpness.cpp"
  if ((hmpprt::gr_btidf()) < n_1 && next_1 < 512)
  {
   # 372 "sharpness.cpp"
   *(tmpa + (hmpprt::gr_btidf())) = *(tmpa + (hmpprt::gr_btidf())) + *(tmpa + next_1);
  }
 }
 # 372 "sharpness.cpp"
 # 372 "sharpness.cpp"
 if ((hmpprt::gr_btidf()) == 0)
 {
  # 372 "sharpness.cpp"
  *scalar = *tmpa + *scalar;
 }
}
#endif // __HIPCC__



# 372 "sharpness.cpp"

#ifdef __HIPCC__

extern "C" __global__ void hmpp_acc_region_main_375_parallel_region_1(hmpprt::s64* gang_prv_red_sum_2, hmpprt::s32 height_5, hmpprt::s16* pEdge_3, hmpprt::s32 width_5)
{
 
 hmpprt::s64 * sum_3 = (hmpprt::s64 *)(((char *)hmpp_sharedmem + 0));
 
 hmpprt::s64 * tmpr_2 = (hmpprt::s64 *)(((char *)hmpp_sharedmem + 0+8));
 
 # 374 "sharpness.cpp"
 hmpprt::s64 tmp_27;
 # 374 "sharpness.cpp"
 if ((hmpprt::gr_btidf()) == 0)
 {
  # 374 "sharpness.cpp"
  *sum_3 = 0;
 }
 # 374 "sharpness.cpp"
 (hmpprt::gr_barrier());
 # 374 "sharpness.cpp"
 {
  # 372 "sharpness.cpp"
  hmpprt::s32 iter_per_gang_5;
  # 372 "sharpness.cpp"
  hmpprt::s32 first_gang_iter_5;
  # 372 "sharpness.cpp"
  hmpprt::s32 last_gang_iter_5;
  # 372 "sharpness.cpp"
  iter_per_gang_5 = ((1 + (height_5 * width_5 - 1) / 192) > 256 ? (1 + (height_5 * width_5 - 1) / 192) : 256);
  # 372 "sharpness.cpp"
  first_gang_iter_5 = (hmpprt::gr_gbidx()) * iter_per_gang_5;
  # 372 "sharpness.cpp"
  last_gang_iter_5 = ((first_gang_iter_5 + iter_per_gang_5 - 1) < (height_5 * width_5 - 1) ? (first_gang_iter_5 + iter_per_gang_5 - 1) : (height_5 * width_5 - 1));
  # 372 "sharpness.cpp"
  hmpprt::s64 sum_2;
  # 378 "sharpness.cpp"
  sum_2 = 0;
  # 378 "sharpness.cpp"
  hmpprt::s32 i_4;
  # 378 "sharpness.cpp"
  # 382 "sharpness.cpp"
  for (i_4 = first_gang_iter_5 + (hmpprt::gr_btidy()) ; i_4 <= last_gang_iter_5 ; i_4 = i_4 + (hmpprt::gr_btnumy()))
  {
   # 378 "sharpness.cpp"
   hmpprt::s32 i_2;
   # 381 "sharpness.cpp"
   hmpprt::s32 j_2;
   # 383 "sharpness.cpp"
   j_2 = i_4 % width_5;
   # 383 "sharpness.cpp"
   i_2 = i_4 / width_5;
   # 383 "sharpness.cpp"
   sum_2 = sum_2 + (hmpprt::s64 ) (*(pEdge_3 + (i_2 * width_5 + j_2)));
  }
  # 372 "sharpness.cpp"
  # 372 "sharpness.cpp"
  *sum_3 = (reduce_s64_add_y(sum_2, tmpr_2, *sum_3));
 }
 # 372 "sharpness.cpp"
 tmp_27 = *sum_3;
 # 372 "sharpness.cpp"
 if ((hmpprt::gr_btidy()) == 0)
 {
  # 372 "sharpness.cpp"
  *(gang_prv_red_sum_2 + (hmpprt::gr_gbidx())) = tmp_27;
 }
 # 372 "sharpness.cpp"
 (hmpprt::gr_barrier());
}
#endif // __HIPCC__



# 372 "sharpness.cpp"

#ifndef __HIPCC__
void hmpp_acc_region_main_375_internal_1(hmpprt::s32 width_64, hmpprt::s32 height_34, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s16>  pEdge_11, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s64>  sum_4)
{
 # 372 "sharpness.cpp"
 hmpprt::DevicePtr<hmpprt::MS_CUDA_SHARED,hmpprt::s64>  sum_1;
 # 372 "sharpness.cpp"
 hmpprt::s64 tmp_28;
 # 372 "sharpness.cpp"
 hmpprt::Context::getInstance()->allocate((void **) (&sum_1), hmpprt::MS_CUDA_SHARED, 8);
 # 372 "sharpness.cpp"
 hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s64>  gang_prv_red_sum_1;
 # 372 "sharpness.cpp"
 hmpprt::Context::getInstance()->allocate((void **) (&gang_prv_red_sum_1), hmpprt::MS_CUDA_GLOB, 1536);
 # 372 "sharpness.cpp"
 hmpprt::DevicePtr<hmpprt::MS_CUDA_SHARED,hmpprt::s64>  tmpr_3;
 # 372 "sharpness.cpp"
 hmpprt::Context::getInstance()->allocate((void **) (&tmpr_3), hmpprt::MS_CUDA_SHARED, 2048);
 # 372 "sharpness.cpp"
 if (1)
 {
  hmpprt::CUDAGridCall __hmppcg_call;
  __hmppcg_call.setSizeX(192);
  __hmppcg_call.setSizeY(1);
  __hmppcg_call.setBlockSizeX(1);
  __hmppcg_call.setBlockSizeY(256);
  __hmppcg_call.addLocalParameter(&gang_prv_red_sum_1, 8, "gang_prv_red_sum_2");
  __hmppcg_call.addLocalParameter((hmpprt::s32) (height_34), "height_5");
  __hmppcg_call.addLocalParameter(&pEdge_11, 8, "pEdge_3");
  __hmppcg_call.addSharedParameter(sum_1, "sum_3");
  __hmppcg_call.addSharedParameter(tmpr_3, "tmpr_2");
  __hmppcg_call.addLocalParameter((hmpprt::s32) (width_64), "width_5");
  __hmppcg_call.launch(hmpp_acc_region_main_375_parallel_region_1, hmpprt::Context::getInstance()->getCUDADevice());
 }
 ;
 # 372 "sharpness.cpp"
 hmpprt::DevicePtr<hmpprt::MS_CUDA_SHARED,hmpprt::s64>  tmpr_1;
 # 372 "sharpness.cpp"
 hmpprt::Context::getInstance()->allocate((void **) (&tmpr_1), hmpprt::MS_CUDA_SHARED, 4096);
 # 372 "sharpness.cpp"
 tmp_28 = 0LL;
 # 372 "sharpness.cpp"
 if (1)
 {
  hmpprt::CUDAGridCall __hmppcg_call;
  __hmppcg_call.setSizeX(1);
  __hmppcg_call.setSizeY(1);
  __hmppcg_call.setBlockSizeX(512);
  __hmppcg_call.setBlockSizeY(1);
  __hmppcg_call.addLocalParameter(&tmp_28, 8, "neutral");
  __hmppcg_call.addLocalParameter(&sum_4, 8, "scalar");
  __hmppcg_call.addSharedParameter(tmpr_1, "tmpa");
  __hmppcg_call.addLocalParameter((hmpprt::s32) (192), "size");
  __hmppcg_call.addLocalParameter(&gang_prv_red_sum_1, 8, "array");
  __hmppcg_call.launch(reduce_s64_add_xy, hmpprt::Context::getInstance()->getCUDADevice());
 }
 ;
 # 372 "sharpness.cpp"
 hmpprt::Context::getInstance()->free((void **) (&gang_prv_red_sum_1));
 # 372 "sharpness.cpp"
 hmpprt::Context::getInstance()->free((void **) (&sum_1));
 # 372 "sharpness.cpp"
 hmpprt::Context::getInstance()->free((void **) (&tmpr_1));
 # 372 "sharpness.cpp"
 hmpprt::Context::getInstance()->free((void **) (&tmpr_3));
}
#endif // __HIPCC__



# 372 "sharpness.cpp"

#ifndef __HIPCC__
extern "C" CDLT_API  void hmpp_acc_region_main_375(hmpprt::s32 width_46, hmpprt::s32 height_23, hmpprt::s16* pEdge_7, hmpprt::s64* sum)
{
 # 391 "sharpness.cpp"
 (hmpp_acc_region_main_375_internal_1(width_46, height_23, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s16> (pEdge_7), hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s64> (sum)));
}
#endif // __HIPCC__



# 391 "sharpness.cpp"

#ifdef __HIPCC__

extern "C" __global__ void hmpp_acc_region_main_411_parallel_region_1(double cutoff, double cutoff2, hmpprt::s32 height_4, double low, double order, double order2, hmpprt::s16* pEdge_2, hmpprt::s16* pError_1, hmpprt::s16* pSharpness, double peak, hmpprt::s32 width_4, hmpprt::u08* yPlaneCSER_1)
{
 # 393 "sharpness.cpp"
 {
  # 414 "sharpness.cpp"
  hmpprt::s32 iter_per_gang_4;
  # 414 "sharpness.cpp"
  hmpprt::s32 first_gang_iter_4;
  # 414 "sharpness.cpp"
  hmpprt::s32 last_gang_iter_4;
  # 414 "sharpness.cpp"
  iter_per_gang_4 = ((1 + (height_4 * width_4 - 1) / 192) > 256 ? (1 + (height_4 * width_4 - 1) / 192) : 256);
  # 414 "sharpness.cpp"
  first_gang_iter_4 = (hmpprt::gr_gbidx()) * iter_per_gang_4;
  # 414 "sharpness.cpp"
  last_gang_iter_4 = ((first_gang_iter_4 + iter_per_gang_4 - 1) < (height_4 * width_4 - 1) ? (first_gang_iter_4 + iter_per_gang_4 - 1) : (height_4 * width_4 - 1));
  # 414 "sharpness.cpp"
  hmpprt::s32 h_24;
  # 414 "sharpness.cpp"
  # 418 "sharpness.cpp"
  for (h_24 = first_gang_iter_4 + (hmpprt::gr_btidy()) ; h_24 <= last_gang_iter_4 ; h_24 = h_24 + (hmpprt::gr_btnumy()))
  {
   # 419 "sharpness.cpp"
   hmpprt::s16 edge;
   # 414 "sharpness.cpp"
   hmpprt::s32 h_19;
   # 417 "sharpness.cpp"
   hmpprt::s32 w_19;
   # 419 "sharpness.cpp"
   w_19 = h_24 % width_4;
   # 419 "sharpness.cpp"
   h_19 = h_24 / width_4;
   # 419 "sharpness.cpp"
   edge = *(pEdge_2 + (h_19 * width_4 + w_19));
   # 420 "sharpness.cpp"
   double weight;
   # 420 "sharpness.cpp"
   weight = (double) 1. / ((double) 1. + pow((double ) (edge) / cutoff2, (double) 2. * order2));
   # 421 "sharpness.cpp"
   double strength;
   # 421 "sharpness.cpp"
   strength = (peak - low) * ((double) 1. - (double) 1. / ((double) 1. + pow((double ) (edge) / cutoff, (double) 2. * order)));
   # 422 "sharpness.cpp"
   strength = strength * weight + low;
   # 424 "sharpness.cpp"
   *(pSharpness + (h_19 * width_4 + w_19)) = (hmpprt::s16 ) (*(yPlaneCSER_1 + (h_19 * width_4 + w_19)));
   # 425 "sharpness.cpp"
   *(pSharpness + (h_19 * width_4 + w_19)) = *(pSharpness + (h_19 * width_4 + w_19)) + (double ) (*(pError_1 + (h_19 * width_4 + w_19))) * strength;
  }
  # 391 "sharpness.cpp"
 }
}
#endif // __HIPCC__



# 391 "sharpness.cpp"

#ifndef __HIPCC__
void hmpp_acc_region_main_411_internal_1(hmpprt::s32 width_63, hmpprt::s32 height_33, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  yPlaneCSER_34, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s16>  pError_5, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s16>  pEdge_6, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s16>  pSharpness_6, double cutoff_2, double peak_2, double cutoff2_2, double order_2, double order2_2, double low_2)
{
 # 391 "sharpness.cpp"
 if (1)
 {
  hmpprt::CUDAGridCall __hmppcg_call;
  __hmppcg_call.setSizeX(192);
  __hmppcg_call.setSizeY(1);
  __hmppcg_call.setBlockSizeX(1);
  __hmppcg_call.setBlockSizeY(256);
  __hmppcg_call.addLocalParameter(&cutoff_2, 8, "cutoff");
  __hmppcg_call.addLocalParameter(&cutoff2_2, 8, "cutoff2");
  __hmppcg_call.addLocalParameter((hmpprt::s32) (height_33), "height_4");
  __hmppcg_call.addLocalParameter(&low_2, 8, "low");
  __hmppcg_call.addLocalParameter(&order_2, 8, "order");
  __hmppcg_call.addLocalParameter(&order2_2, 8, "order2");
  __hmppcg_call.addLocalParameter(&pEdge_6, 8, "pEdge_2");
  __hmppcg_call.addLocalParameter(&pError_5, 8, "pError_1");
  __hmppcg_call.addLocalParameter(&pSharpness_6, 8, "pSharpness");
  __hmppcg_call.addLocalParameter(&peak_2, 8, "peak");
  __hmppcg_call.addLocalParameter((hmpprt::s32) (width_63), "width_4");
  __hmppcg_call.addLocalParameter(&yPlaneCSER_34, 8, "yPlaneCSER_1");
  __hmppcg_call.launch(hmpp_acc_region_main_411_parallel_region_1, hmpprt::Context::getInstance()->getCUDADevice());
 }
 ;
}
#endif // __HIPCC__



# 391 "sharpness.cpp"

#ifndef __HIPCC__
extern "C" CDLT_API  void hmpp_acc_region_main_411(hmpprt::s32 width_45, hmpprt::s32 height_22, hmpprt::u08* yPlaneCSER_16, hmpprt::s16* pError_3, hmpprt::s16* pEdge_10, hmpprt::s16* pSharpness_3, double cutoff_1, double peak_1, double cutoff2_1, double order_1, double order2_1, double low_1)
{
 # 433 "sharpness.cpp"
 (hmpp_acc_region_main_411_internal_1(width_45, height_22, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08> (yPlaneCSER_16), hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s16> (pError_3), hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s16> (pEdge_10), hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s16> (pSharpness_3), cutoff_1, peak_1, cutoff2_1, order_1, order2_1, low_1));
}
#endif // __HIPCC__



# 433 "sharpness.cpp"

#ifdef __HIPCC__

extern "C" __global__ void hmpp_acc_region_main_436_parallel_region_1(float cutoff3, hmpprt::s32 height_3, float low3, float order3, hmpprt::s16* pEdge_1, hmpprt::s16* pSharpness_2, float peak3, hmpprt::s32 width_3, hmpprt::u08* yPlane_3)
{
 # 435 "sharpness.cpp"
 {
  # 439 "sharpness.cpp"
  hmpprt::s32 iter_per_gang_3;
  # 439 "sharpness.cpp"
  hmpprt::s32 first_gang_iter_3;
  # 439 "sharpness.cpp"
  hmpprt::s32 last_gang_iter_3;
  # 439 "sharpness.cpp"
  iter_per_gang_3 = ((1 + (height_3 * width_3 - 1) / 192) > 256 ? (1 + (height_3 * width_3 - 1) / 192) : 256);
  # 439 "sharpness.cpp"
  first_gang_iter_3 = (hmpprt::gr_gbidx()) * iter_per_gang_3;
  # 439 "sharpness.cpp"
  last_gang_iter_3 = ((first_gang_iter_3 + iter_per_gang_3 - 1) < (height_3 * width_3 - 1) ? (first_gang_iter_3 + iter_per_gang_3 - 1) : (height_3 * width_3 - 1));
  # 439 "sharpness.cpp"
  hmpprt::s32 h_25;
  # 439 "sharpness.cpp"
  # 443 "sharpness.cpp"
  for (h_25 = first_gang_iter_3 + (hmpprt::gr_btidy()) ; h_25 <= last_gang_iter_3 ; h_25 = h_25 + (hmpprt::gr_btnumy()))
  {
   # 439 "sharpness.cpp"
   hmpprt::s32 h_20;
   # 442 "sharpness.cpp"
   hmpprt::s32 w_20;
   # 444 "sharpness.cpp"
   w_20 = h_25 % width_3;
   # 444 "sharpness.cpp"
   h_20 = h_25 / width_3;
   # 444 "sharpness.cpp"
   if (!(h_20 == 0 || h_20 == height_3 - 1 || w_20 == 0 || w_20 == width_3 - 1))
   {
    # 446 "sharpness.cpp"
    hmpprt::s32 enhanced;
    # 446 "sharpness.cpp"
    hmpprt::s32 maxValue;
    # 446 "sharpness.cpp"
    hmpprt::s32 minValue;
    # 447 "sharpness.cpp"
    double oscStrength;
    # 448 "sharpness.cpp"
    enhanced = (hmpprt::s32 ) (*(pSharpness_2 + (h_20 * width_3 + w_20)));
    # 450 "sharpness.cpp"
    maxValue = (hmpprt::s32 ) ( ((hmpprt::s32 ) ( ((hmpprt::s32 ) (*(yPlane_3 + ((h_20 - 1) * width_3 + w_20 - 1))) > (hmpprt::s32 ) (*(yPlane_3 + ((h_20 - 1) * width_3 + w_20)))) ? (*(yPlane_3 + ((h_20 - 1) * width_3 + w_20 - 1))) : (*(yPlane_3 + ((h_20 - 1) * width_3 + w_20)))) > (hmpprt::s32 ) (*(yPlane_3 + ((h_20 - 1) * width_3 + w_20 + 1)))) ? ( ((hmpprt::s32 ) (*(yPlane_3 + ((h_20 - 1) * width_3 + w_20 - 1))) > (hmpprt::s32 ) (*(yPlane_3 + ((h_20 - 1) * width_3 + w_20)))) ? (*(yPlane_3 + ((h_20 - 1) * width_3 + w_20 - 1))) : (*(yPlane_3 + ((h_20 - 1) * width_3 + w_20)))) : (*(yPlane_3 + ((h_20 - 1) * width_3 + w_20 + 1))));
    # 451 "sharpness.cpp"
    maxValue = ( (maxValue > (hmpprt::s32 ) ( ((hmpprt::s32 ) ( ((hmpprt::s32 ) (*(yPlane_3 + (h_20 * width_3 + w_20 - 1))) > (hmpprt::s32 ) (*(yPlane_3 + (h_20 * width_3 + w_20)))) ? (*(yPlane_3 + (h_20 * width_3 + w_20 - 1))) : (*(yPlane_3 + (h_20 * width_3 + w_20)))) > (hmpprt::s32 ) (*(yPlane_3 + (h_20 * width_3 + w_20 + 1)))) ? ( ((hmpprt::s32 ) (*(yPlane_3 + (h_20 * width_3 + w_20 - 1))) > (hmpprt::s32 ) (*(yPlane_3 + (h_20 * width_3 + w_20)))) ? (*(yPlane_3 + (h_20 * width_3 + w_20 - 1))) : (*(yPlane_3 + (h_20 * width_3 + w_20)))) : (*(yPlane_3 + (h_20 * width_3 + w_20 + 1))))) ? maxValue : ((hmpprt::s32 ) ( ((hmpprt::s32 ) ( ((hmpprt::s32 ) (*(yPlane_3 + (h_20 * width_3 + w_20 - 1))) > (hmpprt::s32 ) (*(yPlane_3 + (h_20 * width_3 + w_20)))) ? (*(yPlane_3 + (h_20 * width_3 + w_20 - 1))) : (*(yPlane_3 + (h_20 * width_3 + w_20)))) > (hmpprt::s32 ) (*(yPlane_3 + (h_20 * width_3 + w_20 + 1)))) ? ( ((hmpprt::s32 ) (*(yPlane_3 + (h_20 * width_3 + w_20 - 1))) > (hmpprt::s32 ) (*(yPlane_3 + (h_20 * width_3 + w_20)))) ? (*(yPlane_3 + (h_20 * width_3 + w_20 - 1))) : (*(yPlane_3 + (h_20 * width_3 + w_20)))) : (*(yPlane_3 + (h_20 * width_3 + w_20 + 1))))));
    # 452 "sharpness.cpp"
    maxValue = ( (maxValue > (hmpprt::s32 ) ( ((hmpprt::s32 ) ( ((hmpprt::s32 ) (*(yPlane_3 + ((h_20 + 1) * width_3 + w_20 - 1))) > (hmpprt::s32 ) (*(yPlane_3 + ((h_20 + 1) * width_3 + w_20)))) ? (*(yPlane_3 + ((h_20 + 1) * width_3 + w_20 - 1))) : (*(yPlane_3 + ((h_20 + 1) * width_3 + w_20)))) > (hmpprt::s32 ) (*(yPlane_3 + ((h_20 + 1) * width_3 + w_20 + 1)))) ? ( ((hmpprt::s32 ) (*(yPlane_3 + ((h_20 + 1) * width_3 + w_20 - 1))) > (hmpprt::s32 ) (*(yPlane_3 + ((h_20 + 1) * width_3 + w_20)))) ? (*(yPlane_3 + ((h_20 + 1) * width_3 + w_20 - 1))) : (*(yPlane_3 + ((h_20 + 1) * width_3 + w_20)))) : (*(yPlane_3 + ((h_20 + 1) * width_3 + w_20 + 1))))) ? maxValue : ((hmpprt::s32 ) ( ((hmpprt::s32 ) ( ((hmpprt::s32 ) (*(yPlane_3 + ((h_20 + 1) * width_3 + w_20 - 1))) > (hmpprt::s32 ) (*(yPlane_3 + ((h_20 + 1) * width_3 + w_20)))) ? (*(yPlane_3 + ((h_20 + 1) * width_3 + w_20 - 1))) : (*(yPlane_3 + ((h_20 + 1) * width_3 + w_20)))) > (hmpprt::s32 ) (*(yPlane_3 + ((h_20 + 1) * width_3 + w_20 + 1)))) ? ( ((hmpprt::s32 ) (*(yPlane_3 + ((h_20 + 1) * width_3 + w_20 - 1))) > (hmpprt::s32 ) (*(yPlane_3 + ((h_20 + 1) * width_3 + w_20)))) ? (*(yPlane_3 + ((h_20 + 1) * width_3 + w_20 - 1))) : (*(yPlane_3 + ((h_20 + 1) * width_3 + w_20)))) : (*(yPlane_3 + ((h_20 + 1) * width_3 + w_20 + 1))))));
    # 454 "sharpness.cpp"
    minValue = (hmpprt::s32 ) ( ((hmpprt::s32 ) (*(yPlane_3 + ((h_20 - 1) * width_3 + w_20 - 1))) < (hmpprt::s32 ) ( ((hmpprt::s32 ) (*(yPlane_3 + ((h_20 - 1) * width_3 + w_20))) < (hmpprt::s32 ) (*(yPlane_3 + ((h_20 - 1) * width_3 + w_20 + 1)))) ? (*(yPlane_3 + ((h_20 - 1) * width_3 + w_20))) : (*(yPlane_3 + ((h_20 - 1) * width_3 + w_20 + 1))))) ? (*(yPlane_3 + ((h_20 - 1) * width_3 + w_20 - 1))) : ( ((hmpprt::s32 ) (*(yPlane_3 + ((h_20 - 1) * width_3 + w_20))) < (hmpprt::s32 ) (*(yPlane_3 + ((h_20 - 1) * width_3 + w_20 + 1)))) ? (*(yPlane_3 + ((h_20 - 1) * width_3 + w_20))) : (*(yPlane_3 + ((h_20 - 1) * width_3 + w_20 + 1)))));
    # 455 "sharpness.cpp"
    minValue = ( (minValue < (hmpprt::s32 ) ( ((hmpprt::s32 ) (*(yPlane_3 + (h_20 * width_3 + w_20 - 1))) < (hmpprt::s32 ) ( ((hmpprt::s32 ) (*(yPlane_3 + (h_20 * width_3 + w_20))) < (hmpprt::s32 ) (*(yPlane_3 + (h_20 * width_3 + w_20 + 1)))) ? (*(yPlane_3 + (h_20 * width_3 + w_20))) : (*(yPlane_3 + (h_20 * width_3 + w_20 + 1))))) ? (*(yPlane_3 + (h_20 * width_3 + w_20 - 1))) : ( ((hmpprt::s32 ) (*(yPlane_3 + (h_20 * width_3 + w_20))) < (hmpprt::s32 ) (*(yPlane_3 + (h_20 * width_3 + w_20 + 1)))) ? (*(yPlane_3 + (h_20 * width_3 + w_20))) : (*(yPlane_3 + (h_20 * width_3 + w_20 + 1)))))) ? minValue : ((hmpprt::s32 ) ( ((hmpprt::s32 ) (*(yPlane_3 + (h_20 * width_3 + w_20 - 1))) < (hmpprt::s32 ) ( ((hmpprt::s32 ) (*(yPlane_3 + (h_20 * width_3 + w_20))) < (hmpprt::s32 ) (*(yPlane_3 + (h_20 * width_3 + w_20 + 1)))) ? (*(yPlane_3 + (h_20 * width_3 + w_20))) : (*(yPlane_3 + (h_20 * width_3 + w_20 + 1))))) ? (*(yPlane_3 + (h_20 * width_3 + w_20 - 1))) : ( ((hmpprt::s32 ) (*(yPlane_3 + (h_20 * width_3 + w_20))) < (hmpprt::s32 ) (*(yPlane_3 + (h_20 * width_3 + w_20 + 1)))) ? (*(yPlane_3 + (h_20 * width_3 + w_20))) : (*(yPlane_3 + (h_20 * width_3 + w_20 + 1)))))));
    # 456 "sharpness.cpp"
    minValue = ( (minValue < (hmpprt::s32 ) ( ((hmpprt::s32 ) (*(yPlane_3 + ((h_20 + 1) * width_3 + w_20 - 1))) < (hmpprt::s32 ) ( ((hmpprt::s32 ) (*(yPlane_3 + ((h_20 + 1) * width_3 + w_20))) < (hmpprt::s32 ) (*(yPlane_3 + ((h_20 + 1) * width_3 + w_20 + 1)))) ? (*(yPlane_3 + ((h_20 + 1) * width_3 + w_20))) : (*(yPlane_3 + ((h_20 + 1) * width_3 + w_20 + 1))))) ? (*(yPlane_3 + ((h_20 + 1) * width_3 + w_20 - 1))) : ( ((hmpprt::s32 ) (*(yPlane_3 + ((h_20 + 1) * width_3 + w_20))) < (hmpprt::s32 ) (*(yPlane_3 + ((h_20 + 1) * width_3 + w_20 + 1)))) ? (*(yPlane_3 + ((h_20 + 1) * width_3 + w_20))) : (*(yPlane_3 + ((h_20 + 1) * width_3 + w_20 + 1)))))) ? minValue : ((hmpprt::s32 ) ( ((hmpprt::s32 ) (*(yPlane_3 + ((h_20 + 1) * width_3 + w_20 - 1))) < (hmpprt::s32 ) ( ((hmpprt::s32 ) (*(yPlane_3 + ((h_20 + 1) * width_3 + w_20))) < (hmpprt::s32 ) (*(yPlane_3 + ((h_20 + 1) * width_3 + w_20 + 1)))) ? (*(yPlane_3 + ((h_20 + 1) * width_3 + w_20))) : (*(yPlane_3 + ((h_20 + 1) * width_3 + w_20 + 1))))) ? (*(yPlane_3 + ((h_20 + 1) * width_3 + w_20 - 1))) : ( ((hmpprt::s32 ) (*(yPlane_3 + ((h_20 + 1) * width_3 + w_20))) < (hmpprt::s32 ) (*(yPlane_3 + ((h_20 + 1) * width_3 + w_20 + 1)))) ? (*(yPlane_3 + ((h_20 + 1) * width_3 + w_20))) : (*(yPlane_3 + ((h_20 + 1) * width_3 + w_20 + 1)))))));
    # 460 "sharpness.cpp"
    double cutoff2Order;
    # 460 "sharpness.cpp"
    cutoff2Order = pow((double ) (cutoff3), (double ) ((float) 2. * order3));
    # 461 "sharpness.cpp"
    double edge2Order;
    # 461 "sharpness.cpp"
    edge2Order = pow((double ) (*(pEdge_1 + (h_20 * width_3 + w_20))), (double) 2. * (double ) (order3));
    # 462 "sharpness.cpp"
    oscStrength = ((double ) (peak3) * cutoff2Order + (double ) (low3) * edge2Order) / (cutoff2Order + edge2Order);
    # 466 "sharpness.cpp"
    if (enhanced > maxValue)
    {
     # 468 "sharpness.cpp"
     *(pSharpness_2 + (h_20 * width_3 + w_20)) = (hmpprt::s16 ) ( ((double ) (maxValue) + oscStrength * (double ) (enhanced - maxValue) < (double) 255.) ? ((double ) (maxValue) + oscStrength * (double ) (enhanced - maxValue)) : ((double) 255.));
    }
    else
    {
     # 470 "sharpness.cpp"
     if (enhanced < minValue)
     {
      # 472 "sharpness.cpp"
      *(pSharpness_2 + (h_20 * width_3 + w_20)) = (hmpprt::s16 ) ( ((double) 0. > (double ) (minValue) - oscStrength * (double ) (minValue - enhanced)) ? ((double) 0.) : ((double ) (minValue) - oscStrength * (double ) (minValue - enhanced)));
     }
     else
     {
      # 476 "sharpness.cpp"
      *(pSharpness_2 + (h_20 * width_3 + w_20)) = (hmpprt::s16 ) ( (( (enhanced > 0) ? enhanced : 0) < 255) ? ( (enhanced > 0) ? enhanced : 0) : 255);
     }
    }
   }
  }
  # 433 "sharpness.cpp"
 }
}
#endif // __HIPCC__



# 433 "sharpness.cpp"

#ifndef __HIPCC__
void hmpp_acc_region_main_436_internal_1(hmpprt::s32 width_62, hmpprt::s32 height_32, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  yPlane_13, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s16>  pEdge_9, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s16>  pSharpness_8, float cutoff3_2, float order3_2, float peak3_2, float low3_2)
{
 # 433 "sharpness.cpp"
 if (1)
 {
  hmpprt::CUDAGridCall __hmppcg_call;
  __hmppcg_call.setSizeX(192);
  __hmppcg_call.setSizeY(1);
  __hmppcg_call.setBlockSizeX(1);
  __hmppcg_call.setBlockSizeY(256);
  __hmppcg_call.addLocalParameter((float) (cutoff3_2) , "cutoff3");
  __hmppcg_call.addLocalParameter((hmpprt::s32) (height_32), "height_3");
  __hmppcg_call.addLocalParameter((float) (low3_2) , "low3");
  __hmppcg_call.addLocalParameter((float) (order3_2) , "order3");
  __hmppcg_call.addLocalParameter(&pEdge_9, 8, "pEdge_1");
  __hmppcg_call.addLocalParameter(&pSharpness_8, 8, "pSharpness_2");
  __hmppcg_call.addLocalParameter((float) (peak3_2) , "peak3");
  __hmppcg_call.addLocalParameter((hmpprt::s32) (width_62), "width_3");
  __hmppcg_call.addLocalParameter(&yPlane_13, 8, "yPlane_3");
  __hmppcg_call.launch(hmpp_acc_region_main_436_parallel_region_1, hmpprt::Context::getInstance()->getCUDADevice());
 }
 ;
}
#endif // __HIPCC__



# 433 "sharpness.cpp"

#ifndef __HIPCC__
extern "C" CDLT_API  void hmpp_acc_region_main_436(hmpprt::s32 width_43, hmpprt::s32 height_21, hmpprt::u08* yPlane_8, hmpprt::s16* pEdge_5, hmpprt::s16* pSharpness_5, float cutoff3_1, float order3_1, float peak3_1, float low3_1)
{
 # 486 "sharpness.cpp"
 (hmpp_acc_region_main_436_internal_1(width_43, height_21, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08> (yPlane_8), hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s16> (pEdge_5), hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s16> (pSharpness_5), cutoff3_1, order3_1, peak3_1, low3_1));
}
#endif // __HIPCC__



# 486 "sharpness.cpp"

#ifdef __HIPCC__

extern "C" __global__ void hmpp_acc_region_main_490_parallel_region_1(hmpprt::s32 height_2, hmpprt::s16* pSharpness_1, hmpprt::s32 width_2, hmpprt::u08* yPlane_2)
{
 # 488 "sharpness.cpp"
 {
  # 493 "sharpness.cpp"
  hmpprt::s32 iter_per_gang_2;
  # 493 "sharpness.cpp"
  hmpprt::s32 first_gang_iter_2;
  # 493 "sharpness.cpp"
  hmpprt::s32 last_gang_iter_2;
  # 493 "sharpness.cpp"
  iter_per_gang_2 = ((1 + (height_2 * width_2 - 1) / 192) > 256 ? (1 + (height_2 * width_2 - 1) / 192) : 256);
  # 493 "sharpness.cpp"
  first_gang_iter_2 = (hmpprt::gr_gbidx()) * iter_per_gang_2;
  # 493 "sharpness.cpp"
  last_gang_iter_2 = ((first_gang_iter_2 + iter_per_gang_2 - 1) < (height_2 * width_2 - 1) ? (first_gang_iter_2 + iter_per_gang_2 - 1) : (height_2 * width_2 - 1));
  # 493 "sharpness.cpp"
  hmpprt::s32 h_26;
  # 493 "sharpness.cpp"
  # 497 "sharpness.cpp"
  for (h_26 = first_gang_iter_2 + (hmpprt::gr_btidy()) ; h_26 <= last_gang_iter_2 ; h_26 = h_26 + (hmpprt::gr_btnumy()))
  {
   # 493 "sharpness.cpp"
   hmpprt::s32 h_21;
   # 496 "sharpness.cpp"
   hmpprt::s32 w_21;
   # 498 "sharpness.cpp"
   w_21 = h_26 % width_2;
   # 498 "sharpness.cpp"
   h_21 = h_26 / width_2;
   # 498 "sharpness.cpp"
   *(yPlane_2 + (h_21 * width_2 + w_21)) = (hmpprt::u08 ) ( ((hmpprt::s32 ) (*(pSharpness_1 + (h_21 * width_2 + w_21))) > 255) ? 255 : ( ((hmpprt::s32 ) (*(pSharpness_1 + (h_21 * width_2 + w_21))) < 0) ? 0 : ((hmpprt::s32 ) (*(pSharpness_1 + (h_21 * width_2 + w_21))))));
  }
  # 486 "sharpness.cpp"
 }
}
#endif // __HIPCC__



# 486 "sharpness.cpp"

#ifndef __HIPCC__
void hmpp_acc_region_main_490_internal_1(hmpprt::s32 width_61, hmpprt::s32 height_31, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  yPlane_12, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s16>  pSharpness_7)
{
 # 486 "sharpness.cpp"
 if (1)
 {
  hmpprt::CUDAGridCall __hmppcg_call;
  __hmppcg_call.setSizeX(192);
  __hmppcg_call.setSizeY(1);
  __hmppcg_call.setBlockSizeX(1);
  __hmppcg_call.setBlockSizeY(256);
  __hmppcg_call.addLocalParameter((hmpprt::s32) (height_31), "height_2");
  __hmppcg_call.addLocalParameter(&pSharpness_7, 8, "pSharpness_1");
  __hmppcg_call.addLocalParameter((hmpprt::s32) (width_61), "width_2");
  __hmppcg_call.addLocalParameter(&yPlane_12, 8, "yPlane_2");
  __hmppcg_call.launch(hmpp_acc_region_main_490_parallel_region_1, hmpprt::Context::getInstance()->getCUDADevice());
 }
 ;
}
#endif // __HIPCC__



# 486 "sharpness.cpp"

#ifndef __HIPCC__
extern "C" CDLT_API  void hmpp_acc_region_main_490(hmpprt::s32 width_34, hmpprt::s32 height_20, hmpprt::u08* yPlane_7, hmpprt::s16* pSharpness_4)
{
 # 23 "sharpness.cpp"
 (hmpp_acc_region_main_490_internal_1(width_34, height_20, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08> (yPlane_7), hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s16> (pSharpness_4)));
}
#endif // __HIPCC__



# 23 "sharpness.cpp"

#ifdef __HIPCC__

extern "C" __global__ void hmpp_acc_region_main_84_parallel_region_1(hmpprt::s32 height_1, hmpprt::s32 quarterWidth_1, hmpprt::s32 width_1, hmpprt::u08* yPlaneDown_1, hmpprt::u08* yPlane_1)
{
 # 25 "sharpness.cpp"
 {
  # 87 "sharpness.cpp"
  hmpprt::s32 iter_per_gang_1;
  # 87 "sharpness.cpp"
  hmpprt::s32 first_gang_iter_1;
  # 87 "sharpness.cpp"
  hmpprt::s32 last_gang_iter_1;
  # 87 "sharpness.cpp"
  iter_per_gang_1 = ((1 + ((height_1 + 3) / 4 * ((width_1 + 3) / 4) - 1) / 192) > 256 ? (1 + ((height_1 + 3) / 4 * ((width_1 + 3) / 4) - 1) / 192) : 256);
  # 87 "sharpness.cpp"
  first_gang_iter_1 = (hmpprt::gr_gbidx()) * iter_per_gang_1;
  # 87 "sharpness.cpp"
  last_gang_iter_1 = ((first_gang_iter_1 + iter_per_gang_1 - 1) < ((height_1 + 3) / 4 * ((width_1 + 3) / 4) - 1) ? (first_gang_iter_1 + iter_per_gang_1 - 1) : ((height_1 + 3) / 4 * ((width_1 + 3) / 4) - 1));
  # 87 "sharpness.cpp"
  hmpprt::s32 i_5;
  # 87 "sharpness.cpp"
  # 91 "sharpness.cpp"
  for (i_5 = first_gang_iter_1 + (hmpprt::gr_btidy()) ; i_5 <= last_gang_iter_1 ; i_5 = i_5 + (hmpprt::gr_btnumy()))
  {
   # 87 "sharpness.cpp"
   hmpprt::u08 tbt_1;
   # 87 "sharpness.cpp"
   hmpprt::s32 tbt_2;
   # 87 "sharpness.cpp"
   hmpprt::s32 tbt_3;
   # 87 "sharpness.cpp"
   hmpprt::s32 i_3;
   # 90 "sharpness.cpp"
   hmpprt::s32 j_3;
   # 92 "sharpness.cpp"
   j_3 = i_5 % ((width_1 + 3) / 4);
   # 92 "sharpness.cpp"
   i_3 = i_5 / ((width_1 + 3) / 4);
   # 92 "sharpness.cpp"
   tbt_1 = *(yPlane_1 + ((i_3 * 4 + 1) * width_1 + j_3 * 4 + 3));
   # 92 "sharpness.cpp"
   tbt_2 = (hmpprt::s32 ) (*(yPlane_1 + ((i_3 * 4 + 1) * width_1 + j_3 * 4 + 2)));
   # 92 "sharpness.cpp"
   tbt_3 = (hmpprt::s32 ) (*(yPlane_1 + (i_3 * 4 * width_1 + j_3 * 4))) + (hmpprt::s32 ) (*(yPlane_1 + (i_3 * 4 * width_1 + j_3 * 4 + 1))) + (hmpprt::s32 ) (*(yPlane_1 + (i_3 * 4 * width_1 + j_3 * 4 + 2))) + (hmpprt::s32 ) (*(yPlane_1 + (i_3 * 4 * width_1 + j_3 * 4 + 3))) + (hmpprt::s32 ) (*(yPlane_1 + ((i_3 * 4 + 1) * width_1 + j_3 * 4))) + (hmpprt::s32 ) (*(yPlane_1 + ((i_3 * 4 + 1) * width_1 + j_3 * 4 + 1)));
   # 92 "sharpness.cpp"
   *(yPlaneDown_1 + (i_3 * 4 / 4 * quarterWidth_1 + j_3 * 4 / 4)) = (hmpprt::u08 ) (tbt_3 + tbt_2 + (hmpprt::s32 ) (tbt_1) + (hmpprt::s32 ) (*(yPlane_1 + ((i_3 * 4 + 2) * width_1 + j_3 * 4))) + (hmpprt::s32 ) (*(yPlane_1 + ((i_3 * 4 + 2) * width_1 + j_3 * 4 + 1))) + (hmpprt::s32 ) (*(yPlane_1 + ((i_3 * 4 + 2) * width_1 + j_3 * 4 + 2))) + (hmpprt::s32 ) (*(yPlane_1 + ((i_3 * 4 + 2) * width_1 + j_3 * 4 + 3))) + (hmpprt::s32 ) (*(yPlane_1 + ((i_3 * 4 + 3) * width_1 + j_3 * 4))) + (hmpprt::s32 ) (*(yPlane_1 + ((i_3 * 4 + 3) * width_1 + j_3 * 4 + 1))) + (hmpprt::s32 ) (*(yPlane_1 + ((i_3 * 4 + 3) * width_1 + j_3 * 4 + 2))) + (hmpprt::s32 ) (*(yPlane_1 + ((i_3 * 4 + 3) * width_1 + j_3 * 4 + 3))) + 8 >> 4);
  }
  # 23 "sharpness.cpp"
 }
}
#endif // __HIPCC__



# 23 "sharpness.cpp"

#ifndef __HIPCC__
void hmpp_acc_region_main_84_internal_1(hmpprt::s32 width_60, hmpprt::s32 height_30, hmpprt::s32 quarterWidth_23, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  yPlane_11, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08>  yPlaneDown_15)
{
 # 23 "sharpness.cpp"
 if (1)
 {
  hmpprt::CUDAGridCall __hmppcg_call;
  __hmppcg_call.setSizeX(192);
  __hmppcg_call.setSizeY(1);
  __hmppcg_call.setBlockSizeX(1);
  __hmppcg_call.setBlockSizeY(256);
  __hmppcg_call.addLocalParameter((hmpprt::s32) (height_30), "height_1");
  __hmppcg_call.addLocalParameter((hmpprt::s32) (quarterWidth_23), "quarterWidth_1");
  __hmppcg_call.addLocalParameter((hmpprt::s32) (width_60), "width_1");
  __hmppcg_call.addLocalParameter(&yPlaneDown_15, 8, "yPlaneDown_1");
  __hmppcg_call.addLocalParameter(&yPlane_11, 8, "yPlane_1");
  __hmppcg_call.launch(hmpp_acc_region_main_84_parallel_region_1, hmpprt::Context::getInstance()->getCUDADevice());
 }
 ;
}
#endif // __HIPCC__



# 23 "sharpness.cpp"

#ifndef __HIPCC__
extern "C" CDLT_API  void hmpp_acc_region_main_84(hmpprt::s32 width_32, hmpprt::s32 height_16, hmpprt::s32 quarterWidth_9, hmpprt::u08* yPlane_6, hmpprt::u08* yPlaneDown_7)
{
 # 1 "<preprocessor>"
 (hmpp_acc_region_main_84_internal_1(width_32, height_16, quarterWidth_9, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08> (yPlane_6), hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u08> (yPlaneDown_7)));
}
#endif // __HIPCC__



# 1 "<preprocessor>"

#ifdef __HIPCC__
__device__ hmpprt::s64 reduce_s64_add_y(hmpprt::s64 priv, hmpprt::s64* tmpa_1, hmpprt::s64 orig)
{
 # 1 "<preprocessor>"
 (hmpprt::gr_barrier());
 # 1 "<preprocessor>"
 *(tmpa_1 + (hmpprt::gr_btidy()) + (hmpprt::gr_btnumy()) * (hmpprt::gr_btidx())) = priv;
 # 1 "<preprocessor>"
 hmpprt::s32 n;
 # 1 "<preprocessor>"
 # 1 "<preprocessor>"
 for (n = ((hmpprt::gr_btnumy()) - 1 >> 1) + 1 ; n > 0 ; n = (n >> 1))
 {
  # 1 "<preprocessor>"
  hmpprt::s32 next;
  # 1 "<preprocessor>"
  next = (hmpprt::gr_btidy()) + n;
  # 1 "<preprocessor>"
  (hmpprt::gr_barrier());
  # 1 "<preprocessor>"
  if ((hmpprt::gr_btidy()) < n && next < (hmpprt::gr_btnumy()))
  {
   # 1 "<preprocessor>"
   *(tmpa_1 + (hmpprt::gr_btidy()) + (hmpprt::gr_btnumy()) * (hmpprt::gr_btidx())) = *(tmpa_1 + (hmpprt::gr_btidy()) + (hmpprt::gr_btnumy()) * (hmpprt::gr_btidx())) + *(tmpa_1 + next + (hmpprt::gr_btnumy()) * (hmpprt::gr_btidx()));
  }
 }
 # 1 "<preprocessor>"
 # 1 "<preprocessor>"
 (hmpprt::gr_barrier());
 # 1 "<preprocessor>"
 return *(tmpa_1 + (hmpprt::gr_btnumy()) * (hmpprt::gr_btidx())) + orig;
}
#endif // __HIPCC__




#ifndef __HIPCC__
extern "C" const char * hmpprt_cuda_get_gpu_code();

static hmpprt::CUDAModule * hmpprt_module = 0;
static int hmpprt_uses = 0;

extern "C" CDLT_API void * hmpprt_init()
{
  try
  {
    if (hmpprt_uses++ == 0)
    {
      hmpprt_module = new hmpprt::CUDAModule(hmpprt_cuda_get_gpu_code());
      hmpp_acc_region_main_107_parallel_region_1 = new hmpprt::CUDAGrid(hmpprt_module, "hmpp_acc_region_main_107_parallel_region_1");
      hmpp_acc_region_main_119_parallel_region_1 = new hmpprt::CUDAGrid(hmpprt_module, "hmpp_acc_region_main_119_parallel_region_1");
      hmpp_acc_region_main_140_loop1D_1 = new hmpprt::CUDAGrid(hmpprt_module, "hmpp_acc_region_main_140_loop1D_1");
      hmpp_acc_region_main_152_loop1D_1 = new hmpprt::CUDAGrid(hmpprt_module, "hmpp_acc_region_main_152_loop1D_1");
      hmpp_acc_region_main_164_loop1D_1 = new hmpprt::CUDAGrid(hmpprt_module, "hmpp_acc_region_main_164_loop1D_1");
      hmpp_acc_region_main_185_loop1D_1 = new hmpprt::CUDAGrid(hmpprt_module, "hmpp_acc_region_main_185_loop1D_1");
      hmpp_acc_region_main_197_loop1D_1 = new hmpprt::CUDAGrid(hmpprt_module, "hmpp_acc_region_main_197_loop1D_1");
      hmpp_acc_region_main_209_loop1D_1 = new hmpprt::CUDAGrid(hmpprt_module, "hmpp_acc_region_main_209_loop1D_1");
      hmpp_acc_region_main_230_loop1D_1 = new hmpprt::CUDAGrid(hmpprt_module, "hmpp_acc_region_main_230_loop1D_1");
      hmpp_acc_region_main_242_loop1D_1 = new hmpprt::CUDAGrid(hmpprt_module, "hmpp_acc_region_main_242_loop1D_1");
      hmpp_acc_region_main_254_loop1D_1 = new hmpprt::CUDAGrid(hmpprt_module, "hmpp_acc_region_main_254_loop1D_1");
      hmpp_acc_region_main_277_loop1D_1 = new hmpprt::CUDAGrid(hmpprt_module, "hmpp_acc_region_main_277_loop1D_1");
      hmpp_acc_region_main_289_loop1D_1 = new hmpprt::CUDAGrid(hmpprt_module, "hmpp_acc_region_main_289_loop1D_1");
      hmpp_acc_region_main_326_parallel_region_1 = new hmpprt::CUDAGrid(hmpprt_module, "hmpp_acc_region_main_326_parallel_region_1");
      hmpp_acc_region_main_346_parallel_region_1 = new hmpprt::CUDAGrid(hmpprt_module, "hmpp_acc_region_main_346_parallel_region_1");
      reduce_s64_add_xy = new hmpprt::CUDAGrid(hmpprt_module, "reduce_s64_add_xy");
      hmpp_acc_region_main_375_parallel_region_1 = new hmpprt::CUDAGrid(hmpprt_module, "hmpp_acc_region_main_375_parallel_region_1");
      hmpp_acc_region_main_411_parallel_region_1 = new hmpprt::CUDAGrid(hmpprt_module, "hmpp_acc_region_main_411_parallel_region_1");
      hmpp_acc_region_main_436_parallel_region_1 = new hmpprt::CUDAGrid(hmpprt_module, "hmpp_acc_region_main_436_parallel_region_1");
      hmpp_acc_region_main_490_parallel_region_1 = new hmpprt::CUDAGrid(hmpprt_module, "hmpp_acc_region_main_490_parallel_region_1");
      hmpp_acc_region_main_84_parallel_region_1 = new hmpprt::CUDAGrid(hmpprt_module, "hmpp_acc_region_main_84_parallel_region_1");

    }
    hmpprt::Context::getInstance()->getGrouplet()->setTarget(hmpprt::CUDA);
    hmpprt::Context::getInstance()->getGrouplet()->addSignature("hmpp_acc_region_main_107", "prototype hmpp_acc_region_main_107(width: s32, quarterWidth: s32, quarterHeight: s32, yPlaneCSER: ^cudaglob u8, yPlaneDown: ^cudaglob u8)");
    hmpprt::Context::getInstance()->getGrouplet()->addSignature("hmpp_acc_region_main_119", "prototype hmpp_acc_region_main_119(width: s32, quarterHeight: s32, yPlaneCSER: ^cudaglob u8)");
    hmpprt::Context::getInstance()->getGrouplet()->addSignature("hmpp_acc_region_main_140", "prototype hmpp_acc_region_main_140(width: ^host s32, height: s32, yPlaneCSER: ^cudaglob u8)");
    hmpprt::Context::getInstance()->getGrouplet()->addSignature("hmpp_acc_region_main_152", "prototype hmpp_acc_region_main_152(width: ^host s32, quarterWidth: s32, yPlaneCSER: ^cudaglob u8, yPlaneDown: ^cudaglob u8)");
    hmpprt::Context::getInstance()->getGrouplet()->addSignature("hmpp_acc_region_main_164", "prototype hmpp_acc_region_main_164(width: ^host s32, quarterWidth: s32, yPlaneCSER: ^cudaglob u8)");
    hmpprt::Context::getInstance()->getGrouplet()->addSignature("hmpp_acc_region_main_185", "prototype hmpp_acc_region_main_185(width: ^host s32, yPlaneCSER: ^cudaglob u8)");
    hmpprt::Context::getInstance()->getGrouplet()->addSignature("hmpp_acc_region_main_197", "prototype hmpp_acc_region_main_197(width: ^host s32, height: ^host s32, quarterWidth: s32, quarterHeight: s32, yPlaneCSER: ^cudaglob u8, yPlaneDown: ^cudaglob u8)");
    hmpprt::Context::getInstance()->getGrouplet()->addSignature("hmpp_acc_region_main_209", "prototype hmpp_acc_region_main_209(width: ^host s32, height: ^host s32, quarterWidth: s32, yPlaneCSER: ^cudaglob u8)");
    hmpprt::Context::getInstance()->getGrouplet()->addSignature("hmpp_acc_region_main_230", "prototype hmpp_acc_region_main_230(width: ^host s32, height: ^host s32, yPlaneCSER: ^cudaglob u8)");
    hmpprt::Context::getInstance()->getGrouplet()->addSignature("hmpp_acc_region_main_242", "prototype hmpp_acc_region_main_242(width: ^host s32, quarterWidth: s32, quarterHeight: s32, yPlaneCSER: ^cudaglob u8, yPlaneDown: ^cudaglob u8)");
    hmpprt::Context::getInstance()->getGrouplet()->addSignature("hmpp_acc_region_main_254", "prototype hmpp_acc_region_main_254(width: ^host s32, quarterHeight: s32, yPlaneCSER: ^cudaglob u8)");
    hmpprt::Context::getInstance()->getGrouplet()->addSignature("hmpp_acc_region_main_277", "prototype hmpp_acc_region_main_277(width: ^host s32, height: s32, yPlaneCSER: ^cudaglob u8)");
    hmpprt::Context::getInstance()->getGrouplet()->addSignature("hmpp_acc_region_main_289", "prototype hmpp_acc_region_main_289(width: ^host s32, quarterWidth: s32, quarterHeight: s32, yPlaneCSER: ^cudaglob u8, yPlaneDown: ^cudaglob u8)");
    hmpprt::Context::getInstance()->getGrouplet()->addSignature("hmpp_acc_region_main_326", "prototype hmpp_acc_region_main_326(width: s32, height: s32, yPlane: ^cudaglob u8, yPlaneCSER: ^cudaglob u8, pError: ^cudaglob s16)");
    hmpprt::Context::getInstance()->getGrouplet()->addSignature("hmpp_acc_region_main_346", "prototype hmpp_acc_region_main_346(width: s32, height: s32, yPlane: ^cudaglob u8, pEdge: ^cudaglob s16)");
    hmpprt::Context::getInstance()->getGrouplet()->addSignature("hmpp_acc_region_main_375", "prototype hmpp_acc_region_main_375(width: s32, height: s32, pEdge: ^cudaglob s16, sum: ^cudaglob s64)");
    hmpprt::Context::getInstance()->getGrouplet()->addSignature("hmpp_acc_region_main_411", "prototype hmpp_acc_region_main_411(width: s32, height: s32, yPlaneCSER: ^cudaglob u8, pError: ^cudaglob s16, pEdge: ^cudaglob s16, pSharpness: ^cudaglob s16, cutoff: double, peak: double, cutoff2: double, order: double, order2: double, low: double)");
    hmpprt::Context::getInstance()->getGrouplet()->addSignature("hmpp_acc_region_main_436", "prototype hmpp_acc_region_main_436(width: s32, height: s32, yPlane: ^cudaglob u8, pEdge: ^cudaglob s16, pSharpness: ^cudaglob s16, cutoff3: float, order3: float, peak3: float, low3: float)");
    hmpprt::Context::getInstance()->getGrouplet()->addSignature("hmpp_acc_region_main_490", "prototype hmpp_acc_region_main_490(width: s32, height: s32, yPlane: ^cudaglob u8, pSharpness: ^cudaglob s16)");
    hmpprt::Context::getInstance()->getGrouplet()->addSignature("hmpp_acc_region_main_84", "prototype hmpp_acc_region_main_84(width: s32, height: s32, quarterWidth: s32, yPlane: ^cudaglob u8, yPlaneDown: ^cudaglob u8)");

  }
  catch (hmpperr::Error & e)
  {
    return e.clone();
  }
  catch(...)
  {
    fprintf(stderr,"Unexpected error in hmpprt_init()\n");
    abort();
  }
  return 0;
}
#endif // __HIPCC__

#ifndef __HIPCC__
extern "C" CDLT_API void * hmpprt_fini()
{
  try
  {
    if (--hmpprt_uses == 0)
    {
      delete hmpp_acc_region_main_107_parallel_region_1;
      delete hmpp_acc_region_main_119_parallel_region_1;
      delete hmpp_acc_region_main_140_loop1D_1;
      delete hmpp_acc_region_main_152_loop1D_1;
      delete hmpp_acc_region_main_164_loop1D_1;
      delete hmpp_acc_region_main_185_loop1D_1;
      delete hmpp_acc_region_main_197_loop1D_1;
      delete hmpp_acc_region_main_209_loop1D_1;
      delete hmpp_acc_region_main_230_loop1D_1;
      delete hmpp_acc_region_main_242_loop1D_1;
      delete hmpp_acc_region_main_254_loop1D_1;
      delete hmpp_acc_region_main_277_loop1D_1;
      delete hmpp_acc_region_main_289_loop1D_1;
      delete hmpp_acc_region_main_326_parallel_region_1;
      delete hmpp_acc_region_main_346_parallel_region_1;
      delete reduce_s64_add_xy;
      delete hmpp_acc_region_main_375_parallel_region_1;
      delete hmpp_acc_region_main_411_parallel_region_1;
      delete hmpp_acc_region_main_436_parallel_region_1;
      delete hmpp_acc_region_main_490_parallel_region_1;
      delete hmpp_acc_region_main_84_parallel_region_1;

      delete hmpprt_module;
      hmpprt_module = 0;
    }
  }
  catch (hmpperr::Error & e)
  {
    return e.clone();
  }
  catch(...)
  {
    fprintf(stderr,"Unexpected error in hmpprt_fini()\n");
    abort();
  }
  return 0;
}
#endif // __HIPCC__

// footer
